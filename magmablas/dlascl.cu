#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @generated from zlascl.cu normal z -> d, Fri Jan 30 19:00:09 2015


       @author Mark Gates
*/
#include "common_magma.h"

#define NB 64


// each thread block does one NB x n block row of A.
// each thread does one row, starting from left edge and moving right.
__global__ void
dlascl_full(int m, int n, double mul, double* A, int lda)
{
    int ind = blockIdx.x * NB + threadIdx.x;

    A += ind;
    if (ind < m) {
        for(int j=0; j < n; j++ )
            A[j*lda] *= mul;
    }
}


// each thread block does one NB x n block row of A.
// each thread does one row, starting from left edge and moving right to diagonal.
__global__ void
dlascl_lower(int m, int n, double mul, double* A, int lda)
{
    int ind = blockIdx.x * NB + threadIdx.x;

    int break_d = (ind < n) ? ind : n-1;

    A += ind;
    if (ind < m) {
        for(int j=0; j <= break_d; j++ )
            A[j*lda] *= mul;
    }
}


// each thread block does one NB x n block row of A.
// each thread does one row, starting from right edge and moving left to diagonal.
__global__ void
dlascl_upper(int m, int n, double mul, double* A, int lda)
{
    int ind = blockIdx.x * NB + threadIdx.x;

    A += ind;
    if (ind < m) {
        for(int j=n-1; j >= ind; j--)
            A[j*lda] *= mul;
    }
}


/**
    Purpose
    -------
    DLASCL multiplies the M by N real matrix A by the real scalar
    CTO/CFROM.  This is done without over/underflow as long as the final
    result CTO*A(I,J)/CFROM does not over/underflow. TYPE specifies that
    A may be full, upper triangular, lower triangular.

    Arguments
    ---------
    \param[in]
    type    magma_type_t
            TYPE indices the storage type of the input matrix A.
            = MagmaFull:   full matrix.
            = MagmaLower:  lower triangular matrix.
            = MagmaUpper:  upper triangular matrix.
            Other formats that LAPACK supports, MAGMA does not currently support.

    \param[in]
    kl      INTEGER
            Unused, for LAPACK compatability.

    \param[in]
    ku      KU is INTEGER
            Unused, for LAPACK compatability.

    \param[in]
    cfrom   DOUBLE PRECISION

    \param[in]
    cto     DOUBLE PRECISION
    \n
            The matrix A is multiplied by CTO/CFROM. A(I,J) is computed
            without over/underflow if the final result CTO*A(I,J)/CFROM
            can be represented without over/underflow.
            CFROM must be nonzero. CFROM and CTO must not be NAN.

    \param[in]
    m       INTEGER
            The number of rows of the matrix A.  M >= 0.

    \param[in]
    n       INTEGER
            The number of columns of the matrix A.  N >= 0.

    \param[in,out]
    dA      DOUBLE PRECISION array, dimension (LDDA,N)
            The matrix to be multiplied by CTO/CFROM.  See TYPE for the
            storage type.

    \param[in]
    ldda    INTEGER
            The leading dimension of the array A.  LDDA >= max(1,M).

    \param[out]
    info    INTEGER
      -     = 0:  successful exit
      -     < 0:  if INFO = -i, the i-th argument had an illegal value.
    
    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_daux2
    ********************************************************************/
extern "C" void
magmablas_dlascl_q(
    magma_type_t type, magma_int_t kl, magma_int_t ku,
    double cfrom, double cto,
    magma_int_t m, magma_int_t n,
    magmaDouble_ptr dA, magma_int_t ldda,
    magma_queue_t queue,
    magma_int_t *info )
{
    *info = 0;
    if ( type != MagmaLower && type != MagmaUpper && type != MagmaFull )
        *info = -1;
    else if ( cfrom == 0 || isnan(cfrom) )
        *info = -4;
    else if ( isnan(cto) )
        *info = -5;
    else if ( m < 0 )
        *info = -6;
    else if ( n < 0 )
        *info = -3;
    else if ( ldda < max(1,m) )
        *info = -7;
    
    if (*info != 0) {
        magma_xerbla( __func__, -(*info) );
        return;  //info;
    }
    
    dim3 grid( (m + NB - 1)/NB );
    dim3 threads( NB );
    
    double smlnum, bignum, cfromc, ctoc, cto1, cfrom1, mul;
    magma_int_t done = false;
    
    // Uses over/underflow procedure from LAPACK dlascl
    // Get machine parameters
    smlnum = lapackf77_dlamch("s");
    bignum = 1 / smlnum;
    
    cfromc = cfrom;
    ctoc   = cto;
    int cnt = 0;
    while( ! done ) {
        cfrom1 = cfromc*smlnum;
        if( cfrom1 == cfromc ) {
            // cfromc is an inf.  Multiply by a correctly signed zero for
            // finite ctoc, or a nan if ctoc is infinite.
            mul  = ctoc / cfromc;
            done = true;
            cto1 = ctoc;
        }
        else {
            cto1 = ctoc / bignum;
            if( cto1 == ctoc ) {
                // ctoc is either 0 or an inf.  In both cases, ctoc itself
                // serves as the correct multiplication factor.
                mul  = ctoc;
                done = true;
                cfromc = 1;
            }
            else if( fabs(cfrom1) > fabs(ctoc) && ctoc != 0 ) {
                mul  = smlnum;
                done = false;
                cfromc = cfrom1;
            }
            else if( fabs(cto1) > fabs(cfromc) ) {
                mul  = bignum;
                done = false;
                ctoc = cto1;
            }
            else {
                mul  = ctoc / cfromc;
                done = true;
            }
        }
        
        if (type == MagmaLower) {
            dlascl_lower <<< grid, threads, 0, queue >>> (m, n, mul, dA, ldda);
        }
        else if (type == MagmaUpper) {
            dlascl_upper <<< grid, threads, 0, queue >>> (m, n, mul, dA, ldda);
        }
        else if (type == MagmaFull) {
            dlascl_full  <<< grid, threads, 0, queue >>> (m, n, mul, dA, ldda);
        }
     
        cnt += 1;
    }
}


/**
    @see magmablas_dlascl_q
    @ingroup magma_daux2
    ********************************************************************/
extern "C" void
magmablas_dlascl(
    magma_type_t type, magma_int_t kl, magma_int_t ku,
    double cfrom, double cto,
    magma_int_t m, magma_int_t n,
    magmaDouble_ptr dA, magma_int_t ldda,
    magma_int_t *info )
{
    magmablas_dlascl_q( type, kl, ku, cfrom, cto, m, n, dA, ldda, magma_stream, info );
}
