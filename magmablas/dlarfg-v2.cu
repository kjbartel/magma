#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.4.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       December 2013

       @generated d Tue Dec 17 13:18:45 2013

*/
#include "common_magma.h"

// 512 is maximum number of threads for CUDA capability 1.x
#define BLOCK_SIZE 512

#define PRECISION_d


__global__
void magma_dlarfg_gpu_kernel( int n, double* dx0, double* dx,
                              double *dtau, double *dxnorm, double* dAkk)
{
    const int i = threadIdx.x;
    const int j = i + BLOCK_SIZE * blockIdx.x;
    __shared__ double scale;
    double xnorm;

    double dxi;

#if (defined(PRECISION_s) || defined(PRECISION_d))
    if( n <= 1 ) {
#else
    if( n <= 0 ) {
#endif
        *dtau = MAGMA_D_ZERO;
        return;
    }

    if ( j < n-1)
        dxi = dx[j];

    xnorm = *dxnorm;
    double alpha = *dx0;

#if (defined(PRECISION_s) || defined(PRECISION_d))
    if ( xnorm != 0 ) {
       if (i == 0) {  
            double beta  = sqrt( alpha*alpha + xnorm*xnorm );
            beta  = -copysign( beta, alpha );

            // todo: deal with badly scaled vectors (see lapack's larfg)
            *dtau = (beta - alpha) / beta;
            *dAkk  = beta;

            scale = 1. / (alpha - beta);
       }
#else
    double alphar =  MAGMA_D_REAL(alpha), alphai = MAGMA_D_IMAG(alpha);
    if ( xnorm != 0 || alphai != 0) {
       if (i == 0) {
            double beta  = sqrt( alphar*alphar + alphai*alphai + xnorm*xnorm );
            beta  = -copysign( beta, alphar );

            // todo: deal with badly scaled vectors (see lapack's larfg)
            *dtau = MAGMA_D_MAKE((beta - alphar)/beta, -alphai/beta);
            *dAkk = MAGMA_D_MAKE(beta, 0.);

            alpha = MAGMA_D_MAKE( MAGMA_D_REAL(alpha) - beta, MAGMA_D_IMAG(alpha));
            scale = MAGMA_D_DIV( MAGMA_D_ONE, alpha);
      }
#endif

      // scale x
      __syncthreads();
      if ( xnorm != 0 && j < n-1)
          dx[j] = MAGMA_D_MUL(dxi, scale);

    } else
        *dtau = MAGMA_D_ZERO;
}


/*
   Generates Householder elementary reflector H = I - tau v v^T to reduce
     H [ dx0 ] = [ beta ]
       [ dx  ]   [ 0    ]
   with beta = ±norm( [dx0, dx] ) = ±dxnorm[0].
   Stores v over dx; first element of v is 1 and is not stored.
   Stores beta over dx0.
   Stores tau.  

   The difference with LAPACK's dlarfg is that the norm of dx, and hence beta,
   are computed outside the routine and passed to it in dxnorm (array on the GPU).
*/
extern "C" magma_int_t
magma_dlarfg_gpu( magma_int_t n, double *dx0, double *dx,
                  double *dtau, double *dxnorm, double *dAkk)
{
    dim3 blocks((n+BLOCK_SIZE-1) / BLOCK_SIZE);
    dim3 threads( BLOCK_SIZE );

    /* recomputing the norm */
    //magmablas_dnrm2_cols(n, 1, dx0, n, dxnorm);
    magmablas_dnrm2_cols(n-1, 1, dx0+1, n, dxnorm);

    magma_dlarfg_gpu_kernel<<< blocks, threads,
                               0, magma_stream >>>(n, dx0, dx, dtau, dxnorm, dAkk);

    return MAGMA_SUCCESS;
}

