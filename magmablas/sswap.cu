#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.4.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       December 2013

       @generated s Tue Dec 17 13:18:45 2013

*/
#include "common_magma.h"

#define BLOCK_SIZE 64

/*********************************************************
 *
 * SWAP BLAS: permute to set of N elements
 *
 ********************************************************/
/*
 *  First version: line per line
 */
typedef struct {
    float *A1;
    float *A2;
    int n, lda1, lda2;
} magmagpu_sswap_params_t;

__global__ void magmagpu_sswap( magmagpu_sswap_params_t params )
{
    unsigned int x = threadIdx.x + __mul24(blockDim.x, blockIdx.x);
    unsigned int offset1 = __mul24( x, params.lda1);
    unsigned int offset2 = __mul24( x, params.lda2);
    if( x < params.n )
    {
        float *A1  = params.A1 + offset1;
        float *A2  = params.A2 + offset2;
        float temp = *A1;
        *A1 = *A2;
        *A2 = temp;
    }
}

extern "C" void 
magmablas_sswap( magma_int_t n, float *dA1T, magma_int_t lda1, 
                 float *dA2T, magma_int_t lda2)
{
    int blocksize = 64;
    dim3 blocks( (n+blocksize-1) / blocksize, 1, 1);
    magmagpu_sswap_params_t params = { dA1T, dA2T, n, lda1, lda2 };
    magmagpu_sswap<<< blocks, blocksize, 0, magma_stream >>>( params );
}

