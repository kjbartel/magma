#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.5.0-beta3) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date July 2014

       @generated from zswap.cu normal z -> s, Fri Jul 18 17:34:12 2014

*/
#include "common_magma.h"

#define BLOCK_SIZE 64

/*********************************************************
 *
 * SWAP BLAS: permute to set of N elements
 *
 ********************************************************/
/*
 *  First version: line per line
 */
typedef struct {
    float *A1;
    float *A2;
    int n, lda1, lda2;
} magmagpu_sswap_params_t;

__global__ void magmagpu_sswap( magmagpu_sswap_params_t params )
{
    unsigned int x = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned int offset1 = x*params.lda1;
    unsigned int offset2 = x*params.lda2;
    if( x < params.n )
    {
        float *A1  = params.A1 + offset1;
        float *A2  = params.A2 + offset2;
        float temp = *A1;
        *A1 = *A2;
        *A2 = temp;
    }
}

extern "C" void 
magmablas_sswap( magma_int_t n, float *dA1T, magma_int_t lda1, 
                 float *dA2T, magma_int_t lda2)
{
    int blocksize = 64;
    dim3 blocks( (n+blocksize-1) / blocksize, 1, 1);
    magmagpu_sswap_params_t params = { dA1T, dA2T, n, lda1, lda2 };
    magmagpu_sswap<<< blocks, blocksize, 0, magma_stream >>>( params );
}

