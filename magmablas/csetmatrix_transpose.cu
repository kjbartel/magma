#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.4.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       December 2013

       @generated c Tue Dec 17 13:18:45 2013

*/
#include "common_magma.h"
#define PRECISION_c
#include "commonblas.h"


//
//      m, n - dimensions in the source (input) matrix.
//             This routine copies the ha matrix from the CPU
//             to dat on the GPU. In addition, the output matrix
//             is transposed. The routine uses a buffer of size
//             2*lddb*nb pointed to by dB (lddb > m) on the GPU. 
//             Note that lda >= m and lddat >= n.
//
extern "C" void 
magmablas_csetmatrix_transpose( magma_int_t m, magma_int_t n,
                                const magmaFloatComplex  *ha, magma_int_t lda, 
                                magmaFloatComplex       *dat, magma_int_t ldda,
                                magmaFloatComplex        *dB, magma_int_t lddb, magma_int_t nb )
{
    magma_int_t i = 0, j = 0, ib;

    /* Quick return */
    if ( (m == 0) || (n == 0) )
        return;

    if (lda < m || ldda < n || lddb < m){
        printf("Wrong arguments in %s.\n", __func__);
        return;
    }

    magma_queue_t stream[2];
    magma_queue_create( &stream[0] );
    magma_queue_create( &stream[1] );
   
    /* Move data from CPU to GPU in the first panel in the dB buffer */
    ib   = min(n-i, nb);
    magma_csetmatrix_async( m, ib,
                            ha + i*lda,             lda,
                            dB + (j%2) * nb * lddb, lddb, stream[j%2] );
    j++;

    for(i=nb; i<n; i+=nb){
       /* Move data from CPU to GPU in the second panel in the dB buffer */
       ib   = min(n-i, nb);
       magma_csetmatrix_async( m, ib,
                               ha+i*lda,               lda,
                               dB + (j%2) * nb * lddb, lddb, stream[j%2] );
       j++;
  
       /* Note that the previous panel (i.e., j%2) comes through the stream
          for the kernel so there is no need to synchronize.             */
       // magmablas_ctranspose2( dat+i-nb, ldda, dB + (j%2)*nb*lddb, lddb, m, nb);
       magmablas_ctranspose2s( dat+i-nb, ldda, dB + (j%2)*nb*lddb, lddb, m, nb, stream[j%2]);
    }

    /* Transpose the last part of the matrix.                            */
    j++;
    // magmablas_ctranspose2( dat+i-nb, ldda, dB + (j%2)*nb*lddb, lddb, m, ib);
    magmablas_ctranspose2s( dat+i-nb, ldda, dB + (j%2)*nb*lddb, lddb, m, ib, stream[j%2]);

    magma_queue_destroy( stream[0] );
    magma_queue_destroy( stream[1] );
}
