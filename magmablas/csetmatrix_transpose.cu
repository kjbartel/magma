#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.2.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       June 2012

       @generated c Thu Jun 28 12:31:22 2012

*/
#include "common_magma.h"
#define PRECISION_c
#include "commonblas.h"

extern "C" void
magmablas_ctranspose2s(hipFloatComplex *odata, magma_int_t ldo,
                       hipFloatComplex *idata, magma_int_t ldi,
                       magma_int_t m, magma_int_t n, hipStream_t *stream );


//
//      m, n - dimensions in the source (input) matrix.
//             This routine copies the ha matrix from the CPU
//             to dat on the GPU. In addition, the output matrix
//             is transposed. The routine uses a buffer of size
//             2*lddb*nb pointed to by dB (lddb > m) on the GPU. 
//             Note that lda >= m and lddat >= n.
//
extern "C" void 
magmablas_csetmatrix_transpose( magma_int_t m, magma_int_t n,
                                hipFloatComplex  *ha, magma_int_t lda, 
                                hipFloatComplex *dat, magma_int_t ldda,
                                hipFloatComplex  *dB, magma_int_t lddb, magma_int_t nb )
{
    magma_int_t i = 0, j = 0, ib;

    /* Quick return */
    if ( (m == 0) || (n == 0) )
        return;

    if (lda < m || ldda < n || lddb < m){
        printf("Wrong arguments in zhtodt.\n");
        return;
    }

    hipStream_t stream[2];
    magma_queue_create( &stream[0] );
    magma_queue_create( &stream[1] );
   
    /* Move data from CPU to GPU in the first panel in the dB buffer */
    ib   = min(n-i, nb);
    magma_csetmatrix_async( m, ib,
                            ha + i*lda,             lda,
                            dB + (j%2) * nb * lddb, lddb, stream[j%2] );
    j++;

    for(i=nb; i<n; i+=nb){
       /* Move data from CPU to GPU in the second panel in the dB buffer */
       ib   = min(n-i, nb);
       magma_csetmatrix_async( m, ib,
                               ha+i*lda,               lda,
                               dB + (j%2) * nb * lddb, lddb, stream[j%2] );
       j++;
  
       /* Note that the previous panel (i.e., j%2) comes through the stream
          for the kernel so there is no need to synchronize.             */
       // magmablas_ctranspose2( dat+i-nb, ldda, dB + (j%2)*nb*lddb, lddb, m, nb);
       magmablas_ctranspose2s( dat+i-nb, ldda, dB + (j%2)*nb*lddb, lddb, m, nb, &stream[j%2]);
    }

    /* Transpose the last part of the matrix.                            */
    j++;
    // magmablas_ctranspose2( dat+i-nb, ldda, dB + (j%2)*nb*lddb, lddb, m, ib);
    magmablas_ctranspose2s( dat+i-nb, ldda, dB + (j%2)*nb*lddb, lddb, m, ib, &stream[j%2]);

    magma_queue_destroy( stream[0] );
    magma_queue_destroy( stream[1] );
}

//===========================================================================
//  This version is similar to the above but for multiGPUs. The distribution
//  is 1D block cyclic. The input arrays are pointers for the corresponding 
//  GPUs. The streams are passed as argument, in contrast to the single GPU
//  routine.
//  NOTE: see magmablas_csetmatrix_transpose_mgpu.
//===========================================================================
extern "C" void 
magmablas_csetmatrix_transpose2( magma_int_t m, magma_int_t n,
                                 hipFloatComplex  *ha,  magma_int_t  lda, 
                                 hipFloatComplex **dat, magma_int_t *ldda,
                                 hipFloatComplex **dB,  magma_int_t  lddb, magma_int_t nb,
                                 magma_int_t num_gpus, hipStream_t stream[][2] )
{
    magma_int_t i = 0, j[4] = {0, 0, 0, 0}, ib, k = 0;

    /* Quick return */
    if ( (m == 0) || (n == 0) )
        return;

    if (lda < m || lddb < m){
        printf("Wrong arguments in zhtodt2.\n");
        return;
    }

    if (n<num_gpus*nb){
       for(i=0; i<n; i+=nb){
          k = (i/nb)%num_gpus;
          magma_setdevice(k);

          ib = min(n-i, nb);
          magma_csetmatrix_async( m, ib,
                                  ha+i*lda, lda,
                                  dB[k],    lddb, stream[k][0] );
       }
       for(i=0; i<n; i+=nb){
          k = (i/nb)%num_gpus;
          magma_setdevice(k);

          ib = min(n-i, nb);
          //magma_queue_sync( stream[k][0] );
          //magmablas_ctranspose2( dat[k]+ i/(nb*num_gpus)*nb, ldda[k],
          //                       dB[k], lddb, m, ib);
          magmablas_ctranspose2s( dat[k]+ i/(nb*num_gpus)*nb, ldda[k],
                                 dB[k], lddb, m, ib, &stream[k][0]);
       }
    } 
    else
    {
      for(i=0; i<(n + num_gpus*nb); i+=nb){
         k = (i/nb)%num_gpus;
         magma_setdevice(k);

         if (i<n){
            /* Move data from CPU to GPU in the second panel in the dB buffer */
            ib = min(n-i, nb);
            magma_csetmatrix_async( m, ib,
                                    ha+i*lda,                 lda,
                                    dB[k] + (j[k]%2)*nb*lddb, lddb, stream[k][j[k]%2] );
         }
         j[k]++;
  
         if (i> (num_gpus-1)*nb){
            /* Make sure that the previous panel (i.e., j[k]%2) has arrived 
               and transpose it directly into the dat matrix                  */
            //magma_queue_sync( stream[k][ j[k]%2 ] );
            ib = min(n - i + num_gpus*nb, nb);
            //magmablas_ctranspose2( dat[k]+ i/(nb*num_gpus)*nb -nb, ldda[k],
            //                       dB[k] +(j[k]%2)*nb*lddb, lddb, m, ib);
            magmablas_ctranspose2s( dat[k]+ i/(nb*num_gpus)*nb -nb, ldda[k],
                                   dB[k] +(j[k]%2)*nb*lddb, lddb, m, ib, &stream[k][j[k]%2]);

         }
      }
    }
}
