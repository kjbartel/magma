#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.4.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       December 2013

       @generated d Tue Dec 17 13:18:44 2013

*/
#include "common_magma.h"

/* ////////////////////////////////////////////////////////////////////////////
   -- This is an auxiliary routine called from dgehrd.  The routine is called
      in 16 blocks, 32 thread per block and initializes to zero the 1st
      32x32 block of A.
*/

__global__ void dset_to_zero(double *A, int lda)
{
    int ind = blockIdx.x*lda + threadIdx.x;

    A += ind;
    A[0] = MAGMA_D_ZERO;
//   A[16*lda] = 0.;
}

__global__ void dset_nbxnb_to_zero(int nb, double *A, int lda)
{
   int ind = blockIdx.x*lda + threadIdx.x, i, j;

   A += ind;
   for(i=0; i<nb; i+=32) {
     for(j=0; j<nb; j+=32)
         A[j] = MAGMA_D_ZERO;
     A += 32*lda;
   }
}

extern "C"
void dzero_32x32_block(double *A, magma_int_t lda)
{
  // dset_to_zero<<< 16, 32, 0, magma_stream >>>(A, lda);
  dset_to_zero<<< 32, 32, 0, magma_stream >>>(A, lda);
}

extern "C"
void dzero_nbxnb_block(magma_int_t nb, double *A, magma_int_t lda)
{
  dset_nbxnb_to_zero<<< 32, 32, 0, magma_stream >>>(nb, A, lda);
}

/* ////////////////////////////////////////////////////////////////////////////
   -- GPU kernel for initializing a matrix by 0
*/
#define dlaset_threads 64

__global__ void dlaset(int m, int n, double *A, int lda)
{
   int ibx = blockIdx.x * dlaset_threads;
   int iby = blockIdx.y * 32;

   int ind = ibx + threadIdx.x;

   A += ind + __mul24(iby, lda);

   #pragma unroll
   for(int i=0; i<32; i++)
     if (iby+i < n && ind < m)
        A[i*lda] = MAGMA_D_ZERO;
}

__global__ void dlaset_identity(int m, int n, double *A, int lda)
{
   int ibx = blockIdx.x * dlaset_threads;
   int iby = blockIdx.y * 32;

   int ind = ibx + threadIdx.x;

   A += ind + __mul24(iby, lda);

   #pragma unroll
   for(int i=0; i<32; i++)
     if (iby+i < n && ind < m) {
        if (ind != i+iby)
           A[i*lda] = MAGMA_D_ZERO;
        else
           A[i*lda] = MAGMA_D_ONE;
     }
}

__global__ void dlaset_identityonly(int m, int n, double *A, int lda)
{
   int ibx = blockIdx.x * dlaset_threads;
   int iby = blockIdx.y * 32;

   int ind = ibx + threadIdx.x;

   A += ind + __mul24(iby, lda);

   #pragma unroll
   for(int i=0; i<32; i++)
     if (iby+i < n && ind < m) {
        if (ind == i+iby)
           A[i*lda] = MAGMA_D_ONE;
     }
}


__global__ void dlasetlower(int m, int n, double *A, int lda)
{
   int ibx = blockIdx.x * dlaset_threads;
   int iby = blockIdx.y * 32;

   int ind = ibx + threadIdx.x;

   A += ind + __mul24(iby, lda);

   #pragma unroll
   for(int i=0; i<32; i++)
     if (iby+i < n && ind < m && ind > i+iby)
        A[i*lda] = MAGMA_D_ZERO;
}

__global__ void dlasetupper(int m, int n, double *A, int lda)
{
   int ibx = blockIdx.x * dlaset_threads;
   int iby = blockIdx.y * 32;

   int ind = ibx + threadIdx.x;

   A += ind + __mul24(iby, lda);

   #pragma unroll
   for(int i=0; i<32; i++)
     if (iby+i < n && ind < m && ind < i+iby)
        A[i*lda] = MAGMA_D_ZERO;
}


/* ////////////////////////////////////////////////////////////////////////////
   -- Set the m x n matrix pointed by A to 0 on the GPU.
*/
extern "C" void
magmablas_dlaset(char uplo, magma_int_t m, magma_int_t n,
                 double *A, magma_int_t lda)
{
   dim3 threads(dlaset_threads, 1, 1);
   dim3 grid(m/dlaset_threads+(m % dlaset_threads != 0), n/32+(n%32!=0));

   if (m!=0 && n !=0)
     if (uplo == MagmaLower)
        dlasetlower<<< grid, threads, 0, magma_stream >>> (m, n, A, lda);
     else if (uplo == MagmaUpper)
        dlasetupper<<< grid, threads, 0, magma_stream >>> (m, n, A, lda);
     else
        dlaset<<< grid, threads, 0, magma_stream >>> (m, n, A, lda);
}

/* ////////////////////////////////////////////////////////////////////////////
   -- Set the m x n matrix pointed by A to I on the GPU.
*/
extern "C" void
magmablas_dlaset_identity(magma_int_t m, magma_int_t n,
                          double *A, magma_int_t lda)
{
   dim3 threads(dlaset_threads, 1, 1);
   dim3 grid(m/dlaset_threads+(m % dlaset_threads != 0), n/32+(n%32!=0));

   if (m!=0 && n !=0)
      dlaset_identity<<< grid, threads, 0, magma_stream >>> (m, n, A, lda);
}

/* ////////////////////////////////////////////////////////////////////////////
   -- Set the m x n matrix pointed by A to I on the diag without touching the offdiag GPU.
*/
extern "C" void
magmablas_dlaset_identityonly(magma_int_t m, magma_int_t n,
                          double *A, magma_int_t lda)
{
   dim3 threads(dlaset_threads, 1, 1);
   dim3 grid(m/dlaset_threads+(m % dlaset_threads != 0), n/32+(n%32!=0));

   if (m!=0 && n !=0)
      dlaset_identityonly<<< grid, threads, 0, magma_stream >>> (m, n, A, lda);
}

/* ////////////////////////////////////////////////////////////////////////////
   -- Given two matrices, 'a' on the CPU and 'da' on the GPU, this function
      returns the Frobenious norm of the difference of the two matrices.
      The function is used for debugging.
*/
extern "C"
double cpu_gpu_ddiff(
    magma_int_t M, magma_int_t N,
    const double *a,  magma_int_t lda,
    const double *da, magma_int_t ldda )
{
  magma_int_t d_one = 1;
  magma_int_t j;
  double c_neg_one = MAGMA_D_NEG_ONE;
  double  work[1];
  double *ha = (double*)malloc( M * N * sizeof(double));
  double res;

  hipblasGetMatrix(M, N, sizeof(double), da, ldda, ha, M);
  for(j=0; j<N; j++)
    blasf77_daxpy(&M, &c_neg_one, a+j*lda, &d_one, ha+j*M, &d_one);
  res = lapackf77_dlange("f", &M, &N, ha, &M, work);

  free(ha);
  return res;
}

/* ////////////////////////////////////////////////////////////////////////////
 -- GPU kernel for setting 0 in the nb-1 upper subdiagonals and 1 in the diagonal
    @author Raffaele Solca
 */
__global__ void dsetdiag1subdiag0_L(int k, double *A, int lda)
{

  int nb = blockDim.x;
  int ibx = blockIdx.x * nb;

  int ind = ibx + threadIdx.x + 1;

  A += ind - nb + __mul24((ibx), lda);

  double tmp = MAGMA_D_ZERO;
  if(threadIdx.x == nb-1)
    tmp = MAGMA_D_ONE;

#pragma unroll
  for(int i=0; i<nb; i++)
    if (ibx+i < k && ind + i  >= nb) {
      A[i*(lda+1)] = tmp;
    }

}

/* ////////////////////////////////////////////////////////////////////////////
 -- GPU kernel for setting 0 in the nb-1 lower subdiagonals and 1 in the diagonal
    @author Raffaele Solca
 */

__global__ void dsetdiag1subdiag0_U(int k, double *A, int lda)
{

  int nb = blockDim.x;
  int ibx = blockIdx.x * nb;

  int ind = ibx + threadIdx.x;

  A += ind + __mul24((ibx), lda);

  double tmp = MAGMA_D_ZERO;
  if(threadIdx.x == 0)
    tmp = MAGMA_D_ONE;

#pragma unroll
  for(int i=0; i<nb; i++)
    if (ibx+i < k && ind + i < k) {
      A[i*(lda+1)] = tmp;
    }

}

/* ////////////////////////////////////////////////////////////////////////////
 -- Set 1s in the diagonal and 0s in the nb-1 lower (UPLO='U') or
    upper (UPLO='L') subdiagonals.
    stream and no stream interfaces
    @author Raffaele Solca
 */
extern "C" void
magmablas_dsetdiag1subdiag0_stream(char uplo, magma_int_t k, magma_int_t nb,
                 double *A, magma_int_t lda, magma_queue_t stream)
{
  dim3 threads(nb, 1, 1);
  dim3 grid((k-1)/nb+1);
  if(k>lda)
    fprintf(stderr,"wrong second argument of dsetdiag1subdiag0");
  if(uplo == MagmaLower)
    dsetdiag1subdiag0_L<<< grid, threads, 0, stream >>> (k, A, lda);
  else if(uplo == MagmaUpper) {
    dsetdiag1subdiag0_U<<< grid, threads, 0, stream >>> (k, A, lda);
  }
  else
    fprintf(stderr,"wrong first argument of dsetdiag1subdiag0");

  return;
}

extern "C" void
magmablas_dsetdiag1subdiag0(char uplo, magma_int_t k, magma_int_t nb,
                 double *A, magma_int_t lda)
{
  magmablas_dsetdiag1subdiag0_stream(uplo, k, nb, A, lda, magma_stream);
}

