#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.2.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       June 2012

       @generated d Thu Jun 28 12:31:17 2012

*/
#include "common_magma.h"

#define BLOCK_SIZE 32

//#define num_threads 64
#define dgemv_bs 32

#define dlansy_bs 64

#define PRECISION_d
#if (!defined(PRECISION_z)) || (GPUSHMEM >= 200)

__global__ void
l_dlansy_special (int n, double* A, int lda,  double *y){
  int tx = threadIdx.x ; 
  int ty = threadIdx.y ; 
  int ind = blockIdx.x*  dgemv_bs + tx ;
  double res = 0.;

  __shared__ double la[dgemv_bs][dgemv_bs+1];
          
  A += ind;
  A+= ty * lda  ;  
  int break_d  =   blockIdx.x* dgemv_bs ;

  for(int  i=0; i<break_d; i += dgemv_bs ){
    #pragma unroll 8 
    for(int j=0; j < dgemv_bs ; j+=4){
        la[tx][ty+j] = A[j*lda] ;
    }
    __syncthreads();

    #pragma unroll 8 
    for(int j=0; j < 8 ; j++){
       res+=fabs( la[tx][j+ty*8]) ;
    }
    A+=lda* dgemv_bs ;
    __syncthreads(); 
  }

 
  #pragma unroll 8
  for(int j =0; j<dgemv_bs; j+=4)
         la[ty+j][tx] = A[ j * lda];


  A+= dgemv_bs ;
  __syncthreads();
  #pragma unroll 8
  for(int  i=ty*8; i<(1+ty)* dgemv_bs/4 ; i++){
         if ( i < tx )   {
                la[tx][i] = la[i][tx] ; 
         }
         else 
                la[tx][i] = la[tx][i]  ;
  
  }
  __syncthreads();
    #pragma unroll 8
    for(int j=0; j < dgemv_bs/4 ; j++){
     res+=fabs(la[tx][j+ty*8]);
    }
  break_d  += dgemv_bs ; 
  __syncthreads();

  for(int i=break_d; i<n; i += dgemv_bs ){
   #pragma unroll 8
    for(int j=0; j<dgemv_bs; j+=4)
       la[ty+j][tx] = A[ j * lda];
    A+= dgemv_bs ;
      __syncthreads();
    #pragma unroll 8
    for(int j=0; j < dgemv_bs/4;j++){
       res+= fabs(la[tx][j+ty*8]);
    }
      __syncthreads();
  }


  la[tx][ty] = MAGMA_D_MAKE( res, 0. );
   __syncthreads();
   if( ty == 0 ) {
     res = res 
       + MAGMA_D_REAL( la[tx][1] ) 
       + MAGMA_D_REAL( la[tx][2] )
       + MAGMA_D_REAL( la[tx][3] );
     y[ind] = res;
   }

}

__global__ void
l_dlansy_generic(int n, double* A, int lda,  double *y , int m_full_block , 
                 int m_mod_32)
{ 
  int tx = threadIdx.x ; 
  int ty = threadIdx.y ; 

  int ind = blockIdx.x*  dgemv_bs + tx ;
  
  double res = 0.;

  __shared__ double la   [dgemv_bs][dgemv_bs+1];

  if( blockIdx.x == m_full_block ) {
  /************************************************************************
   -- Last block --
   -- We will do something unusual here 
   -- For sufficiently large matrix the overhead will be very low
  *************************************************************************/
       if  ( tx < m_mod_32 ){
                A+= ( blockIdx.x * dgemv_bs + tx ) ;
       }                  
       else{
                A+= ( blockIdx.x * dgemv_bs + m_mod_32 -1) ; 
       }
       A+= ty * lda  ;  
       int break_d  =   blockIdx.x* dgemv_bs ;

          /*----------------------------
                Go Right
          -------------------------------*/

          for(int  i=0; i<break_d; i += dgemv_bs ){
            #pragma unroll 8 
            for(int j=0; j < dgemv_bs ; j+=4){
                la[tx][ty+j] = A[j*lda] ;
            }
            __syncthreads();

            #pragma unroll 8 
            for(int j=0; j < 8 ; j++){
               res+=fabs( la[tx][j+ty*8]);
            }
            A+=lda* dgemv_bs ;
            __syncthreads(); 
          }
          /*
           we don't need to make zero, as those computation will be discarded. 
          */
          if( ty==0  ) {
                /*--------------------------------------------
                        he will compute the triangular parts
                        others will be waiting with values. 
                -----------------------------------------------*/
                int j ;
                int count = 1 ; 
                if( tx < m_mod_32 ) 
                        count = tx ; 
                else
                        count = m_mod_32 ;
                for(j =0;j<=count;j++){
                        res+= fabs( A[j*lda]) ;
                }
                A+=(tx)*lda;
                count = 1 ; 
                for(;j<m_mod_32;j++){
                        res+=fabs( A[count]) ;
                        count++;
                }
          }
          else{
          }
          __syncthreads(); 
          la[tx][ty]= MAGMA_D_MAKE( res, 0. ) ;
          __syncthreads();
         /*--------------------------------------------------------
         The leader accumulates all the results from his peer. 
         ----------------------------------------------------------*/
         if( ty == 0 ) {
           res = res 
             + MAGMA_D_REAL( la[tx][1] ) 
             + MAGMA_D_REAL( la[tx][2] )
             + MAGMA_D_REAL( la[tx][3] );
           if( tx < m_mod_32)
             y[ind] = res;
         }
         
  }

  else{ 
  /***************************************
    -----------------------------------
  -- All the blocks but the last one --
  ****************************************
  -------------------------------------*/
  A += ind;
  A+= ty * lda  ;  
  int break_d  =   blockIdx.x* dgemv_bs ;

  /*----------------------------
        Go Right
  -------------------------------*/
  for(int  i=0; i<break_d; i += dgemv_bs ){
    #pragma unroll 8 
    for(int j=0; j < dgemv_bs ; j+=4){
        la[tx][ty+j] = A[j*lda] ;
    }
    __syncthreads();

    #pragma unroll 8 
    for(int j=0; j < 8 ; j++){
       res+=fabs(la[tx][j+ty*8]);
    }
    A+=lda* dgemv_bs ;
    __syncthreads(); 
  }

 
  /*------------------------------------
        Diagonal 
        Copy + Transpose lower triangle
  --------------------------------------*/
  #pragma unroll 8
  for(int j =0; j<dgemv_bs; j+=4)
         la[ty+j][tx] = A[ j * lda];


  A+= dgemv_bs ;
  __syncthreads();
  /*--------------------------------------------
        Mirror Upper Triangle to Lower triangle
  ---------------------------------------------*/
  #pragma unroll 8
  for(int  i=ty*8; i<(1+ty)* dgemv_bs/4 ; i++){
         if ( i < tx )   {
                la[tx][i] = la[i][tx] ; 
         }
         else 
                la[tx][i] = la[tx][i]  ;
  
  }
  __syncthreads();
  /*--------------------------------
        Do diagonal Computation
  -----------------------------------*/
    #pragma unroll 8
    for(int j=0; j < dgemv_bs/4 ; j++){
     res+= fabs(la[tx][j+ty*8]);
    }
  break_d  += dgemv_bs ; 
  __syncthreads();


  n -= m_mod_32 ;  // @ 
  /*-----------------------------
        Go Down 
  -------------------------------*/
  for(int i=break_d; i<n; i += dgemv_bs ){
   #pragma unroll 8
    for(int j=0; j<dgemv_bs; j+=4)
       la[ty+j][tx] = A[ j * lda];
    A+= dgemv_bs ;
      __syncthreads();
    #pragma unroll 8
    for(int j=0; j < dgemv_bs/4;j++){
       res+=fabs(la[tx][j+ty*8]);
    }
      __syncthreads();
  }

  
  /*---------------------------------------------
        doing m_mod_32 stuffs here.
        Symmetric is giving us benefit .. true
  -----------------------------------------------*/
    A-=tx;
    if( tx < m_mod_32){
        A+=tx;
    }
    else{
        A+=(m_mod_32-1); /* Same as above*/
    }

   #pragma unroll 8
    for(int j=0; j<dgemv_bs; j+=4){
       if( tx < m_mod_32 ) 
         la[ty+j][tx] = MAGMA_D_MUL( MAGMA_D_ONE,  A[ j * lda] );
       else
         la[ty+j][tx] = MAGMA_D_MUL( MAGMA_D_ZERO, A[ j * lda] );
       
    }
    __syncthreads();

    /*----------------------------------------
        What about doing some Zeroing here?
        instead of zeroing before?
    -----------------------------------------*/        
    #pragma unroll 8
    for(int j=0; j < dgemv_bs/4;j++){
       res+=fabs(la[tx][j+ty*8]);
    }
    __syncthreads();
   

    la[tx][ty]= MAGMA_D_MAKE( res, 0. );
   __syncthreads();
   /*--------------------------------------------------------
        The leader accumulates all the results from his peer. 
   ----------------------------------------------------------*/
   if( ty == 0 ) {
     res = res 
       + MAGMA_D_REAL( la[tx][1] ) 
       + MAGMA_D_REAL( la[tx][2] )
       + MAGMA_D_REAL( la[tx][3] );
     y[ind] = res;
   }

  }

}

__global__ void
u_dlansy_generic (int n, double* A, int lda, double *y , int m_full_block , int m_mod_32){

  
  int tx = threadIdx.x ; 
  int ty = threadIdx.y ; 

  int ind = blockIdx.x*  dgemv_bs + tx ;
  
  double res = 0.;


  __shared__ double la   [dgemv_bs][dgemv_bs+1];
  int blockIdxx =  blockIdx.x ;

  if( blockIdx.x == m_full_block ) {

  /************************************************************************
   -- Last block --
   -- We will do something unusual here 
   -- For sufficiently large matrix the overhead will be very low
  *************************************************************************/

  ind =  tx ;
  A+= lda*(n-1) ; 


       if  ( tx < m_mod_32 ){
                A+= (  tx ) ;
       }                  
       else{
                A+= (  m_mod_32 -1) ; 
       }
       A-= ty * lda  ;  
       int break_d  =   (blockIdx.x)* dgemv_bs ;

          /*----------------------------
                Go Right
          -------------------------------*/

          for(int  i=0; i<break_d; i += dgemv_bs ){
            #pragma unroll 8 
            for(int j=0; j < dgemv_bs ; j+=4){
                la[tx][ty+j] = A[-j*lda] ;
            }
            __syncthreads();

            #pragma unroll 8 
            for(int j=0; j < 8 ; j++){
               res+=fabs(la[tx][j+ty*8]);
            }
            A-=lda* dgemv_bs ;
            __syncthreads(); 
          }
          /*
           we don't need to make zero, as those computation will be discarded. 
          */
          if( ty==0  ) {
                /*--------------------------------------------
                        he will compute the triangular parts
                        others will be waiting with values. 
                -----------------------------------------------*/
                int j ;
                int count = 1 ; 
                if( tx < m_mod_32 ) 
                        count =m_mod_32- tx ; 
                else
                        count = m_mod_32 ;
                for(j =0;j<count;j++){
                        res+= fabs( A[-j*lda] );
                }
                A-=(count-1)*lda;
                count = 1 ; 
                for(;j<m_mod_32;j++){
                        res+= fabs( A[-count] );
                        count++;
                }
          }
          else{
          }
          __syncthreads(); 
          la[tx][ty]= MAGMA_D_MAKE( res, 0. );
          __syncthreads();
         /*--------------------------------------------------------
         The leader accumulates all the results from his peer. 
         ----------------------------------------------------------*/
         if( ty == 0 ) {
           res = res 
             + MAGMA_D_REAL( la[tx][1] ) 
             + MAGMA_D_REAL( la[tx][2] )
             + MAGMA_D_REAL( la[tx][3] );
           if( tx < m_mod_32)
             y[ind] = res;
         }
         
  }

  else{ 
  /***************************************
    -----------------------------------
  -- All the blocks but the last one --
  -- By the way this code can be optimized more. 
  ****************************************
  -------------------------------------*/
  ind = blockIdx.x *  dgemv_bs + tx + m_mod_32 ;
  double *A1 = A ; 
  A+= lda*(n-1)  ; 

  A += ind;
  A-= ty * lda  ;  

  int break_d  = (n / dgemv_bs -   blockIdxx-1 )* dgemv_bs ;
  /*----------------------------
        Go Left
  -------------------------------*/
  for(int  i=0; i<break_d; i += dgemv_bs ){
    #pragma unroll 8 
    for(int j=0; j < dgemv_bs ; j+=4){
        la[tx][ty+j] = A[-j*lda] ;
    }
    __syncthreads();

    #pragma unroll 8 
    for(int j=0; j < 8 ; j++){
       res+=fabs( la[tx][j+ty*8]);
    }
    A-=lda* dgemv_bs ;
    __syncthreads(); 
  }

 
  /*------------------------------------
        Diagonal 
        Copy + Transpose lower triangle
  --------------------------------------*/
  #pragma unroll 8
  for(int j =0; j<dgemv_bs; j+=4){
         la[tx][31-ty-j] = A[ -j * lda];
  }

  A-= dgemv_bs ;
  __syncthreads();
  /*--------------------------------------------
        Mirror Upper Triangle to Lower triangle
  ---------------------------------------------*/
  #pragma unroll 8
  for(int  i=ty*8; i<(1+ty)* dgemv_bs/4 ; i++){
         if ( i <tx ){
                la[tx][i] = la[i][tx]; 
         }
         else{ 
                la[tx][i] = la[tx][i]  ;
         }
  }
  __syncthreads();
  /*--------------------------------
        Do diagonal Computation
  -----------------------------------*/
    #pragma unroll 8
    for(int j=0; j < dgemv_bs/4 ; j++){
     res+=fabs(  la[tx][j+ty*8] ) ;
    }
  break_d  += dgemv_bs ; 
  __syncthreads();


  n -= m_mod_32 ;  // @ 
  /*-----------------------------
        Go Up 
  -------------------------------*/
  int i ;
  for( i=break_d; i<n; i+= dgemv_bs ){
   #pragma unroll 8
    for(int j=0; j<dgemv_bs; j+=4){
       la[ty+j][tx] = A[- j * lda];
    }
    A-= dgemv_bs ;
      __syncthreads();
    #pragma unroll 8
    for(int j=0; j < dgemv_bs/4;j++){
       res+=fabs ( la[31-tx][j+ty*8] );
    }
      __syncthreads();
  }
  /*---------------------------------------------
        doing m_mod_32 stuffs here.
        Symmetric is giving us benefit .. true
        Do the other way please......
  -----------------------------------------------*/
   A1 = A1 + m_mod_32 * lda + tx *lda ;  
   if( ty == 0  ) {
        for( int j = 0 ;  j < m_mod_32 ; j++){
                res+=  fabs (  A1[ j + lda * (blockIdx.x) * 32 ] ) ;
        }
   }
    __syncthreads();

    la[tx][ty]= MAGMA_D_MAKE( res, 0);
   __syncthreads();
   /*--------------------------------------------------------
        The leader accumulates all the results from his peer. 
   ----------------------------------------------------------*/
   if( ty == 0 ) {
     res = res 
       + MAGMA_D_REAL( la[tx][1] ) 
       + MAGMA_D_REAL( la[tx][2] )
       + MAGMA_D_REAL( la[tx][3] );
     y[ind] =  res;
   }
  }
}

__global__ void
u_dlansy_special (int n, double* A, int lda, double *y ){
  int tx = threadIdx.x ; 
  int ty = threadIdx.y ; 
  int ind = blockIdx.x*  dgemv_bs + tx ;
  double res = 0.;

  /*
        Reverse Computation ... 
                - Left 
                - Triangle 
                - Up 
  */

  A+= lda*(n-1) ; 
  __shared__ double la   [dgemv_bs][dgemv_bs+1];

  A += ind;
  A-= ty * lda  ;  
  int break_d  = (n / dgemv_bs -   blockIdx.x-1 )* dgemv_bs ;

  for(int  i=0; i<break_d; i += dgemv_bs ){
    #pragma unroll 8 
    for(int j=0; j < dgemv_bs ; j+=4){
        la[tx][ty+j] = A[-j*lda] ;
    }
    __syncthreads();

    #pragma unroll 8 
    for(int j=0; j < 8 ; j++){
       res+=fabs(la[tx][j+ty*8]);
    }
    A-=lda* dgemv_bs ;
    __syncthreads(); 
  }

  #pragma unroll 8
  for(int j =0; j<dgemv_bs; j+=4)
         la[tx][31-ty-j] = A[ -j * lda];
  /*
        Look at the indexing changes
  */

  A-= dgemv_bs ;
  __syncthreads();
  #pragma unroll 8
  for(int  i=ty*8; i<(1+ty)* dgemv_bs/4 ; i++){
         if ( i <tx ){
                la[tx][i] = la[i][tx]; 
         }
         else{ 
                la[tx][i] = la[tx][i]  ;
         }
  
  }
  __syncthreads();
    #pragma unroll 8
    for(int j=0; j < dgemv_bs/4 ; j++){
     res+= fabs(la[tx][j+ty*8]);
    }

  break_d  += dgemv_bs ; 
  __syncthreads();

  for(int i=break_d; i<n; i+= dgemv_bs ){
   #pragma unroll 8
    for(int j=0; j<dgemv_bs; j+=4)
       la[ty+j][tx] = A[ -j * lda];

    A-= dgemv_bs ;
      __syncthreads();
    #pragma unroll 8
    for(int j=0; j < dgemv_bs/4;j++){
       res+=fabs( la[31-tx][j+ty*8]);
    }
      __syncthreads();
  }

  la[tx][ty]= MAGMA_D_MAKE( res, 0. );

   __syncthreads();
   if( ty == 0 ) {
     res = res 
       + MAGMA_D_REAL( la[tx][1] ) 
       + MAGMA_D_REAL( la[tx][2] )
       + MAGMA_D_REAL( la[tx][3] );
     y[ind] =   res;
   }
}


extern "C" void mdlansy (char uplo , int m ,  double *A , int lda ,  double *Y  )
{
/*
Note:
        The UPLO = 'U' Version can be optimized more.
        side is not needed........................... 
*/
    int blocks;
    if (m % dgemv_bs==0)
        blocks = m/ dgemv_bs;
    else
        blocks = m/ dgemv_bs + 1;

    dim3 grid(blocks, 1, 1);
    dim3 threads(32, 4, 1);

    if( m % dgemv_bs == 0 ) {
            if( uplo == 'L' || uplo == 'l'){        
                    l_dlansy_special <<< grid, threads, 0, magma_stream >>> (m, A, lda, Y);
            }
            else{
                    u_dlansy_special <<< grid, threads, 0, magma_stream >>> (m, A, lda,  Y);
            } 
                
    } 
    else{        
            int  m_full_block = (m - m % 32 ) /32 ; 
            int  m_mod_32 = m%32 ;  
            if( uplo == 'L' || uplo == 'l'){
                    l_dlansy_generic <<< grid, threads, 0, magma_stream >>> (m, A, lda, Y , m_full_block , m_mod_32);
            }        
            else{
                    u_dlansy_generic <<< grid, threads, 0, magma_stream >>> (m, A, lda, Y , m_full_block , m_mod_32);
            }        
    }
}

#endif /* (!defined(PRECISION_z)) || (GPUSHMEM >= 200) */

__global__ void
l_dlansy_max (int m, double* A, int lda,  double *y){
    int tx  = threadIdx.x ;
    int ind =  blockIdx.x * dlansy_bs + tx ;
    double res = 0., res1;

    int break_d = blockIdx.x* dlansy_bs;

    if (ind < m)
    {
       A += ind;

       for(int i=0; i<break_d; i += dlansy_bs ){
          #pragma unroll 8
          for(int j=0; j< dlansy_bs; j++){
             res1 = fabs(A[j*lda]);
             res = fmax(res,res1);
          }
    
          A += lda*dlansy_bs;
       }   
  
     
       for(int j=0; j<=tx; j++){
          res1 = fabs(A[j*lda]);
          res = fmax(res,res1);
       }

       y[ind] = res;
    }
}

__global__ void
u_dlansy_max (int m, double* A, int lda,  double *y){
    int ind =  blockIdx.x * dlansy_bs + threadIdx.x ;
    double res = 0.;

    A += ind;
    if (ind < m){
      for(int j=m-1; j>= ind; j--)
         res = fmax(res, fabs(A[j*lda]));
      
      y[ind] = res;
    }
}


extern "C" void dlansy_max (char uplo, int m, double *A , int lda , double *y){
    int blocks;
    if (m % dlansy_bs==0)
        blocks = m/ dlansy_bs;
    else
        blocks = m/ dlansy_bs + 1;

    dim3 grid(blocks, 1, 1);
    dim3 threads(dlansy_bs, 1, 1);

    if( uplo == 'L' || uplo == 'l'){
      l_dlansy_max <<< grid, threads, 0, magma_stream >>> (m, A, lda, y);
    }
    else{
      u_dlansy_max <<< grid, threads, 0, magma_stream >>> (m, A, lda, y);
    }
}
 
extern "C" double 
magmablas_dlansy(char norm, char uplo, magma_int_t n, 
                 double *A, magma_int_t lda, double *WORK )
{
        if (norm == 'I' || norm =='i')  
            {
#if (GPUSHMEM >= 200)
                mdlansy ( uplo , n , A , lda , WORK);
                int val = hipblasIdamax(n,WORK,1);
                double retVal[1];
                hipblasGetMatrix( 1, 1, sizeof( double ), WORK+val-1, 1, retVal, 1 ) ;
                return retVal[0];
#else
                printf("Only normM is available. Exit.\n");
                exit(1);
#endif
            }
        else if (norm == 'M' || norm =='m')
            {  
                dlansy_max ( uplo , n , A , lda , WORK);
                int val = hipblasIdamax(n,WORK,1);
                double retVal[1];
                hipblasGetMatrix( 1, 1, sizeof( double ), WORK+val-1, 1, retVal, 1 ) ;
                return retVal[0];
            }
        else
            {
                printf("Only normI and normM are available. Exit.\n");
                exit(1);
            }
}


