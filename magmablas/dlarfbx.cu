#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.5.0-beta3) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date July 2014

       @generated from zlarfbx.cu normal z -> d, Fri Jul 18 17:34:12 2014

*/
#include "common_magma.h"
#include "magma_templates.h"

// 512 is maximum number of threads for CUDA capability 1.x
#define BLOCK_SIZE 512


__global__ void magma_dtrmv_tkernel(double *T, int ldt, double *v, 
                                    double *y);


//==============================================================================

__global__ void 
magma_dgemv_kernel1(int m, const double * __restrict__ V, int ldv, 
                    const double * __restrict__ c, 
                    double *dwork)
{
    const int i = threadIdx.x;
    const double *dV = V + (blockIdx.x) * ldv;

    __shared__ double sum[ BLOCK_SIZE ];
    double lsum;

    /*  lsum := v' * C  */
    lsum = MAGMA_D_ZERO;
    for( int j = i; j < m; j += BLOCK_SIZE )
       lsum += MAGMA_D_MUL( MAGMA_D_CNJG( dV[j] ), c[j] );
    
    sum[i] = lsum;
    magma_sum_reduce< BLOCK_SIZE >( i, sum );

    __syncthreads();
    if (i==0)
       dwork [blockIdx.x] = sum[0];
}

//==============================================================================
/*  ----------------------------------------------------------------------------- 
    Call 
        magma_dgemv_kernel3<<< n, BLOCK_SIZE>>>(m, V, ldv, c, dwork, tau)
    to compute
        DGEMV( "Conjugate transpose", m, n, -tau[0], V, ldv, c, 1, zero, dwork, 1)
        and to set c[0] to 1.
    i.e., 
        work = -tau[0] V' c
    ----------------------------------------------------------------------------- */
__global__ void
magma_dgemv_kernel3(int m, const double * __restrict__ V, int ldv, double *c,
                    double *dwork, double *tau)
{
    const int i = threadIdx.x;
    const double *dV = V + (blockIdx.x) * ldv;

    __shared__ double sum[ BLOCK_SIZE ];
    double lsum;

    if (i==0)
       c[0] = MAGMA_D_ONE;           

    /*  lsum := v' * C  */
    lsum = MAGMA_D_ZERO;
    for( int j = i; j < m; j += BLOCK_SIZE )
       lsum += MAGMA_D_MUL( MAGMA_D_CNJG( dV[j] ), c[j] );

    sum[i] = lsum;
    magma_sum_reduce< BLOCK_SIZE >( i, sum );

    __syncthreads();
    if (i==0)
       dwork [blockIdx.x] = -tau[0]*sum[0];
}

//==============================================================================

__global__ void
magma_dgemv_kernel2(int m, int n, const double * __restrict__ V, int ldv, 
                    const double * __restrict__ x, double *c)
{
    const int i = threadIdx.x;
    const int j = i + BLOCK_SIZE * blockIdx.x;
    double lsum;

    V += j;

    lsum = MAGMA_D_ZERO;
    if (j < m){
       for(int k=0; k<n; k++)
          lsum += MAGMA_D_MUL( V[k*ldv], x[k]);
       
       c[j] -= lsum;
    }
}

//==============================================================================

/*
    Apply a real block reflector H to a real vector C from the left
    (i.e., C = H C). H is represented in the form
          H = I - V T V'
    where T is the real k-by-k upper triangular matrix in the 
    representation of the block reflector, and V is a real block of
    k elementary reflectors. 
*/
extern "C" void
magma_dlarfbx_gpu(magma_int_t m, magma_int_t k, double *V, magma_int_t ldv,
                  double *T, magma_int_t ldt, double *c,
                  double *dwork)
{
    /* dwork = V' c                   */
    magma_dgemv_kernel1<<< k, BLOCK_SIZE, 0, magma_stream >>>(m, V, ldv, c, dwork); 

    /* dwork = T' dwork               */
    magma_dtrmv_tkernel<<< k, k, 0, magma_stream >>>( T, ldt, dwork, dwork+k);
 
    /* c = c - V dwork                */
    dim3  blocks3( (m + BLOCK_SIZE-1) / BLOCK_SIZE );
    dim3 threads3( BLOCK_SIZE );     
    magma_dgemv_kernel2<<< blocks3, threads3, 0, magma_stream >>>( m, k, V, ldv, dwork+k, c);
}

//==============================================================================
