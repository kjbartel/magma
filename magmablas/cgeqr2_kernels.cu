#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

       @author Azzam Haidar
       @author Tingxing Dong

       @generated from zgeqr2_kernels.cu normal z -> c, Fri Jan 30 19:00:10 2015
*/



#include "common_magma.h"
#include "batched_kernel_param.h"



static    magmaFloatComplex neg_one = MAGMA_C_NEG_ONE;
static    magmaFloatComplex one  = MAGMA_C_ONE;
static    magmaFloatComplex zero  = MAGMA_C_ZERO;

__global__ void
cgeqrf_copy_upper_kernel_batched(                
                  int n, int nb,
                  magmaFloatComplex **dV_array,    int ldv,
                  magmaFloatComplex **dR_array,    int ldr)
{

    magmaFloatComplex *dV = dV_array[blockIdx.x];
    magmaFloatComplex *dR = dR_array[blockIdx.x];

    int tid = threadIdx.x;

    int column = (tid / nb + 1) * nb; 
    
    if( tid < n && column < n) 
    {
       for(int i=column; i<n; i++)
       {
          dR[tid + i * ldr]  =  dV[tid + i * ldv];  
       }
    }
}

void cgeqrf_copy_upper_batched(                
                  magma_int_t n, magma_int_t nb,
                  magmaFloatComplex **dV_array,    magma_int_t ldv,
                  magmaFloatComplex **dR_array,    magma_int_t ldr,
          magma_int_t batchCount, magma_queue_t queue)
{
   /* 
        copy some data in dV to dR
   */

      if( nb >= n) return ;

      cgeqrf_copy_upper_kernel_batched<<<batchCount, n, 0, queue>>>(n, nb, dV_array, ldv, dR_array, ldr);

}



extern "C" magma_int_t
magma_clarfb_cgemm_batched(
                  hipblasHandle_t myhandle,
                  magma_int_t m, magma_int_t n, magma_int_t k,
                  magmaFloatComplex **dV_array,    magma_int_t ldv,
                  magmaFloatComplex **dT_array,    magma_int_t ldt,
                  magmaFloatComplex **dA_array,    magma_int_t lda,
                  magmaFloatComplex **W_array,     magma_int_t ldw,
                  magmaFloatComplex **W2_array,    magma_int_t ldw2,
                  magma_int_t batchCount, magma_queue_t queue)

{

    // W is workspace size of W is nb * n 
    // W = V^H * A. V is stored in A(i:m, i:ib)

    
    if( m <=0 || n <= 0 || k <=0 ) return 1;

#if 1  // CUBLAS is faster than MAGMABLAS by 17GFLOP/S at size 512 batchCount = 2000
    hipblasCgemmBatched(myhandle, HIPBLAS_OP_C, HIPBLAS_OP_N, k, n, m,
                             &one, (const magmaFloatComplex**) dV_array, ldv,
                                    (const magmaFloatComplex**) dA_array, lda,
                             &zero,  W_array, ldw, batchCount );



    // W2 = T^H * W        
    hipblasCgemmBatched(myhandle, HIPBLAS_OP_C, HIPBLAS_OP_N, k, n, k,
                             &one, (const magmaFloatComplex**) dT_array, ldt,
                                    (const magmaFloatComplex**) W_array, ldw,
                             &zero,  W2_array, ldw2, batchCount );

        
    // A = A - V * W2 
    hipblasCgemmBatched(myhandle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k,
                             &neg_one, (const magmaFloatComplex**) dV_array, ldv,
                                    (const magmaFloatComplex**) W2_array, ldw2,
                             &one,  dA_array, lda, batchCount );

#else 

    magmablas_cgemm_batched(MagmaConjTrans, MagmaNoTrans, k, n, m,
                             one, (const magmaFloatComplex**) dV_array, ldv,
                                    (const magmaFloatComplex**) dA_array, lda,
                             zero,  W_array, ldw, batchCount );



    // W2 = T^H * W        
    magmablas_cgemm_batched(MagmaConjTrans, MagmaNoTrans, k, n, k,
                             one, (const magmaFloatComplex**) dT_array, ldt,
                                    (const magmaFloatComplex**) W_array, ldw,
                             zero,  W2_array, ldw2, batchCount );

        
    // A = A - V * W2 
    magmablas_cgemm_batched(MagmaNoTrans, MagmaNoTrans, m, n, k,
                             neg_one, (const magmaFloatComplex**) dV_array, ldv,
                                    (const magmaFloatComplex**) W2_array, ldw2,
                             one,  dA_array, lda, batchCount );
          
#endif       
    return 0;

}



