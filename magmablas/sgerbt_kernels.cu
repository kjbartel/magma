#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2014

       @generated from zgerbt_kernels.cu normal z -> s, Sat Nov 15 19:53:59 2014


       @author Adrien REMY
*/
#include "common_magma.h"


#define block_height  32
#define block_width  4
#define block_length 256
#define NB 64
/////////////////////////////////////////////////////////////////////////////////////////////////////////////
static __device__ void 
magmablas_selementary_multiplication_devfunc(
    magma_int_t n,
    float *dA, magma_int_t ldda, 
    float *du, 
    float *dv)
{    
    magma_int_t idx, idy;

    idx = blockIdx.x * blockDim.x + threadIdx.x;
    idy = blockIdx.y * blockDim.y + threadIdx.y;

    if ((idx < n/2)&&(idy < n/2)){

        dA += idx + idy * ldda;

        float a00, a10, a01, a11, b1, b2, b3, b4;
        __shared__ float u1[block_height], u2[block_height], v1[block_width], v2[block_width];

        du += idx;
        dv += idy;

        u1[threadIdx.x]=du[0];
        u2[threadIdx.x]=du[n/2];
        v1[threadIdx.y]=dv[0];
        v2[threadIdx.y]=dv[n/2];

        __syncthreads();

        a00 = dA[0];
        a01 = dA[ldda*n/2];
        a10 = dA[n/2];
        a11 = dA[ldda*n/2+n/2];

        b1 = a00 + a01;
        b2 = a10 + a11;
        b3 = a00 - a01;
        b4 = a10 - a11;

        dA[0] = u1[threadIdx.x] * v1[threadIdx.y] * (b1 + b2);
        dA[ldda*n/2] = u1[threadIdx.x] * v2[threadIdx.y] * (b3 + b4);
        dA[n/2] = u2[threadIdx.x] * v1[threadIdx.y] * (b1 - b2);
        dA[ldda*n/2+n/2] = u2[threadIdx.x] * v2[threadIdx.y] *(b3 - b4);
    }
}
/////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void 
magmablas_selementary_multiplication_kernel(
    magma_int_t n,
    float *dA, magma_int_t offsetA, magma_int_t ldda, 
    float *du, magma_int_t offsetu, 
    float *dv, magma_int_t offsetv)
{    
    magmablas_selementary_multiplication_devfunc( n, dA+offsetA, ldda, du+offsetu, dv+offsetv);
}
/////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void 
magmablas_selementary_multiplication_kernel_batched(
    magma_int_t n,
    float **dA_array, magma_int_t offsetA, magma_int_t ldda, 
    float *du, magma_int_t offsetu, 
    float *dv, magma_int_t offsetv)
{    
    int batchid = blockIdx.z;
    magmablas_selementary_multiplication_devfunc( n, dA_array[batchid]+offsetA, ldda, du+offsetu, dv+offsetv);
}
/////////////////////////////////////////////////////////////////////////////////////////////////////////////
static __device__ void 
magmablas_sapply_vector_devfunc(
    magma_int_t n,
    float *du, float *db)
{
    magma_int_t idx;

    idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n/2){

        du += idx;
        db += idx;

        float a1,a2;

        a1 = du[0]*db[0];
        a2 = du[n/2]*db[n/2];

        db[0] = a1 + a2;
        db[n/2] = a1 -a2;
    }
}
/////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void 
magmablas_sapply_vector_kernel(
    magma_int_t n,
    float *du, magma_int_t offsetu,  float *db, magma_int_t offsetb )
{
    magmablas_sapply_vector_devfunc(n, du+offsetu, db+offsetb);
}
/////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void 
magmablas_sapply_vector_kernel_batched(
    magma_int_t n,
    float *du, magma_int_t offsetu, float **db_array, magma_int_t offsetb )
{

    int batchid = blockIdx.y;
    magmablas_sapply_vector_devfunc(n, du+offsetu, db_array[batchid]+offsetb);
}
/////////////////////////////////////////////////////////////////////////////////////////////////////////////

static __device__ void 
magmablas_sapply_transpose_vector_devfunc(
    magma_int_t n,
    float *du,float *db )
{
    magma_int_t idx;

    idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n/2){

        du += idx;
        db += idx;

        float a1,a2;

        a1 = db[0] + db[n/2];
        a2 = db[0] - db[n/2];

        db[0] = du[0]*a1;
        db[n/2] = du[n/2]*a2;
    }
}
/////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void 
magmablas_sapply_transpose_vector_kernel(
    magma_int_t n,
    float *du, magma_int_t offsetu, float *db, magma_int_t offsetb )
{
    magmablas_sapply_transpose_vector_devfunc(n, du+offsetu, db+offsetb);
}
/////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void 
magmablas_sapply_transpose_vector_kernel_batched(
    magma_int_t n,
    float *du, magma_int_t offsetu, float **db_array, magma_int_t offsetb )
{

    int batchid = blockIdx.y;
    magmablas_sapply_transpose_vector_devfunc(n, du+offsetu, db_array[batchid]+offsetb);
}
/////////////////////////////////////////////////////////////////////////////////////////////////////////////


