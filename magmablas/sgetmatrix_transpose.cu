#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @generated from zgetmatrix_transpose.cu normal z -> s, Fri Jan 30 19:00:09 2015

*/
#include "common_magma.h"

#define PRECISION_s


//
//      m, n - dimensions in the output (hA) matrix.
//             This routine copies the dAT matrix from the GPU
//             to hA on the CPU. In addition, the output matrix
//             is transposed. The routine uses a buffer of size
//             2*lddwork*nb pointed to by dwork (lddwork > m) on the GPU. 
//             Note that lda >= m and lddat >= n.
//
extern "C" void 
magmablas_sgetmatrix_transpose_q(
    magma_int_t m, magma_int_t n,
    magmaFloat_const_ptr dAT, magma_int_t ldda,
    float          *hA,  magma_int_t lda,
    magmaFloat_ptr       dwork,  magma_int_t lddwork, magma_int_t nb,
    magma_queue_t queues[2] )
{
#define    hA(i_, j_)    (hA + (i_) + (j_)*lda)
#define   dAT(i_, j_)   (dAT + (i_) + (j_)*ldda)
#define dwork(i_, j_) (dwork + (i_) + (j_)*lddwork)

    magma_int_t i = 0, j = 0, ib;

    /* Quick return */
    if ( (m == 0) || (n == 0) )
        return;

    // TODO standard check arguments
    if (lda < m || ldda < n || lddwork < m){
        printf("Wrong arguments in sgetmatrix_transpose.\n");
        return;
    }

    for(i=0; i < n; i += nb) {
        /* Move data from GPU to CPU using 2 buffers; 1st transpose the data on the GPU */
        ib = min(n-i, nb);
        
        magmablas_stranspose_q( ib, m, dAT(i,0), ldda, dwork(0,(j%2)*nb), lddwork, queues[j%2] );
        magma_sgetmatrix_async( m, ib,
                                dwork(0,(j%2)*nb), lddwork,
                                hA(0,i), lda, queues[j%2] );
        j++;
    }
}


// @see magmablas_sgetmatrix_transpose_q
extern "C" void 
magmablas_sgetmatrix_transpose(
    magma_int_t m, magma_int_t n,
    magmaFloat_const_ptr dAT, magma_int_t ldda,
    float          *hA,  magma_int_t lda,
    magmaFloat_ptr       dwork,  magma_int_t lddwork, magma_int_t nb )
{
    magma_queue_t queues[2];
    magma_queue_create( &queues[0] );
    magma_queue_create( &queues[1] );

    magmablas_sgetmatrix_transpose_q( m, n, dAT, ldda, hA, lda, dwork, lddwork, nb, queues );

    magma_queue_destroy( queues[0] );
    magma_queue_destroy( queues[1] );
}
