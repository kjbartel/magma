#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.4.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       December 2013

       @generated s Tue Dec 17 13:18:45 2013
       @author Mark Gates
*/
#include "common_magma.h"
#include <assert.h>

#define NB 64

/*
    Matrix is m x m, and is divided into block rows, each NB x m.
    Each block has NB threads.
    Each thread copies one row, iterating across all columns below diagonal.
    The bottom block of rows may be partially outside the matrix;
    if so, rows outside the matrix (i >= m) are disabled.
*/
__global__ void
ssymmetrize_lower( int m, float *dA, int ldda )
{
    // dA iterates across row i and dAT iterates down column i.
    int i = blockIdx.x*NB + threadIdx.x;
    float *dAT = dA;
    if ( i < m ) {
        dA  += i;
        dAT += i*ldda;
        float *dAend = dA + i*ldda;
        while( dA < dAend ) {
            *dAT = (*dA);  // upper := lower
            dA  += ldda;
            dAT += 1;
        }
    }
}


// only difference with _lower version is direction dA=dAT instead of dAT=dA.
__global__ void
ssymmetrize_upper( int m, float *dA, int ldda )
{
    // dA iterates across row i and dAT iterates down column i.
    int i = blockIdx.x*NB + threadIdx.x;
    float *dAT = dA;
    if ( i < m ) {
        dA  += i;
        dAT += i*ldda;
        float *dAend = dA + i*ldda;
        while( dA < dAend ) {
            *dA = (*dAT);  // lower := upper
            dA  += ldda;
            dAT += 1;
        }
    }
}


extern "C" void
magmablas_ssymmetrize( char uplo, magma_int_t m, float *dA, magma_int_t ldda )
{
/*
    Purpose
    =======
    
    SSYMMETRIZE copies lower triangle to upper triangle, or vice-versa,
    to make dA a general representation of a symmetric matrix.
    
    Arguments
    =========
    
    UPLO    (input) CHARACTER*1
            Specifies the part of the matrix dA that is valid on input.
            = 'U':      Upper triangular part
            = 'L':      Lower triangular part
    
    M       (input) INTEGER
            The number of rows of the matrix dA.  M >= 0.
    
    dA      (input/output) COMPLEX REAL array, dimension (LDDA,N)
            The m by m matrix dA.
    
    LDDA    (input) INTEGER
            The leading dimension of the array dA.  LDDA >= max(1,M).
    
    =====================================================================   */

    //printf( "m %d, grid %d, threads %d\n", m, grid.x, threads.x );
    if ( m == 0 )
        return;
    
    assert( m >= 0 );
    assert( ldda >= m );
    
    dim3 threads( NB );
    dim3 grid( (m + NB - 1)/NB );
    
    if ( (uplo == 'U') || (uplo == 'u') ) {
        ssymmetrize_upper<<< grid, threads, 0, magma_stream >>>( m, dA, ldda );
    }
    else if ( (uplo == 'L') || (uplo == 'l') ) {
        ssymmetrize_lower<<< grid, threads, 0, magma_stream >>>( m, dA, ldda );
    }
    else {
        printf( "uplo has illegal value\n" );
        exit(1);
    }
}
