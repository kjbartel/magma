#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.2.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       June 2012

       @generated d Thu Jun 28 12:31:18 2012

*/
#include "common_magma.h"

#define BLOCK_SIZE 64

typedef struct {
        double *A;
        int n, lda, j0;
        short ipiv[BLOCK_SIZE];
} dlaswp_params_t;

typedef struct {
        double *A;
        int n, lda, j0, npivots;
        short ipiv[BLOCK_SIZE];
} dlaswp_params_t2;

/*********************************************************
 *
 * LAPACK Swap: permute a set of lines following ipiv
 *
 ********************************************************/
typedef struct {
    double *A;
    int n, ldx, ldy, j0, npivots;
    short ipiv[BLOCK_SIZE];
} dlaswpx_params_t;


extern "C" void dlaswp3( dlaswp_params_t2 &params );

extern "C" void 
magmablas_dpermute_long3( double *dAT, magma_int_t lda,
                          magma_int_t *ipiv, magma_int_t nb, magma_int_t ind )
{
        int k;
        for( k = 0; k < nb-BLOCK_SIZE; k += BLOCK_SIZE )
        {
                dlaswp_params_t2 params = { dAT, lda, lda, ind + k, BLOCK_SIZE };
                for( int j = 0; j < BLOCK_SIZE; j++ )
                {
                        params.ipiv[j] = ipiv[ind + k + j] - k - 1 - ind;
                }
                    dlaswp3( params );
        }

        int num_pivots = nb - k;
        dlaswp_params_t2 params = { dAT, lda, lda, ind + k, num_pivots};
        for( int j = 0; j < num_pivots; j++ )
        {
            params.ipiv[j] = ipiv[ind + k + j] - k - 1 - ind;
        }
        dlaswp3( params );
}

#undef BLOCK_SIZE
