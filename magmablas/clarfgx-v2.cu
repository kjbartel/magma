#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @generated from zlarfgx-v2.cu normal z -> c, Fri Jan 30 19:00:09 2015

*/
#include "common_magma.h"
#include "commonblas_c.h"

// 512 is maximum number of threads for CUDA capability 1.x
#define BLOCK_SIZE 512

#define PRECISION_c


//==============================================================================

__global__
void magma_clarfgx_gpu_kernel( int n, magmaFloatComplex* dx0, magmaFloatComplex* dx,
                               magmaFloatComplex *dtau, float *dxnorm,
                               magmaFloatComplex *dA, int it)
{
    const int i = threadIdx.x;
    const int j = i + BLOCK_SIZE * blockIdx.x;
    __shared__ magmaFloatComplex scale;
    __shared__ float xnorm;
  
    magmaFloatComplex dxi;

    if ( j < n-1 )
        dxi = dx[j];
  
    if ( i == 0 ) {
        xnorm = *dxnorm;
#if (defined(PRECISION_s) || defined(PRECISION_d))
        float alpha = *dx0;
        float alphai = MAGMA_C_ZERO;
        if ( (xnorm == 0 && alphai == MAGMA_C_ZERO ) || n == 1 )
#else
        magmaFloatComplex alpha = *dx0;
        float alphar =  MAGMA_C_REAL(alpha), alphai = MAGMA_C_IMAG(alpha);
        if ( (xnorm == 0 && alphai == MAGMA_C_ZERO ) || n == 0 )
#endif
        {
            *dtau = MAGMA_C_ZERO;
            *dA   = *dx0;
        }
        else {

#if (defined(PRECISION_s) || defined(PRECISION_d))
            // no need to compute the norm as it is passed as input
            float beta  = xnorm; // sqrt( alpha*alpha + xnorm*xnorm );
            beta  = -copysign( beta, alpha );
 
            // todo: deal with badly scaled vectors (see lapack's larfg)
            if (j==0){
                *dtau = (beta - alpha) / beta;
                //*dx0  = 1.; //cannot be done here because raise condition all threadblock need to read it for alpha
                *dA   = beta;
            }

            scale = 1. / (alpha - beta);
#else
            // no need to compute the norm as it is passed as input
            float beta  = xnorm; // sqrt( alphar*alphar + alphai*alphai + xnorm*xnorm );
            beta  = -copysign( beta, alphar );

            // todo: deal with badly scaled vectors (see lapack's larfg)
            if (j==0){
                *dtau = MAGMA_C_MAKE((beta - alphar)/beta, -alphai/beta);
                //*dx0  = MAGMA_C_MAKE(  1., 0.); //cannot be done here because raise condition all threadblock need to read it for alpha
                *dA   = MAGMA_C_MAKE(beta, 0.);
            }

            alpha = MAGMA_C_MAKE( MAGMA_C_REAL(alpha) - beta, MAGMA_C_IMAG(alpha));
            scale = MAGMA_C_DIV( MAGMA_C_ONE, alpha);
#endif
        }
    }

    // scale x
    __syncthreads();
    if ( xnorm != 0 && j < n-1)
        dx[j] = MAGMA_C_MUL(dxi, scale);

    if (j<it){
        *( dA-it+j) = *(dx0-it+j);
        *(dx0-it+j) = MAGMA_C_MAKE(0., 0.);
    }
}

//==============================================================================

/*
    Generates Householder elementary reflector H = I - tau v v^T to reduce
        H [ dx0 ] = [ beta ]
          [ dx  ]   [ 0    ]
    with beta = ±norm( [dx0, dx] ) = ±dxnorm[0].
    Stores v over dx; first element of v is 1 and is not stored.
    Stores beta over dx0.
    Stores tau.
    
    The difference with LAPACK's clarfg is that the norm of dx, and hance beta,
    are computed outside the routine and passed to it in dxnorm (array on the GPU).
*/
extern "C" void
magma_clarfgx_gpu(
    magma_int_t n,
    magmaFloatComplex_ptr dx0,
    magmaFloatComplex_ptr dx,
    magmaFloatComplex_ptr dtau,
    magmaFloat_ptr        dxnorm,
    magmaFloatComplex_ptr dA, magma_int_t iter)
{
    dim3 blocks((n+BLOCK_SIZE-1) / BLOCK_SIZE);
    dim3 threads( BLOCK_SIZE );
 
    magma_clarfgx_gpu_kernel<<< blocks, threads, 0, magma_stream >>>( n, dx0, dx, dtau, dxnorm, dA, iter);
}

//==============================================================================

/*
    Generates Householder elementary reflector H = I - tau v v^T to reduce
        H [ dx0 ] = [ beta ]
          [ dx  ]   [ 0    ]
    with beta = ±norm( [dx0, dx] ) = ±dxnorm[0].
    Stores v over dx; first element of v is 1 and is not stored.
    Stores beta over dx0.
    Stores tau.
    
    The difference with LAPACK's clarfg is that the norm of dx, and hance beta,
    are computed outside the routine and passed to it in dxnorm (array on the GPU).
*/
extern "C" void
magma_clarfgtx_gpu(
    magma_int_t n,
    magmaFloatComplex_ptr dx0,
    magmaFloatComplex_ptr dx,
    magmaFloatComplex_ptr dtau,
    magmaFloat_ptr        dxnorm,
    magmaFloatComplex_ptr dA, magma_int_t iter,
    magmaFloatComplex_ptr V,  magma_int_t ldv,
    magmaFloatComplex_ptr T,  magma_int_t ldt,
    magmaFloatComplex_ptr dwork)
{
    /*  Generate the elementary reflector H(iter)  */
    magma_clarfgx_gpu(n, dx0, dx, dtau, dxnorm, dA, iter);
    
    if (iter==0) {
        magmaFloatComplex tt = MAGMA_C_ONE;
        magmablas_clacpy(MagmaUpperLower, 1, 1, dtau, 1, T+iter+iter*ldt, 1);
        magma_csetmatrix(1,1, &tt,1, dx0,1);
    }
    else {
        /* Compute the iter-th column of T */
        magma_cgemv_kernel3<<< iter, BLOCK_SIZE, 0, magma_stream >>>( n, V, ldv, dx0, dwork, dtau );
        magma_ctrmv_kernel2<<< iter, iter,       0, magma_stream >>>( T, ldt, dwork, T+iter*ldt, dtau );
    }
}

//==============================================================================
