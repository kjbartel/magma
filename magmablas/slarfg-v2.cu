#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.4.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       December 2013

       @generated s Tue Dec 17 13:18:45 2013

*/
#include "common_magma.h"

// 512 is maximum number of threads for CUDA capability 1.x
#define BLOCK_SIZE 512

#define PRECISION_s


__global__
void magma_slarfg_gpu_kernel( int n, float* dx0, float* dx,
                              float *dtau, float *dxnorm, float* dAkk)
{
    const int i = threadIdx.x;
    const int j = i + BLOCK_SIZE * blockIdx.x;
    __shared__ float scale;
    float xnorm;

    float dxi;

#if (defined(PRECISION_s) || defined(PRECISION_d))
    if( n <= 1 ) {
#else
    if( n <= 0 ) {
#endif
        *dtau = MAGMA_S_ZERO;
        return;
    }

    if ( j < n-1)
        dxi = dx[j];

    xnorm = *dxnorm;
    float alpha = *dx0;

#if (defined(PRECISION_s) || defined(PRECISION_d))
    if ( xnorm != 0 ) {
       if (i == 0) {  
            float beta  = sqrt( alpha*alpha + xnorm*xnorm );
            beta  = -copysign( beta, alpha );

            // todo: deal with badly scaled vectors (see lapack's larfg)
            *dtau = (beta - alpha) / beta;
            *dAkk  = beta;

            scale = 1. / (alpha - beta);
       }
#else
    float alphar =  MAGMA_S_REAL(alpha), alphai = MAGMA_S_IMAG(alpha);
    if ( xnorm != 0 || alphai != 0) {
       if (i == 0) {
            float beta  = sqrt( alphar*alphar + alphai*alphai + xnorm*xnorm );
            beta  = -copysign( beta, alphar );

            // todo: deal with badly scaled vectors (see lapack's larfg)
            *dtau = MAGMA_S_MAKE((beta - alphar)/beta, -alphai/beta);
            *dAkk = MAGMA_S_MAKE(beta, 0.);

            alpha = MAGMA_S_MAKE( MAGMA_S_REAL(alpha) - beta, MAGMA_S_IMAG(alpha));
            scale = MAGMA_S_DIV( MAGMA_S_ONE, alpha);
      }
#endif

      // scale x
      __syncthreads();
      if ( xnorm != 0 && j < n-1)
          dx[j] = MAGMA_S_MUL(dxi, scale);

    } else
        *dtau = MAGMA_S_ZERO;
}


/*
   Generates Householder elementary reflector H = I - tau v v^T to reduce
     H [ dx0 ] = [ beta ]
       [ dx  ]   [ 0    ]
   with beta = ±norm( [dx0, dx] ) = ±dxnorm[0].
   Stores v over dx; first element of v is 1 and is not stored.
   Stores beta over dx0.
   Stores tau.  

   The difference with LAPACK's slarfg is that the norm of dx, and hence beta,
   are computed outside the routine and passed to it in dxnorm (array on the GPU).
*/
extern "C" magma_int_t
magma_slarfg_gpu( magma_int_t n, float *dx0, float *dx,
                  float *dtau, float *dxnorm, float *dAkk)
{
    dim3 blocks((n+BLOCK_SIZE-1) / BLOCK_SIZE);
    dim3 threads( BLOCK_SIZE );

    /* recomputing the norm */
    //magmablas_snrm2_cols(n, 1, dx0, n, dxnorm);
    magmablas_snrm2_cols(n-1, 1, dx0+1, n, dxnorm);

    magma_slarfg_gpu_kernel<<< blocks, threads,
                               0, magma_stream >>>(n, dx0, dx, dtau, dxnorm, dAkk);

    return MAGMA_SUCCESS;
}

