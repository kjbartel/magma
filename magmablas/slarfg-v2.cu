#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.3.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2012

       @generated s Wed Nov 14 22:53:47 2012

*/
#include "common_magma.h"

// 512 is maximum number of threads for CUDA capability 1.x
#define BLOCK_SIZE 512

#define PRECISION_s

__global__
void magma_slarfg_gpu_kernel( int n, float* dx0, float* dx, 
                              float *dtau, float *dxnorm )
{
    const int i = threadIdx.x;
    const int j = i + BLOCK_SIZE * blockIdx.x;
    __shared__ float scale;
    __shared__ float xnorm;    
  
    float dxi;

    if ( j < n-1)
        dxi = dx[j];
  
    if ( i == 0 ) {
        xnorm = *dxnorm;
        if ( xnorm == 0 ) {
            *dtau = MAGMA_S_ZERO;
        }
        else {

#if (defined(PRECISION_s) || defined(PRECISION_d))
            float alpha = *dx0;
            float beta  = xnorm; // sqrt( alpha*alpha + xnorm*xnorm );
            beta  = -copysign( beta, alpha );

            // todo: deal with badly scaled vectors (see lapack's larfg)
            *dtau = (beta - alpha) / beta;
            *dx0  = beta;

            scale = 1 / (alpha - beta);
#else
            float alpha = *dx0;
            float alphar =  MAGMA_S_REAL(alpha), alphai = MAGMA_S_IMAG(alpha);
            float beta  = xnorm; // sqrt( alphar*alphar + alphai*alphai + xnorm*xnorm );
            beta  = -copysign( beta, alphar );

            // todo: deal with badly scaled vectors (see lapack's larfg)
            *dtau = MAGMA_S_MAKE((beta - alphar)/beta, -alphai/beta);
            *dx0  = MAGMA_S_MAKE(beta, 0.);
            
            alpha = MAGMA_S_MAKE( MAGMA_S_REAL(alpha) - beta, MAGMA_S_IMAG(alpha));
            scale = MAGMA_S_DIV( MAGMA_S_ONE, alpha);
#endif
        }
    }

    // scale x
    __syncthreads();
    if ( xnorm != 0 && j < n-1)
        dx[j] = MAGMA_S_MUL(dxi, scale);
}

/*
   Generates Householder elementary reflector H = I - tau v v^T to reduce
     H [ dx0 ] = [ beta ]
       [ dx  ]   [ 0    ]
   with beta = ±norm( [dx0, dx] ).
   Stores v over dx; first element of v is 1 and is not stored.
   Stores beta over dx0.
   Stores tau.  
*/
extern "C" void
magma_slarfg_gpu(int n, float *dx0, float *dx, 
                 float *dtau, float *dxnorm)
{
    dim3 blocks((n+BLOCK_SIZE-1) / BLOCK_SIZE);
    dim3 threads( BLOCK_SIZE );

    magma_slarfg_gpu_kernel<<< blocks, threads >>>( n, dx0, dx, dtau, dxnorm );
}
