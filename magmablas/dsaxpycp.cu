#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.5.0-beta1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date April 2014

       @generated from zcaxpycp.cu mixed zc -> ds, Fri Apr 25 15:05:17 2014

*/
#include "common_magma.h"

#define BLOCK_SIZE 64

// adds   X += R (including conversion to double)  --and--
// copies W = B
// each thread does one index, X[i] and W[i]
extern "C" __global__ void
dsaxpycp_kernel(
    int M, float *R, double *X,
    const double *B, double *W )
{
    const int i = threadIdx.x + blockIdx.x*BLOCK_SIZE;
    if ( i < M ) {
        X[i] = MAGMA_D_ADD( X[i], (double)( R[i] ) );
        W[i] = B[i];
    }
}


// adds   X += R  --and--
// copies R = B
// each thread does one index, X[i] and R[i]
extern "C" __global__ void
daxpycp_kernel(
    int M, double *R, double *X,
    const double *B)
{
    const int i = threadIdx.x + blockIdx.x*BLOCK_SIZE;
    if ( i < M ) {
        X[i] = MAGMA_D_ADD( X[i], R[i] );
        R[i] = B[i];
    }
}


// adds   X += R (including conversion to double)  --and--
// copies W = B
extern "C" void
magmablas_dsaxpycp(
    magma_int_t M, float *R, double *X,
    const double *B, double *W)
{
    dim3 threads( BLOCK_SIZE );
    dim3 grid( (M + BLOCK_SIZE - 1)/BLOCK_SIZE );
    dsaxpycp_kernel <<< grid, threads, 0, magma_stream >>> ( M, R, X, B, W );
}


// adds   X += R  --and--
// copies R = B
extern "C" void
magmablas_daxpycp(
    magma_int_t M, double *R, double *X,
    const double *B)
{
    dim3 threads( BLOCK_SIZE );
    dim3 grid( (M + BLOCK_SIZE - 1)/BLOCK_SIZE );
    daxpycp_kernel <<< grid, threads, 0, magma_stream >>> ( M, R, X, B );
}
