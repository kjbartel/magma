#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.4.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       December 2013

       @generated ds Tue Dec 17 13:18:44 2013

*/
#include "common_magma.h"

#define BLOCK_SIZE 64

// adds   X += R (including conversion to double)  --and--
// copies W = B
// each thread does one index, X[i] and W[i]
extern "C" __global__ void
dsaxpycp_kernel(
    int M, float *R, double *X,
    const double *B, double *W )
{
    const int i = threadIdx.x + blockIdx.x*BLOCK_SIZE;
    if ( i < M ) {
        X[i] = MAGMA_D_ADD( X[i], (double)( R[i] ) );
        W[i] = B[i];
    }
}


// adds   X += R  --and--
// copies R = B
// each thread does one index, X[i] and R[i]
extern "C" __global__ void
daxpycp_kernel(
    int M, double *R, double *X,
    const double *B)
{
    const int i = threadIdx.x + blockIdx.x*BLOCK_SIZE;
    if ( i < M ) {
        X[i] = MAGMA_D_ADD( X[i], R[i] );
        R[i] = B[i];
    }
}


// adds   X += R (including conversion to double)  --and--
// copies W = B
extern "C" void
magmablas_dsaxpycp(
    magma_int_t M, float *R, double *X,
    const double *B, double *W)
{
    dim3 threads( BLOCK_SIZE );
    dim3 grid( (M + BLOCK_SIZE - 1)/BLOCK_SIZE );
    dsaxpycp_kernel <<< grid, threads, 0, magma_stream >>> ( M, R, X, B, W );
}


// adds   X += R  --and--
// copies R = B
extern "C" void
magmablas_daxpycp(
    magma_int_t M, double *R, double *X,
    const double *B)
{
    dim3 threads( BLOCK_SIZE );
    dim3 grid( (M + BLOCK_SIZE - 1)/BLOCK_SIZE );
    daxpycp_kernel <<< grid, threads, 0, magma_stream >>> ( M, R, X, B );
}
