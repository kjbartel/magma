#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.4.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       December 2013

       @generated d Tue Dec 17 13:18:45 2013

*/

#include "common_magma.h"
#include <cblas.h>

#define PRECISION_d


//#if (GPUSHMEM < 200)
   #define BLOCK_SIZE 512
//#else
//   #define BLOCK_SIZE 768
//#endif

__global__ void magma_dgemv_kernel3(int m, const double * __restrict__ V, int ldv,
                                    double *c, double *dwork,
                                    double *tau);

/* --------------------------------------------------------------------------- */

extern "C" magma_int_t
magma_dlaqps2_gpu(magma_int_t m, magma_int_t n, magma_int_t offset,
             magma_int_t nb, magma_int_t *kb,
             double *A,  magma_int_t lda,
             magma_int_t *jpvt, double *tau, 
             double *vn1, double *vn2,
             double *auxv,
             double *F,  magma_int_t ldf)
{
/*
    -- MAGMA (version 1.4.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       December 2013

    Purpose
    =======
    DLAQPS computes a step of QR factorization with column pivoting
    of a real M-by-N matrix A by using Blas-3.  It tries to factorize
    NB columns from A starting from the row OFFSET+1, and updates all
    of the matrix with Blas-3 xGEMM.

    In some cases, due to catastrophic cancellations, it cannot
    factorize NB columns.  Hence, the actual number of factorized
    columns is returned in KB.

    Block A(1:OFFSET,1:N) is accordingly pivoted, but not factorized.

    Arguments
    =========
    M       (input) INTEGER
            The number of rows of the matrix A. M >= 0.

    N       (input) INTEGER
            The number of columns of the matrix A. N >= 0

    OFFSET  (input) INTEGER
            The number of rows of A that have been factorized in
            previous steps.

    NB      (input) INTEGER
            The number of columns to factorize.

    KB      (output) INTEGER
            The number of columns actually factorized.

    A       (input/output) DOUBLE PRECISION array, dimension (LDA,N)
            On entry, the M-by-N matrix A.
            On exit, block A(OFFSET+1:M,1:KB) is the triangular
            factor obtained and block A(1:OFFSET,1:N) has been
            accordingly pivoted, but no factorized.
            The rest of the matrix, block A(OFFSET+1:M,KB+1:N) has
            been updated.

    LDA     (input) INTEGER
            The leading dimension of the array A. LDA >= max(1,M).

    JPVT    (input/output) INTEGER array, dimension (N)
            JPVT(I) = K <==> Column K of the full matrix A has been
            permuted into position I in AP.

    TAU     (output) DOUBLE PRECISION array, dimension (KB)
            The scalar factors of the elementary reflectors.

    VN1     (input/output) DOUBLE PRECISION array, dimension (N)
            The vector with the partial column norms.

    VN2     (input/output) DOUBLE PRECISION array, dimension (N)
            The vector with the exact column norms.

    AUXV    (input/output) DOUBLE PRECISION array, dimension (NB)
            Auxiliar vector.

    F       (input/output) DOUBLE PRECISION array, dimension (LDF,NB)
            Matrix F' = L*Y'*A.

    LDF     (input) INTEGER
            The leading dimension of the array F. LDF >= max(1,N).

    =====================================================================    */
    
#define  A(i, j) (A  + (i) + (j)*(lda ))
#define  F(i, j) (F  + (i) + (j)*(ldf ))

    double c_zero    = MAGMA_D_MAKE( 0.,0.);
    double c_one     = MAGMA_D_MAKE( 1.,0.);
    double c_neg_one = MAGMA_D_MAKE(-1.,0.);
    magma_int_t ione = 1;
    
    magma_int_t i__1, i__2;
    
    magma_int_t k, rk;
    double tauk;
    magma_int_t pvt, itemp;
    double tol3z;

    double *dAkk = auxv;
    auxv+=nb;

    double lsticc, *lsticcs;
    magma_dmalloc( &lsticcs, 1+256*(n+255)/256 );

    tol3z = magma_dsqrt( lapackf77_dlamch("Epsilon"));

    lsticc = 0;
    k = 0;
    while( k < nb && lsticc == 0 ) {
        rk = offset + k;
        
        /* Determine ith pivot column and swap if necessary */
        pvt = k - 1 + magma_idamax( n-k, &vn1[k], ione );
        
        if (pvt != k) {
            magmablas_dswap( k, F(pvt,0), ldf, F(k,0), ldf);
            itemp     = jpvt[pvt];
            jpvt[pvt] = jpvt[k];
            jpvt[k]   = itemp;
            #if (defined(PRECISION_d) || defined(PRECISION_z))
                //magma_dswap( 1, &vn1[pvt], 1, &vn1[k], 1 );
                //magma_dswap( 1, &vn2[pvt], 1, &vn2[k], 1 );
                magma_dswap( 2, &vn1[pvt], n+offset, &vn1[k], n+offset);
            #else
                //magma_sswap( 1, &vn1[pvt], 1, &vn1[k], 1 );
                //magma_sswap( 1, &vn2[pvt], 1, &vn2[k], 1 );
                magma_sswap(2, &vn1[pvt], n+offset, &vn1[k], n+offset);
            #endif

            magmablas_dswap( m, A(0,pvt), ione, A(0, k), ione );
        }

        /* Apply previous Householder reflectors to column K:
           A(RK:M,K) := A(RK:M,K) - A(RK:M,1:K-1)*F(K,1:K-1)'.
           Optimization: multiply with beta=0; wait for vector and subtract */
        if (k > 0) {
            /*#if (defined(PRECISION_c) || defined(PRECISION_z))
            for (j = 0; j < k; ++j){
                *F(k,j) = MAGMA_D_CNJG( *F(k,j) );
            }
            #endif*/

            magmablas_dgemv( MagmaNoTrans, m-rk, k,
                             c_neg_one, A(rk, 0), lda,
                                        F(k,  0), ldf,
                             c_one,     A(rk, k), ione );

            /*#if (defined(PRECISION_c) || defined(PRECISION_z))
            for (j = 0; j < k; ++j) {
                *F(k,j) = MAGMA_D_CNJG( *F(k,j) );
            }
            #endif*/
        }
        
        /*  Generate elementary reflector H(k). */
        magma_dlarfg_gpu(m-rk, A(rk, k), A(rk + 1, k), &tau[k], &vn1[k], &dAkk[k]);
                
        //Akk = *A(rk, k);
        //*A(rk, k) = c_one;
        //magma_dgetvector( 1, A(rk, k), 1, &Akk,     1 );
        // this needs to be done outside dlarfg to avoid the race condition.
        magma_dsetvector( 1, &c_one,   1, A(rk, k), 1 );

        /* Compute Kth column of F:
           Compute  F(K+1:N,K) := tau(K)*A(RK:M,K+1:N)'*A(RK:M,K) on the GPU */
        if (k < n-1 || k > 0 ) magma_dgetvector( 1, &tau[k], 1, &tauk, 1 );
        if (k < n-1) {
            magmablas_dgemv( MagmaTrans, m-rk, n-k-1,
                         tauk,   A( rk,  k+1 ), lda,
                                 A( rk,  k   ), 1,
                         c_zero, F( k+1, k   ), 1 );
        }
        
        /* Incremental updating of F:
           F(1:N,K) := F(1:N,K) - tau(K)*F(1:N,1:K-1)*A(RK:M,1:K-1)'*A(RK:M,K). 
           F(1:N,K) := tau(K)*A(RK:M,K+1:N)'*A(RK:M,K) - tau(K)*F(1:N,1:K-1)*A(RK:M,1:K-1)'*A(RK:M,K)
                    := tau(K)(A(RK:M,K+1:N)' - F(1:N,1:K-1)*A(RK:M,1:K-1)') A(RK:M,K)  
           so, F is (updated A)*V */
        if (k > 0) {
            /*z__1 = MAGMA_D_NEGATE( tauk );
            magmablas_dgemv( MagmaTrans, m-rk, k,
                             z__1,   A(rk, 0), lda,
                                     A(rk, k), ione,
                             c_zero, auxv, ione );*/

            magma_dgemv_kernel3<<< k, BLOCK_SIZE, 0, magma_stream >>>(m-rk, A(rk, 0), lda,
                                                                      A(rk, k), auxv, tau+k);

            /* I think we only need stricly lower-triangular part */
            magmablas_dgemv( MagmaNoTrans, n-k-1, k,
                             c_one, F(k+1,0), ldf,
                                    auxv,     ione,
                             c_one, F(k+1,k), ione );
        }
        
        /* Update the current row of A:
           A(RK,K+1:N) := A(RK,K+1:N) - A(RK,1:K)*F(K+1:N,1:K)'.               */
        if (k < n-1) {
            i__1 = n - k - 1;
            i__2 = k + 1;
            /* left-looking update of rows,                     *
             * since F=A'v with original A, so no right-looking */
            magma_dgemm( MagmaNoTrans, MagmaTrans, ione, i__1, i__2,
                         c_neg_one, A(rk, 0  ), lda,
                                    F(k+1,0  ), ldf,
                         c_one,     A(rk, k+1), lda ); 
        }
        
        /* Update partial column norms. */
        if (rk < min(m, n+offset)-1){
           magmablas_dnrm2_row_check_adjust(n-k-1, tol3z, &vn1[k+1], 
                                             &vn2[k+1], A(rk,k+1), lda, lsticcs); 

           #if defined(PRECISION_d) || defined(PRECISION_z)
               magma_dgetvector( 1, &lsticcs[0], 1, &lsticc, 1 );
           #else
               magma_sgetvector( 1, &lsticcs[0], 1, &lsticc, 1 );
           #endif
        }

        //*A(rk, k) = Akk;
        //magma_dsetvector( 1, &Akk, 1, A(rk, k), 1 );
        //magmablas_dlacpy(MagmaUpperLower, 1, 1, dAkk, 1, A(rk, k), 1);

        ++k;
    }
    // restore the diagonals
    magma_dcopymatrix( 1, k, dAkk, 1, A(offset, 0), lda+1 );

    // leave k as the last column done
    --k;
    *kb = k + 1;
    rk = offset + *kb - 1;

    /* Apply the block reflector to the rest of the matrix:
       A(OFFSET+KB+1:M,KB+1:N) := A(OFFSET+KB+1:M,KB+1:N) - 
                                  A(OFFSET+KB+1:M,1:KB)*F(KB+1:N,1:KB)'  */
    if (*kb < min(n, m - offset)) {
        i__1 = m - rk - 1;
        i__2 = n - *kb;
        
        magma_dgemm( MagmaNoTrans, MagmaTrans, i__1, i__2, *kb,
                     c_neg_one, A(rk+1, 0  ), lda,
                                F(*kb,  0  ), ldf,
                     c_one,     A(rk+1, *kb), lda );
    }

    /* Recomputation of difficult columns. */
    if( lsticc > 0 ) {
        printf( " -- recompute dnorms --\n" );
        magmablas_dnrm2_check(m-rk-1, n-*kb, A(rk+1,*kb), lda,
                               &vn1[*kb], lsticcs);
#if defined(PRECISION_d) || defined(PRECISION_z)
        magma_dcopymatrix( n-*kb, 1, &vn1[*kb], *kb, &vn2[*kb], *kb);
#else   
        magma_scopymatrix( n-*kb, 1, &vn1[*kb], *kb, &vn2[*kb], *kb);
#endif  
    }
    magma_free(lsticcs);
    
    return MAGMA_SUCCESS;
} /* magma_dlaqps */
