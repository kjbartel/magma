#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

       @author Azzam Haidar
       @author Tingxing Dong

       @generated from zgeqr2_batched.cu normal z -> s, Fri Jan 30 19:00:10 2015
*/

#include "common_magma.h"
#include "magma_templates.h"
#include "batched_kernel_param.h"

#define BLOCK_SIZE 256
#define PRECISION_s

#define dA(a_1,a_2) (dA  + (a_1) + (a_2)*(local_lda))

#define REAL

//==============================================================================
static __device__ void
slarfg_device(
    int n,
    float* dalpha, float* dx, int incx,
    float* dtau,  float* swork, float* sscale, float* scale)
{

    const int tx = threadIdx.x;

    float tmp;
    
    // find max of [dalpha, dx], to use as scaling to avoid unnecesary under- and overflow    

    if ( tx == 0 ) {
        tmp = *dalpha;
        #ifdef COMPLEX
        swork[tx] = max( fabs(real(tmp)), fabs(imag(tmp)) );
        #else
        swork[tx] = fabs(tmp);
        #endif
    }
    else {
        swork[tx] = 0;
    }
    if(tx<BLOCK_SIZE)
    {
        for( int j = tx; j < n-1; j += BLOCK_SIZE ) {
            tmp = dx[j*incx];
            #ifdef COMPLEX
            swork[tx] = max( swork[tx], max( fabs(real(tmp)), fabs(imag(tmp)) ));
            #else
            swork[tx] = max( swork[tx], fabs(tmp) );
            #endif
         }
    }

    magma_max_reduce<BLOCK_SIZE>( tx, swork );

    if ( tx == 0 )
        *sscale = swork[0];
    __syncthreads();
    
    // sum norm^2 of dx/sscale
    // dx has length n-1
    if(tx<BLOCK_SIZE) swork[tx] = 0;
    if ( *sscale > 0 ) {
        if(tx<BLOCK_SIZE)
        {
            for( int j = tx; j < n-1; j += BLOCK_SIZE ) {
                tmp = dx[j*incx] / *sscale;
                swork[tx] += real(tmp)*real(tmp) + imag(tmp)*imag(tmp);
            }
        }
        magma_sum_reduce<BLOCK_SIZE>( tx, swork );

    }
    
    if ( tx == 0 ) {
        float alpha = *dalpha;

        if ( swork[0] == 0 && imag(alpha) == 0 ) {
            // H = I
            *dtau = MAGMA_S_ZERO;
        }
        else {
            // beta = norm( [dalpha, dx] )
            float beta;
            tmp  = alpha / *sscale;
            beta = *sscale * sqrt( real(tmp)*real(tmp) + imag(tmp)*imag(tmp) + swork[0] );
            beta = -copysign( beta, real(alpha) );
            // todo: deal with badly scaled vectors (see lapack's larfg)
            *dtau   = MAGMA_S_MAKE( (beta - real(alpha)) / beta, -imag(alpha) / beta );
            *dalpha = MAGMA_S_MAKE( beta, 0 );
            *scale = 1 / (alpha - beta);
        }
    }
    
    // scale x (if norm was not 0)
    __syncthreads();
    if ( swork[0] != 0 ) {
        if(tx<BLOCK_SIZE)
        {
            for( int j = tx; j < n-1; j += BLOCK_SIZE ) {
                dx[j*incx] *= *scale;
            }
        }
    }

}




//==============================================================================

static __device__
void slarfx_device( int m, int n,  float *v, float *tau,
                         float *dc, magma_int_t ldc, float* sum)
{


    if(n <=0) return ;
    if (MAGMA_S_EQUAL(*tau, MAGMA_S_ZERO) )  return; // check singularity

    const int tx = threadIdx.x;

    float lsum;
       
    for(int k=0;k<n;k++)
    {
        /* perform  w := v' * C  */
        if(tx<BLOCK_SIZE)
        {
            if (tx==0)
                lsum = dc[0+ldc*k]; //since V[0] should be one
            else
                lsum = MAGMA_S_ZERO;
            for( int j = tx+1; j < m; j += BLOCK_SIZE ){
                lsum += MAGMA_S_MUL( MAGMA_S_CNJG( v[j] ), dc[j+ldc*k] );
            }

            sum[tx] = lsum;
        }

        magma_sum_reduce< BLOCK_SIZE >( tx, sum );
        __syncthreads();

        float z__1 = - MAGMA_S_CNJG(*tau) * sum[0];
        /*  C := C - v * w  */
        if(tx<BLOCK_SIZE)
        {    
           for( int j = tx+1; j<m ; j += BLOCK_SIZE )
                 dc[j+ldc*k] += z__1 * v[j];
        }
        if(tx==0) dc[0+ldc*k] += z__1;

        __syncthreads();


    } 
}

//==============================================================================

extern __shared__ float shared_data[];


__global__
void sgeqr2_sm_kernel_batched( int m, int n, float** dA_array, magma_int_t lda,
                               float **dtau_array)
{

    float* dA = dA_array[blockIdx.z];
    float* dtau = dtau_array[blockIdx.z];

    float *sdata = (float*)shared_data;

    const int tx = threadIdx.x;

    __shared__ float scale;
    __shared__ float sum[ BLOCK_SIZE ];

    __shared__ float swork[ BLOCK_SIZE ];
    __shared__ float sscale;
    
    //load data from global to shared memory
    for(int s=0;s<n;s++)
    {
        for( int j = tx; j < m; j += BLOCK_SIZE )
        {
            sdata[j + s * m] = dA[j + s * lda] ;
        }
    }


    __syncthreads();
 
    for(int s=0; s<min(m,n); s++)
    {

       //lapack slarfg, compute the norm, scale and generate the householder vector   

       slarfg_device(m-s, &(sdata[s+s*m]), &(sdata[s+1+s*m]), 1, dtau+s, swork, &sscale, &scale); 
       __syncthreads();

       
       //update the trailing matix with the householder
       slarfx_device(m-s, n-(s+1), &(sdata[s+s*m]), dtau+s,&(sdata[s+(s+1)*m]), m, sum);

    }// end of s

    //copy back to global memory
    for(int s=0;s<n;s++)
    {
        for( int j = tx; j < m; j += BLOCK_SIZE )
        {
            dA[j + s * lda] = sdata[j + s * m];
        }
    }

}




//==============================================================================



static __device__
void sgeqr2_device( magma_int_t m, magma_int_t n, float* dA, magma_int_t lda,
                               float *dtau, 
                               float *sdata,
                               float *sum,
                               float *swork,
                               float *scale,
                               float *sscale)
{

    const int tx = threadIdx.x;


    for(int s=0; s<min(m,n); s++)
    {
       //load one vector in shared memory: sdata
       for( int j = tx; j < m-s; j += BLOCK_SIZE )
       {
           sdata[j] = dA[s + j + s * lda] ;
       }

       __syncthreads();

       //if(tx== 0) printf("m-s=%d",m-s);
       //lapack slarfg, compute the norm, scale and generate the householder vector   
       slarfg_device(m-s, sdata, &(sdata[1]), 1, dtau+s, swork, sscale, scale); 

       __syncthreads();

       //update the trailing matix with the householder
       slarfx_device(m-s, n-(s+1), sdata, dtau+s, &(dA[s+(s+1)*lda]), lda, sum);

       for( int j = tx; j < m-s; j += BLOCK_SIZE )
       {
           dA[s + j + s * lda] = sdata[j];
       }

       __syncthreads();

    }// end of s

}



//==============================================================================

__global__
void sgeqr2_kernel_batched( int m, int n, float** dA_array, magma_int_t lda,
                               float **dtau_array)
{

    float* dA = dA_array[blockIdx.z];
    float* dtau = dtau_array[blockIdx.z];

    float *sdata = (float*)shared_data;


    __shared__ float scale;
    __shared__ float sum[ BLOCK_SIZE ];

    __shared__ float swork[ BLOCK_SIZE ];
    __shared__ float sscale;

    sgeqr2_device(m, n, dA, lda, dtau, sdata, sum, swork, &scale, &sscale); 
 
}




//==============================================================================


/**
    Purpose
    -------
    SGEQR2 computes a QR factorization of a real m by n matrix A:
    A = Q * R.

    This expert routine requires two more arguments than the standard
    sgeqr2, namely, dT and ddA, explained below. The storage for A is
    also not as in the LAPACK's sgeqr2 routine (see below).

    The first is used to output the triangular
    n x n factor T of the block reflector used in the factorization.
    The second holds the diagonal nxn blocks of A, i.e., the diagonal
    submatrices of R.

    This version implements the right-looking QR with non-blocking.

    Arguments
    ---------
    @param[in]
    m       INTEGER
            The number of rows of the matrix A.  M >= 0.

    @param[in]
    n       INTEGER
            The number of columns of the matrix A.  N >= 0.

    @param[in,out]
    dA      REAL array, dimension (LDA,N)
            On entry, the m by n matrix A.
            On exit, the unitary matrix Q as a
            product of elementary reflectors (see Further Details).
    \n
            the elements on and above the diagonal of the array
            contain the min(m,n) by n upper trapezoidal matrix R (R is
            upper triangular if m >= n); the elements below the diagonal,
            with the array TAU, represent the unitary matrix Q as a
            product of elementary reflectors (see Further Details).

    @param[in]
    lda    INTEGER
            The leading dimension of the array A.  LDA >= max(1,M).

    @param[out]
    dtau    REAL array, dimension (min(M,N))
            The scalar factors of the elementary reflectors (see Further
            Details).

    @param[out]
    dT      REAL array, dimension N x N.
            Stores the triangular N x N factor T of the block reflector
            used in the factorization. The lower triangular part is 0.


    @param
    dwork   (workspace) REAL array, dimension (N) * ( sizeof(float) + sizeof(float)) 

    @param[out]
    info    INTEGER
      -     = 0: successful exit
      -     < 0: if INFO = -i, the i-th argument had an illegal value

    Further Details
    ---------------
    The matrix Q is represented as a product of elementary reflectors

       Q = H(1) H(2) . . . H(k), where k = min(m,n).

    Each H(i) has the form

       H(i) = I - tau * v * v'

    where tau is a real scalar, and v is a real vector with
    v(1:i-1) = 0 and v(i) = 1; v(i+1:m) is stored on exit in A(i+1:m,i),
    and tau in TAU(i).

    @ingroup magma_sgeqrf_comp
    ********************************************************************/
extern "C" magma_int_t
magma_sgeqr2_batched(magma_int_t m, magma_int_t n, float **dA_array,
                  magma_int_t lda, float **dtau_array,
                  magma_int_t *info_array, magma_int_t batchCount, magma_queue_t queue)
{
    
    magma_int_t k;

    /* Check arguments */
    magma_int_t arginfo = 0;
    if (m < 0)
        arginfo = -1;
    else if (n < 0)
        arginfo = -2;
    else if (lda < max(1,m))
        arginfo = -4;

    if (arginfo != 0) {
        magma_xerbla( __func__, -(arginfo) );
        return arginfo;
    }


    k = min(m,n);

    dim3 blocks(1, 1, batchCount);
    dim3 threads(BLOCK_SIZE);

    if(sizeof(float)*(m*k) <= 128 /*sizeof(float) * 128 * k*/) // there are some static shared memory besides of dynamic ones 
    {   
        //load panel in shared memory and factorize it and copy back to gloabl memory
        //intend for small panel to avoid overfill of shared memory.
        //this kernel is composed of device routine and thus clean
        sgeqr2_sm_kernel_batched<<< blocks, threads, sizeof(float)*(m*k), queue >>>
                                      (m, k, dA_array, lda, dtau_array);
    }
    else
    {
        //load one column vector in shared memory and householder it and used it to update trailing matrix which is global memory 
        // one vector is normally smaller than  48K shared memory   
        if(sizeof(float)*(m) < 42000)
            sgeqr2_kernel_batched<<< blocks, threads, sizeof(float)*(m), queue >>>
                                      (m, k, dA_array, lda, dtau_array);
        else
            printf("m is too big, kernel launching failed, shared memory is overflowed");
    }


    return arginfo;

} 



//==============================================================================


