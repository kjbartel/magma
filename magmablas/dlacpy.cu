#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2014

       @generated from zlacpy.cu normal z -> d, Sat Nov 15 19:53:59 2014
       @author Mark Gates
       @author Azzam Haidar
*/
#include "common_magma.h"

#define BLK_X 64
#define BLK_Y 32

/*
    Divides matrix into ceil( m/BLK_X ) x ceil( n/BLK_Y ) blocks.
    Each block has BLK_X threads.
    Each thread loops across one row, updating BLK_Y entries.

    Code similar to dlaset.
*/
static __device__
void dlacpy_full_device(
    int m, int n,
    const double *dA, int ldda,
    double       *dB, int lddb )
{
    int ind = blockIdx.x*BLK_X + threadIdx.x;
    int iby = blockIdx.y*BLK_Y;
    /* check if full block-column */
    bool full = (iby + BLK_Y <= n);
    /* do only rows inside matrix */
    if ( ind < m ) {
        dA += ind + iby*ldda;
        dB += ind + iby*lddb;
        if ( full ) {
            // full block-column
            #pragma unroll
            for( int j=0; j < BLK_Y; ++j ) {
                dB[j*lddb] = dA[j*ldda];
            }
        }
        else {
            // partial block-column
            for( int j=0; j < BLK_Y && iby+j < n; ++j ) {
                dB[j*lddb] = dA[j*ldda];
            }
        }
    }
}


/*
    Similar to dlacpy_full, but updates only the diagonal and below.
    Blocks that are fully above the diagonal exit immediately.

    Code similar to dlaset.
*/
static __device__
void dlacpy_lower_device(
    int m, int n,
    const double *dA, int ldda,
    double       *dB, int lddb )
{
    int ind = blockIdx.x*BLK_X + threadIdx.x;
    int iby = blockIdx.y*BLK_Y;
    /* check if full block-column && (below diag) */
    bool full = (iby + BLK_Y <= n && (ind >= iby + BLK_Y));
    /* do only rows inside matrix, and blocks not above diag */
    if ( ind < m && ind + BLK_X > iby ) {
        dA += ind + iby*ldda;
        dB += ind + iby*lddb;
        if ( full ) {
            // full block-column, off-diagonal block
            #pragma unroll
            for( int j=0; j < BLK_Y; ++j ) {
                dB[j*lddb] = dA[j*ldda];
            }
        }
        else {
            // either partial block-column or diagonal block
            for( int j=0; j < BLK_Y && iby+j < n && ind >= iby+j; ++j ) {
                dB[j*lddb] = dA[j*ldda];
            }
        }
    }
}


/*
    Similar to dlacpy_full, but updates only the diagonal and above.
    Blocks that are fully below the diagonal exit immediately.

    Code similar to dlaset.
*/
static __device__
void dlacpy_upper_device(
    int m, int n,
    const double *dA, int ldda,
    double       *dB, int lddb )
{
    int ind = blockIdx.x*BLK_X + threadIdx.x;
    int iby = blockIdx.y*BLK_Y;
    /* check if full block-column && (above diag) */
    bool full = (iby + BLK_Y <= n && (ind + BLK_X <= iby));
    /* do only rows inside matrix, and blocks not below diag */
    if ( ind < m && ind < iby + BLK_Y ) {
        dA += ind + iby*ldda;
        dB += ind + iby*lddb;
        if ( full ) {
            // full block-column, off-diagonal block
            #pragma unroll
            for( int j=0; j < BLK_Y; ++j ) {
                dB[j*lddb] = dA[j*ldda];
            }
        }
        else {
            // either partial block-column or diagonal block
            for( int j=0; j < BLK_Y && iby+j < n; ++j ) {
                if ( ind <= iby+j ) {
                    dB[j*lddb] = dA[j*ldda];
                }
            }
        }
    }
}

/*
    kernel wrapper to call the device function.
*/
__global__
void dlacpy_full_kernel(
    int m, int n,
    const double *dA, int ldda,
    double       *dB, int lddb )
{
    dlacpy_full_device(m, n, dA, ldda, dB, lddb);
}

__global__
void dlacpy_lower_kernel(
    int m, int n,
    const double *dA, int ldda,
    double       *dB, int lddb )
{
    dlacpy_lower_device(m, n, dA, ldda, dB, lddb);
}

__global__
void dlacpy_upper_kernel(
    int m, int n,
    const double *dA, int ldda,
    double       *dB, int lddb )
{
    dlacpy_upper_device(m, n, dA, ldda, dB, lddb);
}


/*
    kernel wrapper to call the device function for the batched routine.
*/
__global__
void dlacpy_full_kernel_batched(
    int m, int n,
    double const * const *dAarray, int ldda,
    double **dBarray, int lddb )
{
    int batchid = blockIdx.z;
    dlacpy_full_device(m, n, dAarray[batchid], ldda, dBarray[batchid], lddb);
}

__global__
void dlacpy_lower_kernel_batched(
    int m, int n,
    double const * const *dAarray, int ldda,
    double **dBarray, int lddb )
{
    int batchid = blockIdx.z;
    dlacpy_lower_device(m, n, dAarray[batchid], ldda, dBarray[batchid], lddb);
}

__global__
void dlacpy_upper_kernel_batched(
    int m, int n,
    double const * const *dAarray, int ldda,
    double **dBarray, int lddb )
{
    int batchid = blockIdx.z;
    dlacpy_upper_device(m, n, dAarray[batchid], ldda, dBarray[batchid], lddb);
}


/**
    Purpose
    -------
    DLACPY_Q copies all or part of a two-dimensional matrix dA to another
    matrix dB.
    
    This is the same as DLACPY, but adds queue argument.
    
    Arguments
    ---------
    
    @param[in]
    uplo    magma_uplo_t
            Specifies the part of the matrix dA to be copied to dB.
      -     = MagmaUpper:      Upper triangular part
      -     = MagmaLower:      Lower triangular part
            Otherwise:  All of the matrix dA
    
    @param[in]
    m       INTEGER
            The number of rows of the matrix dA.  M >= 0.
    
    @param[in]
    n       INTEGER
            The number of columns of the matrix dA.  N >= 0.
    
    @param[in]
    dA      DOUBLE_PRECISION array, dimension (LDDA,N)
            The m by n matrix dA.
            If UPLO = MagmaUpper, only the upper triangle or trapezoid is accessed;
            if UPLO = MagmaLower, only the lower triangle or trapezoid is accessed.
    
    @param[in]
    ldda    INTEGER
            The leading dimension of the array dA.  LDDA >= max(1,M).
    
    @param[out]
    dB      DOUBLE_PRECISION array, dimension (LDDB,N)
            The m by n matrix dB.
            On exit, dB = dA in the locations specified by UPLO.
    
    @param[in]
    lddb    INTEGER
            The leading dimension of the array dB.  LDDB >= max(1,M).
    
    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_daux2
    ********************************************************************/
extern "C" void
magmablas_dlacpy_q(
    magma_uplo_t uplo, magma_int_t m, magma_int_t n,
    magmaDouble_const_ptr dA, magma_int_t ldda,
    magmaDouble_ptr       dB, magma_int_t lddb,
    magma_queue_t queue )
{
    magma_int_t info = 0;
    if ( m < 0 )
        info = -2;
    else if ( n < 0 )
        info = -3;
    else if ( ldda < max(1,m))
        info = -5;
    else if ( lddb < max(1,m))
        info = -7;
    
    if ( info != 0 ) {
        magma_xerbla( __func__, -(info) );
        return;
    }
    
    if ( m == 0 || n == 0 )
        return;
    
    dim3 threads( BLK_X, 1 );
    dim3 grid( (m + BLK_X - 1)/BLK_X, (n + BLK_Y - 1)/BLK_Y );
    
    if ( uplo == MagmaLower ) {
        dlacpy_lower_kernel<<< grid, threads, 0, queue >>> ( m, n, dA, ldda, dB, lddb );
    }
    else if ( uplo == MagmaUpper ) {
        dlacpy_upper_kernel<<< grid, threads, 0, queue >>> ( m, n, dA, ldda, dB, lddb );
    }
    else {
        dlacpy_full_kernel <<< grid, threads, 0, queue >>> ( m, n, dA, ldda, dB, lddb );
    }
}

/**
    @see magmablas_dlacpy_q
    @ingroup magma_daux2
    ********************************************************************/
extern "C" void
magmablas_dlacpy(
    magma_uplo_t uplo, magma_int_t m, magma_int_t n,
    magmaDouble_const_ptr dA, magma_int_t ldda,
    magmaDouble_ptr       dB, magma_int_t lddb )
{
    magmablas_dlacpy_q( uplo, m, n, dA, ldda, dB, lddb, magma_stream );
}


/**
    Purpose
    -------
    DLACPY_BATCHED_Q copies all or part of each two-dimensional matrix
    dAarray[i] to matrix dBarray[i], for 0 <= i < batchcount.
    
    This is the same as DLACPY_BATCHED, but adds queue argument.
    
    Arguments
    ---------
    
    @param[in]
    uplo    magma_uplo_t
            Specifies the part of each matrix dA to be copied to dB.
      -     = MagmaUpper:      Upper triangular part
      -     = MagmaLower:      Lower triangular part
            Otherwise:  All of each matrix dA
    
    @param[in]
    m       INTEGER
            The number of rows of each matrix dA.  M >= 0.
    
    @param[in]
    n       INTEGER
            The number of columns of each matrix dA.  N >= 0.
    
    @param[in]
    dAarray DOUBLE_PRECISION* array, dimension (batchCount)
            array of pointers to the matrices dA, where each dA is of dimension (LDDA,N)
            The m by n matrix dA.
            If UPLO = MagmaUpper, only the upper triangle or trapezoid is accessed;
            if UPLO = MagmaLower, only the lower triangle or trapezoid is accessed.
    
    @param[in]
    ldda    INTEGER
            The leading dimension of each array dA.  LDDA >= max(1,M).
    
    @param[out]
    dBarray DOUBLE_PRECISION* array, dimension (batchCount)
            array of pointers to the matrices dB, where each dB is of dimension (LDDB,N)
            The m by n matrix dB.
            On exit, dB = dA in the locations specified by UPLO.
    
    @param[in]
    lddb    INTEGER
            The leading dimension of each array dB.  LDDB >= max(1,M).
    
    @param[in]
    batchCount  Number of matrices in dAarray and dBarray.
    
    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_daux2
    ********************************************************************/
extern "C" void
magmablas_dlacpy_batched_q(
    magma_uplo_t uplo, magma_int_t m, magma_int_t n,
    magmaDouble_const_ptr const dAarray[], magma_int_t ldda,
    magmaDouble_ptr             dBarray[], magma_int_t lddb,
    magma_int_t batchCount, magma_queue_t queue )
{
    magma_int_t info = 0;
    if ( m < 0 )
        info = -2;
    else if ( n < 0 )
        info = -3;
    else if ( ldda < max(1,m))
        info = -5;
    else if ( lddb < max(1,m))
        info = -7;
    else if ( batchCount < 0 )
        info = -8;
    
    if ( info != 0 ) {
        magma_xerbla( __func__, -(info) );
        return;
    }
    
    if ( m == 0 || n == 0 || batchCount == 0 )
        return;
    
    dim3 threads( BLK_X, 1, 1 );
    dim3 grid( (m + BLK_X - 1)/BLK_X, (n + BLK_Y - 1)/BLK_Y, batchCount );
    
    if ( uplo == MagmaLower ) {
        dlacpy_lower_kernel_batched<<< grid, threads, 0, queue >>> ( m, n, dAarray, ldda, dBarray, lddb );
    }
    else if ( uplo == MagmaUpper ) {
        dlacpy_upper_kernel_batched<<< grid, threads, 0, queue >>> ( m, n, dAarray, ldda, dBarray, lddb );
    }
    else {
        dlacpy_full_kernel_batched <<< grid, threads, 0, queue >>> ( m, n, dAarray, ldda, dBarray, lddb );
    }
}


/**
    @see magmablas_dlacpy_batched_q
    @ingroup magma_daux2
    ********************************************************************/
extern "C" void
magmablas_dlacpy_batched(
    magma_uplo_t uplo, magma_int_t m, magma_int_t n,
    magmaDouble_const_ptr const dAarray[], magma_int_t ldda,
    magmaDouble_ptr             dBarray[], magma_int_t lddb,
    magma_int_t batchCount )
{
    magmablas_dlacpy_batched_q( uplo, m, n, dAarray, ldda, dBarray, lddb, batchCount, magma_stream );
}
