#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @generated from zlacpy_cnjg.cu normal z -> d, Fri Jan 30 19:00:09 2015

*/
#include "common_magma.h"

#define BLOCK_SIZE 64

/*********************************************************
 *
 * SWAP BLAS: permute to set of N elements
 *
 ********************************************************/
/*
 *  First version: line per line
 */
typedef struct {
    double *A1;
    double *A2;
    int n, lda1, lda2;
} magmagpu_dlacpy_cnjg_params_t;

__global__ void magmagpu_dlacpy_cnjg( magmagpu_dlacpy_cnjg_params_t params )
{
    unsigned int x = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned int offset1 = x*params.lda1;
    unsigned int offset2 = x*params.lda2;
    if( x < params.n )
    {
        double *A1  = params.A1 + offset1;
        double *A2  = params.A2 + offset2;
        *A2 = MAGMA_D_CNJG(*A1);
    }
}


extern "C" void 
magmablas_dlacpy_cnjg_q(
    magma_int_t n, double *dA1, magma_int_t lda1, 
    double *dA2, magma_int_t lda2,
    magma_queue_t queue )
{
    int blocksize = 64;
    dim3 blocks( (n+blocksize-1) / blocksize, 1, 1);
    magmagpu_dlacpy_cnjg_params_t params = { dA1, dA2, n, lda1, lda2 };
    magmagpu_dlacpy_cnjg<<< blocks, blocksize, 0, queue >>>( params );
}


extern "C" void 
magmablas_dlacpy_cnjg(
    magma_int_t n, double *dA1, magma_int_t lda1, 
    double *dA2, magma_int_t lda2)
{
    magmablas_dlacpy_cnjg_q( n, dA1, lda1, dA2, lda2, magma_stream );
}
