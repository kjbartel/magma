#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

       @generated from zlarft_kernels.cu normal z -> d, Fri Jan 30 19:00:10 2015
       @author Azzam Haidar
*/

#include "common_magma.h"
#include "magma_templates.h"
#define dgemv_bs 32
#define BLOCK_SIZE 512

#define use_gemm_larft

extern __shared__ double shared_data[];


//===================================================================================================
static __device__
void dlarft_gemvcolwise_device( int m, double *v, double *tau,
                         double *c, int ldc, double *T, int ldt, int step )
{

    const int thblk =  blockIdx.x;
    if (thblk > step)
        return;
    /* if blockIdx.x<step step performs the z = V(tx:n,tx)' * V(tx:n,1:tx-1) used for computing T:*/

    if ( !MAGMA_D_EQUAL(*tau, MAGMA_D_ZERO) ) {
        if(thblk<step){    
            const int tx = threadIdx.x;
            double *dc = c + blockIdx.x * ldc;
           
            __shared__ double sum[ BLOCK_SIZE ];
            double tmp;
           
            /* perform  {T_i}^H := V(:,i)' * V(:,1:i-1)  */
            if (tx==0)
                tmp = dc[0]; //since V[0] should be one
            else
                tmp = MAGMA_D_ZERO;
            for( int j = tx+1; j < m; j += BLOCK_SIZE ){
                tmp +=  MAGMA_D_CNJG( v[j] ) * dc[j];
            }
            sum[tx] = tmp;
            magma_sum_reduce< BLOCK_SIZE >( tx, sum );
            #if defined (use_gemm_larft)
            *(T+thblk) = MAGMA_D_CNJG(sum[0]);
            #else
            tmp = - MAGMA_D_CNJG(*tau) * sum[0]; 
            *(T+thblk) = MAGMA_D_CNJG(tmp); // T = - tau(tx) * V(tx:n,1:tx-1)' * V(tx:n,tx) = tmp'
            //*(T+thblk) = - MAGMA_D_CNJG(sum[0]) * (*tau); // T = - tau(tx) * V(tx:n,1:tx-1)' * V(tx:n,tx) = tmp'
            #endif
        }
        else{
            #if defined (use_gemm_larft)
            *(T+thblk) = MAGMA_D_ONE;
            #else
            *(T+thblk) = *tau;
            #endif
        }
    }// in case tau is zero put the corresponding column of T to zero
    else 
    {
        *(T+thblk) = MAGMA_D_ZERO;
    }
}
//===================================================================================================
__global__
void dlarft_gemvcolwise_kernel( int m, double *v, int ldv, double *tau,
                          double *T, int ldt, int step )
{
    dlarft_gemvcolwise_device(m, v+step+step*ldv, tau+step, v+step, ldv, T+step*ldt, ldt, step);
}
//===================================================================================================
__global__
void dlarft_gemvcolwise_kernel_batched( int m, double **v_array, int ldv, double **tau_array,
                          double **T_array, int ldt, int step )
{
    int batchid = blockIdx.z;
    dlarft_gemvcolwise_device(m, v_array[batchid]+step+step*ldv, tau_array[batchid]+step, v_array[batchid]+step, ldv, T_array[batchid]+step*ldt, ldt, step);
}
//===================================================================================================
extern "C" 
void magmablas_dlarft_gemvcolwise(
    magma_int_t m,  magma_int_t step,
    double *v, magma_int_t ldv, 
    double *T,  magma_int_t ldt,
    double *tau)
{
    dim3 grid( step+1, 1, 1 );
    dim3 threads( BLOCK_SIZE );
    dlarft_gemvcolwise_kernel<<< grid, threads, 0, magma_stream >>>( m, v, ldv, tau, T, ldt, step);

}
//===================================================================================================
extern "C" 
void magmablas_dlarft_gemvcolwise_batched(
    magma_int_t m,  magma_int_t step,
    double **v_array, magma_int_t ldv, 
    double **T_array,  magma_int_t ldt,
    double **tau_array, magma_int_t batchCount, magma_queue_t queue )
{
    dim3 grid( step+1, 1, batchCount );
    dim3 threads( BLOCK_SIZE );
    dlarft_gemvcolwise_kernel_batched<<< grid, threads, 0, queue >>>( m, v_array, ldv, tau_array, T_array, ldt, step);

}
//===================================================================================================




//===================================================================================================
// dgemv(y=alpha*A*x) interface: T/W=tau*v*x, 
static __device__ void
dlarft_gemvrowwise_device(
    int m, int i,
    double *tau, 
    double *v_ptr, int ldv, 
    double *x_ptr, int incx,
    double *T_ptr, int ldt,
    double *W, double* sdata)
{
    int tx = threadIdx.x; 
    int ty = threadIdx.y; 


    if(tx ==0 && ty == 0)
    {
        T_ptr[0] = *tau;
    } 

    if(i <= 0) return;
    
    double res = MAGMA_D_ZERO;

    v_ptr += ldv * ty;
            

   
    if(tx < dgemv_bs)
    {
        for(int s=tx; s<m; s+= dgemv_bs)
        {
            res += MAGMA_D_CNJG (v_ptr[s]) * x_ptr[s*incx];
        }
    
        sdata[ty * dgemv_bs + tx] = res;
    }
    __syncthreads();

    magma_sum_reduce<dgemv_bs>(tx, &(sdata[ty*dgemv_bs+0]));

    #if defined (use_gemm_larft)
    if(tx == 0)
    {
            W[ty] = -sdata[ty * dgemv_bs + 0];
    } 
    #else
    if(tx == 0)
    {
            W[ty] = -sdata[ty * dgemv_bs + 0] * (*tau) ;
    }
    #endif 
}




//T(1:i-1,i) := - tau(i) * V(i:n,1:i-1)' * V(i:n,i)
//T(i,i) = tau(i)
//===================================================================================================
 __global__ void
dlarft_gemvrowwise_kernel(
    int m, int i, 
    double *tau, 
    double *v, int ldv, 
    double *T, int ldt)
{

    double *W =  T +i*ldt;

    double *sdata = (double*)shared_data;

    dlarft_gemvrowwise_device(m, i, tau+i, v+i, ldv,  v+i+i*ldv, 1,  
                           T+i+i*ldt , ldt, W, sdata);
}

//===================================================================================================
__global__ void
dlarft_gemvrowwise_kernel_batched(
    int m, int i,
    double **tau_array, 
    double **v_array, int ldv, 
    double **T_array, int ldt)
{

    int batchid = blockIdx.z;

    double *W =  T_array[batchid] +i*ldt;

    double *sdata = (double*)shared_data;

    dlarft_gemvrowwise_device(m, i, tau_array[batchid]+i, v_array[batchid]+i, ldv,  v_array[batchid]+i+i*ldv, 1,  
                           T_array[batchid] +i+i*ldt , ldt, W, sdata);
}

//===================================================================================================
extern "C"
void magmablas_dlarft_gemvrowwise(
    magma_int_t m, magma_int_t i, 
    double *tau, 
    double *v, magma_int_t ldv, 
    double *T, magma_int_t ldt,
    double *W)
{

    dim3 grid(1);


    dim3 threads(dgemv_bs, max(i,1), 1);


    dlarft_gemvrowwise_kernel <<< grid, threads, sizeof(double)*dgemv_bs*(i+1), magma_stream>>>(m, i, tau, v, ldv, T, ldt);
}
//===================================================================================================
extern "C"
void magmablas_dlarft_gemvrowwise_batched(
    magma_int_t m, magma_int_t i, 
    double **tau_array, 
    double **v_array, magma_int_t ldv, 
    double **T_array, magma_int_t ldt,
    magma_int_t batchCount, magma_queue_t queue)
{

    dim3 grid(1, 1, batchCount);
    dim3 threads(dgemv_bs, max(i,1), 1);

    /*  dgemvrowwise used a bigger shared memory and has more data reuse and performs better
    */
    dlarft_gemvrowwise_kernel_batched <<< grid, threads, sizeof(double)*dgemv_bs*(i+1), queue>>>(m, i,  tau_array, v_array, ldv, T_array, ldt);
}
//===================================================================================================
   


//===================================================================================================
/*
   loop_inside
*/
static __device__ void
dlarft_gemv_loop_inside_device(
    int n, int k, 
    double *tau, 
    double *v, int ldv, 
    double *T, int ldt)
{
    int tx = threadIdx.x; 
    int ty = threadIdx.y; 
    
    int incx = 1;
    double *sdata = (double*)shared_data;

    double res;

    // write the first elment
    if(tx ==0 && ty == 0)
    {
        T[0] = tau[0];
    } 
 
    for(int i=1; i<k;i++)
    {

        int m = n-i; 

        double *v_ptr = v;

        v_ptr += i;

        double *x_ptr = v_ptr + i * ldv;
            
        res = MAGMA_D_ZERO;
            
        if(tx < dgemv_bs && ty < i)
        {
            v_ptr += ldv * ty;

            for(int s=tx; s<m; s+= dgemv_bs)
            {
                res += MAGMA_D_CNJG (v_ptr[s]) * x_ptr[s*incx];
            }
    
            sdata[ty * dgemv_bs + tx] = res;
        }
        __syncthreads();

        magma_sum_reduce<dgemv_bs>(tx, &(sdata[ty*dgemv_bs+0]));
        

       __syncthreads();
       #if defined (use_gemm_larft)
       if(tx < i && ty == 0)
       {
            T[i* ldt + tx] = sdata[tx * dgemv_bs + 0];  
       } 
       // not needed since it is overwritten in trmv
       /*
       if(tx == i && ty == 0)
       {
           T[i * ldt + i] = tau[i];
       }
       */
       #else
       if(tx < i && ty == 0)
       {
           T[i* ldt + tx] = -sdata[tx * dgemv_bs + 0] * (tau[i]) ;  
       } 
      
       if(tx == i && ty == 0)
       {
           T[i * ldt + i] = tau[i];
       }
       #endif
     
       v_ptr -= i;

    }// end of loop k
}
//===================================================================================================
__global__ void
dlarft_gemv_loop_inside_kernel(
    int n, int k, 
    double *tau, 
    double *v, int ldv, 
    double *T, int ldt)
{
    dlarft_gemv_loop_inside_device(n, k, tau, v, ldv, T, ldt);
}
//===================================================================================================
__global__ void
dlarft_gemv_loop_inside_kernel_batched(
    int n, int k, 
    double **tau_array, 
    double **v_array, int ldv, 
    double **T_array, int ldt)
{
    int batchid = blockIdx.z;
    dlarft_gemv_loop_inside_device(n, k, tau_array[batchid], v_array[batchid], ldv, T_array[batchid], ldt);
}
//===================================================================================================
//===================================================================================================
//===================================================================================================
extern "C"
void magmablas_dlarft_gemv_loop_inside(
    int n, int k, 
    double *tau, 
    double *v, int ldv, 
    double *T, int ldt)
{

    dim3 grid(1);
    dim3 threads(dgemv_bs, max(k,1), 1);
    dlarft_gemv_loop_inside_kernel<<<grid, threads, sizeof(double) * (dgemv_bs*(k+1)), magma_stream>>>(n, k, tau, v, ldv, T, ldt); 
}
//===================================================================================================
extern "C"
void magmablas_dlarft_gemv_loop_inside_batched(
    int n, int k, 
    double **tau_array, 
    double **v_array, int ldv, 
    double **T_array, int ldt, magma_int_t batchCount, magma_queue_t queue)
{

    dim3 grid(1, 1, batchCount);
    dim3 threads(dgemv_bs, max(k,1), 1);
    dlarft_gemv_loop_inside_kernel_batched<<<grid, threads, sizeof(double) * (dgemv_bs*(k+1)), queue>>>(n, k, tau_array, v_array, ldv, T_array, ldt); 
}
//===================================================================================================





//===================================================================================================
static  __device__ void 
dlarft_dtrmv_sm32x32_device(
    int n, int k, double *tau,
    double *Tin, int ldtin,  double *Tout, int ldtout )
{
    int tx = threadIdx.x; 
    double *sdata = (double*)shared_data;
    double res;

    // this routine apply a sequence of trmv to update k column of the triangular
    // T starting at n-k to n where T is of size n by n and where the first n-k 
    // columns of T are supposed updated previously.
    // So the routine load all of T nxn to the shared memory 
    // and apply the sequence of trmv.
    // to update a certain column i, threads go in horizontal fashion where
    // every thread read one row and do it gemv(dot) to generate 
    // one element of the column of T then move to the next column

    // read T into shared
    for(int s=0; s<n-k; s++)
    {
        sdata[tx + s*n] = Tin[tx + s * ldtin];
    }
    
#if defined(use_gemm_larft)
    for(int s=n-k; s<n; s++)
    {
        if(tx == s)
            sdata[tx + s*n] = tau[s];
        else
            sdata[tx + s*n] = -tau[s] * Tin[tx + s * ldtin];
    }
#else
    for(int s=n-k; s<n; s++)
    {
        sdata[tx + s*n] = Tin[tx + s * ldtin];
    }
#endif

    // perform trmv
    for(int i=n-k; i<n;i++)
    {
       __syncthreads();  
       res = MAGMA_D_ZERO;
       if(tx < i)
       {
           for(int j=tx; j<i; j++)
           {
               res += sdata[tx + j * n] * sdata[j+ i * n];      
           }
       }       
       __syncthreads();  
       if(tx < i)
       {
           sdata[tx + i * n] = res;
       }
    } 

    __syncthreads();  
    // write back the updated block of k column of T
    for(int s=n-k; s<n; s++)
    {
       Tout[tx + s * ldtout] = sdata[tx + s*n];
    }

}
//===================================================================================================
__global__ void 
dlarft_dtrmv_sm32x32_kernel(
    int n, int k, double *tau,
    double *Tin, int ldtin,  double *Tout, int ldtout )
{
    dlarft_dtrmv_sm32x32_device( n, k, tau, Tin, ldtin, Tout, ldtout);
}
//===================================================================================================
__global__ void 
dlarft_dtrmv_sm32x32_kernel_batched(
    int n, int k, double **tau_array,
    double **Tin_array, int ldtin,  double **Tout_array, int ldtout )
{
    int batchId = blockIdx.z;
    dlarft_dtrmv_sm32x32_device( n, k, tau_array[batchId], Tin_array[batchId], ldtin, Tout_array[batchId], ldtout);
}
//===================================================================================================
//===================================================================================================
extern "C"
void magmablas_dlarft_dtrmv_sm32x32(
    magma_int_t m, magma_int_t n, 
    double *tau, 
    double *Tin, magma_int_t ldtin, 
    double *Tout, magma_int_t ldtout)
{

    dim3 grid(1);
    dim3 threads(max(m,1), 1, 1);
    dlarft_dtrmv_sm32x32_kernel <<< grid, threads, sizeof(double)*(m*m), magma_stream >>> (m, n,  tau, Tin, ldtin, Tout, ldtout);
}
//===================================================================================================
extern "C"
void magmablas_dlarft_dtrmv_sm32x32_batched(
    magma_int_t m, magma_int_t n, 
    double **tau_array, 
    double **Tin_array, magma_int_t ldtin, 
    double **Tout_array, magma_int_t ldtout,
    magma_int_t batchCount, magma_queue_t queue)
{

    dim3 grid(1, 1, batchCount);
    dim3 threads(max(m,1), 1, 1);
    dlarft_dtrmv_sm32x32_kernel_batched <<< grid, threads, sizeof(double)*(m*m), queue >>> (m, n,  tau_array, Tin_array, ldtin, Tout_array, ldtout);
}
//===================================================================================================




//===================================================================================================
//===================================================================================================
static __device__ void 
dlarft_recdtrmv_sm32x32_device(
    int m, int n, double *tau,
    double *Trec, int ldtrec, double *Ttri, int ldttri)
{
    int tx = threadIdx.x; 
    double *sdata = (double*)shared_data;
    double res;

    // to update a certain column i, threads go in horizontal fashion where
    // every thread read one row and do it gemv(dot) to generate 
    // one element of the column of T then move to the next column

    // read T into shared
    for(int s=0; s<n; s++)
    {
        sdata[tx + s*n] = Trec[tx + s * ldtrec];
    }
    __syncthreads();  
    
    // perform sequence of n-1 gemv
    for(int i=0; i<n;i++)
    {
       res = MAGMA_D_ZERO;
       for(int j=0; j<i; j++)
       {
           res += sdata[tx + j * n] * Ttri[j+ i * ldttri];      
       }
       __syncthreads();   // a enlever
       sdata[tx + i * n] = -tau[i] * (sdata[tx + i * n] + res);
       __syncthreads();  
    } 

    // write back the updated block of k column of T  multiplying by -tau
    for(int s=0; s<n; s++)
    {
       Trec[tx + s * ldtrec] = sdata[tx + s*n];
    }

}

//===================================================================================================
__global__ void 
dlarft_recdtrmv_sm32x32_kernel(
    int m, int n, double *tau,
    double *Trec, int ldtrec, double *Ttri, int ldttri)
{
    dlarft_recdtrmv_sm32x32_device(m, n, tau, Trec, ldtrec, Ttri, ldttri);
}
//===================================================================================================
__global__ void 
dlarft_recdtrmv_sm32x32_kernel_batched(
    int m, int n, double **tau_array,
    double **Trec_array, int ldtrec, double **Ttri_array, int ldttri)
{
    int batchId = blockIdx.z;
    dlarft_recdtrmv_sm32x32_device(m, n, tau_array[batchId], Trec_array[batchId], ldtrec, Ttri_array[batchId], ldttri);
}
//===================================================================================================
extern "C"
void magmablas_dlarft_recdtrmv_sm32x32(
    magma_int_t m, magma_int_t n, 
    double *tau, 
    double *Trec, magma_int_t ldtrec, 
    double *Ttri, magma_int_t ldttri)
{

    dim3 grid(1);
    dim3 threads(max(m,1), 1, 1);
    dlarft_recdtrmv_sm32x32_kernel <<< grid, threads, sizeof(double)*(m*n), magma_stream >>> (m, n,  tau, Trec, ldtrec, Ttri, ldttri);
}
//===================================================================================================
extern "C"
void magmablas_dlarft_recdtrmv_sm32x32_batched(
    magma_int_t m, magma_int_t n, 
    double **tau_array, 
    double **Trec_array, magma_int_t ldtrec, 
    double **Ttri_array, magma_int_t ldttri,
    magma_int_t batchCount, magma_queue_t queue)
{

    dim3 grid(1, 1, batchCount);
    dim3 threads(max(m,1), 1, 1);
    dlarft_recdtrmv_sm32x32_kernel_batched <<< grid, threads, sizeof(double)*(m*n), queue >>> (m, n,  tau_array, Trec_array, ldtrec, Ttri_array, ldttri);
}
//===================================================================================================


