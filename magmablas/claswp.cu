#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.3.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2012

       @generated c Wed Nov 14 22:53:48 2012
       
       @author Mathieu Faverge
       @author Ichitaro Yamazaki
       @author Mark Gates
*/
#include "common_magma.h"

// MAX_PIVOTS is maximum number of pivots to apply in each kernel launch
// NTHREADS is number of threads in a block
#define MAX_PIVOTS 32
#define NTHREADS   64

typedef struct {
    hipFloatComplex *dAT;
    int n, lda, j0, npivots;
    int ipiv[MAX_PIVOTS];
} claswp_params_t;


// Matrix A is stored row-wise in dAT.
// Divide matrix A into block-columns of NTHREADS columns each.
// Each GPU block processes one block-column of A.
// Each thread goes down a column of A,
// swapping rows according to pivots stored in params.
__global__ void claswp_kernel( claswp_params_t params )
{
    unsigned int tid = threadIdx.x + blockDim.x*blockIdx.x;
    if( tid < params.n ) {
        int lda = params.lda;
        hipFloatComplex *dAT = params.dAT + tid + params.j0*lda;
        hipFloatComplex *A1  = dAT;
        
        for( int i1 = 0; i1 < params.npivots; ++i1 ) {
            int i2 = params.ipiv[i1];
            hipFloatComplex *A2 = dAT + i2*lda;
            hipFloatComplex temp = *A1;
            *A1 = *A2;
            *A2 = temp;
            A1 += lda;  // A1 = dA + i1*ldx
        }
    }
}


// Launch claswp kernel with ceil( n / NTHREADS ) blocks of NTHREADS threads each.
extern "C" void claswp( claswp_params_t &params )
{
    int blocks = (params.n + NTHREADS - 1) / NTHREADS;
    claswp_kernel<<< blocks, NTHREADS, 0, magma_stream >>>( params );
}


// Swap rows of A, stored row-wise.
// This version updates each entry of ipiv by adding ind.
// It is used in cgetrf, cgetrf_gpu, cgetrf_mgpu, cgetrf_ooc.
extern "C" void
magmablas_cpermute_long2( magma_int_t n, hipFloatComplex *dAT, magma_int_t lda,
                          magma_int_t *ipiv, magma_int_t nb, magma_int_t ind )
{
    for( int k = 0; k < nb; k += MAX_PIVOTS ) {
        int npivots = min( MAX_PIVOTS, nb-k );
        // fields are:             dAT  n  lda  j0       npivots
        claswp_params_t params = { dAT, n, lda, ind + k, npivots };
        for( int j = 0; j < npivots; ++j ) {
            params.ipiv[j] = ipiv[ind + k + j] - k - 1;
            ipiv[ind + k + j] += ind;
        }
        claswp( params );
    }
}


// Swap rows of A, stored row-wise.
// This version assumes ind has already been added to ipiv.
// It is used in cgetrf_mgpu, cgetrf_ooc.
extern "C" void
magmablas_cpermute_long3( hipFloatComplex *dAT, magma_int_t lda,
                          const magma_int_t *ipiv, magma_int_t nb, magma_int_t ind )
{
    for( int k = 0; k < nb; k += MAX_PIVOTS ) {
        int npivots = min( MAX_PIVOTS, nb-k );
        // fields are:             dAT  n    lda  j0       npivots
        claswp_params_t params = { dAT, lda, lda, ind + k, npivots };
        for( int j = 0; j < MAX_PIVOTS; ++j ) {
            params.ipiv[j] = ipiv[ind + k + j] - k - 1 - ind;
        }
        claswp( params );
    }
}


// Swap rows of A, stored row-wise.
// This interface is identical to LAPACK's laswp interface.
// It is used in cgessm, cgetrf_incpiv.
extern "C" void
magmablas_claswp( magma_int_t n, hipFloatComplex *dAT, magma_int_t lda,
                  magma_int_t i1, magma_int_t i2,
                  const magma_int_t *ipiv, magma_int_t inci )
{
    for( int k = i1-1; k < i2; k += MAX_PIVOTS ) {
        int npivots = min( MAX_PIVOTS, i2-k );
        // fields are:             dAT        n  lda  j0 npivots
        claswp_params_t params = { dAT+k*lda, n, lda, 0, npivots };
        for( int j = 0; j < npivots; ++j ) {
            params.ipiv[j] = ipiv[(k+j)*inci] - k - 1;
        }
        claswp( params );
    }
}


// ------------------------------------------------------------
// Extended version has stride in both directions (ldx, ldy)
// to handle both row-wise and column-wise storage.

typedef struct {
    hipFloatComplex *dA;
    int n, ldx, ldy, j0, npivots;
    int ipiv[MAX_PIVOTS];
} claswpx_params_t;


// Matrix A is stored row-wise in dA.
// Divide matrix A into block-columns of NTHREADS columns each.
// Each GPU block processes one block-column of A.
// Each thread goes down a column of A,
// swapping rows according to pivots stored in params.
__global__ void claswpx_kernel( claswpx_params_t params )
{
    unsigned int tid = threadIdx.x + blockDim.x*blockIdx.x;
    if( tid < params.n ) {
        int ldx = params.ldx;
        hipFloatComplex *dA = params.dA + tid*params.ldy + params.j0*ldx;
        hipFloatComplex *A1  = dA;
        
        for( int i1 = 0; i1 < params.npivots; ++i1 ) {
            int i2 = params.ipiv[i1];
            hipFloatComplex *A2 = dA + i2*ldx;
            hipFloatComplex temp = *A1;
            *A1 = *A2;
            *A2 = temp;
            A1 += ldx;  // A1 = dA + i1*ldx
        }
    }
}


// Launch claswpx kernel with ceil( n / NTHREADS ) blocks of NTHREADS threads each.
extern "C" void claswpx( claswpx_params_t &params )
{
    int blocks = (params.n + NTHREADS - 1) / NTHREADS;
    claswpx_kernel<<< blocks, NTHREADS, 0, magma_stream >>>( params );
}


// Swap rows of A.
// For A stored row-wise,    set ldx=lda and ldy=1.
// For A stored column-wise, set ldx=1   and ldy=lda.
// Otherwise, this interface is identical to LAPACK's laswp interface.
extern "C" void
magmablas_claswpx( magma_int_t n, hipFloatComplex *dA, magma_int_t ldx, magma_int_t ldy,
                   magma_int_t i1, magma_int_t i2,
                   const magma_int_t *ipiv, magma_int_t inci )
{
    for( int k = i1-1; k < i2; k += MAX_PIVOTS ) {
        int npivots = min( MAX_PIVOTS, i2-k );
        // fields are:              dA        n  ldx  ldy  j0 npivots
        claswpx_params_t params = { dA+k*ldx, n, ldx, ldy, 0, npivots };
        for( int j = 0; j < npivots; ++j ) {
            params.ipiv[j] = ipiv[(k+j)*inci] - k - 1;
        }
        claswpx( params );
    }
}


// ------------------------------------------------------------
// This version takes d_ipiv on the GPU. Thus it does not pass pivots
// as an argument using a structure, avoiding all the argument size
// limitations of CUDA and OpenCL. It also needs just one kernel launch
// with all the pivots, instead of multiple kernel launches with small
// batches of pivots. On Fermi, it is faster than magmablas_claswp
// (including copying pivots to the GPU).

__global__ void claswp2_kernel( int n, hipFloatComplex *dAT, int lda, int npivots, const magma_int_t* d_ipiv )
{
    unsigned int tid = threadIdx.x + blockDim.x*blockIdx.x;
    if( tid < n ) {
        dAT += tid;
        hipFloatComplex *A1  = dAT;
        
        for( int i1 = 0; i1 < npivots; ++i1 ) {
            int i2 = d_ipiv[i1] - 1;  // Fortran index
            hipFloatComplex *A2 = dAT + i2*lda;
            hipFloatComplex temp = *A1;
            *A1 = *A2;
            *A2 = temp;
            A1 += lda;  // A1 = dA + i1*ldx
        }
    }
}

// Swap rows of A, stored row-wise.
// d_ipiv is vector of pivots stored on the GPU,
// unlike magmablas_claswp where ipiv is stored on the CPU.
// This interface is identical to LAPACK's laswp interface.
extern "C" void
magmablas_claswp2( magma_int_t n, hipFloatComplex* dAT, magma_int_t lda,
                   magma_int_t i1, magma_int_t i2,
                   const magma_int_t *d_ipiv )
{
    int blocks = (n + NTHREADS - 1) / NTHREADS;
    claswp2_kernel<<< blocks, NTHREADS, 0, magma_stream >>>(
        n, dAT + (i1-1)*lda, lda, i2-(i1-1), d_ipiv );
}
