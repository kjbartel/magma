#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       May 2012

       @generated c Tue May 15 18:18:02 2012

*/
#include "common_magma.h"

#define BLOCK_SIZE 64

/*********************************************************/
/*
*  Blocked version: swap several pair of line
 */
typedef struct {
    hipFloatComplex *A1;
    hipFloatComplex *A2;
    int n, lda1, lda2, npivots;
    short ipiv[BLOCK_SIZE];
} magmagpu_cswapblk_params_t;

__global__ void magmagpu_cswapblkrm( magmagpu_cswapblk_params_t params )
{
    unsigned int y = threadIdx.x + blockDim.x*blockIdx.x;
    if( y < params.n )
    {
        hipFloatComplex *A1 = params.A1 + y - params.lda1;
        hipFloatComplex *A2 = params.A2 + y;
      
        for( int i = 0; i < params.npivots; i++ )
        {
            A1 += params.lda1;
            if ( params.ipiv[i] == -1 )
                continue;
            hipFloatComplex tmp1  = *A1;
            hipFloatComplex *tmp2 = A2 + params.ipiv[i]*params.lda2;
            *A1   = *tmp2;
            *tmp2 = tmp1;
        }
    }
}

__global__ void magmagpu_cswapblkcm( magmagpu_cswapblk_params_t params )
{
    unsigned int y = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned int offset1 = __mul24( y, params.lda1);
    unsigned int offset2 = __mul24( y, params.lda2);
    if( y < params.n )
    {
        hipFloatComplex *A1 = params.A1 + offset1 - 1;
        hipFloatComplex *A2 = params.A2 + offset2;
      
        for( int i = 0; i < params.npivots; i++ )
        {
            A1++;
            if ( params.ipiv[i] == -1 )
                continue;
            hipFloatComplex tmp1  = *A1;
            hipFloatComplex *tmp2 = A2 + params.ipiv[i];
            *A1   = *tmp2;
            *tmp2 = tmp1;
        }
    }
    __syncthreads();
}

extern "C" void 
magmablas_cswapblk( char storev, int n, 
                    hipFloatComplex *dA1T, int lda1,
                    hipFloatComplex *dA2T, int lda2,
                    int i1, int i2, int *ipiv, int inci, int offset )
{
    int  blocksize = 64;
    dim3 blocks( (n+blocksize-1) / blocksize, 1, 1);
    int  k, im;

    if ( (storev == 'C') || (storev == 'c') ) {
        for( k=(i1-1); k<i2; k+=BLOCK_SIZE )
        {
            int sb = min(BLOCK_SIZE, i2-k);
            magmagpu_cswapblk_params_t params = { dA1T+k, dA2T, n, lda1, lda2, sb };
            for( int j = 0; j < sb; j++ )
            {
                im = ipiv[(k+j)*inci] - 1;
                if ( (k+j) == im)
                    params.ipiv[j] = -1;
                else
                    params.ipiv[j] = im - offset;
            }
            magmagpu_cswapblkcm<<< blocks, blocksize, 0, magma_stream >>>( params );
        }
    }else {
        for( k=(i1-1); k<i2; k+=BLOCK_SIZE )
        {
            int sb = min(BLOCK_SIZE, i2-k);
            magmagpu_cswapblk_params_t params = { dA1T+k*lda1, dA2T, n, lda1, lda2, sb };
            for( int j = 0; j < sb; j++ )
            {
                im = ipiv[(k+j)*inci] - 1;
                if ( (k+j) == im)
                    params.ipiv[j] = -1;
                else
                    params.ipiv[j] = im - offset;
            }
            magmagpu_cswapblkrm<<< blocks, blocksize, 0, magma_stream >>>( params );
        }
    }
}

