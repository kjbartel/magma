#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.3.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

       @generated c Wed Nov 14 22:53:54 2012
       @author Ichitaro Yamazaki
*/
#include "common_magma.h"
#define PRECISION_c
#include "commonblas.h"

//
//    m, n - dimensions in the output (ha) matrix.
//             This routine copies the dat matrix from the GPU
//             to ha on the CPU. In addition, the output matrix
//             is transposed. The routine uses a buffer of size
//             2*lddb*nb pointed to by dB (lddb > m) on the GPU. 
//             Note that lda >= m and lddat >= n.
//
extern "C" void 
magmablas_cgetmatrix_transpose_mgpu(
                  magma_int_t ngpus, hipStream_t stream[][2],
                  hipFloatComplex **dat, magma_int_t ldda,
                  hipFloatComplex   *ha, magma_int_t lda,
                  hipFloatComplex  **db, magma_int_t lddb,
                  magma_int_t m, magma_int_t n, magma_int_t nb)
{
#define   A(j)     (ha  + (j)*lda)
#define  dB(d, j)  (db[(d)]  + (j)*nb*lddb)
#define  dAT(d, j) (dat[(d)] + (j)*nb)
    int nstreams = 2, j, j_local, d, id, ib;

    /* Quick return */
    if ( (m == 0) || (n == 0) )
        return;

    if (lda < m || ngpus*ldda < n || lddb < m){
        printf( "Wrong arguments in magmablas_cgetmatrix_transpose_mgpu (%d<%d), (%d*%d<%d), or (%d<%d).\n",
                lda, m, ngpus, ldda, n, lddb, m );
        return;
    }
    
    /* Move data from GPU to CPU using two buffers; first transpose the data on the GPU */
    for(j=0; j<n; j+=nb){
       d       = (j/nb)%ngpus;
       j_local = (j/nb)/ngpus;
       id      = j_local%nstreams;
       magma_setdevice(d);

       ib = min(n-j, nb);
       magmablasSetKernelStream(stream[d][id]);
       magmablas_ctranspose2(  dB(d, id),      lddb, 
                              dAT(d, j_local), ldda, 
                              ib, m);
       magma_cgetmatrix_async( m, ib,
                               dB(d, id), lddb,
                               A(j),      lda, 
                               stream[d][id] );
    }
}



