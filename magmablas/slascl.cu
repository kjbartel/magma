#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.4.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       December 2013

       @generated s Tue Dec 17 13:18:45 2013

*/
#include "common_magma.h"

#define slascl_bs 64


__global__ void
l_slascl (int m, int n, float mul, float* A, int lda){
    int ind =  blockIdx.x * slascl_bs + threadIdx.x ;

    int break_d = (ind < n)? ind: n-1;

    A += ind;
    if (ind < m)
       for(int j=0; j<=break_d; j++ )
           A[j*lda] *= mul;
}

__global__ void
u_slascl (int m, int n, float mul, float* A, int lda){
    int ind =  blockIdx.x * slascl_bs + threadIdx.x ;

    A += ind;
    if (ind < m)
      for(int j=n-1; j>= ind; j--)
         A[j*lda] *= mul;
}


extern "C" void
magmablas_slascl(char type, magma_int_t kl, magma_int_t ku, 
                 float cfrom, float cto,
                 magma_int_t m, magma_int_t n, 
                 float *A, magma_int_t lda, magma_int_t *info )
{
    int blocks;
    if (m % slascl_bs==0)
        blocks = m/ slascl_bs;
    else
        blocks = m/ slascl_bs + 1;

    dim3 grid(blocks, 1, 1);
    dim3 threads(slascl_bs, 1, 1);

    /* To do : implment the accuracy procedure */
    float mul = cto / cfrom;

    if (type == 'L' || type =='l')  
       l_slascl <<< grid, threads, 0, magma_stream >>> (m, n, mul, A, lda);
    else if (type == 'U' || type =='u')
       u_slascl <<< grid, threads, 0, magma_stream >>> (m, n, mul, A, lda);  
    else {
       printf("Only type L and U are available in slascl. Exit.\n");
       exit(1);
    }
}


