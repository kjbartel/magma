#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @generated from zswapdblk.cu normal z -> d, Fri Jan 30 19:00:09 2015

*/
#include "common_magma.h"


/*********************************************************/
/*
 *  Swap diagonal blocks of two matrices.
 *  Each thread block swaps one diagonal block.
 *  Each thread iterates across one row of the block.
 */

__global__ void 
dswapdblk_kernel( int nb,
                  double *dA, int ldda, int inca,
                  double *dB, int lddb, int incb )
{
    const int tx = threadIdx.x;
    const int bx = blockIdx.x;

    dA += tx + bx * nb * (ldda + inca);
    dB += tx + bx * nb * (lddb + incb);

    double tmp;

    #pragma unroll
    for( int i = 0; i < nb; i++ ){
        tmp        = dA[i*ldda];
        dA[i*ldda] = dB[i*lddb];
        dB[i*lddb] = tmp;
    }
}


/**
    Purpose
    -------
    dswapdblk swaps diagonal blocks of size nb x nb between matrices
    dA and dB on the GPU. It swaps nblocks = n/nb blocks.
    For i = 1 .. nblocks, submatrices
    dA( i*nb*inca, i*nb ) and
    dB( i*nb*incb, i*nb ) are swapped.
    
    Arguments
    ---------
    @param[in]
    n       INTEGER
            The number of columns of the matrices dA and dB.  N >= 0.

    @param[in]
    nb      INTEGER
            The size of diagonal blocks.
            NB > 0 and NB <= maximum threads per CUDA block (512 or 1024).

    @param[in,out]
    dA      DOUBLE_PRECISION array, dimension (LDDA,N)
            The matrix dA.

    @param[in]
    ldda    INTEGER
            The leading dimension of the array dA.
            LDDA >= (nblocks - 1)*nb*inca + nb.

    @param[in]
    inca    INTEGER
            The row increment between diagonal blocks of dA. inca >= 0. For example,
            inca = 1 means blocks are stored on the diagonal at dA(i*nb, i*nb),
            inca = 0 means blocks are stored side-by-side    at dA(0,    i*nb).

    @param[in,out]
    dB      DOUBLE_PRECISION array, dimension (LDDB,N)
            The matrix dB.

    @param[in]
    lddb    INTEGER
            The leading dimension of the array db.
            LDDB >= (nblocks - 1)*nb*incb + nb.

    @param[in]
    incb    INTEGER
            The row increment between diagonal blocks of dB. incb >= 0. See inca.
    
    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_daux2
    ********************************************************************/
extern "C" void 
magmablas_dswapdblk_q(
    magma_int_t n, magma_int_t nb,
    magmaDouble_ptr dA, magma_int_t ldda, magma_int_t inca,
    magmaDouble_ptr dB, magma_int_t lddb, magma_int_t incb,
    magma_queue_t queue )
{
    magma_int_t nblocks = n / nb;
    
    magma_int_t info = 0;
    if (n < 0) {
        info = -1;
    } else if (nb < 1 || nb > 1024) {
        info = -2;
    } else if (ldda < (nblocks-1)*nb*inca + nb) {
        info = -4;
    } else if (inca < 0) {
        info = -5;
    } else if (lddb < (nblocks-1)*nb*incb + nb) {
        info = -7;
    } else if (incb < 0) {
        info = -8;
    }

    if (info != 0) {
        magma_xerbla( __func__, -(info) );
        return;  //info;
    }

    if ( nblocks > 0 ) {
        dswapdblk_kernel<<< nblocks, nb, 0, queue >>>
            ( nb, dA, ldda, inca,
                  dB, lddb, incb );
    }
}


/**
    @see magmablas_dswapdblk_q
    @ingroup magma_daux2
    ********************************************************************/
extern "C" void 
magmablas_dswapdblk(
    magma_int_t n, magma_int_t nb,
    magmaDouble_ptr dA, magma_int_t ldda, magma_int_t inca,
    magmaDouble_ptr dB, magma_int_t lddb, magma_int_t incb )
{
    magmablas_dswapdblk_q( n, nb, dA, ldda, inca, dB, lddb, incb, magma_stream );
}
