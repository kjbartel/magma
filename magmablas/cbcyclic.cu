#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

       @generated c Sun Nov 13 20:48:41 2011

*/
#include "common_magma.h"
#define PRECISION_c
#include "commonblas.h"

//===========================================================================
//  Set a matrix from CPU to multi-GPUs is 1D block cyclic distribution. 
//  The da arrays are pointers to the matrix data for the corresponding GPUs. 
//===========================================================================
extern "C" void 
magmablas_csetmatrix_1D_bcyclic( int m, int n,
                                 hipFloatComplex  *ha, int lda, 
                                 hipFloatComplex  *da[], int ldda, 
                                 int num_gpus, int nb )
{
    int i, k, nk, cdevice;

    hipGetDevice(&cdevice);

    for(i=0; i<n; i+=nb){
       k = (i/nb)%num_gpus;
       hipSetDevice(k);
         
       nk = min(nb, n-i);
       //hipblasSetMatrix( m, nk, sizeof(hipFloatComplex), ha+i*lda, lda,
       //                 da[k]+i/(nb*num_gpus)*nb*ldda, ldda);
       hipMemcpy2DAsync(da[k]+i/(nb*num_gpus)*nb*ldda, ldda*sizeof(hipFloatComplex),
                         ha + i*lda, lda*sizeof(hipFloatComplex),
                         sizeof(hipFloatComplex)*m, nk,
                         hipMemcpyHostToDevice, NULL);
    }

    hipSetDevice(cdevice);
}


//===========================================================================
//  Get a matrix with 1D block cyclic distribution on multiGPUs to the CPU.
//  The da arrays are pointers to the matrix data for the corresponding GPUs.
//===========================================================================
extern "C" void
magmablas_cgetmatrix_1D_bcyclic( int m, int n,
                                 hipFloatComplex  *da[], int ldda,
                                 hipFloatComplex  *ha, int lda,
                                 int num_gpus, int nb )
{
    int i, k, nk, cdevice;

    hipGetDevice(&cdevice);

    for(i=0; i<n; i+=nb){
       k = (i/nb)%num_gpus;
       hipSetDevice(k);

       nk = min(nb, n-i);
       //hipblasGetMatrix( m, nk, sizeof(hipFloatComplex),
       //                 da[k]+i/(nb*num_gpus)*nb*ldda, ldda,
       //                 ha+i*lda, lda);
       hipMemcpy2DAsync(ha + i*lda, lda*sizeof(hipFloatComplex),
                         da[k]+i/(nb*num_gpus)*nb*ldda, ldda*sizeof(hipFloatComplex),
                         sizeof(hipFloatComplex)*m, nk,
                         hipMemcpyDeviceToHost, NULL);
    }
        
    hipSetDevice(cdevice);
}

