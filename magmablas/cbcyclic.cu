#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.3.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2012

       @author Stan Tomov
       @generated c Wed Nov 14 22:53:53 2012
*/
#include "common_magma.h"
#define PRECISION_c
#include "commonblas.h"

//===========================================================================
//  Set a matrix from CPU to multi-GPUs is 1D block cyclic distribution.
//  The dA arrays are pointers to the matrix data for the corresponding GPUs.
//===========================================================================
extern "C" void
magmablas_csetmatrix_1D_bcyclic( magma_int_t m, magma_int_t n,
                                 const hipFloatComplex *hA,   magma_int_t lda,
                                 hipFloatComplex       *dA[], magma_int_t ldda,
                                 magma_int_t num_gpus, magma_int_t nb )
{
    magma_int_t i, d, nk;
    magma_device_t cdevice;

    magma_getdevice( &cdevice );

    for( i = 0; i < n; i += nb ) {
        d = (i/nb) % num_gpus;
        magma_setdevice( d );
        nk = min(nb, n-i);
        magma_csetmatrix_async( m, nk,
                                hA + i*lda, lda,
                                dA[d] + i/(nb*num_gpus)*nb*ldda, ldda, NULL );
    }

    magma_setdevice( cdevice );
}


//===========================================================================
//  Get a matrix with 1D block cyclic distribution on multiGPUs to the CPU.
//  The dA arrays are pointers to the matrix data for the corresponding GPUs.
//===========================================================================
extern "C" void
magmablas_cgetmatrix_1D_bcyclic( magma_int_t m, magma_int_t n,
                                 hipFloatComplex  *dA[], magma_int_t ldda,
                                 hipFloatComplex  *hA,   magma_int_t lda,
                                 magma_int_t num_gpus, magma_int_t nb )
{
    magma_int_t i, d, nk;
    magma_device_t cdevice;

    magma_getdevice( &cdevice );

    for( i = 0; i < n; i += nb ) {
        d = (i/nb) % num_gpus;
        magma_setdevice( d );
        nk = min(nb, n-i);
        magma_cgetmatrix_async( m, nk,
                                dA[d] + i/(nb*num_gpus)*nb*ldda, ldda,
                                hA + i*lda, lda, NULL );
    }

    magma_setdevice( cdevice );
}
