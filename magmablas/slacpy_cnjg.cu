#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @generated from zlacpy_cnjg.cu normal z -> s, Fri Jan 30 19:00:08 2015

*/
#include "common_magma.h"

#define BLOCK_SIZE 64

/*********************************************************
 *
 * SWAP BLAS: permute to set of N elements
 *
 ********************************************************/
/*
 *  First version: line per line
 */
typedef struct {
    float *A1;
    float *A2;
    int n, lda1, lda2;
} magmagpu_slacpy_cnjg_params_t;

__global__ void magmagpu_slacpy_cnjg( magmagpu_slacpy_cnjg_params_t params )
{
    unsigned int x = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned int offset1 = x*params.lda1;
    unsigned int offset2 = x*params.lda2;
    if( x < params.n )
    {
        float *A1  = params.A1 + offset1;
        float *A2  = params.A2 + offset2;
        *A2 = MAGMA_S_CNJG(*A1);
    }
}


extern "C" void 
magmablas_slacpy_cnjg_q(
    magma_int_t n, float *dA1, magma_int_t lda1, 
    float *dA2, magma_int_t lda2,
    magma_queue_t queue )
{
    int blocksize = 64;
    dim3 blocks( (n+blocksize-1) / blocksize, 1, 1);
    magmagpu_slacpy_cnjg_params_t params = { dA1, dA2, n, lda1, lda2 };
    magmagpu_slacpy_cnjg<<< blocks, blocksize, 0, queue >>>( params );
}


extern "C" void 
magmablas_slacpy_cnjg(
    magma_int_t n, float *dA1, magma_int_t lda1, 
    float *dA2, magma_int_t lda2)
{
    magmablas_slacpy_cnjg_q( n, dA1, lda1, dA2, lda2, magma_stream );
}
