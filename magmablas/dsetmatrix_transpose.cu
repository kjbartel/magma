#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       May 2012

       @generated d Tue May 15 18:18:06 2012

*/
#include "common_magma.h"
#define PRECISION_d
#include "commonblas.h"

extern "C" void
magmablas_dtranspose2s(double *odata, int ldo,
                       double *idata, int ldi,
                       int m, int n, hipStream_t *stream );


//
//      m, n - dimensions in the source (input) matrix.
//             This routine copies the ha matrix from the CPU
//             to dat on the GPU. In addition, the output matrix
//             is transposed. The routine uses a buffer of size
//             2*lddb*nb pointed to by dB (lddb > m) on the GPU. 
//             Note that lda >= m and lddat >= n.
//
extern "C" void 
magmablas_dsetmatrix_transpose( int m, int n,
                                double  *ha, int lda, 
                                double *dat, int ldda,
                                double  *dB, int lddb, int nb )
{
    int i = 0, j = 0, ib;

    /* Quick return */
    if ( (m == 0) || (n == 0) )
        return;

    if (lda < m || ldda < n || lddb < m){
        printf("Wrong arguments in zhtodt.\n");
        return;
    }

    static hipStream_t stream[2];
    magma_queue_create( &stream[0] );
    magma_queue_create( &stream[1] );
   
    /* Move data from CPU to GPU in the first panel in the dB buffer */
    ib   = min(n-i, nb);
    magma_dsetmatrix_async( m, ib,
                            ha + i*lda,             lda,
                            dB + (j%2) * nb * lddb, lddb, stream[j%2] );
    j++;

    for(i=nb; i<n; i+=nb){
       /* Move data from CPU to GPU in the second panel in the dB buffer */
       ib   = min(n-i, nb);
       magma_dsetmatrix_async( m, ib,
                               ha+i*lda,               lda,
                               dB + (j%2) * nb * lddb, lddb, stream[j%2] );
       j++;
  
       /* Note that the previous panel (i.e., j%2) comes through the stream
          for the kernel so there is no need to synchronize.             */
       // magmablas_dtranspose2( dat+i-nb, ldda, dB + (j%2)*nb*lddb, lddb, m, nb);
       magmablas_dtranspose2s( dat+i-nb, ldda, dB + (j%2)*nb*lddb, lddb, m, nb, &stream[j%2]);
    }

    /* Transpose the last part of the matrix.                            */
    j++;
    // magmablas_dtranspose2( dat+i-nb, ldda, dB + (j%2)*nb*lddb, lddb, m, ib);
    magmablas_dtranspose2s( dat+i-nb, ldda, dB + (j%2)*nb*lddb, lddb, m, ib, &stream[j%2]);

    magma_queue_destroy( stream[0] );
    magma_queue_destroy( stream[1] );
}

//===========================================================================
//  This version is similar to the above but for multiGPUs. The distribution
//  is 1D block cyclic. The input arrays are pointers for the corresponding 
//  GPUs. The streams are passed as argument, in contrast to the single GPU
//  routine.
//===========================================================================
extern "C" void 
magmablas_dsetmatrix_transpose2( int m, int n,
                                 double  *ha,  int  lda, 
                                 double **dat, int *ldda,
                                 double **dB,  int  lddb, int nb,
                                 int num_gpus, hipStream_t stream[][2] )
{
    int i = 0, j[4] = {0, 0, 0, 0}, ib, k = 0;

    /* Quick return */
    if ( (m == 0) || (n == 0) )
        return;

    if (lda < m || lddb < m){
        printf("Wrong arguments in zhtodt2.\n");
        return;
    }

    if (n<num_gpus*nb){
       for(i=0; i<n; i+=nb){
          k = (i/nb)%num_gpus;
          magma_setdevice(k);

          ib = min(n-i, nb);
          magma_dsetmatrix_async( m, ib,
                                  ha+i*lda, lda,
                                  dB[k],    lddb, stream[k][0] );
       }
       for(i=0; i<n; i+=nb){
          k = (i/nb)%num_gpus;
          magma_setdevice(k);

          ib = min(n-i, nb);
          //magma_queue_sync( stream[k][0] );
          //magmablas_dtranspose2( dat[k]+ i/(nb*num_gpus)*nb, ldda[k],
          //                       dB[k], lddb, m, ib);
          magmablas_dtranspose2s( dat[k]+ i/(nb*num_gpus)*nb, ldda[k],
                                 dB[k], lddb, m, ib, &stream[k][0]);
       }
    } 
    else
    {
      for(i=0; i<(n + num_gpus*nb); i+=nb){
         k = (i/nb)%num_gpus;
         magma_setdevice(k);

         if (i<n){
            /* Move data from CPU to GPU in the second panel in the dB buffer */
            ib = min(n-i, nb);
            magma_dsetmatrix_async( m, ib,
                                    ha+i*lda,                 lda,
                                    dB[k] + (j[k]%2)*nb*lddb, lddb, stream[k][j[k]%2] );
         }
         j[k]++;
  
         if (i> (num_gpus-1)*nb){
            /* Make sure that the previous panel (i.e., j[k]%2) has arrived 
               and transpose it directly into the dat matrix                  */
            //magma_queue_sync( stream[k][ j[k]%2 ] );
            ib = min(n - i + num_gpus*nb, nb);
            //magmablas_dtranspose2( dat[k]+ i/(nb*num_gpus)*nb -nb, ldda[k],
            //                       dB[k] +(j[k]%2)*nb*lddb, lddb, m, ib);
            magmablas_dtranspose2s( dat[k]+ i/(nb*num_gpus)*nb -nb, ldda[k],
                                   dB[k] +(j[k]%2)*nb*lddb, lddb, m, ib, &stream[k][j[k]%2]);

         }
      }
    }
}
