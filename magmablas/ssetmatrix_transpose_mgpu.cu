#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.4.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       December 2013

       @generated s Tue Dec 17 13:18:45 2013
       @author Ichitaro Yamazaki
*/
#include "common_magma.h"
#define PRECISION_s
#include "commonblas.h"

//
//    m, n - dimensions in the source (input) matrix.
//             This routine copies the ha matrix from the CPU
//             to dat on the GPU. In addition, the output matrix
//             is transposed. The routine uses a buffer of size
//             2*lddb*nb pointed to by dB (lddb > m) on the GPU. 
//             Note that lda >= m and lddat >= n.
//
extern "C" void 
magmablas_ssetmatrix_transpose_mgpu(
                  magma_int_t ngpus, magma_queue_t stream[][2],
                  const float *ha,  magma_int_t lda, 
                  float       *dat[], magma_int_t ldda, 
                  float       *db[],  magma_int_t lddb,
                  magma_int_t m, magma_int_t n, magma_int_t nb)
{
#define   A(j)    (ha       + (j)*lda)
#define  dB(d, j) (db[(d)]  + (j)*nb*lddb)
#define dAT(d, j) (dat[(d)] + (j)*nb)
    magma_int_t nstreams = 2, d, j, j_local, id, ib;

    /* Quick return */
    if ( (m == 0) || (n == 0) )
        return;

    if (lda < m || ngpus*ldda < n || lddb < m){
        printf( "Wrong arguments in magmablas_ssetmatrix_transpose_mgpu (%d<%d), (%d*%d<%d), or (%d<%d).\n",
                (int) lda, (int) m, (int) ngpus, (int) ldda, (int) n, (int) lddb, (int) m );
        return;
    }
    
    /* Move data from CPU to GPU by block columns and transpose it */
    for(j=0; j<n; j+=nb){
       d       = (j/nb)%ngpus;
       j_local = (j/nb)/ngpus;
       id      = j_local%nstreams;
       magma_setdevice(d);

       ib = min(n-j, nb);
       magma_ssetmatrix_async( m, ib,
                               A(j),      lda,
                               dB(d, id), lddb, 
                               stream[d][id] );

       magmablasSetKernelStream(stream[d][id]);
       magmablas_stranspose2(dAT(d, j_local), ldda, 
                             dB(d, id),       lddb, 
                             m, ib);
    }
}

