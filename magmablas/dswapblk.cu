#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.5.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date September 2014

       @generated from zswapblk.cu normal z -> d, Wed Sep 17 15:08:23 2014

*/
#include "common_magma.h"

#define BLOCK_SIZE 64

/*********************************************************/
/*
 *  Blocked version: swap several pairs of lines
 */
typedef struct {
    double *A1;
    double *A2;
    int n, lda1, lda2, npivots;
    short ipiv[BLOCK_SIZE];
} magmagpu_dswapblk_params_t;

__global__ void magmagpu_dswapblkrm( magmagpu_dswapblk_params_t params )
{
    unsigned int y = threadIdx.x + blockDim.x*blockIdx.x;
    if( y < params.n )
    {
        double *A1 = params.A1 + y - params.lda1;
        double *A2 = params.A2 + y;
      
        for( int i = 0; i < params.npivots; i++ )
        {
            A1 += params.lda1;
            if ( params.ipiv[i] == -1 )
                continue;
            double tmp1  = *A1;
            double *tmp2 = A2 + params.ipiv[i]*params.lda2;
            *A1   = *tmp2;
            *tmp2 = tmp1;
        }
    }
}

__global__ void magmagpu_dswapblkcm( magmagpu_dswapblk_params_t params )
{
    unsigned int y = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned int offset1 = y*params.lda1;
    unsigned int offset2 = y*params.lda2;
    if( y < params.n )
    {
        double *A1 = params.A1 + offset1 - 1;
        double *A2 = params.A2 + offset2;
      
        for( int i = 0; i < params.npivots; i++ )
        {
            A1++;
            if ( params.ipiv[i] == -1 )
                continue;
            double tmp1  = *A1;
            double *tmp2 = A2 + params.ipiv[i];
            *A1   = *tmp2;
            *tmp2 = tmp1;
        }
    }
    __syncthreads();
}


/**
    @ingroup magma_dblas2
    ********************************************************************/
extern "C" void 
magmablas_dswapblk_q(
    magma_order_t order, magma_int_t n, 
    double *dA1T, magma_int_t lda1,
    double *dA2T, magma_int_t lda2,
    magma_int_t i1, magma_int_t i2,
    const magma_int_t *ipiv, magma_int_t inci, magma_int_t offset,
    magma_queue_t queue )
{
    magma_int_t  blocksize = 64;
    dim3 blocks( (n+blocksize-1) / blocksize, 1, 1);
    magma_int_t  k, im;
    
    /* Quick return */
    if ( n == 0 )
        return;
    
    if ( order == MagmaColMajor ) {
        for( k=(i1-1); k<i2; k+=BLOCK_SIZE )
        {
            magma_int_t sb = min(BLOCK_SIZE, i2-k);
            magmagpu_dswapblk_params_t params = { dA1T+k, dA2T, n, lda1, lda2, sb };
            for( magma_int_t j = 0; j < sb; j++ )
            {
                im = ipiv[(k+j)*inci] - 1;
                if ( (k+j) == im )
                    params.ipiv[j] = -1;
                else
                    params.ipiv[j] = im - offset;
            }
            magmagpu_dswapblkcm<<< blocks, blocksize, 0, queue >>>( params );
        }
    }
    else {
        for( k=(i1-1); k<i2; k+=BLOCK_SIZE )
        {
            magma_int_t sb = min(BLOCK_SIZE, i2-k);
            magmagpu_dswapblk_params_t params = { dA1T+k*lda1, dA2T, n, lda1, lda2, sb };
            for( magma_int_t j = 0; j < sb; j++ )
            {
                im = ipiv[(k+j)*inci] - 1;
                if ( (k+j) == im )
                    params.ipiv[j] = -1;
                else
                    params.ipiv[j] = im - offset;
            }
            magmagpu_dswapblkrm<<< blocks, blocksize, 0, queue >>>( params );
        }
    }
}


/**
    @see magmablas_dswapblk_q
    @ingroup magma_dblas2
    ********************************************************************/
extern "C" void 
magmablas_dswapblk(
    magma_order_t order, magma_int_t n, 
    double *dA1T, magma_int_t lda1,
    double *dA2T, magma_int_t lda2,
    magma_int_t i1, magma_int_t i2,
    const magma_int_t *ipiv, magma_int_t inci, magma_int_t offset )
{
    magmablas_dswapblk_q(
        order, n, dA1T, lda1, dA2T, lda2, i1, i2, ipiv, inci, offset, magma_stream );
}
