#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @generated from zswapblk.cu normal z -> d, Fri Jan 30 19:00:09 2015

*/
#include "common_magma.h"

#define BLOCK_SIZE 64

/*********************************************************/
/*
 *  Blocked version: swap several pairs of lines
 */
typedef struct {
    double *A;
    double *B;
    int n, ldda, lddb, npivots;
    short ipiv[BLOCK_SIZE];
} magmagpu_dswapblk_params_t;

__global__ void magmagpu_dswapblkrm( magmagpu_dswapblk_params_t params )
{
    unsigned int y = threadIdx.x + blockDim.x*blockIdx.x;
    if( y < params.n )
    {
        double *A = params.A + y - params.ldda;
        double *B = params.B + y;
      
        for( int i = 0; i < params.npivots; i++ )
        {
            A += params.ldda;
            if ( params.ipiv[i] == -1 )
                continue;
            double  tmp1 = *A;
            double *tmp2 = B + params.ipiv[i]*params.lddb;
            *A    = *tmp2;
            *tmp2 =  tmp1;
        }
    }
}

__global__ void magmagpu_dswapblkcm( magmagpu_dswapblk_params_t params )
{
    unsigned int y = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned int offset1 = y*params.ldda;
    unsigned int offset2 = y*params.lddb;
    if( y < params.n )
    {
        double *A = params.A + offset1 - 1;
        double *B = params.B + offset2;
      
        for( int i = 0; i < params.npivots; i++ )
        {
            A++;
            if ( params.ipiv[i] == -1 )
                continue;
            double  tmp1 = *A;
            double *tmp2 = B + params.ipiv[i];
            *A    = *tmp2;
            *tmp2 =  tmp1;
        }
    }
    __syncthreads();
}


/**
    @ingroup magma_dblas2
    ********************************************************************/
extern "C" void 
magmablas_dswapblk_q(
    magma_order_t order, magma_int_t n, 
    magmaDouble_ptr dA, magma_int_t ldda,
    magmaDouble_ptr dB, magma_int_t lddb,
    magma_int_t i1, magma_int_t i2,
    const magma_int_t *ipiv, magma_int_t inci, magma_int_t offset,
    magma_queue_t queue )
{
    magma_int_t  blocksize = 64;
    dim3 blocks( (n+blocksize-1) / blocksize, 1, 1);
    magma_int_t  k, im;
    
    /* Quick return */
    if ( n == 0 )
        return;
    
    if ( order == MagmaColMajor ) {
        for( k=(i1-1); k<i2; k+=BLOCK_SIZE )
        {
            magma_int_t sb = min(BLOCK_SIZE, i2-k);
            magmagpu_dswapblk_params_t params = { dA+k, dB, n, ldda, lddb, sb };
            for( magma_int_t j = 0; j < sb; j++ )
            {
                im = ipiv[(k+j)*inci] - 1;
                if ( (k+j) == im )
                    params.ipiv[j] = -1;
                else
                    params.ipiv[j] = im - offset;
            }
            magmagpu_dswapblkcm<<< blocks, blocksize, 0, queue >>>( params );
        }
    }
    else {
        for( k=(i1-1); k<i2; k+=BLOCK_SIZE )
        {
            magma_int_t sb = min(BLOCK_SIZE, i2-k);
            magmagpu_dswapblk_params_t params = { dA+k*ldda, dB, n, ldda, lddb, sb };
            for( magma_int_t j = 0; j < sb; j++ )
            {
                im = ipiv[(k+j)*inci] - 1;
                if ( (k+j) == im )
                    params.ipiv[j] = -1;
                else
                    params.ipiv[j] = im - offset;
            }
            magmagpu_dswapblkrm<<< blocks, blocksize, 0, queue >>>( params );
        }
    }
}


/**
    @see magmablas_dswapblk_q
    @ingroup magma_dblas2
    ********************************************************************/
extern "C" void 
magmablas_dswapblk(
    magma_order_t order, magma_int_t n, 
    magmaDouble_ptr dA, magma_int_t ldda,
    magmaDouble_ptr dB, magma_int_t lddb,
    magma_int_t i1, magma_int_t i2,
    const magma_int_t *ipiv, magma_int_t inci, magma_int_t offset )
{
    magmablas_dswapblk_q(
        order, n, dA, ldda, dB, lddb, i1, i2, ipiv, inci, offset, magma_stream );
}
