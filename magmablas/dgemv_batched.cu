#include "hip/hip_runtime.h"
/*
   -- MAGMA (version 1.6.1) --
   Univ. of Tennessee, Knoxville
   Univ. of California, Berkeley
   Univ. of Colorado, Denver
   @date January 2015

   @author Azzam Haidar
   @author Tingxing Dong

   @generated from zgemv_batched.cu normal z -> d, Fri Jan 30 19:00:10 2015
 */
#include "common_magma.h"


#define dgemv_bs 32

extern __shared__ double shared_data[];


__global__ void
kernel_dgemvn_batched(
    int m, int n, double alpha,
    double **dA_array, int lda,
    double **x_array, int incx,
    double beta, double  **y_array, int incy)
{

    double *A = dA_array[blockIdx.x];
    double *x = x_array[blockIdx.x];
    double *y = y_array[blockIdx.x];

    int tx = threadIdx.x;

    double res = MAGMA_D_ZERO;

    double *buff = (double*)shared_data;

    if(tx < n)
    {
        buff[tx] = x[tx*incx];
    }
    __syncthreads();
   
    
    if(tx < m )
    {
        for(int j=0; j < n ; j++)
        {
            res += A[tx]*buff[j];
            A += lda;
        }
  
        y[tx*incy] = alpha * res + y[tx*incy] * beta;
    }

}

/*
    Matrix Non-transpose Vector Multiplication
    y := alpha*A*x + beta*y,
*/
extern "C"
void magmablas_dgemvn_batched(
    int m, int n, 
    double alpha, double **dA_array, int lda, 
    double **x_array,  int incx,
    double beta, double **y_array,  int incy, 
    int batchCount, magma_queue_t queue)
{

    if( m > 512 || n > 512)
    {
        fprintf( stderr, "m=%d, n=%d, dgemv_batched nontranspose assume row && column lower than %d. Plz call magmablas_dgemv instead", m, n, 512);
        return ;
    }

    dim3 grid(batchCount, 1, 1);
    dim3 threads(max(m,n), 1, 1);
   
    kernel_dgemvn_batched<<< grid, threads, n * sizeof(double), queue >>>( m, n, alpha,  dA_array, lda, x_array, incx,  
                                                                         beta, y_array, incy);
}



__global__ void
kernel_dgemvt_batched(
    int m, int n, int m1, double alpha,
    double **dA_array, int lda,
    double **x_array, int incx,
    double beta, double  **y_array, int incy)
{
  

    double *A_ptr = dA_array[blockIdx.x];
    double *x_ptr = x_array[blockIdx.x];
    double *y_ptr = y_array[blockIdx.x];

    int tx = threadIdx.x;
    
    double res = MAGMA_D_ZERO;

    if(tx<m)
    {  
        A_ptr += lda * blockIdx.y + tx;
        x_ptr += tx * incx;
    }
        
    __shared__ double sdata[dgemv_bs];

    for(int i=0; i<m1; i+= dgemv_bs)
    {
        res += A_ptr[i] * x_ptr[i*incx];
    }

    if(m > m1)
    {
        if( tx + m1 <  m )
        {
            res  += A_ptr[m1] * x_ptr[m1*incx];
        }
        else
        {
            res  = res;
        }
    }

    sdata[tx] = res;
    __syncthreads();

    for(int s=blockDim.x/2; s>32;s>>=1)
    {
        if(tx<s)
        {
            sdata[tx] += sdata[tx+s];
        } 
        __syncthreads();
    }

    if(dgemv_bs > 32)
    {  
        if(tx<32)
        {
            sdata[tx] += sdata[tx+32];
        }
    }

    if(tx == 0)
    {
        for(int i=1;i<32;i++)
        {
            sdata[tx] += sdata[tx + i];
        }
        
        y_ptr[blockIdx.y * incy] = sdata[0] * alpha + beta * y_ptr[blockIdx.y*incy];
               
    }
}

/*
    Matrix Transpose Vector Multiplication
    y := alpha* A**T *x + beta*y,
*/

extern "C"
void magmablas_dgemvt_batched(
    int m, int n, 
    double alpha, double **dA_array, int lda, 
    double **x_array,  int incx,
    double beta, double **y_array,  int incy, 
    int batchCount, magma_queue_t queue)
{

    dim3 grid(batchCount, n, 1);
    dim3 threads(dgemv_bs, 1, 1);

    int m1 = (m / dgemv_bs) * dgemv_bs;

    kernel_dgemvt_batched <<< grid, threads,0, queue  >>>(m, n, m1, alpha,  dA_array, lda, x_array, incx, beta, y_array, incy);

}
   

#if defined(PRECISION_z) || defined (PRECISION_c)


__global__ void
kernel_dgemvc_batched(
    int m, int n, int m1, double alpha,
    double **dA_array, int lda,
    double **x_array, int incx,
    double beta, double  **y_array, int incy)
{
  

    double *A_ptr = dA_array[blockIdx.x];
    double *x_ptr = x_array[blockIdx.x];
    double *y_ptr = y_array[blockIdx.x];

    int tx = threadIdx.x;
    
    double res = MAGMA_D_ZERO;

    if(tx<m)
    {
        A_ptr += lda * blockIdx.y + tx;
        x_ptr += tx * incx;
    }
        
    __shared__ double sdata[dgemv_bs];

    for(int i=0; i<m1; i+= dgemv_bs)
    {
        res += MAGMA_D_CNJG (A_ptr[i]) * x_ptr[i*incx];
    }

    if(m > m1)
    {
        if( tx + m1 <  m )
        {
            res  += MAGMA_D_CNJG(A_ptr[m1]) * x_ptr[m1*incx];
        }
        else
        {
            res  = res;
        }
    }

    sdata[tx] = res;
    __syncthreads();

    for(int s=blockDim.x/2; s>32;s>>=1)
    {
        if(tx<s)
        {
            sdata[tx] += sdata[tx+s];
        } 
        __syncthreads();
    }

    if(dgemv_bs > 32)
    {  
        if(tx<32)
        {
            sdata[tx] += sdata[tx+32];
        }
    }

    if(tx == 0)
    {
        for(int i=1;i<32;i++)
        {
            sdata[tx] += sdata[tx + i];
        }
        
        y_ptr[blockIdx.y * incy] = sdata[0] * alpha + beta * y_ptr[blockIdx.y*incy];
               
    }
}

/*
    Matrix Conjugate Transpose Vector Multiplication
    y := alpha* A**H *x + beta*y,
*/

extern "C"
void magmablas_dgemvc_batched(
    int m, int n, 
    double alpha, double **dA_array, int lda, 
    double **x_array,  int incx,
    double beta, double **y_array,  int incy, 
    int batchCount, magma_queue_t queue)
{

    dim3 grid(batchCount, n, 1);
    dim3 threads(dgemv_bs, 1, 1);

    int m1 = (m / dgemv_bs) * dgemv_bs;

    kernel_dgemvc_batched <<< grid, threads, 0, queue >>>(m, n, m1, alpha,  dA_array, lda, x_array, incx, beta, y_array, incy);
}
   
#endif // defined(PRECISION_z) || defined (PRECISION_c)


/**
    Purpose
    -------

    This routine computes Y = alpha opt(A) x + beta y, on the GPU, where
    A = dA_array[i],x = x_array[i] and y = y_array[i], i=[0,batchCount-1].
    This is a batched version.

    @param[in]
    trans  CHARACTER*1.
           On entry, TRANS specifies the form of op( A ) to be used in
           the matrix multiplication as follows:
           = 'N':  op( A ) = A.
           = 'T':  op( A ) = A**T.
           = 'C':  op( A ) = A**H.

    @param[in]
    m       INTEGER.
            On entry, M specifies the number of rows of the matrix opt(A).

    @param[in]
    n       INTEGER.
            On entry, N specifies the number of columns of the matrix opt(A)

    @param[in]
    alpha   DOUBLE PRECISION.
            On entry, ALPHA specifies the scalar alpha.

    @param[in]
    dA_array A = dA_array[i] 
            A: DOUBLE PRECISION array of dimension ( LDA, n ) on the GPU.
   
    @param[in]
    lda     INTEGER.
            LDA specifies the leading dimension of A.

    @param[in]
    x_array x = x_array[i]
            x: DOUBLE PRECISION array of dimension.
            n if trans == MagmaNoTrans.
            m if trans == MagmaTrans or MagmaConjTrans.

    @param[in]
    incx    INTEGER.
            incx specifies the increment for the elments of x.
            incx must not be zero.
    
    @param[in]
    beta    DOUBLE PRECISION.
            On entry, BETA specifies the scalar beta.

    @param[out]
    y_array y = y_array[i]:       
            On exit y = alpha opt(A) x + beta y.
            y: DOUBLE PRECISION array of dimension.
            m if trans == MagmaNoTrans.
            n if trans == MagmaTrans or MagmaConjTrans.

    @param[in]
    incy    INTEGER.
            incy specifies the increment for the elments of y.
            incy must not be zero.
    
    @param[in]
    batchCount INTEGER
            number of pointers contained in dA_array, x_array and y_array.

    @ingroup magma_dblas2
    *******************************************************************   */

extern "C"
void magmablas_dgemv_batched(
    magma_trans_t trans, magma_int_t m, magma_int_t n, 
    double alpha,
    magmaDouble_ptr dA_array[], magma_int_t ldda, 
    magmaDouble_ptr dx_array[], magma_int_t incx,
    double beta,
    magmaDouble_ptr dy_array[], magma_int_t incy, 
    magma_int_t batchCount, magma_queue_t queue)
{       
    magma_int_t info = 0;
    if ( trans != MagmaNoTrans && trans != MagmaTrans && trans != MagmaConjTrans )
        info = -1;
    else if ( m < 0 )
        info = -2;
    else if ( n < 0 )
        info = -3;
    else if ( ldda < m )
        info = -6;
    else if ( incx == 0 )
        info = -8;
    else if ( incy == 0 )
        info = -11;

    if (info != 0) {
        magma_xerbla( __func__, -(info) );
        return;  //info;
    }

    if(m==0 || n ==0 ) return;

    if ( trans == MagmaNoTrans ) {

        magmablas_dgemvn_batched(m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue);
            
    }
    else if ( trans == MagmaTrans ) {
        magmablas_dgemvt_batched(m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue);
    }
    else if ( trans == MagmaConjTrans ) {
#if defined(PRECISION_z) || defined (PRECISION_c)
        magmablas_dgemvc_batched(m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue);
#else
        magmablas_dgemvt_batched(m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue);
#endif
    }
    else {
        fprintf( stderr, "trans = %c is invalid\n", lapacke_trans_const(trans) );
    }
}

#undef dgemv_bs 
