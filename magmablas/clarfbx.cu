#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2014

       @generated from zlarfbx.cu normal z -> c, Sat Nov 15 19:53:59 2014

*/
#include "common_magma.h"
#include "commonblas_c.h"
#include "magma_templates.h"

// 512 is maximum number of threads for CUDA capability 1.x
#define BLOCK_SIZE 512




//==============================================================================
extern "C"
__global__ void 
magma_cgemv_kernel1(int m, const magmaFloatComplex * __restrict__ V, int ldv, 
                    const magmaFloatComplex * __restrict__ c, 
                    magmaFloatComplex *dwork)
{
    const int i = threadIdx.x;
    const magmaFloatComplex *dV = V + (blockIdx.x) * ldv;

    __shared__ magmaFloatComplex sum[ BLOCK_SIZE ];
    magmaFloatComplex lsum;

    /*  lsum := v**H * C  */
    lsum = MAGMA_C_ZERO;
    for( int j = i; j < m; j += BLOCK_SIZE )
       lsum += MAGMA_C_MUL( MAGMA_C_CNJG( dV[j] ), c[j] );
    
    sum[i] = lsum;
    magma_sum_reduce< BLOCK_SIZE >( i, sum );

    __syncthreads();
    if (i==0)
       dwork [blockIdx.x] = sum[0];
}

//==============================================================================
/*  ----------------------------------------------------------------------------- 
    Call 
        magma_cgemv_kernel3<<< n, BLOCK_SIZE>>>(m, V, ldv, c, dwork, tau)
    to compute
        CGEMV( "Conjugate transpose", m, n, -tau[0], V, ldv, c, 1, zero, dwork, 1)
        and to set c[0] to 1.
    i.e., 
        work = -tau[0] V**H c
    ----------------------------------------------------------------------------- */
extern "C"
__global__ void
magma_cgemv_kernel3(int m, const magmaFloatComplex * __restrict__ V, int ldv, magmaFloatComplex *c,
                    magmaFloatComplex *dwork, magmaFloatComplex *tau)
{
    const int i = threadIdx.x;
    const magmaFloatComplex *dV = V + (blockIdx.x) * ldv;

    __shared__ magmaFloatComplex sum[ BLOCK_SIZE ];
    magmaFloatComplex lsum;

    if (i==0)
       c[0] = MAGMA_C_ONE;           

    /*  lsum := v**H * C  */
    lsum = MAGMA_C_ZERO;
    for( int j = i; j < m; j += BLOCK_SIZE )
       lsum += MAGMA_C_MUL( MAGMA_C_CNJG( dV[j] ), c[j] );

    sum[i] = lsum;
    magma_sum_reduce< BLOCK_SIZE >( i, sum );

    __syncthreads();
    if (i==0)
       dwork [blockIdx.x] = -tau[0]*sum[0];
}

//==============================================================================
extern "C"
__global__ void
magma_cgemv_kernel2(int m, int n, const magmaFloatComplex * __restrict__ V, int ldv, 
                    const magmaFloatComplex * __restrict__ x, magmaFloatComplex *c)
{
    const int i = threadIdx.x;
    const int j = i + BLOCK_SIZE * blockIdx.x;
    magmaFloatComplex lsum;

    V += j;

    lsum = MAGMA_C_ZERO;
    if (j < m){
       for(int k=0; k<n; k++)
          lsum += MAGMA_C_MUL( V[k*ldv], x[k]);
       
       c[j] -= lsum;
    }
}

//==============================================================================

/*
    Apply a complex block reflector H to a complex vector C from the left
    (i.e., C = H C). H is represented in the form
          H = I - V T V**H
    where T is the complex k-by-k upper triangular matrix in the 
    representation of the block reflector, and V is a complex block of
    k elementary reflectors. 
*/
extern "C" void
magma_clarfbx_gpu(
    magma_int_t m, magma_int_t k,
    magmaFloatComplex_ptr V,  magma_int_t ldv,
    magmaFloatComplex_ptr dT, magma_int_t ldt,
    magmaFloatComplex_ptr c,
    magmaFloatComplex_ptr dwork)
{
    /* dwork = V**H c     */
    magma_cgemv_kernel1<<< k, BLOCK_SIZE, 0, magma_stream >>>(m, V, ldv, c, dwork); 

    /* dwork = T**H dwork */
    magma_ctrmv_tkernel<<< k, k, 0, magma_stream >>>( dT, ldt, dwork, dwork+k);
 
    /* c = c - V dwork    */
    dim3  blocks3( (m + BLOCK_SIZE-1) / BLOCK_SIZE );
    dim3 threads3( BLOCK_SIZE );     
    magma_cgemv_kernel2<<< blocks3, threads3, 0, magma_stream >>>( m, k, V, ldv, dwork+k, c);
}

//==============================================================================
