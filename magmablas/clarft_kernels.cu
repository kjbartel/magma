#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

       @generated from zlarft_kernels.cu normal z -> c, Fri Jan 30 19:00:10 2015
       @author Azzam Haidar
*/

#include "common_magma.h"
#include "magma_templates.h"
#define cgemv_bs 32
#define BLOCK_SIZE 512

#define use_gemm_larft

extern __shared__ magmaFloatComplex shared_data[];


//===================================================================================================
static __device__
void clarft_gemvcolwise_device( int m, magmaFloatComplex *v, magmaFloatComplex *tau,
                         magmaFloatComplex *c, int ldc, magmaFloatComplex *T, int ldt, int step )
{

    const int thblk =  blockIdx.x;
    if (thblk > step)
        return;
    /* if blockIdx.x<step step performs the z = V(tx:n,tx)' * V(tx:n,1:tx-1) used for computing T:*/

    if ( !MAGMA_C_EQUAL(*tau, MAGMA_C_ZERO) ) {
        if(thblk<step){    
            const int tx = threadIdx.x;
            magmaFloatComplex *dc = c + blockIdx.x * ldc;
           
            __shared__ magmaFloatComplex sum[ BLOCK_SIZE ];
            magmaFloatComplex tmp;
           
            /* perform  {T_i}^H := V(:,i)' * V(:,1:i-1)  */
            if (tx==0)
                tmp = dc[0]; //since V[0] should be one
            else
                tmp = MAGMA_C_ZERO;
            for( int j = tx+1; j < m; j += BLOCK_SIZE ){
                tmp +=  MAGMA_C_CNJG( v[j] ) * dc[j];
            }
            sum[tx] = tmp;
            magma_sum_reduce< BLOCK_SIZE >( tx, sum );
            #if defined (use_gemm_larft)
            *(T+thblk) = MAGMA_C_CNJG(sum[0]);
            #else
            tmp = - MAGMA_C_CNJG(*tau) * sum[0]; 
            *(T+thblk) = MAGMA_C_CNJG(tmp); // T = - tau(tx) * V(tx:n,1:tx-1)' * V(tx:n,tx) = tmp'
            //*(T+thblk) = - MAGMA_C_CNJG(sum[0]) * (*tau); // T = - tau(tx) * V(tx:n,1:tx-1)' * V(tx:n,tx) = tmp'
            #endif
        }
        else{
            #if defined (use_gemm_larft)
            *(T+thblk) = MAGMA_C_ONE;
            #else
            *(T+thblk) = *tau;
            #endif
        }
    }// in case tau is zero put the corresponding column of T to zero
    else 
    {
        *(T+thblk) = MAGMA_C_ZERO;
    }
}
//===================================================================================================
__global__
void clarft_gemvcolwise_kernel( int m, magmaFloatComplex *v, int ldv, magmaFloatComplex *tau,
                          magmaFloatComplex *T, int ldt, int step )
{
    clarft_gemvcolwise_device(m, v+step+step*ldv, tau+step, v+step, ldv, T+step*ldt, ldt, step);
}
//===================================================================================================
__global__
void clarft_gemvcolwise_kernel_batched( int m, magmaFloatComplex **v_array, int ldv, magmaFloatComplex **tau_array,
                          magmaFloatComplex **T_array, int ldt, int step )
{
    int batchid = blockIdx.z;
    clarft_gemvcolwise_device(m, v_array[batchid]+step+step*ldv, tau_array[batchid]+step, v_array[batchid]+step, ldv, T_array[batchid]+step*ldt, ldt, step);
}
//===================================================================================================
extern "C" 
void magmablas_clarft_gemvcolwise(
    magma_int_t m,  magma_int_t step,
    magmaFloatComplex *v, magma_int_t ldv, 
    magmaFloatComplex *T,  magma_int_t ldt,
    magmaFloatComplex *tau)
{
    dim3 grid( step+1, 1, 1 );
    dim3 threads( BLOCK_SIZE );
    clarft_gemvcolwise_kernel<<< grid, threads, 0, magma_stream >>>( m, v, ldv, tau, T, ldt, step);

}
//===================================================================================================
extern "C" 
void magmablas_clarft_gemvcolwise_batched(
    magma_int_t m,  magma_int_t step,
    magmaFloatComplex **v_array, magma_int_t ldv, 
    magmaFloatComplex **T_array,  magma_int_t ldt,
    magmaFloatComplex **tau_array, magma_int_t batchCount, magma_queue_t queue )
{
    dim3 grid( step+1, 1, batchCount );
    dim3 threads( BLOCK_SIZE );
    clarft_gemvcolwise_kernel_batched<<< grid, threads, 0, queue >>>( m, v_array, ldv, tau_array, T_array, ldt, step);

}
//===================================================================================================




//===================================================================================================
// cgemv(y=alpha*A*x) interface: T/W=tau*v*x, 
static __device__ void
clarft_gemvrowwise_device(
    int m, int i,
    magmaFloatComplex *tau, 
    magmaFloatComplex *v_ptr, int ldv, 
    magmaFloatComplex *x_ptr, int incx,
    magmaFloatComplex *T_ptr, int ldt,
    magmaFloatComplex *W, magmaFloatComplex* sdata)
{
    int tx = threadIdx.x; 
    int ty = threadIdx.y; 


    if(tx ==0 && ty == 0)
    {
        T_ptr[0] = *tau;
    } 

    if(i <= 0) return;
    
    magmaFloatComplex res = MAGMA_C_ZERO;

    v_ptr += ldv * ty;
            

   
    if(tx < cgemv_bs)
    {
        for(int s=tx; s<m; s+= cgemv_bs)
        {
            res += MAGMA_C_CNJG (v_ptr[s]) * x_ptr[s*incx];
        }
    
        sdata[ty * cgemv_bs + tx] = res;
    }
    __syncthreads();

    magma_sum_reduce<cgemv_bs>(tx, &(sdata[ty*cgemv_bs+0]));

    #if defined (use_gemm_larft)
    if(tx == 0)
    {
            W[ty] = -sdata[ty * cgemv_bs + 0];
    } 
    #else
    if(tx == 0)
    {
            W[ty] = -sdata[ty * cgemv_bs + 0] * (*tau) ;
    }
    #endif 
}




//T(1:i-1,i) := - tau(i) * V(i:n,1:i-1)' * V(i:n,i)
//T(i,i) = tau(i)
//===================================================================================================
 __global__ void
clarft_gemvrowwise_kernel(
    int m, int i, 
    magmaFloatComplex *tau, 
    magmaFloatComplex *v, int ldv, 
    magmaFloatComplex *T, int ldt)
{

    magmaFloatComplex *W =  T +i*ldt;

    magmaFloatComplex *sdata = (magmaFloatComplex*)shared_data;

    clarft_gemvrowwise_device(m, i, tau+i, v+i, ldv,  v+i+i*ldv, 1,  
                           T+i+i*ldt , ldt, W, sdata);
}

//===================================================================================================
__global__ void
clarft_gemvrowwise_kernel_batched(
    int m, int i,
    magmaFloatComplex **tau_array, 
    magmaFloatComplex **v_array, int ldv, 
    magmaFloatComplex **T_array, int ldt)
{

    int batchid = blockIdx.z;

    magmaFloatComplex *W =  T_array[batchid] +i*ldt;

    magmaFloatComplex *sdata = (magmaFloatComplex*)shared_data;

    clarft_gemvrowwise_device(m, i, tau_array[batchid]+i, v_array[batchid]+i, ldv,  v_array[batchid]+i+i*ldv, 1,  
                           T_array[batchid] +i+i*ldt , ldt, W, sdata);
}

//===================================================================================================
extern "C"
void magmablas_clarft_gemvrowwise(
    magma_int_t m, magma_int_t i, 
    magmaFloatComplex *tau, 
    magmaFloatComplex *v, magma_int_t ldv, 
    magmaFloatComplex *T, magma_int_t ldt,
    magmaFloatComplex *W)
{

    dim3 grid(1);


    dim3 threads(cgemv_bs, max(i,1), 1);


    clarft_gemvrowwise_kernel <<< grid, threads, sizeof(magmaFloatComplex)*cgemv_bs*(i+1), magma_stream>>>(m, i, tau, v, ldv, T, ldt);
}
//===================================================================================================
extern "C"
void magmablas_clarft_gemvrowwise_batched(
    magma_int_t m, magma_int_t i, 
    magmaFloatComplex **tau_array, 
    magmaFloatComplex **v_array, magma_int_t ldv, 
    magmaFloatComplex **T_array, magma_int_t ldt,
    magma_int_t batchCount, magma_queue_t queue)
{

    dim3 grid(1, 1, batchCount);
    dim3 threads(cgemv_bs, max(i,1), 1);

    /*  cgemvrowwise used a bigger shared memory and has more data reuse and performs better
    */
    clarft_gemvrowwise_kernel_batched <<< grid, threads, sizeof(magmaFloatComplex)*cgemv_bs*(i+1), queue>>>(m, i,  tau_array, v_array, ldv, T_array, ldt);
}
//===================================================================================================
   


//===================================================================================================
/*
   loop_inside
*/
static __device__ void
clarft_gemv_loop_inside_device(
    int n, int k, 
    magmaFloatComplex *tau, 
    magmaFloatComplex *v, int ldv, 
    magmaFloatComplex *T, int ldt)
{
    int tx = threadIdx.x; 
    int ty = threadIdx.y; 
    
    int incx = 1;
    magmaFloatComplex *sdata = (magmaFloatComplex*)shared_data;

    magmaFloatComplex res;

    // write the first elment
    if(tx ==0 && ty == 0)
    {
        T[0] = tau[0];
    } 
 
    for(int i=1; i<k;i++)
    {

        int m = n-i; 

        magmaFloatComplex *v_ptr = v;

        v_ptr += i;

        magmaFloatComplex *x_ptr = v_ptr + i * ldv;
            
        res = MAGMA_C_ZERO;
            
        if(tx < cgemv_bs && ty < i)
        {
            v_ptr += ldv * ty;

            for(int s=tx; s<m; s+= cgemv_bs)
            {
                res += MAGMA_C_CNJG (v_ptr[s]) * x_ptr[s*incx];
            }
    
            sdata[ty * cgemv_bs + tx] = res;
        }
        __syncthreads();

        magma_sum_reduce<cgemv_bs>(tx, &(sdata[ty*cgemv_bs+0]));
        

       __syncthreads();
       #if defined (use_gemm_larft)
       if(tx < i && ty == 0)
       {
            T[i* ldt + tx] = sdata[tx * cgemv_bs + 0];  
       } 
       // not needed since it is overwritten in trmv
       /*
       if(tx == i && ty == 0)
       {
           T[i * ldt + i] = tau[i];
       }
       */
       #else
       if(tx < i && ty == 0)
       {
           T[i* ldt + tx] = -sdata[tx * cgemv_bs + 0] * (tau[i]) ;  
       } 
      
       if(tx == i && ty == 0)
       {
           T[i * ldt + i] = tau[i];
       }
       #endif
     
       v_ptr -= i;

    }// end of loop k
}
//===================================================================================================
__global__ void
clarft_gemv_loop_inside_kernel(
    int n, int k, 
    magmaFloatComplex *tau, 
    magmaFloatComplex *v, int ldv, 
    magmaFloatComplex *T, int ldt)
{
    clarft_gemv_loop_inside_device(n, k, tau, v, ldv, T, ldt);
}
//===================================================================================================
__global__ void
clarft_gemv_loop_inside_kernel_batched(
    int n, int k, 
    magmaFloatComplex **tau_array, 
    magmaFloatComplex **v_array, int ldv, 
    magmaFloatComplex **T_array, int ldt)
{
    int batchid = blockIdx.z;
    clarft_gemv_loop_inside_device(n, k, tau_array[batchid], v_array[batchid], ldv, T_array[batchid], ldt);
}
//===================================================================================================
//===================================================================================================
//===================================================================================================
extern "C"
void magmablas_clarft_gemv_loop_inside(
    int n, int k, 
    magmaFloatComplex *tau, 
    magmaFloatComplex *v, int ldv, 
    magmaFloatComplex *T, int ldt)
{

    dim3 grid(1);
    dim3 threads(cgemv_bs, max(k,1), 1);
    clarft_gemv_loop_inside_kernel<<<grid, threads, sizeof(magmaFloatComplex) * (cgemv_bs*(k+1)), magma_stream>>>(n, k, tau, v, ldv, T, ldt); 
}
//===================================================================================================
extern "C"
void magmablas_clarft_gemv_loop_inside_batched(
    int n, int k, 
    magmaFloatComplex **tau_array, 
    magmaFloatComplex **v_array, int ldv, 
    magmaFloatComplex **T_array, int ldt, magma_int_t batchCount, magma_queue_t queue)
{

    dim3 grid(1, 1, batchCount);
    dim3 threads(cgemv_bs, max(k,1), 1);
    clarft_gemv_loop_inside_kernel_batched<<<grid, threads, sizeof(magmaFloatComplex) * (cgemv_bs*(k+1)), queue>>>(n, k, tau_array, v_array, ldv, T_array, ldt); 
}
//===================================================================================================





//===================================================================================================
static  __device__ void 
clarft_ctrmv_sm32x32_device(
    int n, int k, magmaFloatComplex *tau,
    magmaFloatComplex *Tin, int ldtin,  magmaFloatComplex *Tout, int ldtout )
{
    int tx = threadIdx.x; 
    magmaFloatComplex *sdata = (magmaFloatComplex*)shared_data;
    magmaFloatComplex res;

    // this routine apply a sequence of trmv to update k column of the triangular
    // T starting at n-k to n where T is of size n by n and where the first n-k 
    // columns of T are supposed updated previously.
    // So the routine load all of T nxn to the shared memory 
    // and apply the sequence of trmv.
    // to update a certain column i, threads go in horizontal fashion where
    // every thread read one row and do it gemv(dot) to generate 
    // one element of the column of T then move to the next column

    // read T into shared
    for(int s=0; s<n-k; s++)
    {
        sdata[tx + s*n] = Tin[tx + s * ldtin];
    }
    
#if defined(use_gemm_larft)
    for(int s=n-k; s<n; s++)
    {
        if(tx == s)
            sdata[tx + s*n] = tau[s];
        else
            sdata[tx + s*n] = -tau[s] * Tin[tx + s * ldtin];
    }
#else
    for(int s=n-k; s<n; s++)
    {
        sdata[tx + s*n] = Tin[tx + s * ldtin];
    }
#endif

    // perform trmv
    for(int i=n-k; i<n;i++)
    {
       __syncthreads();  
       res = MAGMA_C_ZERO;
       if(tx < i)
       {
           for(int j=tx; j<i; j++)
           {
               res += sdata[tx + j * n] * sdata[j+ i * n];      
           }
       }       
       __syncthreads();  
       if(tx < i)
       {
           sdata[tx + i * n] = res;
       }
    } 

    __syncthreads();  
    // write back the updated block of k column of T
    for(int s=n-k; s<n; s++)
    {
       Tout[tx + s * ldtout] = sdata[tx + s*n];
    }

}
//===================================================================================================
__global__ void 
clarft_ctrmv_sm32x32_kernel(
    int n, int k, magmaFloatComplex *tau,
    magmaFloatComplex *Tin, int ldtin,  magmaFloatComplex *Tout, int ldtout )
{
    clarft_ctrmv_sm32x32_device( n, k, tau, Tin, ldtin, Tout, ldtout);
}
//===================================================================================================
__global__ void 
clarft_ctrmv_sm32x32_kernel_batched(
    int n, int k, magmaFloatComplex **tau_array,
    magmaFloatComplex **Tin_array, int ldtin,  magmaFloatComplex **Tout_array, int ldtout )
{
    int batchId = blockIdx.z;
    clarft_ctrmv_sm32x32_device( n, k, tau_array[batchId], Tin_array[batchId], ldtin, Tout_array[batchId], ldtout);
}
//===================================================================================================
//===================================================================================================
extern "C"
void magmablas_clarft_ctrmv_sm32x32(
    magma_int_t m, magma_int_t n, 
    magmaFloatComplex *tau, 
    magmaFloatComplex *Tin, magma_int_t ldtin, 
    magmaFloatComplex *Tout, magma_int_t ldtout)
{

    dim3 grid(1);
    dim3 threads(max(m,1), 1, 1);
    clarft_ctrmv_sm32x32_kernel <<< grid, threads, sizeof(magmaFloatComplex)*(m*m), magma_stream >>> (m, n,  tau, Tin, ldtin, Tout, ldtout);
}
//===================================================================================================
extern "C"
void magmablas_clarft_ctrmv_sm32x32_batched(
    magma_int_t m, magma_int_t n, 
    magmaFloatComplex **tau_array, 
    magmaFloatComplex **Tin_array, magma_int_t ldtin, 
    magmaFloatComplex **Tout_array, magma_int_t ldtout,
    magma_int_t batchCount, magma_queue_t queue)
{

    dim3 grid(1, 1, batchCount);
    dim3 threads(max(m,1), 1, 1);
    clarft_ctrmv_sm32x32_kernel_batched <<< grid, threads, sizeof(magmaFloatComplex)*(m*m), queue >>> (m, n,  tau_array, Tin_array, ldtin, Tout_array, ldtout);
}
//===================================================================================================




//===================================================================================================
//===================================================================================================
static __device__ void 
clarft_recctrmv_sm32x32_device(
    int m, int n, magmaFloatComplex *tau,
    magmaFloatComplex *Trec, int ldtrec, magmaFloatComplex *Ttri, int ldttri)
{
    int tx = threadIdx.x; 
    magmaFloatComplex *sdata = (magmaFloatComplex*)shared_data;
    magmaFloatComplex res;

    // to update a certain column i, threads go in horizontal fashion where
    // every thread read one row and do it gemv(dot) to generate 
    // one element of the column of T then move to the next column

    // read T into shared
    for(int s=0; s<n; s++)
    {
        sdata[tx + s*n] = Trec[tx + s * ldtrec];
    }
    __syncthreads();  
    
    // perform sequence of n-1 gemv
    for(int i=0; i<n;i++)
    {
       res = MAGMA_C_ZERO;
       for(int j=0; j<i; j++)
       {
           res += sdata[tx + j * n] * Ttri[j+ i * ldttri];      
       }
       __syncthreads();   // a enlever
       sdata[tx + i * n] = -tau[i] * (sdata[tx + i * n] + res);
       __syncthreads();  
    } 

    // write back the updated block of k column of T  multiplying by -tau
    for(int s=0; s<n; s++)
    {
       Trec[tx + s * ldtrec] = sdata[tx + s*n];
    }

}

//===================================================================================================
__global__ void 
clarft_recctrmv_sm32x32_kernel(
    int m, int n, magmaFloatComplex *tau,
    magmaFloatComplex *Trec, int ldtrec, magmaFloatComplex *Ttri, int ldttri)
{
    clarft_recctrmv_sm32x32_device(m, n, tau, Trec, ldtrec, Ttri, ldttri);
}
//===================================================================================================
__global__ void 
clarft_recctrmv_sm32x32_kernel_batched(
    int m, int n, magmaFloatComplex **tau_array,
    magmaFloatComplex **Trec_array, int ldtrec, magmaFloatComplex **Ttri_array, int ldttri)
{
    int batchId = blockIdx.z;
    clarft_recctrmv_sm32x32_device(m, n, tau_array[batchId], Trec_array[batchId], ldtrec, Ttri_array[batchId], ldttri);
}
//===================================================================================================
extern "C"
void magmablas_clarft_recctrmv_sm32x32(
    magma_int_t m, magma_int_t n, 
    magmaFloatComplex *tau, 
    magmaFloatComplex *Trec, magma_int_t ldtrec, 
    magmaFloatComplex *Ttri, magma_int_t ldttri)
{

    dim3 grid(1);
    dim3 threads(max(m,1), 1, 1);
    clarft_recctrmv_sm32x32_kernel <<< grid, threads, sizeof(magmaFloatComplex)*(m*n), magma_stream >>> (m, n,  tau, Trec, ldtrec, Ttri, ldttri);
}
//===================================================================================================
extern "C"
void magmablas_clarft_recctrmv_sm32x32_batched(
    magma_int_t m, magma_int_t n, 
    magmaFloatComplex **tau_array, 
    magmaFloatComplex **Trec_array, magma_int_t ldtrec, 
    magmaFloatComplex **Ttri_array, magma_int_t ldttri,
    magma_int_t batchCount, magma_queue_t queue)
{

    dim3 grid(1, 1, batchCount);
    dim3 threads(max(m,1), 1, 1);
    clarft_recctrmv_sm32x32_kernel_batched <<< grid, threads, sizeof(magmaFloatComplex)*(m*n), queue >>> (m, n,  tau_array, Trec_array, ldtrec, Ttri_array, ldttri);
}
//===================================================================================================


