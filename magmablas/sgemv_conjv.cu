#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015
       
       @author Stan Tomov

       @generated from zgemv_conjv.cu normal z -> s, Fri Jan 30 19:00:08 2015
*/
#include "common_magma.h"
#include "commonblas_s.h"

#define PRECISION_s

#define num_threads 256


__global__ void
sgemv_conjv_kernel(
    int m, int n, float alpha,
    const float * __restrict__ A, int lda,
    const float * __restrict__ x, int incx, float beta,
    float *       __restrict__ y, int incy)
{
    int ind = blockIdx.x*num_threads + threadIdx.x;
    
    A += ind;

    if ( ind < m ) {
        float res = MAGMA_S_ZERO;
        
        #pragma unroll
        for( int i=0; i < n; i ++ ) {
            res += A[0] * MAGMA_S_CNJG(x[0]);
            A += lda;
            x += incx;
        }
        
        y[ind*incy] = alpha * res + beta * y[ind*incy];
    }
}


/**
    Purpose
    -------
    SGEMV_CONJV performs the matrix-vector operation
    
        y := alpha*A*conj(x)    + beta*y, 
    
    where alpha and beta are scalars, x and y are vectors and A is an
    m by n matrix.

    Arguments
    ----------
    @param[in]
    m       INTEGER
            On entry, m specifies the number of rows of the matrix A.

    @param[in]
    n       INTEGER
            On entry, n specifies the number of columns of the matrix A

    @param[in]
    alpha   REAL
            On entry, ALPHA specifies the scalar alpha.

    @param[in]
    dA      REAL array of dimension ( LDA, n ) on the GPU.

    @param[in]
    lda     INTEGER
            LDA specifies the leading dimension of A.

    @param[in]
    dx      REAL array of dimension n

    @param[in]
    incx    Specifies the increment for the elements of X.
            INCX must not be zero.

    @param[in]
    beta    DOUBLE REAL
            On entry, BETA specifies the scalar beta. When BETA is
            supplied as zero then Y need not be set on input.

    @param[out]
    dy      REAL array of dimension m

    @param[in]
    incy    Specifies the increment for the elements of Y.
            INCY must not be zero.

    @ingroup magma_sblas2
    ********************************************************************/
extern "C" void
magmablas_sgemv_conjv(
    magma_int_t m, magma_int_t n, float alpha,
    magmaFloat_const_ptr dA, magma_int_t ldda,
    magmaFloat_const_ptr dx, magma_int_t incx,
    float beta,
    magmaFloat_ptr dy, magma_int_t incy)
{
    magma_int_t info = 0;
    if ( m < 0 )
        info = -1;
    else if ( n < 0 )
        info = -2;
    else if ( ldda < m )
        info = -5;
    else if ( incx == 0 )
        info = -7;
    else if ( incy == 0 )
        info = -10;
    
    if (info != 0) {
        magma_xerbla( __func__, -(info) );
        return;  //info;
    }
    
    magma_int_t blocks = (m - 1)/num_threads + 1;
    dim3 grid(blocks, 1, 1);
    dim3 threads(num_threads, 1, 1);

    sgemv_conjv_kernel<<< grid, threads, 0, magma_stream >>>
            (m, n, alpha, dA, ldda, dx, incx, beta, dy, incy);

}

#undef num_threads
