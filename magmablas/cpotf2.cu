#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015
       
       @generated from zpotf2.cu normal z -> c, Fri Jan 30 19:00:10 2015
*/
#include "common_magma.h"

#define PRECISION_c

//#if (GPUSHMEM < 200)
#define cdotc_max_bs 512  // 512 is max threads for 1.x cards
//#else
//#define cdotc_max_bs 1024
//#endif

void cpotf2_csscal(magma_int_t n, magmaFloatComplex *x, magma_int_t incx);
void cpotf2_cdotc(magma_int_t n, magmaFloatComplex *x, magma_int_t incx);

#if defined(PRECISION_z) || defined(PRECISION_c)
void clacgv(magma_int_t n, magmaFloatComplex *x, magma_int_t incx);
#endif

/**
    Purpose
    -------

    cpotf2 computes the Cholesky factorization of a real symmetric
    positive definite matrix A.

    The factorization has the form
        A = U**H * U,  if UPLO = MagmaUpper, or
        A = L  * L**H, if UPLO = MagmaLower,
    where U is an upper triangular matrix and L is lower triangular.

    This is the unblocked version of the algorithm, calling Level 2 BLAS.

    Arguments
    ---------

    @param[in]
    uplo    magma_uplo_t
            Specifies whether the upper or lower triangular part of the
            symmetric matrix A is stored.
      -     = MagmaUpper:  Upper triangular
      -     = MagmaLower:  Lower triangular

    @param[in]
    n       INTEGER
            The order of the matrix A.  N >= 0 and N <= 512.

    @param[in,out]
    dA      COMPLEX array, dimension (LDDA,N)
            On entry, the symmetric matrix A.  If UPLO = MagmaUpper, the leading
            n by n upper triangular part of A contains the upper
            triangular part of the matrix A, and the strictly lower
            triangular part of A is not referenced.  If UPLO = MagmaLower, the
            leading n by n lower triangular part of A contains the lower
            triangular part of the matrix A, and the strictly upper
            triangular part of A is not referenced.
    \n
            On exit, if INFO = 0, the factor U or L from the Cholesky
            factorization A = U**H * U  or A = L * L**H.

    @param[in]
    ldda    INTEGER
            The leading dimension of the array A.  LDDA >= max(1,N).

    @param[out]
    info    INTEGER
      -     = 0: successful exit
      -     < 0: if INFO = -k, the k-th argument had an illegal value
      -     > 0: if INFO = k, the leading minor of order k is not
                 positive definite, and the factorization could not be
                 completed.

    @ingroup magma_cposv_aux
    ********************************************************************/
extern "C" magma_int_t
magma_cpotf2_gpu(
    magma_uplo_t uplo, magma_int_t n,
    magmaFloatComplex_ptr dA, magma_int_t ldda,
    magma_int_t *info )
{
#define dA(i_, j_)  (dA + (i_) + (j_)*ldda)

    magma_int_t j;

    *info = 0;
    if ( uplo != MagmaUpper && uplo != MagmaLower) {
        *info = -1;
    } else if (n < 0 || n > cdotc_max_bs) {
        *info = -2;
    } else if (ldda < max(1,n)) {
        *info = -4;
    }

    if (*info != 0) {
        magma_xerbla( __func__, -(*info) );
        return *info;
    }

    // Quick return if possible
    if (n == 0) {
        return *info;
    }

    magmaFloatComplex alpha = MAGMA_C_NEG_ONE;
    magmaFloatComplex beta  = MAGMA_C_ONE;

    if (uplo == MagmaUpper) {
        for(j = 0; j < n; j++) {
            cpotf2_cdotc(j, dA(0,j), 1); // including cdotc product and update a(j,j)
            if (j < n) {
                #if defined(PRECISION_z) || defined(PRECISION_c)
                clacgv(j, dA(0, j), 1);
                #endif
                magma_cgemv( MagmaTrans, j, n-j-1,
                             alpha, dA(0, j+1), ldda,
                                    dA(0, j),   1,
                             beta,  dA(j, j+1), ldda);

                #if defined(PRECISION_z) || defined(PRECISION_c)
                clacgv(j, dA(0, j), 1);
                #endif
                cpotf2_csscal(n-j, dA(j,j), ldda);
            }
        }
    }
    else {
        for(j = 0; j < n; j++) {
            cpotf2_cdotc(j, dA(j,0), ldda); // including cdotc product and update a(j,j)
            if (j < n) {
                #if defined(PRECISION_z) || defined(PRECISION_c)
                clacgv(j, dA(j, 0), ldda);
                #endif
                magma_cgemv( MagmaNoTrans, n-j-1, j,
                             alpha, dA(j+1, 0), ldda,
                                    dA(j,0),    ldda,
                             beta,  dA(j+1, j), 1 );

                #if defined(PRECISION_z) || defined(PRECISION_c)
                clacgv(j, dA(j, 0), ldda);
                #endif
                cpotf2_csscal(n-j, dA(j,j), 1);
            }
        }
    }

    return *info;
}

#define csscal_bs  32
#define cdotc_bs  512
#define clacgv_bs 512

// dynamically allocated shared memory, set to size number of threads when the kernel is launched.
// See CUDA Guide B.2.3
extern __shared__ float shared_data[];

__global__ void kernel_cdotc(int n, magmaFloatComplex *x, int incx, int threadSize)
{
    int tx = threadIdx.x;

    float *sdata = shared_data;

    magmaFloatComplex res = MAGMA_C_ZERO;

    if (tx < n) {
       res = x[tx*incx];
    }

    sdata[tx] = MAGMA_C_REAL(res * MAGMA_C_CNJG(res));

    __syncthreads();

    for(int s = blockDim.x/2; s > 32; s >>= 1 ) {
        if (tx < s) {
            sdata[tx] += sdata[tx+s];
        }
        __syncthreads();
    }

    if (tx < 32) {
        volatile float* smem = sdata;
        smem[tx] += smem[tx+32];
        smem[tx] += smem[tx+16];
        smem[tx] += smem[tx+8];
        smem[tx] += smem[tx+4];
        smem[tx] += smem[tx+2];
        smem[tx] += smem[tx+1];
    }

    if (tx == 0) {
        float xreal = MAGMA_C_REAL(x[n*incx]);
        x[n*incx] = MAGMA_C_MAKE( sqrt(xreal - sdata[0]), 0 );
    }
}

void cpotf2_cdotc(magma_int_t n, magmaFloatComplex *x, magma_int_t incx)
{
/*
    Specialized Cdotc
    1) performs cdotc sum = x[0:n-1]*conj(x[0:n-1])
    2) updates x[n] = sqrt(x[n]-sum);

*/
    if (n > cdotc_max_bs) {
        fprintf( stderr, "n = %d > %d is not supported in cpotf2_cdotc\n", (int) n, (int) cdotc_max_bs);
        return;
    }
    int threadSize;

    if (n <= 1024 && n > 512) {
        threadSize = 1024;
    }
    else if (n <= 512 && n > 256 ) {
        threadSize = 512;
    }
    else if (n <= 256 && n > 128) {
        threadSize = 256;
    }
    else if (n <= 128 && n > 64) {
        threadSize = 128;
    }
    else {
        threadSize = 64;
    }

    kernel_cdotc<<< 1, threadSize, threadSize * sizeof(float), magma_stream>>> (n, x, incx, threadSize);
}

__global__ void kernel_csscal(int n, magmaFloatComplex *x, int incx)
{
    int id = blockIdx.x * csscal_bs + threadIdx.x;

    __shared__ magmaFloatComplex factor;

    if (threadIdx.x == 0) {
        factor = MAGMA_C_MAKE(1.0/MAGMA_C_REAL(x[0]), 0.0);
    }

    __syncthreads();

    if ( id < n && id >0) {
        x[id*incx] = x[id*incx] * factor;
    }
}


void cpotf2_csscal(magma_int_t n, magmaFloatComplex *x, magma_int_t incx)
{
/*
    Specialized Csscal perform x[1:n-1]/x[0]

*/
    dim3 threads(csscal_bs, 1, 1);
    int num_blocks = (n - 1)/csscal_bs + 1;
    dim3 grid(num_blocks,1);
    kernel_csscal<<< grid, threads, 0, magma_stream >>> (n, x, incx);
}


#if defined(PRECISION_z) || defined(PRECISION_c)

__global__ void kernel_clacgv(int n, magmaFloatComplex *x, int incx)
{
    int id = blockIdx.x * clacgv_bs + threadIdx.x;

    if ( id < n ) {
        x[id*incx] = MAGMA_C_CNJG(x[id*incx]);
    }
}


/**
    Purpose
    -------

    CLACGV conjugates a complex vector of length N.

    Arguments
    ---------

    @param[in]
    n       INTEGER
            The length of the vector X.  N >= 0.

    @param[in,out]
    x       COMPLEX array, dimension
                           (1+(N-1)*abs(INCX))
            On entry, the vector of length N to be conjugated.
            On exit, X is overwritten with conjg(X).

    @param[in]
    incx    INTEGER
            The spacing between successive elements of X.

    @ingroup magma_cposv_aux
    ********************************************************************/
void clacgv(magma_int_t n, magmaFloatComplex *x, magma_int_t incx)
{
    dim3 threads(clacgv_bs, 1, 1);
    int num_blocks = (n - 1)/clacgv_bs + 1;
    dim3 grid(num_blocks,1);
    kernel_clacgv<<< grid, threads, 0, magma_stream >>> (n, x, incx);
}

#endif // defined(PRECISION_z) || defined(PRECISION_c)
