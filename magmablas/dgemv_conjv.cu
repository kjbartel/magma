#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015
       
       @author Stan Tomov

       @generated from zgemv_conjv.cu normal z -> d, Fri Jan 30 19:00:08 2015
*/
#include "common_magma.h"
#include "commonblas_d.h"

#define PRECISION_d

#define num_threads 256


__global__ void
dgemv_conjv_kernel(
    int m, int n, double alpha,
    const double * __restrict__ A, int lda,
    const double * __restrict__ x, int incx, double beta,
    double *       __restrict__ y, int incy)
{
    int ind = blockIdx.x*num_threads + threadIdx.x;
    
    A += ind;

    if ( ind < m ) {
        double res = MAGMA_D_ZERO;
        
        #pragma unroll
        for( int i=0; i < n; i ++ ) {
            res += A[0] * MAGMA_D_CNJG(x[0]);
            A += lda;
            x += incx;
        }
        
        y[ind*incy] = alpha * res + beta * y[ind*incy];
    }
}


/**
    Purpose
    -------
    DGEMV_CONJV performs the matrix-vector operation
    
        y := alpha*A*conj(x)    + beta*y, 
    
    where alpha and beta are scalars, x and y are vectors and A is an
    m by n matrix.

    Arguments
    ----------
    @param[in]
    m       INTEGER
            On entry, m specifies the number of rows of the matrix A.

    @param[in]
    n       INTEGER
            On entry, n specifies the number of columns of the matrix A

    @param[in]
    alpha   DOUBLE_PRECISION
            On entry, ALPHA specifies the scalar alpha.

    @param[in]
    dA      DOUBLE_PRECISION array of dimension ( LDA, n ) on the GPU.

    @param[in]
    lda     INTEGER
            LDA specifies the leading dimension of A.

    @param[in]
    dx      DOUBLE_PRECISION array of dimension n

    @param[in]
    incx    Specifies the increment for the elements of X.
            INCX must not be zero.

    @param[in]
    beta    DOUBLE REAL
            On entry, BETA specifies the scalar beta. When BETA is
            supplied as zero then Y need not be set on input.

    @param[out]
    dy      DOUBLE PRECISION array of dimension m

    @param[in]
    incy    Specifies the increment for the elements of Y.
            INCY must not be zero.

    @ingroup magma_dblas2
    ********************************************************************/
extern "C" void
magmablas_dgemv_conjv(
    magma_int_t m, magma_int_t n, double alpha,
    magmaDouble_const_ptr dA, magma_int_t ldda,
    magmaDouble_const_ptr dx, magma_int_t incx,
    double beta,
    magmaDouble_ptr dy, magma_int_t incy)
{
    magma_int_t info = 0;
    if ( m < 0 )
        info = -1;
    else if ( n < 0 )
        info = -2;
    else if ( ldda < m )
        info = -5;
    else if ( incx == 0 )
        info = -7;
    else if ( incy == 0 )
        info = -10;
    
    if (info != 0) {
        magma_xerbla( __func__, -(info) );
        return;  //info;
    }
    
    magma_int_t blocks = (m - 1)/num_threads + 1;
    dim3 grid(blocks, 1, 1);
    dim3 threads(num_threads, 1, 1);

    dgemv_conjv_kernel<<< grid, threads, 0, magma_stream >>>
            (m, n, alpha, dA, ldda, dx, incx, beta, dy, incy);

}

#undef num_threads
