#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.5.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date September 2014

       @generated from dznrm2.cu normal z -> c, Wed Sep 17 15:08:23 2014

*/
#include "common_magma.h"
#include "commonblas_c.h"
#include "magma_templates.h"

// 512 is maximum number of threads for CUDA capability 1.x
#define BLOCK_SIZE  512
#define BLOCK_SIZEx  32
#define BLOCK_SIZEy  16

#define PRECISION_c


//==============================================================================

__global__ void
magmablas_scnrm2_kernel( int m, magmaFloatComplex *da, int ldda, float *dxnorm )
{
    const int tx = threadIdx.x;
    magmaFloatComplex *dx = da + blockIdx.x * ldda;

    __shared__ float sum[ BLOCK_SIZE ];
    float re, lsum;

    // get norm of dx
    lsum = 0;
    for( int j = tx; j < m; j += BLOCK_SIZE ) {
        #if (defined(PRECISION_s) || defined(PRECISION_d))
        re = dx[j];
        lsum += re*re;
        #else
        re = MAGMA_C_REAL( dx[j] );
        float im = MAGMA_C_IMAG( dx[j] );
        lsum += re*re + im*im;
        #endif
    }
    sum[tx] = lsum;
    magma_sum_reduce< BLOCK_SIZE >( tx, sum );
    
    if (tx==0)
        dxnorm[blockIdx.x] = sqrt(sum[0]);
}


//==============================================================================
__global__ void
magmablas_scnrm2_check_kernel( int m, magmaFloatComplex *da, int ldda, float *dxnorm, 
                               float *lsticc )
{
    const int tx = threadIdx.x;
    magmaFloatComplex *dx = da + blockIdx.x * ldda;

    __shared__ float sum[ BLOCK_SIZE ];
    float re, lsum;

    // get norm of dx only if lsticc[blockIdx+1] != 0
    if ( lsticc[blockIdx.x + 1] == 0 )
        return;

    lsum = 0;
    for( int j = tx; j < m; j += BLOCK_SIZE ) {
        #if (defined(PRECISION_s) || defined(PRECISION_d))
        re = dx[j];
        lsum += re*re;
        #else
        re = MAGMA_C_REAL( dx[j] );
        float im = MAGMA_C_IMAG( dx[j] );
        lsum += re*re + im*im;
        #endif
    }
    sum[tx] = lsum;
    magma_sum_reduce< BLOCK_SIZE >( tx, sum );
    
    if (tx==0)
        dxnorm[blockIdx.x] = sqrt(sum[0]);
}

extern "C" void
magmablas_scnrm2_check(
    magma_int_t m, magma_int_t n, magmaFloatComplex *da, magma_int_t ldda, 
    float *dxnorm, float *lsticc) 
{
    dim3  blocks( n );
    dim3 threads( BLOCK_SIZE );
    
    magmablas_scnrm2_check_kernel<<< blocks, threads >>>( m, da, ldda, dxnorm, lsticc );
}


//==============================================================================
__global__ void
magmablas_scnrm2_smkernel( int m, int n, magmaFloatComplex *da, int ldda,
                           float *dxnorm )
{
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    __shared__ float sum[ BLOCK_SIZEx ][ BLOCK_SIZEy + 1];
    float re, lsum;

    for( int k = ty; k < n; k += BLOCK_SIZEy ) {
        magmaFloatComplex *dx = da + k * ldda;

        // get norm of dx
        lsum = 0;
        for( int j = tx; j < m; j += BLOCK_SIZEx ) {
            #if (defined(PRECISION_s) || defined(PRECISION_d))
            re = dx[j];
            lsum += re*re;
            #else
            re = MAGMA_C_REAL( dx[j] );
            float im = MAGMA_C_IMAG( dx[j] );
            lsum += re*re + im*im;
            #endif
        }
        sum[tx][ty] = lsum;
        magma_sum_reduce_2d< BLOCK_SIZEx, BLOCK_SIZEy+1 >( tx, ty, sum );

        if (tx == 0)
            dxnorm[k] = sqrt(sum[0][ty]);
        __syncthreads();
    }
}


//==============================================================================
/*
    Compute the scnrm2 of each column of m-by-n matrix dA.
    The resulting norms are written in the dxnorm array.
    This routine uses only one SM (block).
*/
extern "C" void
magmablas_scnrm2_sm(
    magma_int_t m, magma_int_t n, magmaFloatComplex *da, magma_int_t ldda,
    float *dxnorm)
{
    dim3  blocks( 1 );
    dim3 threads( BLOCK_SIZEx, BLOCK_SIZEy );

    magmablas_scnrm2_smkernel<<< blocks, threads, 0, magma_stream >>>( m, n, da, ldda, dxnorm );
}

//==============================================================================
extern "C"
__global__ void
magma_scnrm2_adjust_kernel(float *xnorm, magmaFloatComplex *c)
{
    const int tx = threadIdx.x;

    __shared__ float sum[ BLOCK_SIZE ];
    float temp;

    temp = MAGMA_C_ABS( c[tx] ) / xnorm[0];
    sum[tx] = -temp * temp;
    magma_sum_reduce_n( blockDim.x, tx, sum );

    __syncthreads();
    if (tx == 0)
        xnorm[0] = xnorm[0] * sqrt(1+sum[0]);
}


/*
    Adjust the norm of c to give the norm of c[k+1:], assumin that
    c was changed with orthogonal transformations.
*/
extern "C" void
magmablas_scnrm2_adjust(magma_int_t k, float *xnorm, magmaFloatComplex *c)
{
    magma_scnrm2_adjust_kernel<<< 1, k, 0, magma_stream >>> (xnorm, c);
}

//==============================================================================

#define BS 256

__global__ void
magma_scnrm2_row_check_adjust_kernel(int n, float tol, float *xnorm, float *xnorm2, 
                                     magmaFloatComplex *c, int ldc, float *lsticc)
{
    const int tx = threadIdx.x + blockIdx.x*BS;
    lsticc[tx+1] = 0;

    if (tx < n) {
        float temp = MAGMA_C_ABS( c[tx*ldc] ) / xnorm[tx];
        temp = max( 0.0, ((1.0 + temp) * (1.0 - temp)) );
        
        
        float temp2 = xnorm[tx] / xnorm2[tx];
        temp2 = temp * (temp2 * temp2);
        
        if (temp2 <= tol) {
            lsticc[tx+1] = 1;
        } else {
            xnorm[tx] *= sqrt(temp);
        }
    }
    if (tx == 0)
        lsticc[0] = 0;
    magma_sum_reduce_n( blockDim.x, tx, lsticc );
}

/*
    Adjust the norm of c[,1:k] to give the norm of c[k+1:,1:k], assuming that
    c was changed with orthogonal transformations.
    It also do checks for QP3
*/
extern "C" void
magmablas_scnrm2_row_check_adjust(
    magma_int_t k, float tol, float *xnorm, float *xnorm2, 
    magmaFloatComplex *c, magma_int_t ldc, float *lsticc)
{
    int nblocks = (k+BS-1)/BS;
    magma_scnrm2_row_check_adjust_kernel<<< nblocks, BS >>> (k, tol, xnorm, xnorm2, c, ldc, lsticc);
}

//==============================================================================

/*
    Compute the scnrm2 of each column of m-by-n matrix dA.
    The resulting norms are written in the dxnorm array. 
    The computation can be done using n blocks (default) or on one SM (commented).
*/
extern "C" void
magmablas_scnrm2_cols(
    magma_int_t m, magma_int_t n,
    magmaFloatComplex *da, magma_int_t ldda, 
    float *dxnorm) 
{
    dim3  blocks( n );
    dim3 threads( BLOCK_SIZE );
    
    magmablas_scnrm2_kernel<<< blocks, threads, 0, magma_stream >>>( m, da, ldda, dxnorm );

    // The following would do the computation on one SM
    // magmablas_scnrm2_sm(m, n, da, ldda, dxnorm);
}

//==============================================================================
