#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.4.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       December 2013

       @generated d Tue Dec 17 13:18:45 2013

*/
#include "common_magma.h"

// 512 is maximum number of threads for CUDA capability 1.x
//#if (GPUSHMEM < 200)
   #define BLOCK_SIZE 512
//#else
//   #define BLOCK_SIZE 768
//#endif

#define BLOCK_SIZEx  32
#define BLOCK_SIZEy  16

// ----------------------------------------
// Does sum reduction of array x, leaving total in x[0].
// Contents of x are destroyed in the process.
// With k threads, can reduce array up to 2*k in size.
// Assumes number of threads <= 1024 (which is max number of threads up to CUDA capability 3.0)
// Having n as template parameter allows compiler to evaluate some conditions at compile time.
template< int n >
__device__ void sum_reduce( /*int n,*/ int i, double* x )
{
    __syncthreads();
    if ( n > 1024 ) { if ( i < 1024 && i + 1024 < n ) { x[i] += x[i+1024]; }  __syncthreads(); }
    if ( n >  512 ) { if ( i <  512 && i +  512 < n ) { x[i] += x[i+ 512]; }  __syncthreads(); }
    if ( n >  256 ) { if ( i <  256 && i +  256 < n ) { x[i] += x[i+ 256]; }  __syncthreads(); }
    if ( n >  128 ) { if ( i <  128 && i +  128 < n ) { x[i] += x[i+ 128]; }  __syncthreads(); }
    if ( n >   64 ) { if ( i <   64 && i +   64 < n ) { x[i] += x[i+  64]; }  __syncthreads(); }
    if ( n >   32 ) { if ( i <   32 && i +   32 < n ) { x[i] += x[i+  32]; }  __syncthreads(); }
    // probably don't need __syncthreads for < 16 threads
    // because of implicit warp level synchronization.
    if ( n >   16 ) { if ( i <   16 && i +   16 < n ) { x[i] += x[i+  16]; }  __syncthreads(); }
    if ( n >    8 ) { if ( i <    8 && i +    8 < n ) { x[i] += x[i+   8]; }  __syncthreads(); }
    if ( n >    4 ) { if ( i <    4 && i +    4 < n ) { x[i] += x[i+   4]; }  __syncthreads(); }
    if ( n >    2 ) { if ( i <    2 && i +    2 < n ) { x[i] += x[i+   2]; }  __syncthreads(); }
    if ( n >    1 ) { if ( i <    1 && i +    1 < n ) { x[i] += x[i+   1]; }  __syncthreads(); }
}
// end sum_reduce

static
__device__ void zsum_reduce( int n, int i, double* x )
{
    __syncthreads();
    if ( n >  128 ) { if ( i <  128 && i +  128 < n ) { x[i] += x[i+ 128]; }  __syncthreads(); }
    if ( n >   64 ) { if ( i <   64 && i +   64 < n ) { x[i] += x[i+  64]; }  __syncthreads(); }
    if ( n >   32 ) { if ( i <   32 && i +   32 < n ) { x[i] += x[i+  32]; }  __syncthreads(); }
    // probably don't need __syncthreads for < 16 threads
    // because of implicit warp level synchronization.
    if ( n >   16 ) { if ( i <   16 && i +   16 < n ) { x[i] += x[i+  16]; }  __syncthreads(); }
    if ( n >    8 ) { if ( i <    8 && i +    8 < n ) { x[i] += x[i+   8]; }  __syncthreads(); }
    if ( n >    4 ) { if ( i <    4 && i +    4 < n ) { x[i] += x[i+   4]; }  __syncthreads(); }
    if ( n >    2 ) { if ( i <    2 && i +    2 < n ) { x[i] += x[i+   2]; }  __syncthreads(); }
    if ( n >    1 ) { if ( i <    1 && i +    1 < n ) { x[i] += x[i+   1]; }  __syncthreads(); }
}


//==============================================================================

__global__
void magma_dlarfx_kernel( int m, double *v, double *tau,
                         double *c, int ldc, double *xnorm,
                         double *T, int it )
{
    if ( !MAGMA_D_EQUAL(*tau, MAGMA_D_ZERO) ) {
        const int i = threadIdx.x;
        //double *dc = c + (blockIdx.x-it-1) * ldc;
        double *dc = c + (blockIdx.x) * ldc;

        __shared__ double sum[ BLOCK_SIZE ];
        double lsum;

        /*  w := v' * C  */
        lsum = MAGMA_D_ZERO;
        for( int j = i; j < m; j += BLOCK_SIZE ){
            if (j==0){
               lsum += MAGMA_D_MUL( MAGMA_D_ONE, dc[j] );
               v[j] = MAGMA_D_ONE;
            }
            else
               lsum += MAGMA_D_MUL( MAGMA_D_CNJG( v[j] ), dc[j] );
        }
        sum[i] = lsum;
        sum_reduce< BLOCK_SIZE >( i, sum );

        /*  C := C - v * w  */
        __syncthreads();
        double z__1 = - MAGMA_D_CNJG(*tau) * sum[0];
        if (blockIdx.x>it){
           for( int j = m-i-1; j>=0 ; j -= BLOCK_SIZE )
                 dc[j] += z__1 * v[j];
           __syncthreads();

           /* Adjust the rest of the column norms */
           if (i==0){
             double temp = MAGMA_D_ABS( dc[0] ) / xnorm[blockIdx.x-it-1];
             temp = (temp + 1.) * (1. - temp);
             xnorm[blockIdx.x-it-1] = xnorm[blockIdx.x-it-1] * sqrt(temp); 
           }
        }
        else
        {
           if (blockIdx.x==it)
              *(T+it) = *tau;
           else
              *(T+blockIdx.x) = MAGMA_D_CNJG(z__1);
        }
    }
}

//==============================================================================

__global__
void magma_dtrmv_kernel(const double *T, int ldt, double *t)
{
   const int i = threadIdx.x;
   T += i;

   __shared__ double tlocal[ BLOCK_SIZE ];
   double res = MAGMA_D_MAKE(0., 0.);

   tlocal[i] = t[i];
   __syncthreads();

   #pragma unroll
   for(int j=0; j<blockDim.x; j++)
      res +=  T[j*ldt]*tlocal[j];

   t[i] = res;
}

__global__
void magma_dtrmv_kernel2(const double *T, int ldt, double *t, 
                         double *y, double *tau)
{
   const int i = threadIdx.x;
   T += blockIdx.x;

   __shared__ double sum[ 128 ];

   sum[i] = T[i*ldt]*t[i];
   zsum_reduce(blockDim.x, i, sum);

   __syncthreads();

   if (i==0){
      y[blockIdx.x] = sum[0];
      if (blockIdx.x==0)
         y[gridDim.x] = tau[0];
   }
}

//==============================================================================

__global__
void magma_dtrmv_tkernel(double *T, int ldt, double *t, double *y)
{
   const int i = threadIdx.x;
   T += blockIdx.x*ldt;

   __shared__ double sum[ 128 ];

   sum[i] = MAGMA_D_CNJG(T[i])*t[i];
   zsum_reduce(blockDim.x, i, sum);

   __syncthreads();

   if (i==0)
      y[blockIdx.x] = sum[0];
}

//==============================================================================

/*
    Apply a real elementary reflector H to a real M-by-N
    matrix C from the left. H is represented in the form
          H = I - tau * v * v'
    where tau is a real scalar and v is a real vector.
    If tau = 0, then H is taken to be the unit matrix.

    To apply H' (the conjugate transpose of H), supply conjg(tau) 
    instead tau.

    The norms of v(:, 1:n) are given as input in xnorm(1:n). On exit, the norms
    are adjusted to hold the norms of v(2:m,2:n). This is a difference with the 
    LAPACK's dlarf routine. 
 */
extern "C" void
magma_dlarfx_gpu(magma_int_t m, magma_int_t n, double *v, double *tau,
                double *c, magma_int_t ldc, double *xnorm, 
                double *T, magma_int_t i, double *work )
{
    magma_int_t N = n + i + 1;

    if (i==0)
        magma_dlarfx_kernel<<< N, BLOCK_SIZE, 0, magma_stream >>>( m, v, tau, c, ldc, xnorm, T+i*N, i);
    else
        magma_dlarfx_kernel<<< N, BLOCK_SIZE, 0, magma_stream >>>( m, v, tau, c, ldc, xnorm, work, i);

    if (i > 0){
        //magma_dtrmv_kernel<<< 1, i, 0, magma_stream >>>( T, N, T+i*N);
        magma_dtrmv_kernel2<<< i, i, 0, magma_stream  >>>( T, N, work, T+i*N, tau);
    }
}

//==============================================================================
