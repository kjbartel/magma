#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @generated from zclaswp.cu mixed zc -> ds, Fri Jan 30 19:00:08 2015

*/
#include "common_magma.h"

#define NB 64

// TODO check precision, as in dlag2s?

__global__ void
dslaswp_kernel(int n, double *A, int lda, float *SA, int m, const magma_int_t *ipiv)
{
    int ind = blockIdx.x*NB + threadIdx.x;
    int newind;
    float res;
    
    if (ind < m) {
        SA   += ind;
        ipiv += ind;
        
        newind = ipiv[0];
        
        for(int i=0; i < n; i++) {
            res = MAGMA_S_MAKE( (float)(A[newind+i*lda]),
                                (float)(A[newind+i*lda]) );
            SA[i*lda] = res; 
        }
    }
}

__global__ void
dslaswp_inv_kernel(int n, double *A, int lda, float *SA, int m, const magma_int_t *ipiv)
{
    int ind = blockIdx.x*NB + threadIdx.x;
    int newind;
    double res;

    if (ind < m) {
        A    += ind;
        ipiv += ind;

        newind = ipiv[0];

        for(int i=0; i < n; i++) {
            res = MAGMA_D_MAKE( (double)(SA[newind+i*lda]),
                                (double)(SA[newind+i*lda]) );
            A[i*lda] = res;
        }
    }
}


/**
    Purpose
    -------
    Row i of  A is cast to single precision in row ipiv[i] of SA (incx > 0), or
    row i of SA is cast to double precision in row ipiv[i] of  A (incx < 0),
    for 0 <= i < M.

    @param[in]
    n       INTEGER.
            On entry, N specifies the number of columns of the matrix A.

    @param[in,out]
    A       DOUBLE PRECISION array on the GPU, dimension (LDA,N)
            On entry, the M-by-N matrix to which the row interchanges will be applied.
            TODO update docs

    @param[in]
    lda     INTEGER.
            LDA specifies the leading dimension of A.

    @param[in,out]
    SA      REAL array on the GPU, dimension (LDA,N)
            On exit, the single precision, permuted matrix.
            TODO update docs
        
    @param[in]
    m       The number of rows to be interchanged.

    @param[in]
    ipiv    INTEGER array on the GPU, dimension (M)
            The vector of pivot indices. Row i of A is cast to single 
            precision in row ipiv[i] of SA, for 0 <= i < m. 

    @param[in]
    incx    INTEGER
            If INCX is negative, the pivots are applied in reverse order,
            otherwise in straight-forward order.
    
    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_daux2
    ********************************************************************/
extern "C" void
magmablas_dslaswp_q(
    magma_int_t n,
    magmaDouble_ptr A, magma_int_t lda,
    magmaFloat_ptr SA, magma_int_t m,
    const magma_int_t *ipiv, magma_int_t incx,
    magma_queue_t queue )
{
    int blocks = (m - 1)/NB + 1;
    dim3 grid(blocks, 1, 1);
    dim3 threads(NB, 1, 1);

    if (incx >= 0)
        dslaswp_kernel<<< grid, threads, 0, queue >>>(n, A, lda, SA, m, ipiv);
    else
        dslaswp_inv_kernel<<< grid, threads, 0, queue >>>(n, A, lda, SA, m, ipiv);
}


/**
    @see magmablas_dslaswp_q
    @ingroup magma_daux2
    ********************************************************************/
extern "C" void
magmablas_dslaswp(
    magma_int_t n,
    magmaDouble_ptr A, magma_int_t lda,
    magmaFloat_ptr SA, magma_int_t m,
    const magma_int_t *ipiv, magma_int_t incx )
{
    magmablas_dslaswp_q( n, A, lda, SA, m, ipiv, incx, magma_stream );
}
