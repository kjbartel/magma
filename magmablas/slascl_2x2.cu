#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2014

       @generated from zlascl_2x2.cu normal z -> s, Sat Nov 15 19:53:59 2014

       @author Ichitaro Yamazaki
*/
#include "common_magma.h"

#define NB 64
#define A(i,j) (A[(i) + (j)*lda])
#define W(i,j) (W[(i) + (j)*ldw])


// each thread block does one NB x n block row of A.
// each thread does one row, starting from left edge and moving right.
__global__ void
slascl_2x2_full(int m, const float* W, int ldw, float* A, int lda)
{
    int ind = blockIdx.x * NB + threadIdx.x;

    float D21 = W( 1, 0 );
    float D11 = MAGMA_S_DIV( W( 1, 1 ), D21 );
    float D22 = MAGMA_S_DIV( W( 0, 0 ), MAGMA_S_CNJG( D21 ) );
    float T = 1.0 / ( MAGMA_S_REAL( D11*D22 ) - 1.0 );
    D21 = MAGMA_S_DIV( MAGMA_S_MAKE(T,0.0), D21 );

    if (ind < m) {
        A( ind, 0 ) = MAGMA_S_CNJG( D21 )*( D11*W( 2+ind, 0 )-W( 2+ind, 1 ) );
        A( ind, 1 ) = D21*( D22*W( 2+ind, 1 )-W( 2+ind, 0 ) );
    }
}

// each thread block does one NB x n block row of A.
// each thread does one row, starting from left edge and moving right to diagonal.
__global__ void
slascl_2x2_lower(int m, const float* W, int ldw, float* A, int lda)
{
    int ind = blockIdx.x * NB + threadIdx.x;

    float D21 = W( 1, 0 );
    float D11 = MAGMA_S_DIV( W( 1, 1 ), D21 );
    float D22 = MAGMA_S_DIV( W( 0, 0 ), MAGMA_S_CNJG( D21 ) );
    float T = 1.0 / ( MAGMA_S_REAL( D11*D22 ) - 1.0 );
    D21 = MAGMA_S_DIV( MAGMA_S_MAKE(T,0.0), D21 );

    if (ind < m) {
        A( ind, 0 ) = MAGMA_S_CNJG( D21 )*( D11*W( 2+ind, 0 )-W( 2+ind, 1 ) );
        A( ind, 1 ) = D21*( D22*W( 2+ind, 1 )-W( 2+ind, 0 ) );
    }
}


// each thread block does one NB x n block row of A.
// each thread does one row, starting from right edge and moving left to diagonal.
__global__ void
slascl_2x2_upper(int m, const float *W, int ldw, float* A, int lda)
{
    int ind = blockIdx.x * NB + threadIdx.x;

    float D21 = W( m, 1 );
    float D11 = MAGMA_S_DIV( W( m+1, 1 ), MAGMA_S_CNJG( D21 ) );
    float D22 = MAGMA_S_DIV( W( m, 0 ), D21 );
    float T = 1.0 / ( MAGMA_S_REAL( D11*D22 ) - 1.0 );
    D21 = MAGMA_S_DIV( MAGMA_S_MAKE(T,0.0), D21 );

    if (ind < m) {
        A( ind, 0 ) = D21*( D11*W( ind, 0 )-W( ind, 1 ) );
        A( ind, 1 ) = MAGMA_S_CNJG( D21 )*( D22*W( ind, 1 )-W( ind, 0 ) );
    }
}


/**
    Purpose
    -------
    SLASCL2 scales the M by N real matrix A by the real diagonal matrix dD.
    TYPE specifies that A may be full, upper triangular, lower triangular.

    Arguments
    ---------
    \param[in]
    type    magma_type_t
            TYPE indices the storage type of the input matrix A.
            = MagmaFull:   full matrix.
            = MagmaLower:  lower triangular matrix.
            = MagmaUpper:  upper triangular matrix.
            Other formats that LAPACK supports, MAGMA does not currently support.

    \param[in]
    m       INTEGER
            The number of rows of the matrix A.  M >= 0.

    \param[in]
    n       INTEGER
            The number of columns of the matrix A.  N >= 0.

    \param[in]
    dD      REAL vector, dimension (M)
            The diagonal matrix containing the scalar factors. Stored as a vector.

    \param[in,out]
    dA      REAL array, dimension (LDDA,N)
            The matrix to be scaled by dD.  See TYPE for the
            storage type.

    \param[in]
    ldda    INTEGER
            The leading dimension of the array A.  LDDA >= max(1,M).

    \param[out]
    info    INTEGER
      -     = 0:  successful exit
      -     < 0:  if INFO = -i, the i-th argument had an illegal value.

    @ingroup magma_saux2
    ********************************************************************/
extern "C" void
magmablas_slascl_2x2_q(
    magma_type_t type, magma_int_t m, 
    const float *dW, magma_int_t lddw, 
    float *dA, magma_int_t ldda, 
    magma_int_t *info, magma_queue_t queue )
{
    *info = 0;
    if ( type != MagmaLower && type != MagmaUpper && type != MagmaFull )
        *info = -1;
    else if ( m < 0 )
        *info = -2;
    else if ( ldda < max(1,m) )
        *info = -4;
    
    if (*info != 0) {
        magma_xerbla( __func__, -(*info) );
        return;  //info;
    }
    
    dim3 grid( (m + NB - 1)/NB );
    dim3 threads( NB );
    
    if (type == MagmaLower) {
        slascl_2x2_lower <<< grid, threads, 0, queue >>> (m, dW, lddw, dA, ldda);
    }
    else {
        slascl_2x2_upper <<< grid, threads, 0, queue >>> (m, dW, lddw, dA, ldda);
    }
}


/**
    @see magmablas_slascl2_q
    @ingroup magma_saux2
    ********************************************************************/
extern "C" void
magmablas_slascl_2x2(
    magma_type_t type, magma_int_t m, 
    float *dW, magma_int_t lddw, 
    float *dA, magma_int_t ldda, 
    magma_int_t *info )
{
    magmablas_slascl_2x2_q( type, m, dW, lddw, dA, ldda, info, magma_stream );
}
