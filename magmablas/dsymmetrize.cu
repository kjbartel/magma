#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.2.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       June 2012

       @generated d Thu Jun 28 12:31:20 2012
       @author Mark Gates
*/
#include "common_magma.h"

/*
    Matrix is divided into 64 x m block rows.
    Each block has 64 threads.
    Each thread copies one row, iterating across all columns below diagonal.
    The bottom block of rows may be partially outside the matrix;
    if so, rows outside the matrix (i >= m) are disabled.
*/
__global__ void
dsymmetrize_kernel_lower( int m, double *A, int lda )
{
    // A iterates across row i and AT iterates down column i.
    int i = blockIdx.x*64 + threadIdx.x;
    double *AT = A;
    if ( i < m ) {
        A  += i;
        AT += i*lda;
        double *Aend = A + i*lda;
        while( A < Aend ) {
            *AT = (*A);
            A  += lda;
            AT += 1;
        }
    }
}


// only difference with _lower version is direction A=AT instead of AT=A.
__global__ void
dsymmetrize_kernel_upper( int m, double *A, int lda )
{
    // A iterates across row i and AT iterates down column i.
    int i = blockIdx.x*64 + threadIdx.x;
    double *AT = A;
    if ( i < m ) {
        A  += i;
        AT += i*lda;
        double *Aend = A + i*lda;
        while( A < Aend ) {
            *A = (*AT);
            A  += lda;
            AT += 1;
        }
    }
}


extern "C" void
magmablas_dsymmetrize( char uplo, int m, double *A, int lda )
{
/*
  Purpose
  =======

  DSYMMETRIZE copies lower triangle to upper triangle, or vice-versa,
  to make A a general representation of a symmetric matrix.

  Arguments
  =========

  UPLO    (input) CHARACTER*1
          Specifies the part of the matrix A that is valid on input.
          = 'U':      Upper triangular part
          = 'L':      Lower triangular part

  M       (input) INTEGER
          The number of rows of the matrix A.  M >= 0.

  A       (input/output) COMPLEX DOUBLE PRECISION array, dimension (LDA,N)
          The m by m matrix A.

  LDA     (input) INTEGER
          The leading dimension of the array A.  LDA >= max(1,M).

  =====================================================================   */

    dim3 threads( 64 );
    dim3 grid( m/64 + (m%64 != 0) );
    
    //printf( "m %d, grid %d, threads %d\n", m, grid.x, threads.x );
    if ( m == 0 )
        return;
    
    if ( (uplo == 'U') || (uplo == 'u') ) {
        dsymmetrize_kernel_upper<<< grid, threads, 0, magma_stream >>>( m, A, lda );
    }
    else if ( (uplo == 'L') || (uplo == 'l') ) {
        dsymmetrize_kernel_lower<<< grid, threads, 0, magma_stream >>>( m, A, lda );
    }
    else {
        printf( "uplo has illegal value\n" );
        exit(1);
    }
}
