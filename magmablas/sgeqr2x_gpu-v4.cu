#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.4.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       December 2013

       @generated s Tue Dec 17 13:18:45 2013

*/
#include "common_magma.h"

//#if (GPUSHMEM < 200)
   #define BLOCK_SIZE 512
//#else
//   #define BLOCK_SIZE 768
//#endif

__global__ void 
magma_strmv_kernel2(const float *T, int ldt,
                    float *v, float *y, float *tau);

__global__ void 
magma_sgemv_kernel3(int m, const float * __restrict__ V, int ldv,
                    float *c, float *dwork,
                    float *tau);


//////////////////////////////////////////////////////////////////////////////

__global__ void
magma_sgemv_kernel1(int m, const float * __restrict__ V, int ldv,
                    const float * __restrict__ c,
                    float *dwork);
__global__ void
magma_sgemv_kernel2(int m, int n, const float * __restrict__ V, int ldv,
                    const float * __restrict__ x, float *c);
__global__ void 
magma_strmv_tkernel(float *T, int ldt, float *v,
                                    float *y);
__global__ void
magma_snrm2_adjust_kernel(float *xnorm, float *c);

extern "C" magma_int_t
magma_sgeqr2x4_gpu(magma_int_t *m, magma_int_t *n, float *dA, 
                   magma_int_t *ldda, float *dtau,
                   float *dT, float *ddA,
                   float *dwork, magma_int_t *info, magma_queue_t stream)
{
/*  -- MAGMA (version 1.4.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       December 2013

    Purpose   
    =======   
    SGEQR2 computes a QR factorization of a real m by n matrix A:   
    A = Q * R.

    This expert routine requires two more arguments than the standard 
    sgeqr2, namely, dT and ddA, explained below. The storage for A is 
    also not as in the LAPACK's sgeqr2 routine (see below). 

    The first is used to output the triangular 
    n x n factor T of the block reflector used in the factorization. 
    The second holds the diagonal nxn blocks of A, i.e., the diagonal
    submatrices of R. This routine implements the left looking QR.

    This version adds internal blocking.

    Arguments   
    =========   
    M       (input) INTEGER   
            The number of rows of the matrix A.  M >= 0.   

    N       (input) INTEGER   
            The number of columns of the matrix A.  N >= 0.   

    A       (input/output) REAL array, dimension (LDA,N)   
            On entry, the m by n matrix A.   
            On exit, the unitary matrix Q as a
            product of elementary reflectors (see Further Details).

            the elements on and above the diagonal of the array   
            contain the min(m,n) by n upper trapezoidal matrix R (R is   
            upper triangular if m >= n); the elements below the diagonal,   
            with the array TAU, represent the unitary matrix Q as a   
            product of elementary reflectors (see Further Details).   

    LDA     (input) INTEGER   
            The leading dimension of the array A.  LDA >= max(1,M).   

    TAU     (output) REAL array, dimension (min(M,N))   
            The scalar factors of the elementary reflectors (see Further   
            Details).   

    dT      (output) REAL array, dimension N x N.
            Stores the triangular N x N factor T of the block reflector 
            used in the factorization. The lower triangular part is 0.

    ddA     (output) REAL array, dimension N x N.
            Stores the elements of the upper N x N diagonal block of A.
            LAPACK stores this array in A. There are 0s below the diagonal.

    RWORK   (workspace) DOUBLE_PRECISION array, dimension (3 N)

    INFO    (output) INTEGER   
            = 0: successful exit   
            < 0: if INFO = -i, the i-th argument had an illegal value   

    Further Details   
    ===============   
    The matrix Q is represented as a product of elementary reflectors   

       Q = H(1) H(2) . . . H(k), where k = min(m,n).   

    Each H(i) has the form   

       H(i) = I - tau * v * v'   

    where tau is a real scalar, and v is a real vector with   
    v(1:i-1) = 0 and v(i) = 1; v(i+1:m) is stored on exit in A(i+1:m,i),   
    and tau in TAU(i).   
    =====================================================================    */

    #define da_ref(a_1,a_2) ( dA+(a_2)*(*ldda) + (a_1))
    #define dt_ref(a_1,a_2) ( dT+(a_2)*(k) + (a_1))
    #define BS 32

    magma_int_t i, k;

    float *dnorm = (float *)dwork;
    float *work = (float *)(dwork+2*(*n));

    magma_queue_t cstream;
    magmablasGetKernelStream(&cstream);
    magmablasSetKernelStream(stream);

    *info = 0;
    if (*m < 0) {
        *info = -1;
    } else if (*n < 0) {
        *info = -2;
    } else if (*ldda < max(1,*m)) {
        *info = -4;
    }
    if (*info != 0) {
        magma_xerbla( __func__, -(*info) );
        return *info;
    }

    /* Compute the norms of the trailing columns */
    k = min(*m,*n);
    magmablas_snrm2_cols(*m, k, da_ref(0,0), *ldda, dnorm);

    for (magma_int_t b=0; b < k; b += BS) {
        for (i = b; i < min(k, b+BS); ++i) {

            /*   Apply H' to A(:,i) from the left                           */    
            if ( i-b > 0){
                magma_sgemv_kernel3<<< i-1, BLOCK_SIZE, 0, magma_stream >>>( *m-i+1, da_ref(i-1,0), *ldda,
                                                    da_ref(i-1, i-1), work, dtau+i-1);
                magma_strmv_kernel2<<< i-1, i-1, 0, magma_stream >>>( dt_ref(0,0), k, work,
                                                    dt_ref(0,i-1), dtau+i-1);

                /* dwork = V' c                   */
                magma_sgemv_kernel1<<< i-b, BLOCK_SIZE, 0, magma_stream >>>(*m-b, da_ref(b, b), 
                             *ldda, da_ref(b,i), work);

                /* dwork = T' work                */
                magma_strmv_tkernel<<< i-b, i-b, 0, magma_stream >>>(dt_ref(b,b), k, work, work+i-b);

                /* c = c - V work                 */
                dim3  blocks3( (*m-b + BLOCK_SIZE-1) / BLOCK_SIZE );
                dim3 threads3( BLOCK_SIZE );
                magma_sgemv_kernel2<<< blocks3, threads3, 0, magma_stream >>>(*m-b, i-b, da_ref(b,b), *ldda, 
                                   work+i-b, da_ref(b, i));
            }

            /*   Adjust the dnorm[i] to hold the norm of A(i:m,i)           */ 
            if ( i > 0 )
                magma_snrm2_adjust_kernel<<< 1, i, 0, magma_stream >>> (dnorm+i, da_ref(0, i));
            
            /*  Generate elementary reflector H(i) to annihilate A(i+1:m,i) 
                1. 1 is not yet put on the diagonal of A
                2. Elements above the diagonal are copied in ddA and
                   the ones in A are set to zero                                         
                3. update T                                                 */
            magma_slarfgx_gpu(*m-i, da_ref(i, i), da_ref(min(i+1,*m),i), dtau+i, 
                              dnorm+i, ddA + i + i*(*n), i);

            if (i==0){
              float tt = MAGMA_S_ONE;
              magmablas_slacpy(MagmaUpperLower, 1, 1, dtau, 1, dt_ref(0,0), 1);
              magma_ssetmatrix(1,1, &tt,1, da_ref(i, i),1);
            }
/*
            else
             {
                // Compute the i-th column of T.
                //   Set da_ref(i, i) = 1.                                    
                magma_sgemv_kernel3<<< i, BLOCK_SIZE, 0, magma_stream >>>( *m-i, da_ref(i,0), *ldda, 
                                          da_ref(i, i), work, dtau+i);
                magma_strmv_kernel2<<< i, i, 0, magma_stream          >>>( dt_ref(0,0), k, work, 
                                                          dt_ref(0,i), dtau+i);
              }
*/

        }
        magma_sgemv_kernel3<<< i-1, BLOCK_SIZE, 0, magma_stream >>>( *m-i+1, da_ref(i-1,0), *ldda,
                                                    da_ref(i-1, i-1), work, dtau+i-1);
        magma_strmv_kernel2<<< i-1, i-1, 0, magma_stream >>>( dt_ref(0,0), k, work,
                                                    dt_ref(0,i-1), dtau+i-1);

        
        /* Apply the transformations to the trailing matrix. */
        //magma_slarfb2_gpu( MagmaLeft, MagmaTrans, MagmaForward, MagmaColumnwise,
        magma_slarfb2_gpu(
                           *m-b, k-i, BS,
                           da_ref(b, b), *ldda, dT+b+b*k, k,
                           da_ref(b, i), *ldda, work, k-i);
    }

    magmablasSetKernelStream(cstream);

    return *info;
} /* magma_sgeqr2 */
