#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.5.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date September 2014

       @generated from zgeadd.cu normal z -> d, Wed Sep 17 15:08:23 2014
       @author Mark Gates
*/
#include "common_magma.h"

#define BLK_X 64
#define BLK_Y 32

/*
    Divides matrix into ceil( m/BLK_X ) x ceil( n/BLK_Y ) blocks.
    Each block has BLK_X threads.
    Each thread loops across one row, updating BLK_Y entries.

    Code similar to dlaset.
*/
__global__
void dgeadd_full(
    int m, int n,
    double alpha,
    const double *dA, int ldda,
    double       *dB, int lddb )
{
    int ind = blockIdx.x*BLK_X + threadIdx.x;
    int iby = blockIdx.y*BLK_Y;
    /* check if full block-column */
    bool full = (iby + BLK_Y <= n);
    /* do only rows inside matrix */
    if ( ind < m ) {
        dA += ind + iby*ldda;
        dB += ind + iby*lddb;
        if ( full ) {
            // full block-column
            #pragma unroll
            for( int j=0; j < BLK_Y; ++j ) {
                dB[j*lddb] = alpha*dA[j*ldda] + dB[j*lddb];
            }
        }
        else {
            // partial block-column
            for( int j=0; j < BLK_Y && iby+j < n; ++j ) {
                dB[j*lddb] = alpha*dA[j*ldda] + dB[j*lddb];
            }
        }
    }
}


/**
    Purpose
    -------
    ZGEADD adds two matrices, dB = alpha*dA + dB.
    
    Arguments
    ---------
    
    @param[in]
    m       INTEGER
            The number of rows of the matrix dA.  M >= 0.
    
    @param[in]
    n       INTEGER
            The number of columns of the matrix dA.  N >= 0.
    
    @param[in]
    alpha   DOUBLE_PRECISION
            The scalar alpha.
            
    @param[in]
    dA      DOUBLE_PRECISION array, dimension (LDDA,N)
            The m by n matrix dA.
    
    @param[in]
    ldda    INTEGER
            The leading dimension of the array dA.  LDDA >= max(1,M).
            
    @param[in,out]
    dB      DOUBLE_PRECISION array, dimension (LDDB,N)
            The m by n matrix dB.
    
    @param[in]
    lddb    INTEGER
            The leading dimension of the array dB.  LDDB >= max(1,M).
    
    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_daux2
    ********************************************************************/
extern "C" void
magmablas_dgeadd_q(
    magma_int_t m, magma_int_t n,
    double alpha,
    const double *dA, magma_int_t ldda,
    double       *dB, magma_int_t lddb,
    magma_queue_t queue )
{
    magma_int_t info = 0;
    if ( m < 0 )
        info = -1;
    else if ( n < 0 )
        info = -2;
    else if ( ldda < max(1,m))
        info = -5;
    else if ( lddb < max(1,m))
        info = -7;
    
    if ( info != 0 ) {
        magma_xerbla( __func__, -(info) );
        return;
    }
    
    if ( m == 0 || n == 0 )
        return;
    
    dim3 threads( BLK_X );
    dim3 grid( (m + BLK_X - 1)/BLK_X, (n + BLK_Y - 1)/BLK_Y );
    
    dgeadd_full<<< grid, threads, 0, queue >>>
        ( m, n, alpha, dA, ldda, dB, lddb );
}


/**
    @see magmablas_dgeadd_q
    @ingroup magma_daux2
    ********************************************************************/
extern "C" void
magmablas_dgeadd(
    magma_int_t m, magma_int_t n,
    double alpha,
    const double *dA, magma_int_t ldda,
    double       *dB, magma_int_t lddb )
{
    magmablas_dgeadd_q( m, n, alpha, dA, ldda, dB, lddb, magma_stream );
}
