#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.4.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       December 2013

       @generated s Tue Dec 17 13:18:45 2013
       
       @author Stan Tomov
       @author Mathieu Faverge
       @author Ichitaro Yamazaki
       @author Mark Gates
*/
#include "common_magma.h"

// MAX_PIVOTS is maximum number of pivots to apply in each kernel launch
// NTHREADS is number of threads in a block
// 64 and 256 are better on Kepler; 
//#define MAX_PIVOTS 64
//#define NTHREADS   256
#define MAX_PIVOTS 32
#define NTHREADS   64

typedef struct {
    float *dAT;
    int n, lda, j0, npivots;
    int ipiv[MAX_PIVOTS];
} slaswp_params_t;


// Matrix A is stored row-wise in dAT.
// Divide matrix A into block-columns of NTHREADS columns each.
// Each GPU block processes one block-column of A.
// Each thread goes down a column of A,
// swapping rows according to pivots stored in params.
__global__ void slaswp_kernel( slaswp_params_t params )
{
    unsigned int tid = threadIdx.x + blockDim.x*blockIdx.x;
    if( tid < params.n ) {
        int lda = params.lda;
        float *dAT = params.dAT + tid + params.j0*lda;
        float *A1  = dAT;
        
        for( int i1 = 0; i1 < params.npivots; ++i1 ) {
            int i2 = params.ipiv[i1];
            float *A2 = dAT + i2*lda;
            float temp = *A1;
            *A1 = *A2;
            *A2 = temp;
            A1 += lda;  // A1 = dA + i1*ldx
        }
    }
}


// Launch slaswp kernel with ceil( n / NTHREADS ) blocks of NTHREADS threads each.
extern "C" void slaswp_launch( slaswp_params_t &params )
{
    int blocks = (params.n + NTHREADS - 1) / NTHREADS;
    slaswp_kernel<<< blocks, NTHREADS, 0, magma_stream >>>( params );
}


// Swap rows of A, stored row-wise.
// This version updates each entry of ipiv by adding ind.
// It is used in sgetrf, sgetrf_gpu, sgetrf_mgpu, sgetrf_ooc.
extern "C" void
magmablas_spermute_long2( magma_int_t n, float *dAT, magma_int_t lda,
                          magma_int_t *ipiv, magma_int_t nb, magma_int_t ind )
{
    for( int k = 0; k < nb; k += MAX_PIVOTS ) {
        int npivots = min( MAX_PIVOTS, nb-k );
        // fields are:             dAT  n  lda  j0       npivots
        slaswp_params_t params = { dAT, n, lda, ind + k, npivots };
        for( int j = 0; j < npivots; ++j ) {
            params.ipiv[j] = ipiv[ind + k + j] - k - 1;
            ipiv[ind + k + j] += ind;
        }
        slaswp_launch( params );
    }
}


// Swap rows of A, stored row-wise.
// This version assumes ind has already been added to ipiv.
// It is used in sgetrf_mgpu, sgetrf_ooc.
extern "C" void
magmablas_spermute_long3( float *dAT, magma_int_t lda,
                          const magma_int_t *ipiv, magma_int_t nb, magma_int_t ind )
{
    for( int k = 0; k < nb; k += MAX_PIVOTS ) {
        int npivots = min( MAX_PIVOTS, nb-k );
        // fields are:             dAT  n    lda  j0       npivots
        slaswp_params_t params = { dAT, lda, lda, ind + k, npivots };
        for( int j = 0; j < MAX_PIVOTS; ++j ) {
            params.ipiv[j] = ipiv[ind + k + j] - k - 1 - ind;
        }
        slaswp_launch( params );
    }
}


// Swap rows of A, stored row-wise.
// This interface is identical to LAPACK's laswp interface.
// It is used in sgessm, sgetrf_incpiv.
extern "C" void
magmablas_slaswp( magma_int_t n, float *dAT, magma_int_t lda,
                  magma_int_t i1, magma_int_t i2,
                  const magma_int_t *ipiv, magma_int_t inci )
{
    for( int k = i1-1; k < i2; k += MAX_PIVOTS ) {
        int npivots = min( MAX_PIVOTS, i2-k );
        // fields are:             dAT        n  lda  j0 npivots
        slaswp_params_t params = { dAT+k*lda, n, lda, 0, npivots };
        for( int j = 0; j < npivots; ++j ) {
            params.ipiv[j] = ipiv[(k+j)*inci] - k - 1;
        }
        slaswp_launch( params );
    }
}


// ------------------------------------------------------------
// Extended version has stride in both directions (ldx, ldy)
// to handle both row-wise and column-wise storage.

typedef struct {
    float *dA;
    int n, ldx, ldy, j0, npivots;
    int ipiv[MAX_PIVOTS];
} slaswpx_params_t;


// Matrix A is stored row-wise in dA.
// Divide matrix A into block-columns of NTHREADS columns each.
// Each GPU block processes one block-column of A.
// Each thread goes down a column of A,
// swapping rows according to pivots stored in params.
__global__ void slaswpx_kernel( slaswpx_params_t params )
{
    unsigned int tid = threadIdx.x + blockDim.x*blockIdx.x;
    if( tid < params.n ) {
        int ldx = params.ldx;
        float *dA = params.dA + tid*params.ldy + params.j0*ldx;
        float *A1  = dA;
        
        for( int i1 = 0; i1 < params.npivots; ++i1 ) {
            int i2 = params.ipiv[i1];
            float *A2 = dA + i2*ldx;
            float temp = *A1;
            *A1 = *A2;
            *A2 = temp;
            A1 += ldx;  // A1 = dA + i1*ldx
        }
    }
}


// Launch slaswpx kernel with ceil( n / NTHREADS ) blocks of NTHREADS threads each.
extern "C" void slaswpx( slaswpx_params_t &params )
{
    int blocks = (params.n + NTHREADS - 1) / NTHREADS;
    slaswpx_kernel<<< blocks, NTHREADS, 0, magma_stream >>>( params );
}


// Swap rows of A.
// For A stored row-wise,    set ldx=lda and ldy=1.
// For A stored column-wise, set ldx=1   and ldy=lda.
// Otherwise, this interface is identical to LAPACK's laswp interface.
extern "C" void
magmablas_slaswpx( magma_int_t n, float *dA, magma_int_t ldx, magma_int_t ldy,
                   magma_int_t i1, magma_int_t i2,
                   const magma_int_t *ipiv, magma_int_t inci )
{
    for( int k = i1-1; k < i2; k += MAX_PIVOTS ) {
        int npivots = min( MAX_PIVOTS, i2-k );
        // fields are:              dA        n  ldx  ldy  j0 npivots
        slaswpx_params_t params = { dA+k*ldx, n, ldx, ldy, 0, npivots };
        for( int j = 0; j < npivots; ++j ) {
            params.ipiv[j] = ipiv[(k+j)*inci] - k - 1;
        }
        slaswpx( params );
    }
}


// ------------------------------------------------------------
// This version takes d_ipiv on the GPU. Thus it does not pass pivots
// as an argument using a structure, avoiding all the argument size
// limitations of CUDA and OpenCL. It also needs just one kernel launch
// with all the pivots, instead of multiple kernel launches with small
// batches of pivots. On Fermi, it is faster than magmablas_slaswp
// (including copying pivots to the GPU).

__global__ void slaswp2_kernel( int n, float *dAT, int lda, int npivots, const magma_int_t* d_ipiv )
{
    unsigned int tid = threadIdx.x + blockDim.x*blockIdx.x;
    if( tid < n ) {
        dAT += tid;
        float *A1  = dAT;
        
        for( int i1 = 0; i1 < npivots; ++i1 ) {
            int i2 = d_ipiv[i1] - 1;  // Fortran index
            float *A2 = dAT + i2*lda;
            float temp = *A1;
            *A1 = *A2;
            *A2 = temp;
            A1 += lda;  // A1 = dA + i1*ldx
        }
    }
}

// Swap rows of A, stored row-wise.
// d_ipiv is vector of pivots stored on the GPU,
// unlike magmablas_slaswp where ipiv is stored on the CPU.
// This interface is identical to LAPACK's laswp interface.
extern "C" void
magmablas_slaswp2( magma_int_t n, float* dAT, magma_int_t lda,
                   magma_int_t i1, magma_int_t i2,
                   const magma_int_t *d_ipiv )
{
    int blocks = (n + NTHREADS - 1) / NTHREADS;
    slaswp2_kernel<<< blocks, NTHREADS, 0, magma_stream >>>(
        n, dAT + (i1-1)*lda, lda, i2-(i1-1), d_ipiv );
}
