#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.5.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date September 2014
       
       csymv.cu is nearly identical to chemv.cu, just change names and drop hipConjf.
       
       @generated from zhemv.cu normal z -> c, Wed Sep 17 15:08:23 2014
       
       @author Mark Gates
*/
#include "common_magma.h"

#define PRECISION_c

#define NB_X         64
#define NB_Y          4
#define bank_shift   33
#define quarter_NB_X 16
#define half_NB_X    32


/*******************************************************************************
    Lower case, compute block multiply, work = A*x, for any size n:
    
    [ A11*x1   A12*x2             A13*x3                    ]   [ A11 A12 A13 ]   [ x1 ]
    [  ---    (A21*x1 + A22*x2)   A23*x3                    ] = [ A21 A22 A23 ] * [ x2 ]
    [  ---      ---              (A31*x1 + A32*x2 + A33*x3) ]   [ A31 A32 A33 ]   [ x3 ]
    
    Uses a 64x4 thread block.
    For     diagonal tiles, covers a 64x64 tile using three 32x32 tiles (plus one gets transposed).
    For off-diagonal tiles, covers a 64x64 tile using four  64x16 tiles.
    In both cases, each thread multiplies 4 elements.
    
    For rows past the bottom of the matrix, the A pointer is adjusted to be the
    last valid row of A, which multiple threads will read.
    Extra rows are ignored when saving results to work.
    Columns past the right edge are explicitly ignored when loading.
    x values past the bottom are set to zero, thus, extra columns are zeroed
    when multiplying.
    ********************************************************************/
__global__ void
chemv_kernel_L(
    int n,
    const magmaFloatComplex * __restrict__ A, int lda,
    const magmaFloatComplex * __restrict__ x, int incx,
    magmaFloatComplex * __restrict__ work)
{
#if defined(PRECISION_s) || defined(PRECISION_d) || defined(PRECISION_c) || (__CUDA_ARCH__ >= 200)

    // treats sA as 16x64 block
    #define sA16(i_, j_) (sA[(i_)][(j_)])  // i.e., sA[ (i_)*(NB_X+3) + (j_) ]
    
    // treats sA as 32x32 block
    #define sA32(i_, j_) (sA[0][(i_) + bank_shift*(j_)])
    
    // 64x4 thread block
    const int tx  = threadIdx.x;
    const int ty  = threadIdx.y;
    const int blk = blockIdx.x;
    const int blk_ind = NB_X * blk;
    const int td  = NB_X * ty + tx;

    // 32x8 thread block
    const int tx2 = td % half_NB_X;
    const int ty2 = td / half_NB_X;

    // If this blk has fewer than NB_X rows, partial is the number of valid rows,
    // so tx = 0, ..., partial-1 are valid rows, and tx >= partial are invalid.
    // Else, partial == 0.
    const int partial = (blk == gridDim.x - 1 ? (n % NB_X) : 0);
    
    magmaFloatComplex psum, psum2;
    magmaFloatComplex total = MAGMA_C_ZERO;

    // sA is used as a 32x32 block, sA32(i,j),
    // and as a 16x64 block, sA16(i,j), in different parts of the code.
    // sA must be at least half_NB_X*bank_shift = 32x33 = 1056;
    // quarter_NB_X*(NB_X + 2) = 16*(64 + 2) = 1056
    __shared__ magmaFloatComplex sA [quarter_NB_X][NB_X + 3]; /* Why +3? seems it only needs +2. Does +3 reduce bank conflicts? */
    __shared__ magmaFloatComplex sx [NB_X];  // for x[ blk ]
    __shared__ magmaFloatComplex sx2[NB_X];  // for x[ blk2 ], which cycles over all blocks left of diag

    magmaFloatComplex rA[4];
    magmaFloatComplex psums[4];

    // --------------------
    // load 64x1 block x(blk_ind + 0:63) into sx
    x += (blk_ind + tx)*incx;  // x is x(blk_ind + tx)
    if ( ty == 0 ) {
        if ( partial && tx >= partial ) {
            sx[tx] = MAGMA_C_ZERO;
        }
        else {
            sx[tx] = x[0];
        }
    }

    // --------------------
    // move to 32x32 diag block
    A += blk_ind * (lda + 1);  // A is A(blk_ind, blk_ind)
    A += ty2*lda + tx2;        // A is A(blk_ind + tx2, blk_ind + ty2)

    // load 32x32 diag block A(blk_ind + 0:31, blk_ind + 0:31) into sA,
    // as four 32x8 sections one after another:
    // columns 0:7, then 8:15, then 16:23, then 24:31
    if ( partial ) {
        if ( tx2 >= partial ) {
            A = A - tx2 + (partial - 1);
        }
        #pragma unroll
        for(int j=0; j < half_NB_X; j += 8) {
            if ( ty2+j < partial ) {
                sA32(tx2, ty2 + j) = A[j*lda];
            }
        }
        if ( tx2 >= partial ) {
            A = A + tx2 - (partial - 1);
        }
    }
    else {
        #pragma unroll
        for(int j=0; j < half_NB_X; j += 8) {
            sA32(tx2, ty2 + j) = A[j*lda];
        }
    }
    __syncthreads();

    // symmetrize 32x32 diag block, copying lower to upper triangle,
    // as four 32x8 sections in parallel:
    // columns 0,4,8,12,16,20,24,28; then 1,5,...,29; then 2,6,...,30, then 3,7,...,31
    #pragma unroll
    for(int j=ty2*4; j < ty2*4 + 4; j++) {
        if ( j < tx2 )
            sA32(j, tx2) = hipConjf( sA32(tx2, j) );
    }
    __syncthreads();

    // multiply 32x32 diag block * x
    // each thread does partial row sA(tx2, ty2*4 : ty2*4 + 3)
    psum = MAGMA_C_ZERO;
    #pragma unroll
    for(int j=0; j < 4; j++) {
        psum += sA32(tx2, ty2*4 + j) * sx[ty2*4 + j];
    }
    __syncthreads();

    // store partial row sums
    sA32(ty2, tx2) = psum;
    __syncthreads();

    // sum up partial row sums, so thread (tx2,0) has total for row (blk_ind + tx2)
    if ( ty2 == 0 ) {
        total = sA32(0, tx2) + sA32(1, tx2)
              + sA32(2, tx2) + sA32(3, tx2)
              + sA32(4, tx2) + sA32(5, tx2)
              + sA32(6, tx2) + sA32(7, tx2);
    }
    __syncthreads();

    // --------------------
    // move to next 32x32 diag block, then repeat steps from first diag block
    A += half_NB_X + half_NB_X*lda;  // A is A(blk_ind + NB/2 + tx2, blk_ind + NB/2 + ty2)

    // load 32x32 diag block A[block + 0:31, block + 0:31] into sA
    if ( partial ) {
        if ( tx2 + half_NB_X >= partial ) {
            A = A - (tx2 + half_NB_X) + (partial - 1);
        }
        #pragma unroll
        for(int j=0; j < half_NB_X; j += 8) {
            if ( ty2+j + half_NB_X < partial ) {
                sA32(tx2, ty2 + j) = A[j*lda];
            }
        }
        if ( tx2 + half_NB_X >= partial ) {
            A = A + (tx2 + half_NB_X) - (partial - 1);
        }
    }
    else {
        #pragma unroll
        for(int j=0; j < half_NB_X; j += 8) {
            sA32(tx2, ty2 + j) = A[j*lda];
        }
    }
    __syncthreads();

    // symmetrize 32x32 diag block, copying lower to upper triangle
    #pragma unroll
    for(int j=ty2*4; j < ty2*4 + 4; j++) {
        if ( j < tx2 )
            sA32(j, tx2) = hipConjf( sA32(tx2, j) );
    }
    __syncthreads();

    // multiply 32x32 diag block * x
    psum = MAGMA_C_ZERO;
    #pragma unroll
    for(int j=0; j < 4; j++) {
        psum += sA32(tx2, ty2*4 + j) * sx[half_NB_X + ty2*4 + j];
    }
    __syncthreads();
    
    // store partial row sums
    sA32(ty2, tx2) = psum;
    __syncthreads();

    // sum up partial row sums, so thread (tx2,1) has total for row (blk_ind + NB/2 + tx2)
    if ( ty2 == 1 ) {
        total = sA32(0, tx2) + sA32(1, tx2)
              + sA32(2, tx2) + sA32(3, tx2)
              + sA32(4, tx2) + sA32(5, tx2)
              + sA32(6, tx2) + sA32(7, tx2);
    }
    __syncthreads();

    // --------------------
    // move to off-diag 32x32 block
    A -= half_NB_X*lda;  // A is A(blk_ind + NB/2 + tx2, blk_ind + ty2)

    // load 32x32 block of A into sA,
    // as four 32x8 sections one after another:
    // columns 0:7, then 8:15, then 16:23, then 24:31
    if ( partial ) {
        if ( tx2 + half_NB_X >= partial ) {
            A = A - (tx2 + half_NB_X) + (partial - 1);
        }
        #pragma unroll
        for(int j=0; j < half_NB_X; j += 8) {
            if ( ty2+j < partial ) {
                sA32(tx2, ty2 + j) = A[j*lda];
            }
        }
        if ( tx2 + half_NB_X >= partial ) {
            A = A + (tx2 + half_NB_X) - (partial - 1);
        }
    }
    else {
        #pragma unroll
        for(int j=0; j < half_NB_X; j += 8) {
            sA32(tx2, ty2 + j) = A[j*lda];
        }
    }
    __syncthreads();

    // multiply 32x32 block (below diag)
    psum = MAGMA_C_ZERO;
    #pragma unroll
    for(int j=0; j < 4; j++) {
        psum += sA32(tx2, ty2 + j*8) * sx[j*8 + ty2];
    }
    //__syncthreads();  // no sync needed here

    // multiply transposed 32x32 block (above diag)
    psum2 = MAGMA_C_ZERO;
    #pragma unroll
    for(int j=0; j < 4; j++) {
        psum2 += hipConjf( sA32(ty2*4 + j, tx2) ) * sx[half_NB_X + ty2*4 + j];
    }
    __syncthreads();

    // store partial sums for non-transposed 32x32 block
    sA32(ty2, tx2) = psum;
    __syncthreads();
    
    // sum up partial row sums, so thread (tx2,1) has total for row (blk_ind + NB/2 + tx2)
    if ( ty2 == 1 ) {
        total = total
              + sA32(0, tx2) + sA32(1, tx2)
              + sA32(2, tx2) + sA32(3, tx2)
              + sA32(4, tx2) + sA32(5, tx2)
              + sA32(6, tx2) + sA32(7, tx2);
    }
    __syncthreads();

    // store partial sums for transposed 32x32 block
    sA32(ty2, tx2) = psum2;
    __syncthreads();
    
    // sum up partial row sums, so thread (tx2,0) has total for row (blk_ind + tx2)
    if ( ty2 == 0 ) {
        total = total
              + sA32(0, tx2) + sA32(1, tx2)
              + sA32(2, tx2) + sA32(3, tx2)
              + sA32(4, tx2) + sA32(5, tx2)
              + sA32(6, tx2) + sA32(7, tx2);
    }
    __syncthreads();
    
    // --------------------
    // move to left most 64x64 block in block row, and
    // switch thread offset from (tx2,ty2) 32x8 block to (tx,ty) 64x4 block
    A -= half_NB_X;       // A is A(blk_ind + tx2, blk_ind + ty2)
    A -= ty2*lda + tx2;   // A is A(blk_ind, blk_ind)
    A -= blk_ind*lda;     // A is A(blk_ind, 0)
    A += 4*ty*lda + tx;   // A is A(blk_ind + tx, 4*ty)
    
    if ( partial && tx >= partial ) {
        A = A - tx + (partial - 1);
    }
    
    x -= blk_ind * incx;  // x is x(tx)

    // 16x16 thread block
    const int tx4 = td % quarter_NB_X;
    const int ty4 = td / quarter_NB_X;

    work += blk*lda + tx4;  // work is work(tx4, blk)
    
    for(int blk2=0; blk2 < blk; ++blk2) {
        // load 64x1 block x(blk2_ind + 0:63) into sx2
        // since this block is left of diagonal, x cannot be partial rows
        if ( ty == 0 ) {
            sx2[tx] = x[blk2*NB_X*incx];
        }
        __syncthreads();

        for( int k=0; k < 4; k++ ) {
            // load 64x16 block of A into rA, 4 elements per thread,
            // as four 64x4 sections in parallel:
            // columns 0,4,8,12; then 1,5,9,13; then 2,6,10,14; then 3,7,11,15
            // since this block is left of diagonal, it cannot be partial columns
            #pragma unroll
            for(int j=0; j < 4; j++) {
                rA[j] = A[j*lda];
            }

            // 1) multiply 64x16 block A * x2
            //    each thread does partial row rA(tx + 16*k, ty*4 + 16*k : ty*4 + 3 + 16*k)
            // 2) multiply transposed 16x64 block A**H * x,
            //    storing each product Aji*xi to sA(j,i)
            #pragma unroll
            for(int j=0; j < 4; j++) {
                total += rA[j] * sx2[quarter_NB_X*k + ty*4 + j];
                sA16(ty*4 + j, tx) = hipConjf( rA[j] ) * sx[tx];
            }
            __syncthreads();

            // do partial row sums for transposed 16x64 result
            // use 16x16 thread grid (tx4, ty4) instead of 64x4 (tx, ty)
            // sum sixteen 16x4 sections in parallel:
            // columns 0,4,8,...,60; then 1,5,...,61; then 2,6,...,62; then 3,7,...,63
            psum2 = MAGMA_C_ZERO;
            #pragma unroll
            for(int j=0; j < 4; j++) {
                psum2 += sA16(tx4, ty4*4 + j);
            }
            __syncthreads();

            // store partial row sums (locally)
            psums[k] = psum2;

            // move to next 64x16 block
            A += lda * quarter_NB_X;  // A is A(blk_ind + tx#, blk2*NB_x + k*NB_X/4 + 4*ty), # or partial
        }

        // store partial row sums
        #pragma unroll
        for(int k=0; k < 4; k++) {
            sA16(tx4, ty4 + quarter_NB_X*k) = psums[k];
        }
        __syncthreads();
        
        // sum up partial row sums and store final total to workspace
        // thread (tx4,ty4) where ty4 < 4 sums row tx4 + ty4*16
        // since this is the transposed block above the diagonal, it cannot be partial rows
        if ( ty4 < 4 ) {
            int k = ty4*quarter_NB_X;
            psum2 = sA16(tx4,  0 + k) + sA16(tx4,  1 + k)
                  + sA16(tx4,  2 + k) + sA16(tx4,  3 + k)
                  + sA16(tx4,  4 + k) + sA16(tx4,  5 + k)
                  + sA16(tx4,  6 + k) + sA16(tx4,  7 + k)
                  + sA16(tx4,  8 + k) + sA16(tx4,  9 + k)
                  + sA16(tx4, 10 + k) + sA16(tx4, 11 + k)
                  + sA16(tx4, 12 + k) + sA16(tx4, 13 + k)
                  + sA16(tx4, 14 + k) + sA16(tx4, 15 + k);
            work[blk2*NB_X + k] = psum2;  // store at work( blk2*NB_X + tx4 + ty4*16, blk )
        }
        __syncthreads();
    }

    work -= tx4;  // work is work(blk_ind)
    work += tx;   // work is work(blk_ind + tx)

    // store row sums
    sA16(ty, tx) = total;
    __syncthreads();
    
    // sum up final total for row tx
    if ( ty == 0 && (partial == 0 || tx < partial) ) {
        total = sA16(0, tx) + sA16(1, tx) + sA16(2, tx) + sA16(3, tx);
        work[blk*NB_X] = total;  // store at work( blk*NB_X + tx, blk )
    }
#endif  /* PRECISION_[sdc] || (__CUDA_ARCH__ >= 200) */
}


/**************************************************************
    Lower case, sum up final results
    
    On input:
           [ A11*x1   A12*x2             A13*x3                    ]
    work = [  ---    (A21*x1 + A22*x2)   A23*x3                    ]
           [  ---      ---              (A31*x1 + A32*x2 + A33*x3) ]
    
    On output:
              [ A11*x1 + A12*x2 + A13*x3 ]
    y = alpha*[ A11*x1 + A22*x2 + A23*x3 ] + beta*y
              [ A21*x1 + A22*x2 + A33*x3 ]
    
    
    Previously:
           [ A11*x1    ---                                         ]
    work = [ A12*x2  (A21*x1 + A22*x2)    ---                      ]
           [ A13*x3   A23*x3            (A31*x1 + A32*x2 + A33*x3) ]
    which doesn't work as well because A13*x3 has 64 rows,
    while A31*x1 has only n % NB rows. This is why it used to need
    lwork = lda*(blocks + 1) instead of lda*blocks.
    ********************************************************************/
__global__ void
chemv_kernel_L_sum(
    int n, magmaFloatComplex alpha,
    int lda,
    magmaFloatComplex beta,
    magmaFloatComplex * __restrict__ y, int incy,
    magmaFloatComplex * __restrict__ work )
{
    int tx  = threadIdx.x;
    int blk = blockIdx.x;
    int blk_ind = blk * NB_X;
    int ind     = blk_ind + tx;
    
    if ( ind < n ) {
        work += ind + blk*lda;
        magmaFloatComplex Ax = MAGMA_C_ZERO;
        for(int i = blk_ind; i < n; i += NB_X) {
            Ax += work[0];
            work += lda;
        }
        y[ind * incy] = beta*y[ind * incy] + alpha*Ax;
    }
}


/**************************************************************
 *  Lower case, launch kernels
 */
extern "C"
void magmablas_chemv_L(
    magma_int_t n, magmaFloatComplex alpha,
    const magmaFloatComplex *A, magma_int_t lda,
    const magmaFloatComplex *x, magma_int_t incx,
    magmaFloatComplex beta,
    magmaFloatComplex *y, magma_int_t incy,
    magmaFloatComplex *dwork)
{
    magma_int_t blocks = (n - 1)/NB_X + 1;
    dim3 grid( blocks, 1, 1 );

    dim3 threads( NB_X, NB_Y, 1 );
    chemv_kernel_L<<< grid, threads, 0, magma_stream >>>
        (n, A, lda, x, incx, dwork);

    dim3 threads_sum( NB_X, 1, 1 );
    chemv_kernel_L_sum<<< grid, threads_sum, 0, magma_stream >>>
        (n, alpha, lda, beta, y, incy, dwork);
}


/**
    Purpose
    -------
    magmablas_chemv_work performs the matrix-vector operation:

        y := alpha*A*x + beta*y,

    where alpha and beta are scalars, x and y are n element vectors and
    A is an n by n Hermitian matrix.

    Arguments
    ----------
    @param[in]
    uplo    magma_uplo_t.
            On entry, UPLO specifies whether the upper or lower
            triangular part of the array A is to be referenced as
            follows:
      -     = MagmaUpper:  Only the upper triangular part of A is to be referenced.
      -     = MagmaLower:  Only the lower triangular part of A is to be referenced.

    @param[in]
    n       INTEGER.
            On entry, N specifies the order of the matrix A.
            N must be at least zero.

    @param[in]
    alpha   COMPLEX.
            On entry, ALPHA specifies the scalar alpha.

    @param[in]
    A       COMPLEX array of DIMENSION ( LDA, n ).
            Before entry with UPLO = MagmaUpper, the leading n by n
            upper triangular part of the array A must contain the upper
            triangular part of the Hermitian matrix and the strictly
            lower triangular part of A is not referenced.
            Before entry with UPLO = MagmaLower, the leading n by n
            lower triangular part of the array A must contain the lower
            triangular part of the Hermitian matrix and the strictly
            upper triangular part of A is not referenced.
            Note that the imaginary parts of the diagonal elements need
            not be set and are assumed to be zero.

    @param[in]
    lda     INTEGER.
            On entry, LDA specifies the first dimension of A as declared
            in the calling (sub) program. LDA must be at least
            max( 1, n ).
            It is recommended that lda is multiple of 16. Otherwise
            performance would be deteriorated as the memory accesses
            would not be fully coalescent.

    @param[in]
    x       COMPLEX array of dimension at least
            ( 1 + ( n - 1 )*abs( INCX ) ).
            Before entry, the incremented array X must contain the n
            element vector x.

    @param[in]
    incx    INTEGER.
            On entry, INCX specifies the increment for the elements of
            X. INCX must not be zero.

    @param[in]
    beta    COMPLEX.
            On entry, BETA specifies the scalar beta. When BETA is
            supplied as zero then Y need not be set on input.

    @param[in, out]
    y       COMPLEX array of dimension at least
            ( 1 + ( n - 1 )*abs( INCY ) ).
            Before entry, the incremented array Y must contain the n
            element vector y. On exit, Y is overwritten by the updated
            vector y.

    @param[in]
    incy    INTEGER.
            On entry, INCY specifies the increment for the elements of
            Y. INCY must not be zero.

    @param[in]
    dwork   (workspace) COMPLEX array on the GPU, dimension (MAX(1, LWORK)),

    @param[in]
    lwork   INTEGER.
            The dimension of the array DWORK. LWORK >= LDA * ceil( N / NB_X ),
            where NB_X = 64.

    MAGMA implements chemv through two steps:
    1)  perform the multiplication in each thread block and put the
        intermediate value in dwork.
    2)  sum the intermediate values and store the final result in y.
    
    magamblas_chemv_work requires users to provide a workspace, while
    magmablas_chemv is a wrapper routine allocating the workspace inside the
    routine and provides the same interface as cublas.
    
    If users need to call chemv frequently, we suggest using
    magmablas_chemv_work instead of magmablas_chemv. As the overhead to
    allocate and free in device memory in magmablas_chemv would hurt performance.
    Our tests show that this penalty is about 10 Gflop/s when the matrix
    size is around 10000.

    @ingroup magma_cblas2
    ********************************************************************/
extern "C"
magma_int_t
magmablas_chemv_work(
    magma_uplo_t uplo, magma_int_t n,
    magmaFloatComplex alpha,
    const magmaFloatComplex *A, magma_int_t lda,
    const magmaFloatComplex *x, magma_int_t incx,
    magmaFloatComplex beta,
    magmaFloatComplex *y, magma_int_t incy,
    magmaFloatComplex *dwork, magma_int_t lwork)
{
#if defined(PRECISION_z)
    // z precision requires CUDA ARCH 2.x; call CUBLAS version instead.
    magma_int_t arch = magma_getdevice_arch();
    if ( arch < 200 ) {
        magma_chemv( uplo, n, alpha, A, lda, x, incx, beta, y, incy );
        return MAGMA_SUCCESS;
    }
#endif

    // --------------------
    // [sdc] precisions, or z precision with CUDA ARCH 2.x
    int upper = (uplo == MagmaUpper);

    magma_int_t blocks = (n - 1)/NB_X + 1;
    magma_int_t lwmin  = lda*blocks;

    /*
     * Test the input parameters.
     */
    magma_int_t info = 0;
    if ((! upper) && (uplo != MagmaLower)) {
        info = -1;
    } else if ( n < 0 ) {
        info = -2;
    } else if ( lda < max(1, n) ) {
        info = -5;
    } else if ( incx == 0 ) {
        info = -7;
    } else if ( incy == 0 ) {
        info = -10;
    } else if ( lwork < lwmin ) {
        info = -12;
    }
    
    if (info != 0) {
        magma_xerbla( __func__, -(info) );
        return info;
    }

    /*
     * Quick return if possible.
     */
    if ( (n == 0) || ( MAGMA_C_EQUAL(alpha, MAGMA_C_ZERO) && MAGMA_C_EQUAL(beta, MAGMA_C_ONE) ) )
        return info;

    /* TODO: Upper case is not implemented in MAGMA */
    if ( upper ) {
        magma_chemv( uplo, n, alpha, A, lda, x, incx, beta, y, incy);
    }
    else {
        magmablas_chemv_L(n, alpha, A, lda, x, incx, beta, y, incy, dwork);
    }
    return info;
}


/**
    Purpose
    -------
    magmablas_chemv performs the matrix-vector operation:

        y := alpha*A*x + beta*y,

    where alpha and beta are scalars, x and y are n element vectors and
    A is an n by n Hermitian matrix.

    Arguments
    ----------
    @param[in]
    uplo    magma_uplo_t.
            On entry, UPLO specifies whether the upper or lower
            triangular part of the array A is to be referenced as
            follows:
      -     = MagmaUpper:  Only the upper triangular part of A is to be referenced.
      -     = MagmaLower:  Only the lower triangular part of A is to be referenced.

    @param[in]
    n       INTEGER.
            On entry, N specifies the order of the matrix A.
            N must be at least zero.

    @param[in]
    alpha   COMPLEX.
            On entry, ALPHA specifies the scalar alpha.

    @param[in]
    A       COMPLEX array of DIMENSION ( LDA, n ).
            Before entry with UPLO = MagmaUpper, the leading n by n
            upper triangular part of the array A must contain the upper
            triangular part of the Hermitian matrix and the strictly
            lower triangular part of A is not referenced.
            Before entry with UPLO = MagmaLower, the leading n by n
            lower triangular part of the array A must contain the lower
            triangular part of the Hermitian matrix and the strictly
            upper triangular part of A is not referenced.
            Note that the imaginary parts of the diagonal elements need
            not be set and are assumed to be zero.

    @param[in]
    lda     INTEGER.
            On entry, LDA specifies the first dimension of A as declared
            in the calling (sub) program. LDA must be at least
            max( 1, n ).
            It is recommended that lda is multiple of 16. Otherwise
            performance would be deteriorated as the memory accesses
            would not be fully coalescent.

    @param[in]
    x       COMPLEX array of dimension at least
            ( 1 + ( n - 1 )*abs( INCX ) ).
            Before entry, the incremented array X must contain the n
            element vector x.

    @param[in]
    incx    INTEGER.
            On entry, INCX specifies the increment for the elements of
            X. INCX must not be zero.

    @param[in]
    beta    COMPLEX.
            On entry, BETA specifies the scalar beta. When BETA is
            supplied as zero then Y need not be set on input.

    @param[in, out]
    y       COMPLEX array of dimension at least
            ( 1 + ( n - 1 )*abs( INCY ) ).
            Before entry, the incremented array Y must contain the n
            element vector y. On exit, Y is overwritten by the updated
            vector y.

    @param[in]
    incy    INTEGER.
            On entry, INCY specifies the increment for the elements of
            Y. INCY must not be zero.

    @ingroup magma_cblas2
    ********************************************************************/
extern "C"
magma_int_t
magmablas_chemv(
    magma_uplo_t uplo, magma_int_t n,
    magmaFloatComplex alpha,
    const magmaFloatComplex *A, magma_int_t lda,
    const magmaFloatComplex *x, magma_int_t incx,
    magmaFloatComplex beta,
    magmaFloatComplex *y, magma_int_t incy)
{
#if defined(PRECISION_z)
    // z precision requires CUDA ARCH 2.x; call CUBLAS version instead.
    magma_int_t arch = magma_getdevice_arch();
    if ( arch < 200 ) {
        magma_chemv( uplo, n, alpha, A, lda, x, incx, beta, y, incy );
        return MAGMA_SUCCESS;
    }
#endif

    // --------------------
    // [sdc] precisions, or z precision with CUDA ARCH 2.x
    int upper = (uplo == MagmaUpper);

    /*
     * Test the input parameters.
     */
    magma_int_t info = 0;
    if ((! upper) && (uplo != MagmaLower)) {
        info = -1;
    } else if ( n < 0 ) {
        info = -2;
    } else if ( lda < max(1, n) ) {
        info = -5;
    } else if ( incx == 0 ) {
        info = -7;
    } else if ( incy == 0 ) {
        info = -10;
    }
    
    if (info != 0) {
        magma_xerbla( __func__, -(info) );
        return info;
    }

    /*
     * Quick return if possible.
     */
    if ( (n == 0) || ( MAGMA_C_EQUAL(alpha, MAGMA_C_ZERO) && MAGMA_C_EQUAL(beta, MAGMA_C_ONE) ) )
        return info;

    /* TODO: Upper case is not implemented in MAGMA */
    if ( upper ) {
        magma_chemv( uplo, n, alpha, A, lda, x, incx, beta, y, incy);
    }
    else {
        magmaFloatComplex *dwork;
        magma_int_t blocks = (n - 1)/NB_X + 1;
        magma_int_t lwork  = lda*blocks;

        magma_cmalloc( &dwork, lwork );
        if ( dwork == NULL ) {
            info = MAGMA_ERR_DEVICE_ALLOC;
            magma_xerbla( __func__, -(info) );
        }
        else {
            magmablas_chemv_L(n, alpha, A, lda, x, incx, beta, y, incy, dwork);
        }
        magma_free( dwork );
    }
    return info;
}
