#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @generated from zlascl2.cu normal z -> d, Fri Jan 30 19:00:09 2015

       @author Theo Mary
*/
#include "common_magma.h"

#define NB 64


// each thread block does one NB x n block row of A.
// each thread does one row, starting from left edge and moving right.
__global__ void
dlascl2_full(int m, int n, const double* D, double* A, int lda)
{
    int ind = blockIdx.x * NB + threadIdx.x;

    double mul = D[ind];
    A += ind;
    if (ind < m) {
        for(int j=0; j < n; j++ )
            A[j*lda] *= mul;
    }
}


// each thread block does one NB x n block row of A.
// each thread does one row, starting from left edge and moving right to diagonal.
__global__ void
dlascl2_lower(int m, int n, const double* D, double* A, int lda)
{
    int ind = blockIdx.x * NB + threadIdx.x;

    int break_d = (ind < n) ? ind : n-1;

    double mul = D[ind];
    A += ind;
    if (ind < m) {
        for(int j=0; j <= break_d; j++ )
            A[j*lda] *= mul;
    }
}


// each thread block does one NB x n block row of A.
// each thread does one row, starting from right edge and moving left to diagonal.
__global__ void
dlascl2_upper(int m, int n, const double *D, double* A, int lda)
{
    int ind = blockIdx.x * NB + threadIdx.x;

    double mul = D[ind];
    A += ind;
    if (ind < m) {
        for(int j=n-1; j >= ind; j--)
            A[j*lda] *= mul;
    }
}


/**
    Purpose
    -------
    DLASCL2 scales the M by N real matrix A by the real diagonal matrix dD.
    TYPE specifies that A may be full, upper triangular, lower triangular.

    Arguments
    ---------
    \param[in]
    type    magma_type_t
            TYPE indices the storage type of the input matrix A.
            = MagmaFull:   full matrix.
            = MagmaLower:  lower triangular matrix.
            = MagmaUpper:  upper triangular matrix.
            Other formats that LAPACK supports, MAGMA does not currently support.

    \param[in]
    m       INTEGER
            The number of rows of the matrix A.  M >= 0.

    \param[in]
    n       INTEGER
            The number of columns of the matrix A.  N >= 0.

    \param[in]
    dD      DOUBLE PRECISION vector, dimension (M)
            The diagonal matrix containing the scalar factors. Stored as a vector.

    \param[in,out]
    dA      DOUBLE PRECISION array, dimension (LDDA,N)
            The matrix to be scaled by dD.  See TYPE for the
            storage type.

    \param[in]
    ldda    INTEGER
            The leading dimension of the array A.  LDDA >= max(1,M).

    \param[out]
    info    INTEGER
      -     = 0:  successful exit
      -     < 0:  if INFO = -i, the i-th argument had an illegal value.
    
    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_daux2
    ********************************************************************/
extern "C" void
magmablas_dlascl2_q(
    magma_type_t type, magma_int_t m, magma_int_t n,
    magmaDouble_const_ptr dD,
    magmaDouble_ptr dA, magma_int_t ldda,
    magma_queue_t queue,
    magma_int_t *info )
{
    *info = 0;
    if ( type != MagmaLower && type != MagmaUpper && type != MagmaFull )
        *info = -1;
    else if ( m < 0 )
        *info = -2;
    else if ( n < 0 )
        *info = -3;
    else if ( ldda < max(1,m) )
        *info = -5;
    
    if (*info != 0) {
        magma_xerbla( __func__, -(*info) );
        return;  //info;
    }
    
    dim3 grid( (m + NB - 1)/NB );
    dim3 threads( NB );
    
    if (type == MagmaLower) {
        dlascl2_lower <<< grid, threads, 0, queue >>> (m, n, dD, dA, ldda);
    }
    else if (type == MagmaUpper) {
        dlascl2_upper <<< grid, threads, 0, queue >>> (m, n, dD, dA, ldda);
    }
    else if (type == MagmaFull) {
        dlascl2_full  <<< grid, threads, 0, queue >>> (m, n, dD, dA, ldda);
    }
}


/**
    @see magmablas_dlascl2_q
    @ingroup magma_daux2
    ********************************************************************/
extern "C" void
magmablas_dlascl2(
    magma_type_t type, magma_int_t m, magma_int_t n,
    magmaDouble_const_ptr dD,
    magmaDouble_ptr dA, magma_int_t ldda, magma_int_t *info )
{
    magmablas_dlascl2_q( type, m, n, dD, dA, ldda, magma_stream, info );
}
