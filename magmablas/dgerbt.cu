#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2014

       @generated from zgerbt.cu normal z -> d, Sat Nov 15 19:53:59 2014


       @author Adrien REMY
*/
#include "common_magma.h"
#include "dgerbt.h"


#define block_height  32
#define block_width  4
#define block_length 256
#define NB 64
/////////////////////////////////////////////////////////////////////////////////////////////////////////////
/**
    Purpose
    -------
    DPRBT_MVT compute B = UTB to randomize B
    
    Arguments
    ---------
    @param[in]
    n       INTEGER
            The number of values of db.  n >= 0.

    @param[in]
    du     DOUBLE_PRECISION array, dimension (n,2)
            The 2*n vector representing the random butterfly matrix V
    
    @param[in,out]
    db     DOUBLE_PRECISION array, dimension (n)
            The n vector db computed by DGESV_NOPIV_GPU
            On exit db = du*db
    
    @param[in]
    queue   magma_queue_t
            Queue to execute in.
    ********************************************************************/
extern "C" void
magmablas_dprbt_mtv_q(
    magma_int_t n, 
    double *du, double *db,
    magma_queue_t queue)
{
    /*

     */
    magma_int_t threads = block_length;
    magma_int_t grid = n/(4*block_length) + ((n%(4*block_length))!=0);

    magmablas_dapply_transpose_vector_kernel<<< grid, threads, 0, queue >>>(n/2, du, n, db, 0);
    magmablas_dapply_transpose_vector_kernel<<< grid, threads, 0, queue >>>(n/2, du, n+n/2, db, n/2);

    threads = block_length;
    grid = n/(2*block_length) + ((n%(2*block_length))!=0);
    magmablas_dapply_transpose_vector_kernel<<< grid, threads, 0, queue >>>(n, du, 0, db, 0);
}

/**
    @see magmablas_dprbt_mtv_q
    ********************************************************************/
extern "C" void
magmablas_dprbt_mtv(
    magma_int_t n, 
    double *du, double *db)
{
    magmablas_dprbt_mtv_q(n, du, db, magma_stream);
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////

/**
    Purpose
    -------
    DPRBT_MV compute B = VB to obtain the non randomized solution
    
    Arguments
    ---------
    @param[in]
    n       INTEGER
            The number of values of db.  n >= 0.
    
    @param[in,out]
    db      DOUBLE_PRECISION array, dimension (n)
            The n vector db computed by DGESV_NOPIV_GPU
            On exit db = dv*db
    
    @param[in]
    dv      DOUBLE_PRECISION array, dimension (n,2)
            The 2*n vector representing the random butterfly matrix V
    
    @param[in]
    queue   magma_queue_t
            Queue to execute in.
    ********************************************************************/
extern "C" void
magmablas_dprbt_mv_q(
    magma_int_t n, 
    double *dv, double *db,
    magma_queue_t queue)
{

    magma_int_t threads = block_length;
    magma_int_t grid = n/(2*block_length) + ((n%(2*block_length))!=0);

    magmablas_dapply_vector_kernel<<< grid, threads, 0, queue >>>(n, dv, 0, db, 0);


    threads = block_length;
    grid = n/(4*block_length) + ((n%(4*block_length))!=0);

    magmablas_dapply_vector_kernel<<< grid, threads, 0, queue >>>(n/2, dv, n, db, 0);
    magmablas_dapply_vector_kernel<<< grid, threads, 0, queue >>>(n/2, dv, n+n/2, db, n/2);
}

/**
    @see magmablas_dprbt_mtv_q
    ********************************************************************/
extern "C" void
magmablas_dprbt_mv(
    magma_int_t n, 
    double *dv, double *db)
{
    magmablas_dprbt_mv_q(n, dv, db, magma_stream);
}
/////////////////////////////////////////////////////////////////////////////////////////////////////////////
/**
    Purpose
    -------
    DPRBT randomize a square general matrix using partial randomized transformation
    
    Arguments
    ---------
    @param[in]
    n       INTEGER
            The number of columns and rows of the matrix dA.  n >= 0.
    
    @param[in,out]
    dA      DOUBLE_PRECISION array, dimension (n,ldda)
            The n-by-n matrix dA
            On exit dA = duT*dA*d_V
    
    @param[in]
    ldda    INTEGER
            The leading dimension of the array dA.  LDA >= max(1,n).
    
    @param[in]
    du      DOUBLE_PRECISION array, dimension (n,2)
            The 2*n vector representing the random butterfly matrix U
    
    @param[in]
    dv      DOUBLE_PRECISION array, dimension (n,2)
            The 2*n vector representing the random butterfly matrix V
    
    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    ********************************************************************/
extern "C" void 
magmablas_dprbt_q(
    magma_int_t n, 
    double *dA, magma_int_t ldda, 
    double *du, double *dv,
    magma_queue_t queue)
{
    du += ldda;
    dv += ldda;

    dim3 threads(block_height, block_width);
    dim3 grid(n/(4*block_height) + ((n%(4*block_height))!=0), 
            n/(4*block_width)  + ((n%(4*block_width))!=0));

    magmablas_delementary_multiplication_kernel<<< grid, threads, 0, queue >>>(n/2, dA,            0, ldda, du,   0, dv,   0);
    magmablas_delementary_multiplication_kernel<<< grid, threads, 0, queue >>>(n/2, dA,     ldda*n/2, ldda, du,   0, dv, n/2);
    magmablas_delementary_multiplication_kernel<<< grid, threads, 0, queue >>>(n/2, dA,          n/2, ldda, du, n/2, dv,   0);
    magmablas_delementary_multiplication_kernel<<< grid, threads, 0, queue >>>(n/2, dA, ldda*n/2+n/2, ldda, du, n/2, dv, n/2);

    dim3 threads2(block_height, block_width);
    dim3 grid2(n/(2*block_height) + ((n%(2*block_height))!=0), 
            n/(2*block_width)  + ((n%(2*block_width))!=0));
    magmablas_delementary_multiplication_kernel<<< grid2, threads2, 0, queue >>>(n, dA, 0, ldda, du, -ldda, dv, -ldda);
}


/**
    @see magmablas_dprbt_q
    ********************************************************************/
extern "C" void 
magmablas_dprbt(
    magma_int_t n, 
    double *dA, magma_int_t ldda, 
    double *du, double *dv)
{
    magmablas_dprbt_q(n, dA, ldda, du, dv, magma_stream);
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////

// adds   x += r  --and--
// copies r = b
// each thread does one index, x[i] and r[i]
__global__ void
daxpycp2_kernel(
    int m, double *r, double *x,
    const double *b)
{
    const int i = threadIdx.x + blockIdx.x*NB;
    if ( i < m ) {
        x[i] = MAGMA_D_ADD( x[i], r[i] );
        r[i] = b[i];
    }
}


// ----------------------------------------------------------------------
// adds   x += r  --and--
// copies r = b
extern "C" void
magmablas_daxpycp2_q(
    magma_int_t m, double *r, double *x,
    const double *b,
    magma_queue_t queue )
{
    dim3 threads( NB );
    dim3 grid( (m + NB - 1)/NB );
    daxpycp2_kernel <<< grid, threads, 0, queue >>> ( m, r, x, b );
}


extern "C" void
magmablas_daxpycp2(
    magma_int_t m, double *r, double *x,
    const double *b)
{
    magmablas_daxpycp2_q( m, r, x, b, magma_stream );
}
