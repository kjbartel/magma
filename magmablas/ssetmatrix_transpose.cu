#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2014

       @generated from zsetmatrix_transpose.cu normal z -> s, Sat Nov 15 19:53:59 2014

*/
#include "common_magma.h"

#define PRECISION_s


//
//      m, n - dimensions in the source (input) matrix.
//             This routine copies the hA matrix from the CPU
//             to dAT on the GPU. In addition, the output matrix
//             is transposed. The routine uses a buffer of size
//             2*lddwork*nb pointed to by dwork (lddwork > m) on the GPU. 
//             Note that lda >= m and lddat >= n.
//
extern "C" void 
magmablas_ssetmatrix_transpose_q(
    magma_int_t m, magma_int_t n,
    const float     *hA, magma_int_t lda, 
    magmaFloat_ptr       dAT, magma_int_t ldda,
    magmaFloat_ptr     dwork, magma_int_t lddwork, magma_int_t nb,
    magma_queue_t queues[2] )
{
#define    hA(i_, j_)    (hA + (i_) + (j_)*lda)
#define   dAT(i_, j_)   (dAT + (i_) + (j_)*ldda)
#define dwork(i_, j_) (dwork + (i_) + (j_)*lddwork)

    magma_int_t i = 0, j = 0, ib;

    /* Quick return */
    if ( (m == 0) || (n == 0) )
        return;

    // TODO standard check arguments
    if (lda < m || ldda < n || lddwork < m){
        printf("Wrong arguments in %s.\n", __func__);
        return;
    }

    /* Move data from CPU to GPU in the first panel in the dwork buffer */
    ib = min(n-i, nb);
    magma_ssetmatrix_async( m, ib,
                            hA(0,i), lda,
                            dwork(0,(j%2)*nb), lddwork, queues[j%2] );
    j++;

    for(i=nb; i < n; i += nb) {
        /* Move data from CPU to GPU in the second panel in the dwork buffer */
        ib = min(n-i, nb);
        magma_ssetmatrix_async( m, ib,
                                hA(0,i), lda,
                                dwork(0,(j%2)*nb), lddwork, queues[j%2] );
        j++;
        
        /* Note that the previous panel (i.e., j%2) comes through the queue
           for the kernel so there is no need to synchronize.             */
        // TODO should this be ib not nb?
        magmablas_stranspose_q( m, nb, dwork(0,(j%2)*nb), lddwork, dAT(i-nb,0), ldda, queues[j%2] );
    }

    /* Transpose the last part of the matrix.                            */
    j++;
    magmablas_stranspose_q( m, ib, dwork(0,(j%2)*nb), lddwork, dAT(i-nb,0), ldda, queues[j%2] );
}


// @see magmablas_ssetmatrix_transpose_q
extern "C" void 
magmablas_ssetmatrix_transpose(
    magma_int_t m, magma_int_t n,
    const float     *hA, magma_int_t lda, 
    magmaFloat_ptr       dAT, magma_int_t ldda,
    magmaFloat_ptr     dwork, magma_int_t lddwork, magma_int_t nb )
{
    magma_queue_t queues[2];
    magma_queue_create( &queues[0] );
    magma_queue_create( &queues[1] );

    magmablas_ssetmatrix_transpose_q( m, n, hA, lda, dAT, ldda, dwork, lddwork, nb, queues );
    
    magma_queue_destroy( queues[0] );
    magma_queue_destroy( queues[1] );
}
