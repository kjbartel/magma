#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

       @generated c Sun Nov 13 20:48:40 2011

*/
#include "common_magma.h"
#define PRECISION_c
#include "commonblas.h"

extern "C" void
magmablas_ctranspose2s(hipFloatComplex *odata, int ldo,
                       hipFloatComplex *idata, int ldi,
                       int m, int n, hipStream_t *stream );


//
//      m, n - dimensions in the output (ha) matrix.
//             This routine copies the dat matrix from the GPU
//             to ha on the CPU. In addition, the output matrix
//             is transposed. The routine uses a buffer of size
//             2*lddb*nb pointed to by dB (lddb > m) on the GPU. 
//             Note that lda >= m and lddat >= n.
//
extern "C" void 
magmablas_cgetmatrix_transpose( int m, int n,
                                hipFloatComplex *dat, int ldda,
                                hipFloatComplex  *ha, int lda,
                                hipFloatComplex  *dB, int lddb, int nb )
{
    int i = 0, j = 0, ib;

    /* Quick return */
    if ( (m == 0) || (n == 0) )
        return;

    if (lda < m || ldda < n || lddb < m){
        printf("Wrong arguments in zdtoht.\n");
        return;
    }

    static hipStream_t stream[2];
    hipStreamCreate(&stream[0]);
    hipStreamCreate(&stream[1]);

    for(i=0; i<n; i+=nb){
       /* Move data from GPU to CPU using 2 buffers; 1st transpose the data on the GPU */
       ib   = min(n-i, nb);

       //magmablas_ctranspose2 ( dB + (j%2)*nb*lddb, lddb, dat+i, ldda, ib, m);
       magmablas_ctranspose2s( dB + (j%2)*nb*lddb, lddb, dat+i, ldda, ib, m, &stream[j%2]);
       hipMemcpy2DAsync(ha+i*lda, lda*sizeof(hipFloatComplex),
                         dB + (j%2) * nb * lddb, lddb*sizeof(hipFloatComplex),
                         sizeof(hipFloatComplex)*m, ib, 
                         hipMemcpyDeviceToHost, stream[j%2]);
       j++;
    }

    hipStreamDestroy( stream[0] );
    hipStreamDestroy( stream[1] );
}

//===========================================================================
//  This version is similar to the above but for multiGPUs. The distribution
//  is 1D block cyclic. The input arrays are pointers for the corresponding
//  GPUs. The streams are passed as argument, in contrast to the single GPU
//  routine.
//===========================================================================
extern "C" void
magmablas_cgetmatrix_transpose2( int m, int n,
                                 hipFloatComplex **dat, int *ldda,
                                 hipFloatComplex  *ha,  int  lda,
                                 hipFloatComplex **dB,  int  lddb, int nb,
                                 int num_gpus, hipStream_t stream[][2] )
{
    int i = 0, j[4] = {0, 0, 0, 0}, ib, k;

    /* Quick return */
    if ( (m == 0) || (n == 0) )
        return;

    if (lda < m || lddb < m){
        printf("Wrong arguments in zdtoht.\n");
        return;
    }

    for(i=0; i<n; i+=nb){
       /* Move data from GPU to CPU using 2 buffers; 1st transpose the data on the GPU */
       k = (i/nb)%num_gpus;
       ib   = min(n-i, nb);
       hipSetDevice(k);

       //hipStreamSynchronize(stream[k][j[k]%2]);
       //magmablas_ctranspose2( dB[k] + (j[k]%2)*nb*lddb, lddb, 
       //                       dat[k]+i/(nb*num_gpus)*nb, ldda[k], ib, m);
       magmablas_ctranspose2s(dB[k] + (j[k]%2)*nb*lddb, lddb,
                              dat[k]+i/(nb*num_gpus)*nb, ldda[k], 
                              ib, m, &stream[k][j[k]%2]);
       hipMemcpy2DAsync(ha+i*lda, lda*sizeof(hipFloatComplex),
                         dB[k] + (j[k]%2) * nb * lddb, lddb*sizeof(hipFloatComplex),
                         sizeof(hipFloatComplex)*m, ib,
                         hipMemcpyDeviceToHost,
                         stream[k][j[k]%2]);
       j[k]++;
    }
}

