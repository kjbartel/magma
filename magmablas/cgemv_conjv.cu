#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015
       
       @author Stan Tomov

       @generated from zgemv_conjv.cu normal z -> c, Fri Jan 30 19:00:08 2015
*/
#include "common_magma.h"
#include "commonblas_c.h"

#define PRECISION_c

#define num_threads 256


__global__ void
cgemv_conjv_kernel(
    int m, int n, magmaFloatComplex alpha,
    const magmaFloatComplex * __restrict__ A, int lda,
    const magmaFloatComplex * __restrict__ x, int incx, magmaFloatComplex beta,
    magmaFloatComplex *       __restrict__ y, int incy)
{
    int ind = blockIdx.x*num_threads + threadIdx.x;
    
    A += ind;

    if ( ind < m ) {
        magmaFloatComplex res = MAGMA_C_ZERO;
        
        #pragma unroll
        for( int i=0; i < n; i ++ ) {
            res += A[0] * MAGMA_C_CNJG(x[0]);
            A += lda;
            x += incx;
        }
        
        y[ind*incy] = alpha * res + beta * y[ind*incy];
    }
}


/**
    Purpose
    -------
    CGEMV_CONJV performs the matrix-vector operation
    
        y := alpha*A*conj(x)    + beta*y, 
    
    where alpha and beta are scalars, x and y are vectors and A is an
    m by n matrix.

    Arguments
    ----------
    @param[in]
    m       INTEGER
            On entry, m specifies the number of rows of the matrix A.

    @param[in]
    n       INTEGER
            On entry, n specifies the number of columns of the matrix A

    @param[in]
    alpha   COMPLEX
            On entry, ALPHA specifies the scalar alpha.

    @param[in]
    dA      COMPLEX array of dimension ( LDA, n ) on the GPU.

    @param[in]
    lda     INTEGER
            LDA specifies the leading dimension of A.

    @param[in]
    dx      COMPLEX array of dimension n

    @param[in]
    incx    Specifies the increment for the elements of X.
            INCX must not be zero.

    @param[in]
    beta    DOUBLE REAL
            On entry, BETA specifies the scalar beta. When BETA is
            supplied as zero then Y need not be set on input.

    @param[out]
    dy      REAL array of dimension m

    @param[in]
    incy    Specifies the increment for the elements of Y.
            INCY must not be zero.

    @ingroup magma_cblas2
    ********************************************************************/
extern "C" void
magmablas_cgemv_conjv(
    magma_int_t m, magma_int_t n, magmaFloatComplex alpha,
    magmaFloatComplex_const_ptr dA, magma_int_t ldda,
    magmaFloatComplex_const_ptr dx, magma_int_t incx,
    magmaFloatComplex beta,
    magmaFloatComplex_ptr dy, magma_int_t incy)
{
    magma_int_t info = 0;
    if ( m < 0 )
        info = -1;
    else if ( n < 0 )
        info = -2;
    else if ( ldda < m )
        info = -5;
    else if ( incx == 0 )
        info = -7;
    else if ( incy == 0 )
        info = -10;
    
    if (info != 0) {
        magma_xerbla( __func__, -(info) );
        return;  //info;
    }
    
    magma_int_t blocks = (m - 1)/num_threads + 1;
    dim3 grid(blocks, 1, 1);
    dim3 threads(num_threads, 1, 1);

    cgemv_conjv_kernel<<< grid, threads, 0, magma_stream >>>
            (m, n, alpha, dA, ldda, dx, incx, beta, dy, incy);

}

#undef num_threads
