#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.5.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date September 2014

       @generated from zlarf.cu normal z -> c, Wed Sep 17 15:08:23 2014
       @author Azzam Haidar

*/
#include "common_magma.h"
#include "magma_templates.h"

// 512 is maximum number of threads for CUDA capability 1.x
#define BLOCK_SIZE 512

#define BLOCK_SIZEx  32
#define BLOCK_SIZEy  16


//==============================================================================
//==============================================================================

__global__
void magma_clarf_kernel( int m, const magmaFloatComplex *dv, const magmaFloatComplex *dtau,
                         magmaFloatComplex *dc, int lddc )
{
    if ( !MAGMA_C_EQUAL(*dtau, MAGMA_C_ZERO) ) {
        const int tx = threadIdx.x;
        dc = dc + blockIdx.x * lddc;

        __shared__ magmaFloatComplex sum[ BLOCK_SIZE ];
        magmaFloatComplex tmp;

        /* perform  w := v' * C  */
        if (tx==0)
            tmp = dc[0]; //since V[0] should be one
        else
            tmp = MAGMA_C_ZERO;
        for( int j = tx+1; j < m; j += BLOCK_SIZE ){
            tmp += MAGMA_C_MUL( MAGMA_C_CNJG( dv[j] ), dc[j] );
        }
        sum[tx] = tmp;
        magma_sum_reduce< BLOCK_SIZE >( tx, sum );

        /*  C := C - v * w  */
        __syncthreads();
        tmp = - MAGMA_C_CNJG(*dtau) * sum[0];
        for( int j = m-tx-1; j>0 ; j -= BLOCK_SIZE )
             dc[j] += tmp * dv[j];

        if(tx==0) dc[0] += tmp;
    }
}

//==============================================================================
//==============================================================================

__global__
void magma_clarf_smkernel( int m, int n, magmaFloatComplex *dv, magmaFloatComplex *dtau,
                           magmaFloatComplex *dc, int lddc )
{
    if ( ! MAGMA_C_EQUAL(*dtau, MAGMA_C_ZERO) ) {
        const int i = threadIdx.x, col= threadIdx.y;

        for( int k = col; k < n; k += BLOCK_SIZEy ) {
            dc = dc + k * lddc;
    
            __shared__ magmaFloatComplex sum[ BLOCK_SIZEx ][ BLOCK_SIZEy + 1];
            magmaFloatComplex lsum;
    
            /*  w := v' * C  */
            lsum = MAGMA_C_ZERO;
            for( int j = i; j < m; j += BLOCK_SIZEx ){
                if (j==0)
                   lsum += MAGMA_C_MUL( MAGMA_C_ONE, dc[j] );
                else
                   lsum += MAGMA_C_MUL( MAGMA_C_CNJG( dv[j] ), dc[j] );
            }
            sum[i][col] = lsum;
            magma_sum_reduce_2d< BLOCK_SIZEx, BLOCK_SIZEy+1 >( i, col, sum );
    
            /*  C := C - v * w  */
            __syncthreads();
            magmaFloatComplex z__1 = - MAGMA_C_CNJG(*dtau) * sum[0][col];
            for( int j = m-i-1; j>=0 ; j -= BLOCK_SIZEx ) {
                 if (j==0)
                    dc[j] += z__1;
                 else
                    dc[j] += z__1 * dv[j];
            }
        }
    }
}

//==============================================================================

/*
    Apply a complex elementary reflector H to a complex M-by-N
    matrix C from the left. H is represented in the form
          H = I - tau * v * v'
    where tau is a complex scalar and v is a complex vector.
    If tau = 0, then H is taken to be the unit matrix.

    To apply H' (the conjugate transpose of H), supply conjg(tau)
    instead tau.

    This routine uses only one SM (block).
 */
extern "C" void
magma_clarf_sm(magma_int_t m, magma_int_t n, magmaFloatComplex *dv, magmaFloatComplex *dtau,
               magmaFloatComplex *dc, magma_int_t lddc)
{
    dim3  blocks( 1 );
    dim3 threads( BLOCK_SIZEx, BLOCK_SIZEy );

    magma_clarf_smkernel<<< blocks, threads, 0, magma_stream >>>( m, n, dv, dtau, dc, lddc );
}
//==============================================================================
/*
    Apply a complex elementary reflector H to a complex M-by-N
    matrix C from the left. H is represented in the form
          H = I - tau * v * v'
    where tau is a complex scalar and v is a complex vector.
    If tau = 0, then H is taken to be the unit matrix.

    To apply H' (the conjugate transpose of H), supply conjg(tau) 
    instead tau.

 */

extern "C" magma_int_t
magma_clarf_gpu(
    magma_int_t m,  magma_int_t n,
    const magmaFloatComplex *dv, const magmaFloatComplex *dtau,
    magmaFloatComplex *dc,  magma_int_t lddc)
{
    dim3 grid( n, 1, 1 );
    dim3 threads( BLOCK_SIZE );
    if ( n>0 ){
        magma_clarf_kernel<<< grid, threads, 0, magma_stream >>>( m, dv, dtau, dc, lddc);
    }

    // The computation can be done on 1 SM with the following routine.
    // magma_clarf_sm(m, n, dv, dtau, dc, lddc);

    return MAGMA_SUCCESS;
}

//==============================================================================
