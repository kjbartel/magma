#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.5.0-beta3) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date July 2014

       @generated from zlarf.cu normal z -> c, Fri Jul 18 17:34:12 2014

*/
#include "common_magma.h"
#include "magma_templates.h"

// 512 is maximum number of threads for CUDA capability 1.x
#define BLOCK_SIZE 512

#define BLOCK_SIZEx  32
#define BLOCK_SIZEy  16


//==============================================================================

__global__
void magma_clarf_kernel( int m, magmaFloatComplex *v, magmaFloatComplex *tau,
                         magmaFloatComplex *c, int ldc, float *xnorm )
{
    if ( !MAGMA_C_EQUAL(*tau, MAGMA_C_ZERO) ) {
        const int i = threadIdx.x;
        magmaFloatComplex *dc = c + blockIdx.x * ldc;

        __shared__ magmaFloatComplex sum[ BLOCK_SIZE ];
        magmaFloatComplex lsum;

        /*  w := v' * C  */
        lsum = MAGMA_C_ZERO;
        for( int j = i; j < m; j += BLOCK_SIZE ){
            if (j==0)
               lsum += MAGMA_C_MUL( MAGMA_C_ONE, dc[j] );
            else
               lsum += MAGMA_C_MUL( MAGMA_C_CNJG( v[j] ), dc[j] );
        }
        sum[i] = lsum;
        magma_sum_reduce< BLOCK_SIZE >( i, sum );

        /*  C := C - v * w  */
        __syncthreads();
        magmaFloatComplex z__1 = - MAGMA_C_CNJG(*tau) * sum[0];
        for( int j = m-i-1; j>=0 ; j -= BLOCK_SIZE ) {
             if (j==0)
                dc[j] += z__1;
             else
                dc[j] += z__1 * v[j];
        }
        __syncthreads();

        /* Adjust the rest of the column norms */
        //if (i==0){
        //    float temp = MAGMA_C_ABS( dc[0] ) / xnorm[blockIdx.x];
        //    temp = (temp + 1.) * (1. - temp);
        //    xnorm[blockIdx.x] = xnorm[blockIdx.x] * sqrt(temp); 
        //}
    }
}

//==============================================================================

__global__
void magma_clarf_smkernel( int m, int n, magmaFloatComplex *v, magmaFloatComplex *tau,
                           magmaFloatComplex *c, int ldc, float *xnorm )
{
    if ( ! MAGMA_C_EQUAL(*tau, MAGMA_C_ZERO) ) {
        const int i = threadIdx.x, col= threadIdx.y;

        for( int k = col; k < n; k += BLOCK_SIZEy ) {
            magmaFloatComplex *dc = c + k * ldc;
    
            __shared__ magmaFloatComplex sum[ BLOCK_SIZEx ][ BLOCK_SIZEy + 1];
            magmaFloatComplex lsum;
    
            /*  w := v' * C  */
            lsum = MAGMA_C_ZERO;
            for( int j = i; j < m; j += BLOCK_SIZEx ){
                if (j==0)
                   lsum += MAGMA_C_MUL( MAGMA_C_ONE, dc[j] );
                else
                   lsum += MAGMA_C_MUL( MAGMA_C_CNJG( v[j] ), dc[j] );
            }
            sum[i][col] = lsum;
            magma_sum_reduce_2d< BLOCK_SIZEx, BLOCK_SIZEy+1 >( i, col, sum );
    
            /*  C := C - v * w  */
            __syncthreads();
            magmaFloatComplex z__1 = - MAGMA_C_CNJG(*tau) * sum[0][col];
            for( int j = m-i-1; j>=0 ; j -= BLOCK_SIZEx ) {
                 if (j==0)
                    dc[j] += z__1;
                 else
                    dc[j] += z__1 * v[j];
            }
            __syncthreads();
    
            /* Adjust the rest of the column norms */
            // if (i==0){
            //    float temp = MAGMA_C_ABS( dc[0] ) / xnorm[k];
            //    temp = (temp + 1.) * (1. - temp);
            //    xnorm[k] = xnorm[k] * sqrt(temp);
            // }
        }
    }
}

//==============================================================================

/*
    Apply a complex elementary reflector H to a complex M-by-N
    matrix C from the left. H is represented in the form
          H = I - tau * v * v'
    where tau is a complex scalar and v is a complex vector.
    If tau = 0, then H is taken to be the unit matrix.

    To apply H' (the conjugate transpose of H), supply conjg(tau)
    instead tau.

    This routine uses only one SM (block).
 */
extern "C" void
magma_clarf_sm(int m, int n, magmaFloatComplex *v, magmaFloatComplex *tau,
               magmaFloatComplex *c, int ldc, float *xnorm)
{
    dim3  blocks( 1 );
    dim3 threads( BLOCK_SIZEx, BLOCK_SIZEy );

    magma_clarf_smkernel<<< blocks, threads, 0, magma_stream >>>( m, n, v, tau, c, ldc, xnorm);
}

//==============================================================================
/*
    Apply a complex elementary reflector H to a complex M-by-N
    matrix C from the left. H is represented in the form
          H = I - tau * v * v'
    where tau is a complex scalar and v is a complex vector.
    If tau = 0, then H is taken to be the unit matrix.

    To apply H' (the conjugate transpose of H), supply conjg(tau) 
    instead tau.

    The norms of v(:, 1:n) are given as input in xnorm(1:n). On exit, the norms
    are adjusted to hold the norms of v(2:m,2:n). This is a difference with the 
    LAPACK's clarf routine. 
 */

extern "C" magma_int_t
magma_clarf_gpu(
    magma_int_t m,  magma_int_t n,
    magmaFloatComplex *v, magmaFloatComplex *tau,
    magmaFloatComplex *c,  magma_int_t ldc, float *xnorm)
{
    dim3  blocks( n );
    dim3 threads( BLOCK_SIZE );

    magma_clarf_kernel<<< blocks, threads, 0, magma_stream >>>( m, v, tau, c, ldc, xnorm);

    // The computation can be done on 1 SM with the following routine.
    // magma_clarf_sm(m, n, v, tau, c, ldc, xnorm);

    return MAGMA_SUCCESS;
}

//==============================================================================
