#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.5.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date September 2014

       @generated from zlaswp.cu normal z -> d, Wed Sep 17 15:08:23 2014
       
       @author Stan Tomov
       @author Mathieu Faverge
       @author Ichitaro Yamazaki
       @author Mark Gates
*/
#include "common_magma.h"

// MAX_PIVOTS is maximum number of pivots to apply in each kernel launch
// NTHREADS is number of threads in a block
// 64 and 256 are better on Kepler; 
//#define MAX_PIVOTS 64
//#define NTHREADS   256
#define MAX_PIVOTS 32
#define NTHREADS   64

typedef struct {
    double *dAT;
    int n, lda, j0, npivots;
    int ipiv[MAX_PIVOTS];
} dlaswp_params_t;


// Matrix A is stored row-wise in dAT.
// Divide matrix A into block-columns of NTHREADS columns each.
// Each GPU block processes one block-column of A.
// Each thread goes down a column of A,
// swapping rows according to pivots stored in params.
__global__ void dlaswp_kernel( dlaswp_params_t params )
{
    unsigned int tid = threadIdx.x + blockDim.x*blockIdx.x;
    if( tid < params.n ) {
        int lda = params.lda;
        double *dAT = params.dAT + tid + params.j0*lda;
        double *A1  = dAT;
        
        for( int i1 = 0; i1 < params.npivots; ++i1 ) {
            int i2 = params.ipiv[i1];
            double *A2 = dAT + i2*lda;
            double temp = *A1;
            *A1 = *A2;
            *A2 = temp;
            A1 += lda;  // A1 = dA + i1*ldx
        }
    }
}


// Launch dlaswp kernel with ceil( n / NTHREADS ) blocks of NTHREADS threads each.
extern "C" void dlaswp_launch( dlaswp_params_t &params, magma_queue_t queue )
{
    int blocks = (params.n + NTHREADS - 1) / NTHREADS;
    dlaswp_kernel<<< blocks, NTHREADS, 0, queue >>>( params );
}


// @deprecated
// Swap rows of A, stored row-wise.
// This version updates each entry of ipiv by adding ind.
// (In contrast, LAPACK applies laswp, then updates ipiv.)
// It is used in dgetrf, dgetrf_gpu, dgetrf_mgpu, dgetrf_ooc.
extern "C" void
magmablas_dpermute_long2( magma_int_t n, double *dAT, magma_int_t lda,
                          magma_int_t *ipiv, magma_int_t nb, magma_int_t ind )
{
    for( int k = 0; k < nb; k += MAX_PIVOTS ) {
        int npivots = min( MAX_PIVOTS, nb-k );
        // fields are:             dAT  n  lda  j0       npivots
        dlaswp_params_t params = { dAT, n, lda, ind + k, npivots };
        for( int j = 0; j < npivots; ++j ) {
            params.ipiv[j] = ipiv[ind + k + j] - k - 1;
            ipiv[ind + k + j] += ind;
        }
        dlaswp_launch( params, magma_stream );
    }
}


// @deprecated
// Swap rows of A, stored row-wise.
// This version assumes ind has already been added to ipiv.
// (In contrast, LAPACK applies laswp, then updates ipiv.)
// It is used in dgetrf_mgpu, dgetrf_ooc.
extern "C" void
magmablas_dpermute_long3( double *dAT, magma_int_t lda,
                          const magma_int_t *ipiv, magma_int_t nb, magma_int_t ind )
{
    for( int k = 0; k < nb; k += MAX_PIVOTS ) {
        int npivots = min( MAX_PIVOTS, nb-k );
        // fields are:             dAT  n    lda  j0       npivots
        dlaswp_params_t params = { dAT, lda, lda, ind + k, npivots };
        for( int j = 0; j < MAX_PIVOTS; ++j ) {
            params.ipiv[j] = ipiv[ind + k + j] - k - 1 - ind;
        }
        dlaswp_launch( params, magma_stream );
    }
}


/**
    Purpose:
    =============
    DLASWP performs a series of row interchanges on the matrix A.
    One row interchange is initiated for each of rows K1 through K2 of A.
    
    ** Unlike LAPACK, here A is stored row-wise (hence dAT). **
    Otherwise, this is identical to LAPACK's interface.
    
    Arguments:
    ==========
    \param[in]
    n        INTEGER
             The number of columns of the matrix A.
    
    \param[in,out]
    dAT      DOUBLE PRECISION array on GPU, stored row-wise, dimension (LDA,N)
             On entry, the matrix of column dimension N to which the row
             interchanges will be applied.
             On exit, the permuted matrix.
    
    \param[in]
    lda      INTEGER
             The leading dimension of the array A. lda >= n.
    
    \param[in]
    k1       INTEGER
             The first element of IPIV for which a row interchange will
             be done. (Fortran one-based index: 1 <= k1 <= n.)
    
    \param[in]
    k2       INTEGER
             The last element of IPIV for which a row interchange will
             be done. (Fortran one-based index: 1 <= k2 <= n.)
    
    \param[in]
    ipiv     INTEGER array, on CPU, dimension (K2*abs(INCI))
             The vector of pivot indices.  Only the elements in positions
             K1 through K2 of IPIV are accessed.
             IPIV(K) = L implies rows K and L are to be interchanged.
    
    \param[in]
    inci     INTEGER
             The increment between successive values of IPIV.
             Currently, IPIV > 0.
             TODO: If IPIV is negative, the pivots are applied in reverse order.

    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_daux2
    ********************************************************************/
// It is used in dgessm, dgetrf_incpiv.
extern "C" void
magmablas_dlaswp_q(
    magma_int_t n, double *dAT, magma_int_t lda,
    magma_int_t k1, magma_int_t k2,
    const magma_int_t *ipiv, magma_int_t inci,
    magma_queue_t queue )
{
    magma_int_t info = 0;
    if ( n < 0 )
        info = -1;
    else if ( k1 < 1 || k1 > n )
        info = -4;
    else if ( k2 < 1 || k2 > n )
        info = -5;
    else if ( inci <= 0 )
        info = -7;

    if (info != 0) {
        magma_xerbla( __func__, -(info) );
        return;  //info;
    }
    
    for( int k = k1-1; k < k2; k += MAX_PIVOTS ) {
        int npivots = min( MAX_PIVOTS, k2-k );
        // fields are:             dAT        n  lda  j0 npivots
        dlaswp_params_t params = { dAT+k*lda, n, lda, 0, npivots };
        for( int j = 0; j < npivots; ++j ) {
            params.ipiv[j] = ipiv[(k+j)*inci] - k - 1;
        }
        dlaswp_launch( params, queue );
    }
}


/**
    @see magmablas_dlaswp_q
    @ingroup magma_daux2
    ********************************************************************/
extern "C" void
magmablas_dlaswp( magma_int_t n, double *dAT, magma_int_t lda,
                  magma_int_t k1, magma_int_t k2,
                  const magma_int_t *ipiv, magma_int_t inci )
{
    magmablas_dlaswp_q( n, dAT, lda, k1, k2, ipiv, inci, magma_stream );
}






// ------------------------------------------------------------
// Extended version has stride in both directions (ldx, ldy)
// to handle both row-wise and column-wise storage.

typedef struct {
    double *dA;
    int n, ldx, ldy, j0, npivots;
    int ipiv[MAX_PIVOTS];
} dlaswpx_params_t;


// Matrix A is stored row or column-wise in dA.
// Divide matrix A into block-columns of NTHREADS columns each.
// Each GPU block processes one block-column of A.
// Each thread goes down a column of A,
// swapping rows according to pivots stored in params.
__global__ void dlaswpx_kernel( dlaswpx_params_t params )
{
    unsigned int tid = threadIdx.x + blockDim.x*blockIdx.x;
    if( tid < params.n ) {
        int ldx = params.ldx;
        double *dA = params.dA + tid*params.ldy + params.j0*ldx;
        double *A1  = dA;
        
        for( int i1 = 0; i1 < params.npivots; ++i1 ) {
            int i2 = params.ipiv[i1];
            double *A2 = dA + i2*ldx;
            double temp = *A1;
            *A1 = *A2;
            *A2 = temp;
            A1 += ldx;  // A1 = dA + i1*ldx
        }
    }
}


// Launch dlaswpx kernel with ceil( n / NTHREADS ) blocks of NTHREADS threads each.
extern "C" void dlaswpx( dlaswpx_params_t &params, magma_queue_t queue )
{
    int blocks = (params.n + NTHREADS - 1) / NTHREADS;
    dlaswpx_kernel<<< blocks, NTHREADS, 0, queue >>>( params );
}


/**
    Purpose:
    =============
    DLASWPX performs a series of row interchanges on the matrix A.
    One row interchange is initiated for each of rows K1 through K2 of A.
    
    ** Unlike LAPACK, here A is stored either row-wise or column-wise,
       depending on ldx and ldy. **
    Otherwise, this is identical to LAPACK's interface.
    
    Arguments:
    ==========
    \param[in]
    n        INTEGER
             The number of columns of the matrix A.
    
    \param[in,out]
    dA       DOUBLE PRECISION array on GPU, dimension (*,*)
             On entry, the matrix of column dimension N to which the row
             interchanges will be applied.
             On exit, the permuted matrix.
    
    \param[in]
    ldx      INTEGER
             Stride between elements in same column.
    
    \param[in]
    ldy      INTEGER
             Stride between elements in same row.
             For A stored row-wise,    set ldx=lda and ldy=1.
             For A stored column-wise, set ldx=1   and ldy=lda.
    
    \param[in]
    k1       INTEGER
             The first element of IPIV for which a row interchange will
             be done. (One based index.)
    
    \param[in]
    k2       INTEGER
             The last element of IPIV for which a row interchange will
             be done. (One based index.)
    
    \param[in]
    ipiv     INTEGER array, on CPU, dimension (K2*abs(INCI))
             The vector of pivot indices.  Only the elements in positions
             K1 through K2 of IPIV are accessed.
             IPIV(K) = L implies rows K and L are to be interchanged.
    
    \param[in]
    inci     INTEGER
             The increment between successive values of IPIV.
             Currently, IPIV > 0.
             TODO: If IPIV is negative, the pivots are applied in reverse order.

    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_daux2
    ********************************************************************/
extern "C" void
magmablas_dlaswpx_q(
    magma_int_t n, double *dA, magma_int_t ldx, magma_int_t ldy,
    magma_int_t k1, magma_int_t k2,
    const magma_int_t *ipiv, magma_int_t inci,
    magma_queue_t queue )
{
    magma_int_t info = 0;
    if ( n < 0 )
        info = -1;
    else if ( k1 < 0 )
        info = -4;  
    else if ( k2 < 0 || k2 < k1 )
        info = -5;
    else if ( inci <= 0 )
        info = -7;

    if (info != 0) {
        magma_xerbla( __func__, -(info) );
        return;  //info;
    }
    
    for( int k = k1-1; k < k2; k += MAX_PIVOTS ) {
        int npivots = min( MAX_PIVOTS, k2-k );
        // fields are:              dA        n  ldx  ldy  j0 npivots
        dlaswpx_params_t params = { dA+k*ldx, n, ldx, ldy, 0, npivots };
        for( int j = 0; j < npivots; ++j ) {
            params.ipiv[j] = ipiv[(k+j)*inci] - k - 1;
        }
        dlaswpx( params, queue );
    }
}


/**
    @see magmablas_dlaswpx_q
    @ingroup magma_daux2
    ********************************************************************/
extern "C" void
magmablas_dlaswpx( magma_int_t n, double *dA, magma_int_t ldx, magma_int_t ldy,
                   magma_int_t k1, magma_int_t k2,
                   const magma_int_t *ipiv, magma_int_t inci )
{
    return magmablas_dlaswpx_q( n, dA, ldx, ldy, k1, k2, ipiv, inci, magma_stream );
}






// ------------------------------------------------------------
// This version takes d_ipiv on the GPU. Thus it does not pass pivots
// as an argument using a structure, avoiding all the argument size
// limitations of CUDA and OpenCL. It also needs just one kernel launch
// with all the pivots, instead of multiple kernel launches with small
// batches of pivots. On Fermi, it is faster than magmablas_dlaswp
// (including copying pivots to the GPU).

__global__ void dlaswp2_kernel(
    int n, double *dAT, int lda, int npivots,
    const magma_int_t* d_ipiv, magma_int_t inci )
{
    unsigned int tid = threadIdx.x + blockDim.x*blockIdx.x;
    if( tid < n ) {
        dAT += tid;
        double *A1  = dAT;
        
        for( int i1 = 0; i1 < npivots; ++i1 ) {
            int i2 = d_ipiv[i1*inci] - 1;  // Fortran index
            double *A2 = dAT + i2*lda;
            double temp = *A1;
            *A1 = *A2;
            *A2 = temp;
            A1 += lda;  // A1 = dA + i1*ldx
        }
    }
}


/**
    Purpose:
    =============
    DLASWP2 performs a series of row interchanges on the matrix A.
    One row interchange is initiated for each of rows K1 through K2 of A.
    
    ** Unlike LAPACK, here A is stored row-wise (hence dAT). **
    Otherwise, this is identical to LAPACK's interface.
    
    Here, d_ipiv is passed in GPU memory.
    
    Arguments:
    ==========
    \param[in]
    n        INTEGER
             The number of columns of the matrix A.
    
    \param[in,out]
    dAT      DOUBLE PRECISION array on GPU, stored row-wise, dimension (LDA,*)
             On entry, the matrix of column dimension N to which the row
             interchanges will be applied.
             On exit, the permuted matrix.
    
    \param[in]
    lda      INTEGER
             The leading dimension of the array A.
             (I.e., stride between elements in a column.)
    
    \param[in]
    k1       INTEGER
             The first element of IPIV for which a row interchange will
             be done. (One based index.)
    
    \param[in]
    k2       INTEGER
             The last element of IPIV for which a row interchange will
             be done. (One based index.)
    
    \param[in]
    d_ipiv   INTEGER array, on GPU, dimension (K2*abs(INCI))
             The vector of pivot indices.  Only the elements in positions
             K1 through K2 of IPIV are accessed.
             IPIV(K) = L implies rows K and L are to be interchanged.
    
    \param[in]
    inci     INTEGER
             The increment between successive values of IPIV.
             Currently, IPIV > 0.
             TODO: If IPIV is negative, the pivots are applied in reverse order.

    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_daux2
    ********************************************************************/
extern "C" void
magmablas_dlaswp2_q(
    magma_int_t n, double* dAT, magma_int_t lda,
    magma_int_t k1, magma_int_t k2,
    const magma_int_t *d_ipiv, magma_int_t inci,
    magma_queue_t queue )
{
    magma_int_t info = 0;
    if ( n < 0 )
        info = -1;
    else if ( k1 < 0 )
        info = -4;  
    else if ( k2 < 0 || k2 < k1 )
        info = -5;
    else if ( inci <= 0 )
        info = -7;

    if (info != 0) {
        magma_xerbla( __func__, -(info) );
        return;  //info;
    }
    
    int blocks = (n + NTHREADS - 1) / NTHREADS;
    dlaswp2_kernel<<< blocks, NTHREADS, 0, queue >>>(
        n, dAT + (k1-1)*lda, lda, k2-(k1-1), d_ipiv, inci );
}


/**
    @see magmablas_dlaswp2_q
    @ingroup magma_daux2
    ********************************************************************/
extern "C" void
magmablas_dlaswp2( magma_int_t n, double* dAT, magma_int_t lda,
                   magma_int_t k1, magma_int_t k2,
                   const magma_int_t *d_ipiv, magma_int_t inci )
{
    magmablas_dlaswp2_q( n, dAT, lda, k1, k2, d_ipiv, inci, magma_stream );
}
