#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.5.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date September 2014

       @generated from zswap.cu normal z -> d, Wed Sep 17 15:08:23 2014

*/
#include "common_magma.h"

#define BLOCK_SIZE 64

/*********************************************************
 *
 * SWAP BLAS: permute to set of N elements
 *
 ********************************************************/
/*
 *  First version: line per line
 */
typedef struct {
    double *A1;
    double *A2;
    int n, lda1, lda2;
} magmagpu_dswap_params_t;

__global__ void magmagpu_dswap( magmagpu_dswap_params_t params )
{
    unsigned int x = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned int offset1 = x*params.lda1;
    unsigned int offset2 = x*params.lda2;
    if( x < params.n )
    {
        double *A1  = params.A1 + offset1;
        double *A2  = params.A2 + offset2;
        double temp = *A1;
        *A1 = *A2;
        *A2 = temp;
    }
}


extern "C" void 
magmablas_dswap_q(
    magma_int_t n, double *dA1T, magma_int_t lda1, 
    double *dA2T, magma_int_t lda2,
    magma_queue_t queue )
{
    int blocksize = 64;
    dim3 blocks( (n+blocksize-1) / blocksize, 1, 1);
    magmagpu_dswap_params_t params = { dA1T, dA2T, n, lda1, lda2 };
    magmagpu_dswap<<< blocks, blocksize, 0, queue >>>( params );
}


extern "C" void 
magmablas_dswap(
    magma_int_t n, double *dA1T, magma_int_t lda1, 
    double *dA2T, magma_int_t lda2)
{
    magmablas_dswap_q( n, dA1T, lda1, dA2T, lda2, magma_stream );
}
