#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.4.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       December 2013

       @generated d Tue Dec 17 13:18:45 2013

*/
#include "common_magma.h"

#define BLOCK_SIZE 64

/*********************************************************
 *
 * SWAP BLAS: permute to set of N elements
 *
 ********************************************************/
/*
 *  First version: line per line
 */
typedef struct {
    double *A1;
    double *A2;
    int n, lda1, lda2;
} magmagpu_dswap_params_t;

__global__ void magmagpu_dswap( magmagpu_dswap_params_t params )
{
    unsigned int x = threadIdx.x + __mul24(blockDim.x, blockIdx.x);
    unsigned int offset1 = __mul24( x, params.lda1);
    unsigned int offset2 = __mul24( x, params.lda2);
    if( x < params.n )
    {
        double *A1  = params.A1 + offset1;
        double *A2  = params.A2 + offset2;
        double temp = *A1;
        *A1 = *A2;
        *A2 = temp;
    }
}

extern "C" void 
magmablas_dswap( magma_int_t n, double *dA1T, magma_int_t lda1, 
                 double *dA2T, magma_int_t lda2)
{
    int blocksize = 64;
    dim3 blocks( (n+blocksize-1) / blocksize, 1, 1);
    magmagpu_dswap_params_t params = { dA1T, dA2T, n, lda1, lda2 };
    magmagpu_dswap<<< blocks, blocksize, 0, magma_stream >>>( params );
}

