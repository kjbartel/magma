#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.2.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       June 2012

       @generated s Thu Jun 28 12:31:18 2012

*/
#include "common_magma.h"

#define BLOCK_SIZE 64

typedef struct {
        float *A;
        int n, lda, j0;
        short ipiv[BLOCK_SIZE];
} slaswp_params_t;

typedef struct {
        float *A;
        int n, lda, j0, npivots;
        short ipiv[BLOCK_SIZE];
} slaswp_params_t2;

/*********************************************************
 *
 * LAPACK Swap: permute a set of lines following ipiv
 *
 ********************************************************/
typedef struct {
    float *A;
    int n, ldx, ldy, j0, npivots;
    short ipiv[BLOCK_SIZE];
} slaswpx_params_t;

__global__ void myslaswpx( slaswpx_params_t params )
{
    unsigned int y = threadIdx.x + __mul24(blockDim.x, blockIdx.x);
    unsigned int offset1 = __mul24( y, params.ldy);
    if( y < params.n )
    {
        int ldx = params.ldx;
        float *A = params.A + offset1 + ldx * params.j0;
        float *Ai = A;
        
        for( int i = 0; i < params.npivots; i++ )
        {
            int j = params.ipiv[i];
            float *p2 = A + j*ldx;
            float temp = *Ai;
            *Ai = *p2;
            *p2 = temp;
            Ai += ldx;
        }
    }
}

extern "C" void slaswpx( slaswpx_params_t &params )
{
         int blocksize = 64;
        dim3 blocks = (params.n+blocksize-1) / blocksize;
        myslaswpx<<< blocks, blocksize, 0, magma_stream >>>( params );
}

/*
 * Old version
 */
__global__ void myslaswp2( slaswp_params_t2 params )
{
        unsigned int tid = threadIdx.x + __mul24(blockDim.x, blockIdx.x);
        if( tid < params.n )
        {
                int lda = params.lda;
                float *A = params.A + tid + lda * params.j0;

                for( int i = 0; i < params.npivots; i++ )
                {
                         int j = params.ipiv[i];
                        float *p1 = A + i*lda;
                        float *p2 = A + j*lda;
                        float temp = *p1;
                        *p1 = *p2;
                        *p2 = temp;
                }
        }
}

extern "C" void slaswp2( slaswp_params_t &params );

extern "C" void slaswp3( slaswp_params_t2 &params )
{
         int blocksize = 64;
        dim3 blocks = (params.n+blocksize-1) / blocksize;
        myslaswp2<<< blocks, blocksize, 0, magma_stream >>>( params );
}


extern "C" void 
magmablas_spermute_long2( magma_int_t n, float *dAT, magma_int_t lda,
                          magma_int_t *ipiv, magma_int_t nb, magma_int_t ind )
{
        int k;

        for( k = 0; k < nb-BLOCK_SIZE; k += BLOCK_SIZE )
        {
                //slaswp_params_t params = { dAT, lda, lda, ind + k };
                slaswp_params_t2 params = { dAT, n, lda, ind + k, BLOCK_SIZE };
                for( int j = 0; j < BLOCK_SIZE; j++ )
                {
                        params.ipiv[j] = ipiv[ind + k + j] - k - 1;
                        ipiv[ind + k + j] += ind;
                }
                //slaswp2( params );
                slaswp3( params );
        }

        int num_pivots = nb - k;

        slaswp_params_t2 params = { dAT, n, lda, ind + k, num_pivots};
        for( int j = 0; j < num_pivots; j++ )
        {
            params.ipiv[j] = ipiv[ind + k + j] - k - 1;
            ipiv[ind + k + j] += ind;
        }
        slaswp3( params );
}

extern "C" void 
magmablas_slaswp( magma_int_t n, float *dAT, magma_int_t lda, 
                  magma_int_t i1, magma_int_t i2, magma_int_t *ipiv, magma_int_t inci )
{
  int k;
  
  for( k=(i1-1); k<i2; k+=BLOCK_SIZE )
    {
      int sb = min(BLOCK_SIZE, i2-k);
      //slaswp_params_t params = { dAT, lda, lda, ind + k };
      slaswp_params_t2 params = { dAT+k*lda, n, lda, 0, sb };
      for( int j = 0; j < sb; j++ )
        {
          params.ipiv[j] = ipiv[(k+j)*inci] - k - 1;
        }
      slaswp3( params );
    }
}

extern "C" void 
magmablas_slaswpx( magma_int_t n, float *dAT, magma_int_t ldx, magma_int_t ldy, 
                   magma_int_t i1, magma_int_t i2, magma_int_t *ipiv, magma_int_t inci )
{
  int k;
  
  for( k=(i1-1); k<i2; k+=BLOCK_SIZE )
    {
      int sb = min(BLOCK_SIZE, i2-k);
      //slaswp_params_t params = { dAT, lda, lda, ind + k };
      slaswpx_params_t params = { dAT+k*ldx, n, ldx, ldy, 0, sb };
      for( int j = 0; j < sb; j++ )
        {
          params.ipiv[j] = ipiv[(k+j)*inci] - k - 1;
        }
      slaswpx( params );
    }
}

#undef BLOCK_SIZE
