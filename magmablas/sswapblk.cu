#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

       @generated s Sun Nov 13 20:48:38 2011

*/
#include "common_magma.h"

#define BLOCK_SIZE 64

/*********************************************************/
/*
*  Blocked version: swap several pair of line
 */
typedef struct {
    float *A1;
    float *A2;
    int n, lda1, lda2, npivots;
    short ipiv[BLOCK_SIZE];
} magmagpu_sswapblk_params_t;

__global__ void magmagpu_sswapblkrm( magmagpu_sswapblk_params_t params )
{
    unsigned int y = threadIdx.x + blockDim.x*blockIdx.x;
    if( y < params.n )
    {
        float *A1 = params.A1 + y - params.lda1;
        float *A2 = params.A2 + y;
      
        for( int i = 0; i < params.npivots; i++ )
        {
            A1 += params.lda1;
            if ( params.ipiv[i] == -1 )
                continue;
            float tmp1  = *A1;
            float *tmp2 = A2 + params.ipiv[i]*params.lda2;
            *A1   = *tmp2;
            *tmp2 = tmp1;
        }
    }
}

__global__ void magmagpu_sswapblkcm( magmagpu_sswapblk_params_t params )
{
    unsigned int y = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned int offset1 = __mul24( y, params.lda1);
    unsigned int offset2 = __mul24( y, params.lda2);
    if( y < params.n )
    {
        float *A1 = params.A1 + offset1 - 1;
        float *A2 = params.A2 + offset2;
      
        for( int i = 0; i < params.npivots; i++ )
        {
            A1++;
            if ( params.ipiv[i] == -1 )
                continue;
            float tmp1  = *A1;
            float *tmp2 = A2 + params.ipiv[i];
            *A1   = *tmp2;
            *tmp2 = tmp1;
        }
    }
    __syncthreads();
}

extern "C" void 
magmablas_sswapblk( char storev, int n, 
                    float *dA1T, int lda1,
                    float *dA2T, int lda2,
                    int i1, int i2, int *ipiv, int inci, int offset )
{
    int  blocksize = 64;
    dim3 blocks( (n+blocksize-1) / blocksize, 1, 1);
    int  k, im;

    if ( (storev == 'C') || (storev == 'c') ) {
        for( k=(i1-1); k<i2; k+=BLOCK_SIZE )
        {
            int sb = min(BLOCK_SIZE, i2-k);
            magmagpu_sswapblk_params_t params = { dA1T+k, dA2T, n, lda1, lda2, sb };
            for( int j = 0; j < sb; j++ )
            {
                im = ipiv[(k+j)*inci] - 1;
                if ( (k+j) == im)
                    params.ipiv[j] = -1;
                else
                    params.ipiv[j] = im - offset;
            }
            magmagpu_sswapblkcm<<< blocks, blocksize, 0, magma_stream >>>( params );
        }
    }else {
        for( k=(i1-1); k<i2; k+=BLOCK_SIZE )
        {
            int sb = min(BLOCK_SIZE, i2-k);
            magmagpu_sswapblk_params_t params = { dA1T+k*lda1, dA2T, n, lda1, lda2, sb };
            for( int j = 0; j < sb; j++ )
            {
                im = ipiv[(k+j)*inci] - 1;
                if ( (k+j) == im)
                    params.ipiv[j] = -1;
                else
                    params.ipiv[j] = im - offset;
            }
            magmagpu_sswapblkrm<<< blocks, blocksize, 0, magma_stream >>>( params );
        }
    }
}

