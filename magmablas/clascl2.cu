#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @generated from zlascl2.cu normal z -> c, Fri Jan 30 19:00:09 2015

       @author Theo Mary
*/
#include "common_magma.h"

#define NB 64


// each thread block does one NB x n block row of A.
// each thread does one row, starting from left edge and moving right.
__global__ void
clascl2_full(int m, int n, const float* D, magmaFloatComplex* A, int lda)
{
    int ind = blockIdx.x * NB + threadIdx.x;

    float mul = D[ind];
    A += ind;
    if (ind < m) {
        for(int j=0; j < n; j++ )
            A[j*lda] *= mul;
    }
}


// each thread block does one NB x n block row of A.
// each thread does one row, starting from left edge and moving right to diagonal.
__global__ void
clascl2_lower(int m, int n, const float* D, magmaFloatComplex* A, int lda)
{
    int ind = blockIdx.x * NB + threadIdx.x;

    int break_d = (ind < n) ? ind : n-1;

    float mul = D[ind];
    A += ind;
    if (ind < m) {
        for(int j=0; j <= break_d; j++ )
            A[j*lda] *= mul;
    }
}


// each thread block does one NB x n block row of A.
// each thread does one row, starting from right edge and moving left to diagonal.
__global__ void
clascl2_upper(int m, int n, const float *D, magmaFloatComplex* A, int lda)
{
    int ind = blockIdx.x * NB + threadIdx.x;

    float mul = D[ind];
    A += ind;
    if (ind < m) {
        for(int j=n-1; j >= ind; j--)
            A[j*lda] *= mul;
    }
}


/**
    Purpose
    -------
    CLASCL2 scales the M by N complex matrix A by the real diagonal matrix dD.
    TYPE specifies that A may be full, upper triangular, lower triangular.

    Arguments
    ---------
    \param[in]
    type    magma_type_t
            TYPE indices the storage type of the input matrix A.
            = MagmaFull:   full matrix.
            = MagmaLower:  lower triangular matrix.
            = MagmaUpper:  upper triangular matrix.
            Other formats that LAPACK supports, MAGMA does not currently support.

    \param[in]
    m       INTEGER
            The number of rows of the matrix A.  M >= 0.

    \param[in]
    n       INTEGER
            The number of columns of the matrix A.  N >= 0.

    \param[in]
    dD      REAL vector, dimension (M)
            The diagonal matrix containing the scalar factors. Stored as a vector.

    \param[in,out]
    dA      COMPLEX array, dimension (LDDA,N)
            The matrix to be scaled by dD.  See TYPE for the
            storage type.

    \param[in]
    ldda    INTEGER
            The leading dimension of the array A.  LDDA >= max(1,M).

    \param[out]
    info    INTEGER
      -     = 0:  successful exit
      -     < 0:  if INFO = -i, the i-th argument had an illegal value.
    
    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_caux2
    ********************************************************************/
extern "C" void
magmablas_clascl2_q(
    magma_type_t type, magma_int_t m, magma_int_t n,
    magmaFloat_const_ptr dD,
    magmaFloatComplex_ptr dA, magma_int_t ldda,
    magma_queue_t queue,
    magma_int_t *info )
{
    *info = 0;
    if ( type != MagmaLower && type != MagmaUpper && type != MagmaFull )
        *info = -1;
    else if ( m < 0 )
        *info = -2;
    else if ( n < 0 )
        *info = -3;
    else if ( ldda < max(1,m) )
        *info = -5;
    
    if (*info != 0) {
        magma_xerbla( __func__, -(*info) );
        return;  //info;
    }
    
    dim3 grid( (m + NB - 1)/NB );
    dim3 threads( NB );
    
    if (type == MagmaLower) {
        clascl2_lower <<< grid, threads, 0, queue >>> (m, n, dD, dA, ldda);
    }
    else if (type == MagmaUpper) {
        clascl2_upper <<< grid, threads, 0, queue >>> (m, n, dD, dA, ldda);
    }
    else if (type == MagmaFull) {
        clascl2_full  <<< grid, threads, 0, queue >>> (m, n, dD, dA, ldda);
    }
}


/**
    @see magmablas_clascl2_q
    @ingroup magma_caux2
    ********************************************************************/
extern "C" void
magmablas_clascl2(
    magma_type_t type, magma_int_t m, magma_int_t n,
    magmaFloat_const_ptr dD,
    magmaFloatComplex_ptr dA, magma_int_t ldda, magma_int_t *info )
{
    magmablas_clascl2_q( type, m, n, dD, dA, ldda, magma_stream, info );
}
