#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.5.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date September 2014

       @generated from zlarf.cu normal z -> s, Wed Sep 17 15:08:23 2014
       @author Azzam Haidar

*/
#include "common_magma.h"
#include "magma_templates.h"

// 512 is maximum number of threads for CUDA capability 1.x
#define BLOCK_SIZE 512

#define BLOCK_SIZEx  32
#define BLOCK_SIZEy  16


//==============================================================================
//==============================================================================

__global__
void magma_slarf_kernel( int m, const float *dv, const float *dtau,
                         float *dc, int lddc )
{
    if ( !MAGMA_S_EQUAL(*dtau, MAGMA_S_ZERO) ) {
        const int tx = threadIdx.x;
        dc = dc + blockIdx.x * lddc;

        __shared__ float sum[ BLOCK_SIZE ];
        float tmp;

        /* perform  w := v' * C  */
        if (tx==0)
            tmp = dc[0]; //since V[0] should be one
        else
            tmp = MAGMA_S_ZERO;
        for( int j = tx+1; j < m; j += BLOCK_SIZE ){
            tmp += MAGMA_S_MUL( MAGMA_S_CNJG( dv[j] ), dc[j] );
        }
        sum[tx] = tmp;
        magma_sum_reduce< BLOCK_SIZE >( tx, sum );

        /*  C := C - v * w  */
        __syncthreads();
        tmp = - MAGMA_S_CNJG(*dtau) * sum[0];
        for( int j = m-tx-1; j>0 ; j -= BLOCK_SIZE )
             dc[j] += tmp * dv[j];

        if(tx==0) dc[0] += tmp;
    }
}

//==============================================================================
//==============================================================================

__global__
void magma_slarf_smkernel( int m, int n, float *dv, float *dtau,
                           float *dc, int lddc )
{
    if ( ! MAGMA_S_EQUAL(*dtau, MAGMA_S_ZERO) ) {
        const int i = threadIdx.x, col= threadIdx.y;

        for( int k = col; k < n; k += BLOCK_SIZEy ) {
            dc = dc + k * lddc;
    
            __shared__ float sum[ BLOCK_SIZEx ][ BLOCK_SIZEy + 1];
            float lsum;
    
            /*  w := v' * C  */
            lsum = MAGMA_S_ZERO;
            for( int j = i; j < m; j += BLOCK_SIZEx ){
                if (j==0)
                   lsum += MAGMA_S_MUL( MAGMA_S_ONE, dc[j] );
                else
                   lsum += MAGMA_S_MUL( MAGMA_S_CNJG( dv[j] ), dc[j] );
            }
            sum[i][col] = lsum;
            magma_sum_reduce_2d< BLOCK_SIZEx, BLOCK_SIZEy+1 >( i, col, sum );
    
            /*  C := C - v * w  */
            __syncthreads();
            float z__1 = - MAGMA_S_CNJG(*dtau) * sum[0][col];
            for( int j = m-i-1; j>=0 ; j -= BLOCK_SIZEx ) {
                 if (j==0)
                    dc[j] += z__1;
                 else
                    dc[j] += z__1 * dv[j];
            }
        }
    }
}

//==============================================================================

/*
    Apply a real elementary reflector H to a real M-by-N
    matrix C from the left. H is represented in the form
          H = I - tau * v * v'
    where tau is a real scalar and v is a real vector.
    If tau = 0, then H is taken to be the unit matrix.

    To apply H' (the conjugate transpose of H), supply conjg(tau)
    instead tau.

    This routine uses only one SM (block).
 */
extern "C" void
magma_slarf_sm(magma_int_t m, magma_int_t n, float *dv, float *dtau,
               float *dc, magma_int_t lddc)
{
    dim3  blocks( 1 );
    dim3 threads( BLOCK_SIZEx, BLOCK_SIZEy );

    magma_slarf_smkernel<<< blocks, threads, 0, magma_stream >>>( m, n, dv, dtau, dc, lddc );
}
//==============================================================================
/*
    Apply a real elementary reflector H to a real M-by-N
    matrix C from the left. H is represented in the form
          H = I - tau * v * v'
    where tau is a real scalar and v is a real vector.
    If tau = 0, then H is taken to be the unit matrix.

    To apply H' (the conjugate transpose of H), supply conjg(tau) 
    instead tau.

 */

extern "C" magma_int_t
magma_slarf_gpu(
    magma_int_t m,  magma_int_t n,
    const float *dv, const float *dtau,
    float *dc,  magma_int_t lddc)
{
    dim3 grid( n, 1, 1 );
    dim3 threads( BLOCK_SIZE );
    if ( n>0 ){
        magma_slarf_kernel<<< grid, threads, 0, magma_stream >>>( m, dv, dtau, dc, lddc);
    }

    // The computation can be done on 1 SM with the following routine.
    // magma_slarf_sm(m, n, dv, dtau, dc, lddc);

    return MAGMA_SUCCESS;
}

//==============================================================================
