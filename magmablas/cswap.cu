#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.4.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       December 2013

       @generated c Tue Dec 17 13:18:45 2013

*/
#include "common_magma.h"

#define BLOCK_SIZE 64

/*********************************************************
 *
 * SWAP BLAS: permute to set of N elements
 *
 ********************************************************/
/*
 *  First version: line per line
 */
typedef struct {
    magmaFloatComplex *A1;
    magmaFloatComplex *A2;
    int n, lda1, lda2;
} magmagpu_cswap_params_t;

__global__ void magmagpu_cswap( magmagpu_cswap_params_t params )
{
    unsigned int x = threadIdx.x + __mul24(blockDim.x, blockIdx.x);
    unsigned int offset1 = __mul24( x, params.lda1);
    unsigned int offset2 = __mul24( x, params.lda2);
    if( x < params.n )
    {
        magmaFloatComplex *A1  = params.A1 + offset1;
        magmaFloatComplex *A2  = params.A2 + offset2;
        magmaFloatComplex temp = *A1;
        *A1 = *A2;
        *A2 = temp;
    }
}

extern "C" void 
magmablas_cswap( magma_int_t n, magmaFloatComplex *dA1T, magma_int_t lda1, 
                 magmaFloatComplex *dA2T, magma_int_t lda2)
{
    int blocksize = 64;
    dim3 blocks( (n+blocksize-1) / blocksize, 1, 1);
    magmagpu_cswap_params_t params = { dA1T, dA2T, n, lda1, lda2 };
    magmagpu_cswap<<< blocks, blocksize, 0, magma_stream >>>( params );
}

