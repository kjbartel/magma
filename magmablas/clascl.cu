#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.4.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       December 2013

       @generated c Tue Dec 17 13:18:45 2013

*/
#include "common_magma.h"

#define clascl_bs 64


__global__ void
l_clascl (int m, int n, float mul, magmaFloatComplex* A, int lda){
    int ind =  blockIdx.x * clascl_bs + threadIdx.x ;

    int break_d = (ind < n)? ind: n-1;

    A += ind;
    if (ind < m)
       for(int j=0; j<=break_d; j++ )
           A[j*lda] *= mul;
}

__global__ void
u_clascl (int m, int n, float mul, magmaFloatComplex* A, int lda){
    int ind =  blockIdx.x * clascl_bs + threadIdx.x ;

    A += ind;
    if (ind < m)
      for(int j=n-1; j>= ind; j--)
         A[j*lda] *= mul;
}


extern "C" void
magmablas_clascl(char type, magma_int_t kl, magma_int_t ku, 
                 float cfrom, float cto,
                 magma_int_t m, magma_int_t n, 
                 magmaFloatComplex *A, magma_int_t lda, magma_int_t *info )
{
    int blocks;
    if (m % clascl_bs==0)
        blocks = m/ clascl_bs;
    else
        blocks = m/ clascl_bs + 1;

    dim3 grid(blocks, 1, 1);
    dim3 threads(clascl_bs, 1, 1);

    /* To do : implment the accuracy procedure */
    float mul = cto / cfrom;

    if (type == 'L' || type =='l')  
       l_clascl <<< grid, threads, 0, magma_stream >>> (m, n, mul, A, lda);
    else if (type == 'U' || type =='u')
       u_clascl <<< grid, threads, 0, magma_stream >>> (m, n, mul, A, lda);  
    else {
       printf("Only type L and U are available in clascl. Exit.\n");
       exit(1);
    }
}


