#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @generated from ztrtri_diag.cu normal z -> d, Fri Jan 30 19:00:09 2015

       @author Peng Du
       @author Tingxing Dong
       @author Mark Gates
       @author Azzam Haidar
       
       File named dtrtri_diag.cu to avoid name conflict with src/dtrtri.o
       in the library. The actual kernels are in dtrtri_lower.cu and dtrtri_upper.cu
*/

#include "common_magma.h"
#include "dtrtri.h"


/**
    Inverts the NB x NB diagonal blocks of a triangular matrix.
    This routine is used in dtrsm.
    
    Same as dtrtri_diag, but adds queue argument.
    
    @ingroup magma_dblas3
    ********************************************************************/
/**
    Purpose
    -------
    dtrtri_diag inverts the NB x NB diagonal blocks of A.

    Arguments
    ----------
    @param[in]
    uplo    magma_uplo_t.
            On entry, uplo specifies whether the matrix A is an upper or
            lower triangular matrix as follows:
      -     = MagmaUpper:  A is an upper triangular matrix.
      -     = MagmaLower:  A is a  lower triangular matrix.

    @param[in]
    diag    magma_diag_t.
            On entry, diag specifies whether or not A is unit triangular
            as follows:
      -     = MagmaUnit:     A is assumed to be unit triangular.
      -     = MagmaNonUnit:  A is not assumed to be unit triangular.

    @param[in]
    n       INTEGER.
            On entry, n specifies the order of the matrix A. N >= 0.

    @param[in]
    dA      DOUBLE_PRECISION array of dimension ( ldda, n )
            The triangular matrix A.
    \n
            If UPLO = 'U', the leading N-by-N upper triangular part of A
            contains the upper triangular matrix, and the strictly lower
            triangular part of A is not referenced.
    \n
            If UPLO = 'L', the leading N-by-N lower triangular part of A
            contains the lower triangular matrix, and the strictly upper
            triangular part of A is not referenced.
    \n
            If DIAG = 'U', the diagonal elements of A are also not referenced
            and are assumed to be 1.

    @param[in]
    ldda    INTEGER.
            The leading dimension of the array A.  LDDA >= max(1,N).

    @param[out]
    d_dinvA DOUBLE_PRECISION array of dimension (NB, ((n+NB-1)/NB)*NB),
            where NB = 128.
            On exit, contains inverses of the NB-by-NB diagonal blocks of A.

    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_dblas3
    ********************************************************************/
extern "C" void
magmablas_dtrtri_diag_q(
    magma_uplo_t uplo, magma_diag_t diag, magma_int_t n,
    magmaDouble_const_ptr dA, magma_int_t ldda,
    magmaDouble_ptr d_dinvA,
    magma_queue_t queue)
{
    magma_int_t info = 0;
    if (uplo != MagmaLower && uplo != MagmaUpper)
        info = -1;
    else if (diag != MagmaNonUnit && diag != MagmaUnit)
        info = -2;
    else if (n < 0)
        info = -3;
    else if (ldda < n)
        info = -5;

    if (info != 0) {
        magma_xerbla( __func__, -(info) );
        return;  //info
    }
    
    int nblocks = (n + IB - 1)/IB;

    hipMemset( d_dinvA, 0, ((n+NB-1)/NB)*NB*NB * sizeof(double) );
    
    if ( uplo == MagmaLower ) {
        // invert diagonal IB x IB inner blocks
        dtrtri_diag_lower_kernel<<< nblocks, IB, 0, queue >>>( diag, n, dA, ldda, d_dinvA );

        // build up NB x NB blocks (assuming IB=16 here):
        // use   16 x 16  blocks to build  32 x 32  blocks,  1 x (1 x npages) grid,  4 x 4 threads;
        // then  32 x 32  blocks to build  64 x 64  blocks,  1 x (2 x npages) grid,  8 x 4 threads;
        // then  64 x 64  blocks to build 128 x 128 blocks,  1 x (4 x npages) grid, 16 x 4 threads;
        // then 128 x 128 blocks to build 256 x 256 blocks,  2 x (8 x npages) grid, 16 x 4 threads.
        for( int jb=IB; jb < NB; jb *= 2 ) {
            int kb = jb*2;
            int npages = (n + kb - 1)/kb;
            dim3 threads( (jb <= 32 ? jb/4 : 16), 4 );
            dim3 grid( jb/(threads.x*threads.y), npages*(jb/16) );  // emulate 3D grid: NX * (NY*npages), for CUDA ARCH 1.x
            
            //printf( "n %d, jb %d, grid %d x %d (%d x %d)\n", n, jb, grid.x, grid.y, grid.y / npages, npages );
            switch (jb) {
                case 16:
                    triple_dgemm16_part1_lower_kernel<<< grid, threads, 0, queue >>>( n, dA, ldda, d_dinvA, jb, npages );
                    triple_dgemm16_part2_lower_kernel<<< grid, threads, 0, queue >>>( n, dA, ldda, d_dinvA, jb, npages );
                    break;
                case 32:
                    triple_dgemm32_part1_lower_kernel<<< grid, threads, 0, queue >>>( n, dA, ldda, d_dinvA, jb, npages );
                    triple_dgemm32_part2_lower_kernel<<< grid, threads, 0, queue >>>( n, dA, ldda, d_dinvA, jb, npages );
                    break;
                case 64:
                    triple_dgemm64_part1_lower_kernel<<< grid, threads, 0, queue >>>( n, dA, ldda, d_dinvA, jb, npages );
                    triple_dgemm64_part2_lower_kernel<<< grid, threads, 0, queue >>>( n, dA, ldda, d_dinvA, jb, npages );
                    break;
                default:
                    triple_dgemm_above64_part1_lower_kernel<<< grid, threads, 0, queue >>>( n, dA, ldda, d_dinvA, jb, npages );
                    triple_dgemm_above64_part2_lower_kernel<<< grid, threads, 0, queue >>>( n, dA, ldda, d_dinvA, jb, npages );
                    triple_dgemm_above64_part3_lower_kernel<<< grid, threads, 0, queue >>>( n, dA, ldda, d_dinvA, jb, npages );
                    break;
            }
            if ( kb >= n ) break;
        }
    }
    else {
        dtrtri_diag_upper_kernel<<< nblocks, IB, 0, queue >>>( diag, n, dA, ldda, d_dinvA );

        // update the inverse up to the size of IB
        for( int jb=IB; jb < NB; jb*=2 ) {
            int kb = jb*2;
            int npages = (n + kb - 1)/kb;
            dim3 threads( (jb <= 32 ? jb/4 : 16), 4 );
            dim3 grid( jb/(threads.x*threads.y), npages*(jb/16) );  // emulate 3D grid: NX * (NY*npages), for CUDA ARCH 1.x
            
            switch (jb) {
                case 16:
                    triple_dgemm16_part1_upper_kernel<<< grid, threads, 0, queue >>>( n, dA, ldda, d_dinvA, jb, npages );
                    triple_dgemm16_part2_upper_kernel<<< grid, threads, 0, queue >>>( n, dA, ldda, d_dinvA, jb, npages );
                    break;
                case 32:
                    triple_dgemm32_part1_upper_kernel<<< grid, threads, 0, queue >>>( n, dA, ldda, d_dinvA, jb, npages );
                    triple_dgemm32_part2_upper_kernel<<< grid, threads, 0, queue >>>( n, dA, ldda, d_dinvA, jb, npages );
                    break;
                case 64:
                    triple_dgemm64_part1_upper_kernel<<< grid, threads, 0, queue >>>( n, dA, ldda, d_dinvA, jb, npages );
                    triple_dgemm64_part2_upper_kernel<<< grid, threads, 0, queue >>>( n, dA, ldda, d_dinvA, jb, npages );
                    break;
                default:
                    triple_dgemm_above64_part1_upper_kernel<<< grid, threads, 0, queue >>>( n, dA, ldda, d_dinvA, jb, npages );
                    triple_dgemm_above64_part2_upper_kernel<<< grid, threads, 0, queue >>>( n, dA, ldda, d_dinvA, jb, npages );
                    triple_dgemm_above64_part3_upper_kernel<<< grid, threads, 0, queue >>>( n, dA, ldda, d_dinvA, jb, npages );
                    break;
            }
            if ( kb >= n ) break;
        }
    }
}

/**
    @see magmablas_dtrtri_diag_q
    @ingroup magma_dblas3
    ********************************************************************/
extern "C" void
magmablas_dtrtri_diag(
    magma_uplo_t uplo, magma_diag_t diag, magma_int_t n,
    magmaDouble_const_ptr dA, magma_int_t ldda,
    magmaDouble_ptr d_dinvA)
{
    magmablas_dtrtri_diag_q( uplo, diag, n, dA, ldda, d_dinvA, magma_stream );
}
