#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @generated from zsymmetrize.cu normal z -> c, Fri Jan 30 19:00:09 2015
       @author Mark Gates
*/
#include "common_magma.h"

#define NB 64

/*
    Matrix is m x m, and is divided into block rows, each NB x m.
    Each block has NB threads.
    Each thread copies one row, iterating across all columns below diagonal.
    The bottom block of rows may be partially outside the matrix;
    if so, rows outside the matrix (i >= m) are disabled.
*/
__global__ void
csymmetrize_lower( int m, magmaFloatComplex *dA, int ldda )
{
    // dA iterates across row i and dAT iterates down column i.
    int i = blockIdx.x*NB + threadIdx.x;
    magmaFloatComplex *dAT = dA;
    if ( i < m ) {
        dA  += i;
        dAT += i*ldda;
        magmaFloatComplex *dAend = dA + i*ldda;
        while( dA < dAend ) {
            *dAT = hipConjf(*dA);  // upper := lower
            dA  += ldda;
            dAT += 1;
        }
    }
}


// only difference with _lower version is direction dA=dAT instead of dAT=dA.
__global__ void
csymmetrize_upper( int m, magmaFloatComplex *dA, int ldda )
{
    // dA iterates across row i and dAT iterates down column i.
    int i = blockIdx.x*NB + threadIdx.x;
    magmaFloatComplex *dAT = dA;
    if ( i < m ) {
        dA  += i;
        dAT += i*ldda;
        magmaFloatComplex *dAend = dA + i*ldda;
        while( dA < dAend ) {
            *dA = hipConjf(*dAT);  // lower := upper
            dA  += ldda;
            dAT += 1;
        }
    }
}


/**
    Purpose
    -------
    
    CSYMMETRIZE copies lower triangle to upper triangle, or vice-versa,
    to make dA a general representation of a symmetric matrix.
    
    Arguments
    ---------
    
    @param[in]
    uplo    magma_uplo_t
            Specifies the part of the matrix dA that is valid on input.
      -     = MagmaUpper:      Upper triangular part
      -     = MagmaLower:      Lower triangular part
    
    @param[in]
    m       INTEGER
            The number of rows of the matrix dA.  M >= 0.
    
    @param[in,out]
    dA      COMPLEX array, dimension (LDDA,N)
            The m by m matrix dA.
    
    @param[in]
    ldda    INTEGER
            The leading dimension of the array dA.  LDDA >= max(1,M).
    
    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_caux2
    ********************************************************************/
extern "C" void
magmablas_csymmetrize_q(
    magma_uplo_t uplo, magma_int_t m,
    magmaFloatComplex_ptr dA, magma_int_t ldda,
    magma_queue_t queue )
{
    magma_int_t info = 0;
    if ( uplo != MagmaLower && uplo != MagmaUpper )
        info = -1;
    else if ( m < 0 )
        info = -2;
    else if ( ldda < max(1,m) )
        info = -4;
    
    if ( info != 0 ) {
        magma_xerbla( __func__, -(info) );
        return;
    }
    
    if ( m == 0 )
        return;
    
    
    dim3 threads( NB );
    dim3 grid( (m + NB - 1)/NB );
    
    if ( uplo == MagmaUpper ) {
        csymmetrize_upper<<< grid, threads, 0, queue >>>( m, dA, ldda );
    }
    else {
        csymmetrize_lower<<< grid, threads, 0, queue >>>( m, dA, ldda );
    }
}


/**
    @see magmablas_csymmetrize_q
    @ingroup magma_caux2
    ********************************************************************/
extern "C" void
magmablas_csymmetrize(
    magma_uplo_t uplo, magma_int_t m,
    magmaFloatComplex_ptr dA, magma_int_t ldda )
{
    magmablas_csymmetrize_q( uplo, m, dA, ldda, magma_stream );
}
