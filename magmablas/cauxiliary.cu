#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

       @generated c Sun Nov 13 20:48:36 2011

*/
#include "common_magma.h"

/* ////////////////////////////////////////////////////////////////////////////
   -- This is an auxiliary routine called from cgehrd.  The routine is called
      in 16 blocks, 32 thread per block and initializes to zero the 1st 
      32x32 block of A.
*/

__global__ void cset_to_zero(hipFloatComplex *A, int lda){
    int ind = blockIdx.x*lda + threadIdx.x;
    
    A += ind;
    A[0] = MAGMA_C_ZERO;
//   A[16*lda] = 0.;
}

__global__ void cset_nbxnb_to_zero(int nb, hipFloatComplex *A, int lda){
   int ind = blockIdx.x*lda + threadIdx.x, i, j;

   A += ind;
   for(i=0; i<nb; i+=32){
     for(j=0; j<nb; j+=32)
         A[j] = MAGMA_C_ZERO;
     A += 32*lda;
   }
}

void czero_32x32_block(hipFloatComplex *A, int lda)
{
  // cset_to_zero<<< 16, 32, 0, magma_stream >>>(A, lda);
  cset_to_zero<<< 32, 32, 0, magma_stream >>>(A, lda);
}

void czero_nbxnb_block(int nb, hipFloatComplex *A, int lda)
{
  cset_nbxnb_to_zero<<< 32, 32, 0, magma_stream >>>(nb, A, lda);
}

/* ////////////////////////////////////////////////////////////////////////////
   -- GPU kernel for initializing a matrix by 0
*/
#define claset_threads 64

__global__ void claset(int m, int n, hipFloatComplex *A, int lda){
   int ibx = blockIdx.x * claset_threads;
   int iby = blockIdx.y * 32;

   int ind = ibx + threadIdx.x;

   A += ind + __mul24(iby, lda);

   #pragma unroll
   for(int i=0; i<32; i++)
     if (iby+i < n && ind < m)
        A[i*lda] = MAGMA_C_ZERO;
}

__global__ void clasetlower(int m, int n, hipFloatComplex *A, int lda){
   int ibx = blockIdx.x * claset_threads;
   int iby = blockIdx.y * 32;

   int ind = ibx + threadIdx.x;

   A += ind + __mul24(iby, lda);

   #pragma unroll
   for(int i=0; i<32; i++)
     if (iby+i < n && ind < m && ind > i+iby)
        A[i*lda] = MAGMA_C_ZERO;
}

__global__ void clasetupper(int m, int n, hipFloatComplex *A, int lda){
   int ibx = blockIdx.x * claset_threads;
   int iby = blockIdx.y * 32;

   int ind = ibx + threadIdx.x;

   A += ind + __mul24(iby, lda);

   #pragma unroll
   for(int i=0; i<32; i++)
     if (iby+i < n && ind < m && ind < i+iby)
        A[i*lda] = MAGMA_C_ZERO;
}


/* ////////////////////////////////////////////////////////////////////////////
   -- Set the m x n matrix pointed by A to 0 on the GPU.
*/
extern "C" void
magmablas_claset(char uplo, magma_int_t m, magma_int_t n, 
                 hipFloatComplex *A, magma_int_t lda)
{
   dim3 threads(claset_threads, 1, 1);
   dim3 grid(m/claset_threads+(m % claset_threads != 0), n/32+(n%32!=0));

   if (m!=0 && n !=0)
     if (uplo == MagmaLower)
        clasetlower<<< grid, threads, 0, magma_stream >>> (m, n, A, lda);        
     else if (uplo == MagmaUpper)
        clasetupper<<< grid, threads, 0, magma_stream >>> (m, n, A, lda);
     else
        claset<<< grid, threads, 0, magma_stream >>> (m, n, A, lda);
}

/* ////////////////////////////////////////////////////////////////////////////
   -- Given two matrices, 'a' on the CPU and 'da' on the GPU, this function
      returns the Frobenious norm of the difference of the two matrices.
      The function is used for debugging.
*/
float cpu_gpu_cdiff(int M, int N, hipFloatComplex * a, int lda, hipFloatComplex *da, int ldda)
{
  int one = 1, j;
  hipFloatComplex mone = MAGMA_C_NEG_ONE;
  float  work[1];
  hipFloatComplex *ha = (hipFloatComplex*)malloc( M * N * sizeof(hipFloatComplex));
  float res;

  hipblasGetMatrix(M, N, sizeof(hipFloatComplex), da, ldda, ha, M);
  for(j=0; j<N; j++)
    blasf77_caxpy(&M, &mone, a+j*lda, &one, ha+j*M, &one);
  res = lapackf77_clange("f", &M, &N, ha, &M, work);

  free(ha);
  return res;
}

/* ////////////////////////////////////////////////////////////////////////////
 -- GPU kernel for setting 0 in the nb-1 upper subdiagonals and 1 in the diagonal
    @author Raffaele Solca
 */
__global__ void csetdiag1subdiag0_L(int k, hipFloatComplex *A, int lda){
  
  int nb = blockDim.x;
  int ibx = blockIdx.x * nb;
  
  int ind = ibx + threadIdx.x + 1;
  
  A += ind - nb + __mul24((ibx), lda);
  
  hipFloatComplex tmp = MAGMA_C_ZERO;
  if(threadIdx.x == nb-1)
    tmp = MAGMA_C_ONE;
  
#pragma unroll
  for(int i=0; i<nb; i++)
    if (ibx+i < k && ind + i  >= nb){
      A[i*(lda+1)] = tmp;
    }
  
}

/* ////////////////////////////////////////////////////////////////////////////
 -- GPU kernel for setting 0 in the nb-1 lower subdiagonals and 1 in the diagonal
    @author Raffaele Solca
 */

__global__ void csetdiag1subdiag0_U(int k, hipFloatComplex *A, int lda){

  int nb = blockDim.x;
  int ibx = blockIdx.x * nb;

  int ind = ibx + threadIdx.x;

  A += ind + __mul24((ibx), lda);

  hipFloatComplex tmp = MAGMA_C_ZERO;
  if(threadIdx.x == 0)
    tmp = MAGMA_C_ONE;

#pragma unroll
  for(int i=0; i<nb; i++)
    if (ibx+i < k && ind + i < k){
      A[i*(lda+1)] = tmp;
    }

}

/* ////////////////////////////////////////////////////////////////////////////
 -- Set 1s in the diagonal and 0s in the nb-1 lower (UPLO='U') or 
    upper (UPLO='L') subdiagonals 
    @author Raffaele Solca
 */
extern "C" void
magmablas_csetdiag1subdiag0(char uplo, magma_int_t k, magma_int_t nb,
                 hipFloatComplex *A, magma_int_t lda)
{
  
  dim3 threads(nb, 1, 1);
  dim3 grid((k-1)/nb+1);
  if(k>lda)  
    fprintf(stderr,"wrong second argument of csetdiag1subdiag0");
  if(uplo == MagmaLower)
    csetdiag1subdiag0_L<<< grid, threads, 0, magma_stream >>> (k, A, lda);
  else if(uplo == MagmaUpper){
    csetdiag1subdiag0_U<<< grid, threads, 0, magma_stream >>> (k, A, lda);
  }
  else 
    fprintf(stderr,"wrong first argument of csetdiag1subdiag0");

  return;
}
