#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.3.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2012

       @author Stan Tomov
       @generated d Wed Nov 14 22:53:53 2012
*/
#include "common_magma.h"
#define PRECISION_d
#include "commonblas.h"

//===========================================================================
//  Set a matrix from CPU to multi-GPUs is 1D block cyclic distribution.
//  The dA arrays are pointers to the matrix data for the corresponding GPUs.
//===========================================================================
extern "C" void
magmablas_dsetmatrix_1D_bcyclic( magma_int_t m, magma_int_t n,
                                 const double *hA,   magma_int_t lda,
                                 double       *dA[], magma_int_t ldda,
                                 magma_int_t num_gpus, magma_int_t nb )
{
    magma_int_t i, d, nk;
    magma_device_t cdevice;

    magma_getdevice( &cdevice );

    for( i = 0; i < n; i += nb ) {
        d = (i/nb) % num_gpus;
        magma_setdevice( d );
        nk = min(nb, n-i);
        magma_dsetmatrix_async( m, nk,
                                hA + i*lda, lda,
                                dA[d] + i/(nb*num_gpus)*nb*ldda, ldda, NULL );
    }

    magma_setdevice( cdevice );
}


//===========================================================================
//  Get a matrix with 1D block cyclic distribution on multiGPUs to the CPU.
//  The dA arrays are pointers to the matrix data for the corresponding GPUs.
//===========================================================================
extern "C" void
magmablas_dgetmatrix_1D_bcyclic( magma_int_t m, magma_int_t n,
                                 double  *dA[], magma_int_t ldda,
                                 double  *hA,   magma_int_t lda,
                                 magma_int_t num_gpus, magma_int_t nb )
{
    magma_int_t i, d, nk;
    magma_device_t cdevice;

    magma_getdevice( &cdevice );

    for( i = 0; i < n; i += nb ) {
        d = (i/nb) % num_gpus;
        magma_setdevice( d );
        nk = min(nb, n-i);
        magma_dgetmatrix_async( m, nk,
                                dA[d] + i/(nb*num_gpus)*nb*ldda, ldda,
                                hA + i*lda, lda, NULL );
    }

    magma_setdevice( cdevice );
}
