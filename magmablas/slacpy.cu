#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.5.0-beta2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2014

       @generated from zlacpy.cu normal z -> s, Fri May 30 10:40:40 2014
       @author Mark Gates
*/
#include "common_magma.h"
#include <assert.h>

#define NB 64

/* =====================================================================
    Matrix is m x n, and is divided into block rows, each NB x n.
    Each CUDA block has NB threads to handle one block row.
    Each thread copies one row, iterating across all columns.
    The bottom block of rows may be partially outside the matrix;
    if so, rows outside the matrix (i >= m) are disabled.
*/
__global__ void
slacpy_kernel(
    int m, int n,
    const float *dA, int ldda,
    float       *dB, int lddb )
{
    // dA and dB iterate across row i
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if ( i < m ) {
        dA += i;
        dB += i;
        const float *dAend = dA + n*ldda;
        while( dA < dAend ) {
            *dB = *dA;
            dA += ldda;
            dB += lddb;
        }
    }
}


/* ===================================================================== */
/**
    Note
    --------
    - UPLO Parameter is disabled
    - Do we want to provide a generic function to the user with all the options?
    
    Purpose
    -------
    SLACPY copies all or part of a two-dimensional matrix dA to another
    matrix dB.
    
    Arguments
    ---------
    
    @param[in]
    uplo    magma_uplo_t
            Specifies the part of the matrix dA to be copied to dB.
      -     = MagmaUpper:      Upper triangular part
      -     = MagmaLower:      Lower triangular part
            Otherwise:  All of the matrix dA
    
    @param[in]
    m       INTEGER
            The number of rows of the matrix dA.  M >= 0.
    
    @param[in]
    n       INTEGER
            The number of columns of the matrix dA.  N >= 0.
    
    @param[in]
    dA      COMPLEX REAL array, dimension (LDDA,N)
            The m by n matrix dA.
            If UPLO = MagmaUpper, only the upper triangle or trapezoid is accessed;
            if UPLO = MagmaLower, only the lower triangle or trapezoid is accessed.
    
    @param[in]
    ldda    INTEGER
            The leading dimension of the array dA.  LDDA >= max(1,M).
    
    @param[out]
    dB      COMPLEX REAL array, dimension (LDDB,N)
            The m by n matrix dB.
            On exit, dB = dA in the locations specified by UPLO.
    
    @param[in]
    lddb    INTEGER
            The leading dimension of the array dB.  LDDB >= max(1,M).
    

    @ingroup magma_saux2
    ********************************************************************/
extern "C" void
magmablas_slacpy(
    magma_uplo_t uplo, magma_int_t m, magma_int_t n,
    const float *dA, magma_int_t ldda,
    float       *dB, magma_int_t lddb )
{
    magma_int_t info = 0;
    if ( m < 0 )
        info = -2;
    else if ( n < 0 )
        info = -3;
    else if ( ldda < max(1,m))
        info = -5;
    else if ( lddb < max(1,m))
        info = -7;
    
    if ( info != 0 ) {
        magma_xerbla( __func__, -(info) );
        return;
    }
    
    if ( m == 0 || n == 0 )
        return;
    
    dim3 threads( NB );
    dim3 grid( (m + NB - 1)/NB );
    
    if ( uplo == MagmaUpper ) {
        fprintf(stderr, "lacpy upper is not implemented\n");
    }
    else if ( uplo == MagmaLower ) {
        fprintf(stderr, "lacpy lower is not implemented\n");
    }
    else {
        slacpy_kernel<<< grid, threads, 0, magma_stream >>>(
            m, n, dA, ldda, dB, lddb );
    }
}
