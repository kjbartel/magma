#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       May 2012

       @generated s Tue May 15 18:18:00 2012

*/
#include "common_magma.h"

/* ////////////////////////////////////////////////////////////////////////////
   -- This is an auxiliary routine called from sgehrd.  The routine is called
      in 16 blocks, 32 thread per block and initializes to zero the 1st
      32x32 block of A.
*/

__global__ void sset_to_zero(float *A, int lda){
    int ind = blockIdx.x*lda + threadIdx.x;

    A += ind;
    A[0] = MAGMA_S_ZERO;
//   A[16*lda] = 0.;
}

__global__ void sset_nbxnb_to_zero(int nb, float *A, int lda){
   int ind = blockIdx.x*lda + threadIdx.x, i, j;

   A += ind;
   for(i=0; i<nb; i+=32){
     for(j=0; j<nb; j+=32)
         A[j] = MAGMA_S_ZERO;
     A += 32*lda;
   }
}

void szero_32x32_block(float *A, int lda)
{
  // sset_to_zero<<< 16, 32, 0, magma_stream >>>(A, lda);
  sset_to_zero<<< 32, 32, 0, magma_stream >>>(A, lda);
}

void szero_nbxnb_block(int nb, float *A, int lda)
{
  sset_nbxnb_to_zero<<< 32, 32, 0, magma_stream >>>(nb, A, lda);
}

/* ////////////////////////////////////////////////////////////////////////////
   -- GPU kernel for initializing a matrix by 0
*/
#define slaset_threads 64

__global__ void slaset(int m, int n, float *A, int lda){
   int ibx = blockIdx.x * slaset_threads;
   int iby = blockIdx.y * 32;

   int ind = ibx + threadIdx.x;

   A += ind + __mul24(iby, lda);

   #pragma unroll
   for(int i=0; i<32; i++)
     if (iby+i < n && ind < m)
        A[i*lda] = MAGMA_S_ZERO;
}

__global__ void slaset_identity(int m, int n, float *A, int lda){
   int ibx = blockIdx.x * slaset_threads;
   int iby = blockIdx.y * 32;

   int ind = ibx + threadIdx.x;

   A += ind + __mul24(iby, lda);

   #pragma unroll
   for(int i=0; i<32; i++)
     if (iby+i < n && ind < m) {
        if (ind != i+iby)
           A[i*lda] = MAGMA_S_ZERO;
        else
           A[i*lda] = MAGMA_S_ONE;
     }
}

__global__ void slaset_identityonly(int m, int n, float *A, int lda){
   int ibx = blockIdx.x * slaset_threads;
   int iby = blockIdx.y * 32;

   int ind = ibx + threadIdx.x;

   A += ind + __mul24(iby, lda);

   #pragma unroll
   for(int i=0; i<32; i++)
     if (iby+i < n && ind < m) {
        if (ind == i+iby)
           A[i*lda] = MAGMA_S_ONE;
     }
}


__global__ void slasetlower(int m, int n, float *A, int lda){
   int ibx = blockIdx.x * slaset_threads;
   int iby = blockIdx.y * 32;

   int ind = ibx + threadIdx.x;

   A += ind + __mul24(iby, lda);

   #pragma unroll
   for(int i=0; i<32; i++)
     if (iby+i < n && ind < m && ind > i+iby)
        A[i*lda] = MAGMA_S_ZERO;
}

__global__ void slasetupper(int m, int n, float *A, int lda){
   int ibx = blockIdx.x * slaset_threads;
   int iby = blockIdx.y * 32;

   int ind = ibx + threadIdx.x;

   A += ind + __mul24(iby, lda);

   #pragma unroll
   for(int i=0; i<32; i++)
     if (iby+i < n && ind < m && ind < i+iby)
        A[i*lda] = MAGMA_S_ZERO;
}


/* ////////////////////////////////////////////////////////////////////////////
   -- Set the m x n matrix pointed by A to 0 on the GPU.
*/
extern "C" void
magmablas_slaset(char uplo, magma_int_t m, magma_int_t n,
                 float *A, magma_int_t lda)
{
   dim3 threads(slaset_threads, 1, 1);
   dim3 grid(m/slaset_threads+(m % slaset_threads != 0), n/32+(n%32!=0));

   if (m!=0 && n !=0)
     if (uplo == MagmaLower)
        slasetlower<<< grid, threads, 0, magma_stream >>> (m, n, A, lda);
     else if (uplo == MagmaUpper)
        slasetupper<<< grid, threads, 0, magma_stream >>> (m, n, A, lda);
     else
        slaset<<< grid, threads, 0, magma_stream >>> (m, n, A, lda);
}

/* ////////////////////////////////////////////////////////////////////////////
   -- Set the m x n matrix pointed by A to I on the GPU.
*/
extern "C" void
magmablas_slaset_identity(magma_int_t m, magma_int_t n,
                          float *A, magma_int_t lda)
{
   dim3 threads(slaset_threads, 1, 1);
   dim3 grid(m/slaset_threads+(m % slaset_threads != 0), n/32+(n%32!=0));

   if (m!=0 && n !=0)
      slaset_identity<<< grid, threads, 0, magma_stream >>> (m, n, A, lda);
}

/* ////////////////////////////////////////////////////////////////////////////
   -- Set the m x n matrix pointed by A to I on the diag without touching the offdiag GPU.
*/
extern "C" void
magmablas_slaset_identityonly(magma_int_t m, magma_int_t n,
                          float *A, magma_int_t lda)
{
   dim3 threads(slaset_threads, 1, 1);
   dim3 grid(m/slaset_threads+(m % slaset_threads != 0), n/32+(n%32!=0));

   if (m!=0 && n !=0)
      slaset_identityonly<<< grid, threads, 0, magma_stream >>> (m, n, A, lda);
}

/* ////////////////////////////////////////////////////////////////////////////
   -- Given two matrices, 'a' on the CPU and 'da' on the GPU, this function
      returns the Frobenious norm of the difference of the two matrices.
      The function is used for debugging.
*/
float cpu_gpu_sdiff(int M, int N, float * a, int lda, float *da, int ldda)
{
  int one = 1, j;
  float mone = MAGMA_S_NEG_ONE;
  float  work[1];
  float *ha = (float*)malloc( M * N * sizeof(float));
  float res;

  hipblasGetMatrix(M, N, sizeof(float), da, ldda, ha, M);
  for(j=0; j<N; j++)
    blasf77_saxpy(&M, &mone, a+j*lda, &one, ha+j*M, &one);
  res = lapackf77_slange("f", &M, &N, ha, &M, work);

  free(ha);
  return res;
}

/* ////////////////////////////////////////////////////////////////////////////
 -- GPU kernel for setting 0 in the nb-1 upper subdiagonals and 1 in the diagonal
    @author Raffaele Solca
 */
__global__ void ssetdiag1subdiag0_L(int k, float *A, int lda){

  int nb = blockDim.x;
  int ibx = blockIdx.x * nb;

  int ind = ibx + threadIdx.x + 1;

  A += ind - nb + __mul24((ibx), lda);

  float tmp = MAGMA_S_ZERO;
  if(threadIdx.x == nb-1)
    tmp = MAGMA_S_ONE;

#pragma unroll
  for(int i=0; i<nb; i++)
    if (ibx+i < k && ind + i  >= nb){
      A[i*(lda+1)] = tmp;
    }

}

/* ////////////////////////////////////////////////////////////////////////////
 -- GPU kernel for setting 0 in the nb-1 lower subdiagonals and 1 in the diagonal
    @author Raffaele Solca
 */

__global__ void ssetdiag1subdiag0_U(int k, float *A, int lda){

  int nb = blockDim.x;
  int ibx = blockIdx.x * nb;

  int ind = ibx + threadIdx.x;

  A += ind + __mul24((ibx), lda);

  float tmp = MAGMA_S_ZERO;
  if(threadIdx.x == 0)
    tmp = MAGMA_S_ONE;

#pragma unroll
  for(int i=0; i<nb; i++)
    if (ibx+i < k && ind + i < k){
      A[i*(lda+1)] = tmp;
    }

}

/* ////////////////////////////////////////////////////////////////////////////
 -- Set 1s in the diagonal and 0s in the nb-1 lower (UPLO='U') or
    upper (UPLO='L') subdiagonals.
    stream and no stream interfaces
    @author Raffaele Solca
 */
extern "C" void
magmablas_ssetdiag1subdiag0_stream(char uplo, magma_int_t k, magma_int_t nb,
                 float *A, magma_int_t lda, hipStream_t stream)
{
  dim3 threads(nb, 1, 1);
  dim3 grid((k-1)/nb+1);
  if(k>lda)
    fprintf(stderr,"wrong second argument of ssetdiag1subdiag0");
  if(uplo == MagmaLower)
    ssetdiag1subdiag0_L<<< grid, threads, 0, stream >>> (k, A, lda);
  else if(uplo == MagmaUpper){
    ssetdiag1subdiag0_U<<< grid, threads, 0, stream >>> (k, A, lda);
  }
  else
    fprintf(stderr,"wrong first argument of ssetdiag1subdiag0");

  return;
}

extern "C" void
magmablas_ssetdiag1subdiag0(char uplo, magma_int_t k, magma_int_t nb,
                 float *A, magma_int_t lda)
{
  magmablas_ssetdiag1subdiag0_stream(uplo, k, nb, A, lda, magma_stream);
}

