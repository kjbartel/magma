#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2015

       @generated from zlobpcg_maxpy.cu normal z -> c, Sun May  3 11:22:58 2015

*/

#include "common_magma.h"

// 512 is maximum number of threads for CUDA capability 1.x
#define BLOCK_SIZE  512



__global__ void
magma_clobpcg_maxpy_kernel( 
    magma_int_t num_rows, 
    magma_int_t num_vecs, 
    magmaFloatComplex * X, 
    magmaFloatComplex * Y)
{

    int row = blockIdx.x * blockDim.x + threadIdx.x; // global row index

    if( row<num_rows ){
        for( int i=0; i<num_vecs; i++ ){ 

            Y[ row + i*num_rows ] += X[ row + i*num_rows ];
        }
    }
}




/**
    Purpose
    -------
    
    This routine computes a axpy for a mxn matrix:
        
        Y = X + Y
        
    It replaces:
            magma_caxpy(m*n, c_one, Y, 1, X, 1);


        / x1[0] x2[0] x3[0] \
        | x1[1] x2[1] x3[1] |
    X = | x1[2] x2[2] x3[2] | = x1[0] x1[1] x1[2] x1[3] x1[4] x2[0] x2[1] .
        | x1[3] x2[3] x3[3] |
        \ x1[4] x2[4] x3[4] /
    
    Arguments
    ---------

    @param[in]
    num_rows    magma_int_t
                number of rows

    @param[in]
    num_vecs    magma_int_t
                number of vectors

    @param[in]
    X           magmaFloatComplex_ptr 
                input vector X

    @param[in/out]
    Y           magmaFloatComplex_ptr 
                input/output vector Y

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_cgegpuk
    ********************************************************************/

extern "C" magma_int_t
magma_clobpcg_maxpy(
    magma_int_t num_rows,
    magma_int_t num_vecs, 
    magmaFloatComplex_ptr X,
    magmaFloatComplex_ptr Y,
    magma_queue_t queue )
{
    // every thread handles one row

    magma_int_t block_size = BLOCK_SIZE;
     magma_int_t threads = BLOCK_SIZE;
    dim3 block( block_size );
    dim3 grid( magma_ceildiv( num_rows, block_size ) );

    magma_clobpcg_maxpy_kernel<<< grid, threads, 0, queue >>>
                                ( num_rows, num_vecs, X, Y );


    return MAGMA_SUCCESS;
}



