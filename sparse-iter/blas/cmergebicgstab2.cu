#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.5.0-beta2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2014

       @generated from zmergebicgstab2.cu normal z -> c, Fri May 30 10:41:37 2014
       @author Hartwig Anzt

*/
#include "common_magma.h"
#include "../include/magmasparse.h"

#define BLOCK_SIZE 512

#define PRECISION_c


// These routines merge multiple kernels from cmergebicgstab into one
// This is the code used for the ASHES2014 paper
// "Accelerating Krylov Subspace Solvers on Graphics Processing Units".
// notice that only CSR format is supported so far.


// accelerated reduction for one vector
__global__ void 
magma_creduce_kernel_spmv1(    int Gs,
                               int n, 
                               magmaFloatComplex *vtmp,
                               magmaFloatComplex *vtmp2 ){

    extern __shared__ magmaFloatComplex temp[];    
    int Idx = threadIdx.x;
    int blockSize = 128;
    int gridSize = blockSize  * 2 * gridDim.x; 
    temp[Idx] = MAGMA_C_MAKE( 0.0, 0.0);
    int i = blockIdx.x * ( blockSize * 2 ) + Idx;   
    while (i < Gs ) {
        temp[ Idx  ] += vtmp[ i ]; 
        temp[ Idx  ] += ( i + blockSize < Gs ) ? vtmp[ i + blockSize ] 
                                                : MAGMA_C_MAKE( 0.0, 0.0); 
        i += gridSize;
    }
    __syncthreads();
    if ( Idx < 64 ){
        temp[ Idx ] += temp[ Idx + 64 ];
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ){
            temp[ Idx ] += temp[ Idx + 32 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 16 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 8 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 4 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 2 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 1 ];__syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    if ( Idx == 0 ){
        vtmp2[ blockIdx.x ] = temp[ 0 ];
    }
}


__global__ void 
magma_cbicgmerge_spmv1_kernel(  
                 int n,
                 magmaFloatComplex *d_val, 
                 magma_index_t *d_rowptr, 
                 magma_index_t *d_colind,
                 magmaFloatComplex *p,
                 magmaFloatComplex *r,
                 magmaFloatComplex *v,
                 magmaFloatComplex *vtmp
                                            ){

    extern __shared__ magmaFloatComplex temp[]; 
    int Idx = threadIdx.x;   
    int i   = blockIdx.x * blockDim.x + Idx;
    int j;

    if( i<n ){
        magmaFloatComplex dot = MAGMA_C_ZERO;
        int start = d_rowptr[ i ];
        int end = d_rowptr[ i+1 ];
        for( j=start; j<end; j++)
            dot += d_val[ j ] * p[ d_colind[j] ];
        v[ i ] =  dot;
    }

    __syncthreads(); 

    temp[ Idx ] = ( i < n ) ? v[ i ] * r[ i ] : MAGMA_C_MAKE( 0.0, 0.0);
    __syncthreads();
    if ( Idx < 128 ){
        temp[ Idx ] += temp[ Idx + 128 ];
    }
    __syncthreads();
    if ( Idx < 64 ){
        temp[ Idx ] += temp[ Idx + 64 ];
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ){
            temp[ Idx ] += temp[ Idx + 32 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 16 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 8 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 4 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 2 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 1 ];__syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif

    if ( Idx == 0 ){
            vtmp[ blockIdx.x ] = temp[ 0 ];
    }
}

__global__ void 
magma_cbicgstab_alphakernel(  
                    magmaFloatComplex *skp ){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if( i==0 ){
        magmaFloatComplex tmp = skp[0];
        skp[0] = skp[4]/tmp;
    }
}

/*  -- MAGMA (version 1.5.0-beta2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2014

    Purpose
    =======

    Merges the first SpmV using CSR with the dot product 
    and the computation of alpha

    Arguments
    =========

    magma_c_sparse_matrix A             system matrix
    magmaFloatComplex *d1              temporary vector
    magmaFloatComplex *d2              temporary vector
    magmaFloatComplex *d_p             input vector p
    magmaFloatComplex *d_r             input vector r
    magmaFloatComplex *d_v             output vector v
    magmaFloatComplex *skp             array for parameters ( skp[0]=alpha )

    ========================================================================  */

extern "C" magma_int_t
magma_cbicgmerge_spmv1(  magma_c_sparse_matrix A,
                         magmaFloatComplex *d1,
                         magmaFloatComplex *d2,
                         magmaFloatComplex *d_p,
                         magmaFloatComplex *d_r,
                         magmaFloatComplex *d_v,
                         magmaFloatComplex *skp ){

    int n = A.num_rows;
    int local_block_size=256;
    dim3 Bs( local_block_size );
    dim3 Gs( (n+local_block_size-1)/local_block_size );
    dim3 Gs_next;
    int Ms =  local_block_size * sizeof( magmaFloatComplex ); 
    magmaFloatComplex *aux1 = d1, *aux2 = d2;
    int b = 1;        

    if( A.storage_type == Magma_CSR)
        magma_cbicgmerge_spmv1_kernel<<<Gs, Bs, Ms>>>
                    ( n, A.val, A.row, A.col, d_p, d_r, d_v, d1 );
    else
        printf("error: only CSR format supported.\n");

    while( Gs.x > 1 ){
        Gs_next.x = ( Gs.x+Bs.x-1 )/ Bs.x ;
        if( Gs_next.x == 1 ) Gs_next.x = 2;
        magma_creduce_kernel_spmv1<<< Gs_next.x/2, Bs.x/2, Ms/2 >>> 
                            ( Gs.x, n, aux1, aux2 );
        Gs_next.x = Gs_next.x /2;
        Gs.x = Gs_next.x;
        b = 1 - b;
        if( b ){ aux1 = d1; aux2 = d2; }
        else   { aux2 = d1; aux1 = d2; }
    }


    hipMemcpy( skp, aux1, sizeof( magmaFloatComplex ), 
                                        hipMemcpyDeviceToDevice );
    dim3 Bs2( 2 );
    dim3 Gs2( 1 );
    magma_cbicgstab_alphakernel<<<Gs2, Bs2, 0>>>( skp );

   return MAGMA_SUCCESS;
}

/* -------------------------------------------------------------------------- */

// accelerated block reduction for multiple vectors
__global__ void 
magma_creduce_kernel_spmv2( int Gs,
                           int n, 
                           magmaFloatComplex *vtmp,
                           magmaFloatComplex *vtmp2 ){

    extern __shared__ magmaFloatComplex temp[];    
    int Idx = threadIdx.x;
    int blockSize = 128;
    int gridSize = blockSize  * 2 * gridDim.x; 
    int j;

    for( j=0; j<2; j++){
        int i = blockIdx.x * ( blockSize * 2 ) + Idx;   
        temp[Idx+j*(blockSize)] = MAGMA_C_MAKE( 0.0, 0.0);
        while (i < Gs ) {
            temp[ Idx+j*(blockSize)  ] += vtmp[ i+j*n ]; 
            temp[ Idx+j*(blockSize)  ] += 
                ( i + (blockSize) < Gs ) ? vtmp[ i+j*n + (blockSize) ] 
                : MAGMA_C_MAKE( 0.0, 0.0); 
            i += gridSize;
        }
    }
    __syncthreads();
    if ( Idx < 64 ){
        for( j=0; j<2; j++){
            temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 64 ];
        }
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ){
            for( j=0; j<2; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 32 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 16 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 8 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 4 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 2 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 1 ];
                __syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            for( j=0; j<2; j++){
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 32 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 16 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 8 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 4 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 2 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 1 ];
            }
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            for( j=0; j<2; j++){
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 32 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 16 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 8 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 4 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 2 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 1 ];
            }
        }
    #endif
    if ( Idx == 0 ){
        for( j=0; j<2; j++){
            vtmp2[ blockIdx.x+j*n ] = temp[ j*(blockSize) ];
        }
    }
}

__global__ void 
magma_cbicgmerge_spmv2_kernel(  
                 int n,
                 magmaFloatComplex *d_val, 
                 magma_index_t *d_rowptr, 
                 magma_index_t *d_colind,
                 magmaFloatComplex *s,
                 magmaFloatComplex *t,
                 magmaFloatComplex *vtmp
                                            ){

    extern __shared__ magmaFloatComplex temp[]; 
    int Idx = threadIdx.x;   
    int i   = blockIdx.x * blockDim.x + Idx;
    int j;

    if( i<n ){
        magmaFloatComplex dot = MAGMA_C_ZERO;
        int start = d_rowptr[ i ];
        int end = d_rowptr[ i+1 ];
        for( j=start; j<end; j++)
            dot += d_val[ j ] * s[ d_colind[j] ];
        t[ i ] =  dot;
    }

    __syncthreads(); 

    // 2 vectors 
    if (i<n){
            magmaFloatComplex tmp2 = t[i];
            temp[Idx] = s[i] * tmp2;
            temp[Idx+blockDim.x] = tmp2 * tmp2;
    }
    else{
        for( j=0; j<2; j++)
            temp[Idx+j*blockDim.x] =MAGMA_C_MAKE( 0.0, 0.0);
    }
    __syncthreads();
    if ( Idx < 128 ){
        for( j=0; j<2; j++){
            temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 128 ];
        }
    }
    __syncthreads();
    if ( Idx < 64 ){
        for( j=0; j<2; j++){
            temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 64 ];
        }
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ){
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 32 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 16 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 8 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 4 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 2 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 1 ];
                __syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            for( j=0; j<2; j++){
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 32 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 16 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 8 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 4 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 2 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 1 ];
            }
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            for( j=0; j<2; j++){
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 32 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 16 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 8 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 4 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 2 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 1 ];
            }
        }
    #endif
    if ( Idx == 0 ){
        for( j=0; j<2; j++){
            vtmp[ blockIdx.x+j*n ] = temp[ j*blockDim.x ];
        }
    }
}

__global__ void 
magma_cbicgstab_omegakernel(  
                    magmaFloatComplex *skp ){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if( i==0 ){
        skp[2] = skp[6]/skp[7];
        skp[3] = skp[4];
    }
}

/*  -- MAGMA (version 1.5.0-beta2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2014

    Purpose
    =======

    Merges the second SpmV using CSR with the dot product 
    and the computation of omega

    Arguments
    =========

    int n                               dimension n
    int n                               dimension n
    magmaFloatComplex *d1              temporary vector
    magmaFloatComplex *d2              temporary vector
    magmaFloatComplex *d_val           matrix values
    int *d_rowptr                       matrix row pointer
    int *d_colind                       matrix column indices
    magmaFloatComplex *d_s             input vector s
    magmaFloatComplex *d_t             output vector t
    magmaFloatComplex *skp             array for parameters

    ========================================================================  */

extern "C" magma_int_t
magma_cbicgmerge_spmv2(  
                 magma_c_sparse_matrix A,
                 magmaFloatComplex *d1,
                 magmaFloatComplex *d2,
                 magmaFloatComplex *d_s,
                 magmaFloatComplex *d_t,
                 magmaFloatComplex *skp ){

    int n = A.num_rows;
    int local_block_size=256;
    dim3 Bs( local_block_size );
    dim3 Gs( (n+local_block_size-1)/local_block_size );
    dim3 Gs_next;
    int Ms =  2*local_block_size * sizeof( magmaFloatComplex ); 
    magmaFloatComplex *aux1 = d1, *aux2 = d2;
    int b = 1;        
    if( A.storage_type == Magma_CSR)
        magma_cbicgmerge_spmv2_kernel<<<Gs, Bs, Ms>>>
                    ( n, A.val, A.row, A.col, d_s, d_t, d1 );
    else
        printf("error: only CSR format supported.\n");

    while( Gs.x > 1 ){
        Gs_next.x = ( Gs.x+Bs.x-1 )/ Bs.x ;
        if( Gs_next.x == 1 ) Gs_next.x = 2;
        magma_creduce_kernel_spmv2<<< Gs_next.x/2, Bs.x/2, Ms/2 >>> 
                    ( Gs.x, n, aux1, aux2 );
        Gs_next.x = Gs_next.x /2;
        Gs.x = Gs_next.x;
        b = 1 - b;
        if( b ){ aux1 = d1; aux2 = d2; }
        else   { aux2 = d1; aux1 = d2; }
    }


    hipMemcpy( skp+6, aux1, sizeof( magmaFloatComplex ), 
                                    hipMemcpyDeviceToDevice );
    hipMemcpy( skp+7, aux1+n, sizeof( magmaFloatComplex ), 
                                    hipMemcpyDeviceToDevice );
    dim3 Bs2( 2 );
    dim3 Gs2( 1 );
    magma_cbicgstab_omegakernel<<<Gs2, Bs2, 0>>>( skp );

   return MAGMA_SUCCESS;
}

/* -------------------------------------------------------------------------- */

__global__ void 
magma_cbicgmerge_xrbeta_kernel(  
                    int n, 
                    magmaFloatComplex *rr,
                    magmaFloatComplex *r,
                    magmaFloatComplex *p,
                    magmaFloatComplex *s,
                    magmaFloatComplex *t,
                    magmaFloatComplex *x, 
                    magmaFloatComplex *skp,
                    magmaFloatComplex *vtmp
                                            ){

    extern __shared__ magmaFloatComplex temp[]; 
    int Idx = threadIdx.x;   
    int i   = blockIdx.x * blockDim.x + Idx;
    int j;

    magmaFloatComplex alpha=skp[0];
    magmaFloatComplex omega=skp[2];

    if( i<n ){
        magmaFloatComplex sl;
        sl = s[i];
        x[i] = x[i] + alpha * p[i] + omega * sl;
        r[i] = sl - omega * t[i];
    }

    __syncthreads(); 

    // 2 vectors 
    if (i<n){
            magmaFloatComplex tmp2 = r[i];
            temp[Idx] = rr[i] * tmp2;
            temp[Idx+blockDim.x] = tmp2 * tmp2;
    }
    else{
        for( j=0; j<2; j++)
            temp[Idx+j*blockDim.x] =MAGMA_C_MAKE( 0.0, 0.0);
    }
    __syncthreads();
    if ( Idx < 128 ){
        for( j=0; j<2; j++){
            temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 128 ];
        }
    }
    __syncthreads();
    if ( Idx < 64 ){
        for( j=0; j<2; j++){
            temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 64 ];
        }
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ){
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 32 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 16 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 8 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 4 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 2 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 1 ];
                __syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            for( j=0; j<2; j++){
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 32 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 16 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 8 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 4 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 2 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 1 ];
            }
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            for( j=0; j<2; j++){
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 32 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 16 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 8 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 4 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 2 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 1 ];
            }
        }
    #endif
    if ( Idx == 0 ){
        for( j=0; j<2; j++){
            vtmp[ blockIdx.x+j*n ] = temp[ j*blockDim.x ];
        }
    }
}

__global__ void 
magma_cbicgstab_betakernel(  
                    magmaFloatComplex *skp ){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if( i==0 ){
        magmaFloatComplex tmp1 = skp[4]/skp[3];
        magmaFloatComplex tmp2 = skp[0] / skp[2];
        skp[1] =  tmp1*tmp2;
    }
}

/*  -- MAGMA (version 1.5.0-beta2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2014

    Purpose
    =======

    Merges the second SpmV using CSR with the dot product 
    and the computation of omega

    Arguments
    =========

    int n                               dimension n
    int n                               dimension n
    magmaFloatComplex *d1              temporary vector
    magmaFloatComplex *d2              temporary vector
    magmaFloatComplex *d_rr            input vector rr
    magmaFloatComplex *d_r             input/output vector r
    magmaFloatComplex *d_p             input vector p
    magmaFloatComplex *d_s             input vector s
    magmaFloatComplex *d_t             input vector t
    magmaFloatComplex *d_x             output vector x
    magmaFloatComplex *skp             array for parameters

    ========================================================================  */

extern "C" magma_int_t
magma_cbicgmerge_xrbeta(  
                 int n,
                 magmaFloatComplex *d1,
                 magmaFloatComplex *d2,
                 magmaFloatComplex *rr,
                 magmaFloatComplex *r,
                 magmaFloatComplex *p,
                 magmaFloatComplex *s,
                 magmaFloatComplex *t,
                 magmaFloatComplex *x, 
                 magmaFloatComplex *skp ){

    int local_block_size=256;
    dim3 Bs( local_block_size );
    dim3 Gs( (n+local_block_size-1)/local_block_size );
    dim3 Gs_next;
    int Ms =  2*local_block_size * sizeof( magmaFloatComplex ); 
    magmaFloatComplex *aux1 = d1, *aux2 = d2;
    int b = 1;        
    magma_cbicgmerge_xrbeta_kernel<<<Gs, Bs, Ms>>>
                    ( n, rr, r, p, s, t, x, skp, d1);  

    while( Gs.x > 1 ){
        Gs_next.x = ( Gs.x+Bs.x-1 )/ Bs.x ;
        if( Gs_next.x == 1 ) Gs_next.x = 2;
        magma_creduce_kernel_spmv2<<< Gs_next.x/2, Bs.x/2, Ms/2 >>> 
                            ( Gs.x, n, aux1, aux2 );
        Gs_next.x = Gs_next.x /2;
        Gs.x = Gs_next.x;
        b = 1 - b;
        if( b ){ aux1 = d1; aux2 = d2; }
        else   { aux2 = d1; aux1 = d2; }
    }


    hipMemcpy( skp+4, aux1, sizeof( magmaFloatComplex ), 
                                        hipMemcpyDeviceToDevice );
    hipMemcpy( skp+5, aux1+n, sizeof( magmaFloatComplex ), 
                                        hipMemcpyDeviceToDevice );
    dim3 Bs2( 2 );
    dim3 Gs2( 1 );
    magma_cbicgstab_betakernel<<<Gs2, Bs2, 0>>>( skp );

   return MAGMA_SUCCESS;
}

/* -------------------------------------------------------------------------- */

