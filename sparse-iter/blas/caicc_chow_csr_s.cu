#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.5.0-beta3) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date July 2014

       @generated from zaicc_chow_csr_s.cu normal z -> c, Fri Jul 18 17:34:28 2014

*/

#include "common_magma.h"
#include "../include/magmasparse_c.h"
#include "../../include/magma.h"


// includes CUDA
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <hipsparse.h>
#include "sm_32_intrinsics.h"

// 512 is maximum number of threads for CUDA capability 1.x
#define BLOCK_SIZE  64


#define PRECISION_c



// every row is handled by one threadblock
__global__ void 
magma_caic_csr_s_kernel( magma_int_t num_rows, 
                         magma_int_t nnz,
                         const magmaFloatComplex * __restrict__ A_val, 
                         magmaFloatComplex *val,
                         magma_index_t *rowptr,
                         magma_index_t *rowidx, 
                         magma_index_t *colidx,
                         magmaFloatComplex *A2 ){

    int i, j;
    int k = (blockDim.x * blockIdx.x + threadIdx.x);// % nnz;
    magmaFloatComplex zero = MAGMA_C_MAKE(0.0, 0.0);
    magmaFloatComplex s, sp;
    int il, iu, jl, ju;

    if (k < nnz)
    {     
        i = rowidx[k];
        j = colidx[k];

#if (__CUDA_ARCH__ >= 350) && (defined(PRECISION_d) || defined(PRECISION_s))
        s = __ldg( A_val+k );
#else
        s = A_val[k];
#endif

        il = rowptr[i];
        iu = rowptr[j];

        while (il < rowptr[i+1] && iu < rowptr[j+1])
        {
            sp = zero;
            jl = colidx[il];
            ju = colidx[iu];

            // avoid branching
            sp = ( jl == ju ) ? val[il] * val[iu] : sp;
            s = ( jl == ju ) ? s-sp : s;
            il = ( jl <= ju ) ? il+1 : il;
            iu = ( jl >= ju ) ? iu+1 : iu;
/*
            if (jl < ju)
                il++;
            else if (ju < jl)
                iu++;
            else
            {
                // we are going to modify this u entry
                sp = val[il] * val[iu];
                s -= sp;
                il++;
                iu++;
            }
*/
        }
        // undo the last operation (it must be the last)
        s += sp;
        __syncthreads();

        // modify entry
        if (i == j)
            A2[k] = MAGMA_C_MAKE(sqrt(abs(MAGMA_C_REAL(s))), 0.0);
        else
            A2[k] =  s / val[rowptr[j+1]-1];
    }

}// kernel 










/**
    Purpose
    -------
    
    This routine computes the IC approximation of a matrix iteratively. 
    The idea is according to Edmond Chow's presentation at SIAM 2014.
    The input format of the matrix is Magma_CSRCOO. 

    Arguments
    ---------

    @param
    A           magma_c_sparse_matrix
                input matrix A - initial guess (lower triangular)

    @param
    A_CSR       magma_c_sparse_matrix
                input/output matrix containing the IC approximation

    @ingroup magmasparse_csygpuk
    ********************************************************************/

extern "C" magma_int_t
magma_caic_csr_s( magma_c_sparse_matrix A,
                 magma_c_sparse_matrix A_CSR ){



    
    int blocksize1 = 1;
    int blocksize2 = 1;

    int dimgrid1 = ( A.nnz + blocksize1 -1 ) / blocksize1;
    int dimgrid2 = 1;
    int dimgrid3 = 1;

    magma_c_vector A2;


            // init DEV vectors
    magma_c_vinit( &A2, Magma_DEV, A.nnz, MAGMA_C_ONE );    

    dim3 grid( dimgrid1, dimgrid2, dimgrid3 );
    dim3 block( blocksize1, blocksize2, 1 );
    magma_caic_csr_s_kernel<<< grid, block, 0, magma_stream >>>
            ( A.num_rows, A.nnz,  A.val, A_CSR.val, A_CSR.row, 
                                    A_CSR.rowidx,  A_CSR.col, A2.val );

    magma_ccopy( A.nnz, A2.val, 1, A_CSR.val, 1 );                            // rr = b

    magma_c_vfree(&A2);

    return MAGMA_SUCCESS;
}



