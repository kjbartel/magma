#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.5.0-beta2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2014

       @generated from zmergebicgstab2.cu normal z -> d, Fri May 30 10:41:37 2014
       @author Hartwig Anzt

*/
#include "common_magma.h"
#include "../include/magmasparse.h"

#define BLOCK_SIZE 512

#define PRECISION_d


// These routines merge multiple kernels from dmergebicgstab into one
// This is the code used for the ASHES2014 paper
// "Accelerating Krylov Subspace Solvers on Graphics Processing Units".
// notice that only CSR format is supported so far.


// accelerated reduction for one vector
__global__ void 
magma_dreduce_kernel_spmv1(    int Gs,
                               int n, 
                               double *vtmp,
                               double *vtmp2 ){

    extern __shared__ double temp[];    
    int Idx = threadIdx.x;
    int blockSize = 128;
    int gridSize = blockSize  * 2 * gridDim.x; 
    temp[Idx] = MAGMA_D_MAKE( 0.0, 0.0);
    int i = blockIdx.x * ( blockSize * 2 ) + Idx;   
    while (i < Gs ) {
        temp[ Idx  ] += vtmp[ i ]; 
        temp[ Idx  ] += ( i + blockSize < Gs ) ? vtmp[ i + blockSize ] 
                                                : MAGMA_D_MAKE( 0.0, 0.0); 
        i += gridSize;
    }
    __syncthreads();
    if ( Idx < 64 ){
        temp[ Idx ] += temp[ Idx + 64 ];
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ){
            temp[ Idx ] += temp[ Idx + 32 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 16 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 8 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 4 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 2 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 1 ];__syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ){
            volatile double *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    if ( Idx == 0 ){
        vtmp2[ blockIdx.x ] = temp[ 0 ];
    }
}


__global__ void 
magma_dbicgmerge_spmv1_kernel(  
                 int n,
                 double *d_val, 
                 magma_index_t *d_rowptr, 
                 magma_index_t *d_colind,
                 double *p,
                 double *r,
                 double *v,
                 double *vtmp
                                            ){

    extern __shared__ double temp[]; 
    int Idx = threadIdx.x;   
    int i   = blockIdx.x * blockDim.x + Idx;
    int j;

    if( i<n ){
        double dot = MAGMA_D_ZERO;
        int start = d_rowptr[ i ];
        int end = d_rowptr[ i+1 ];
        for( j=start; j<end; j++)
            dot += d_val[ j ] * p[ d_colind[j] ];
        v[ i ] =  dot;
    }

    __syncthreads(); 

    temp[ Idx ] = ( i < n ) ? v[ i ] * r[ i ] : MAGMA_D_MAKE( 0.0, 0.0);
    __syncthreads();
    if ( Idx < 128 ){
        temp[ Idx ] += temp[ Idx + 128 ];
    }
    __syncthreads();
    if ( Idx < 64 ){
        temp[ Idx ] += temp[ Idx + 64 ];
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ){
            temp[ Idx ] += temp[ Idx + 32 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 16 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 8 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 4 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 2 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 1 ];__syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ){
            volatile double *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif

    if ( Idx == 0 ){
            vtmp[ blockIdx.x ] = temp[ 0 ];
    }
}

__global__ void 
magma_dbicgstab_alphakernel(  
                    double *skp ){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if( i==0 ){
        double tmp = skp[0];
        skp[0] = skp[4]/tmp;
    }
}

/*  -- MAGMA (version 1.5.0-beta2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2014

    Purpose
    =======

    Merges the first SpmV using CSR with the dot product 
    and the computation of alpha

    Arguments
    =========

    magma_d_sparse_matrix A             system matrix
    double *d1              temporary vector
    double *d2              temporary vector
    double *d_p             input vector p
    double *d_r             input vector r
    double *d_v             output vector v
    double *skp             array for parameters ( skp[0]=alpha )

    ========================================================================  */

extern "C" magma_int_t
magma_dbicgmerge_spmv1(  magma_d_sparse_matrix A,
                         double *d1,
                         double *d2,
                         double *d_p,
                         double *d_r,
                         double *d_v,
                         double *skp ){

    int n = A.num_rows;
    int local_block_size=256;
    dim3 Bs( local_block_size );
    dim3 Gs( (n+local_block_size-1)/local_block_size );
    dim3 Gs_next;
    int Ms =  local_block_size * sizeof( double ); 
    double *aux1 = d1, *aux2 = d2;
    int b = 1;        

    if( A.storage_type == Magma_CSR)
        magma_dbicgmerge_spmv1_kernel<<<Gs, Bs, Ms>>>
                    ( n, A.val, A.row, A.col, d_p, d_r, d_v, d1 );
    else
        printf("error: only CSR format supported.\n");

    while( Gs.x > 1 ){
        Gs_next.x = ( Gs.x+Bs.x-1 )/ Bs.x ;
        if( Gs_next.x == 1 ) Gs_next.x = 2;
        magma_dreduce_kernel_spmv1<<< Gs_next.x/2, Bs.x/2, Ms/2 >>> 
                            ( Gs.x, n, aux1, aux2 );
        Gs_next.x = Gs_next.x /2;
        Gs.x = Gs_next.x;
        b = 1 - b;
        if( b ){ aux1 = d1; aux2 = d2; }
        else   { aux2 = d1; aux1 = d2; }
    }


    hipMemcpy( skp, aux1, sizeof( double ), 
                                        hipMemcpyDeviceToDevice );
    dim3 Bs2( 2 );
    dim3 Gs2( 1 );
    magma_dbicgstab_alphakernel<<<Gs2, Bs2, 0>>>( skp );

   return MAGMA_SUCCESS;
}

/* -------------------------------------------------------------------------- */

// accelerated block reduction for multiple vectors
__global__ void 
magma_dreduce_kernel_spmv2( int Gs,
                           int n, 
                           double *vtmp,
                           double *vtmp2 ){

    extern __shared__ double temp[];    
    int Idx = threadIdx.x;
    int blockSize = 128;
    int gridSize = blockSize  * 2 * gridDim.x; 
    int j;

    for( j=0; j<2; j++){
        int i = blockIdx.x * ( blockSize * 2 ) + Idx;   
        temp[Idx+j*(blockSize)] = MAGMA_D_MAKE( 0.0, 0.0);
        while (i < Gs ) {
            temp[ Idx+j*(blockSize)  ] += vtmp[ i+j*n ]; 
            temp[ Idx+j*(blockSize)  ] += 
                ( i + (blockSize) < Gs ) ? vtmp[ i+j*n + (blockSize) ] 
                : MAGMA_D_MAKE( 0.0, 0.0); 
            i += gridSize;
        }
    }
    __syncthreads();
    if ( Idx < 64 ){
        for( j=0; j<2; j++){
            temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 64 ];
        }
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ){
            for( j=0; j<2; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 32 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 16 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 8 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 4 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 2 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 1 ];
                __syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ){
            volatile double *temp2 = temp;
            for( j=0; j<2; j++){
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 32 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 16 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 8 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 4 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 2 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 1 ];
            }
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            for( j=0; j<2; j++){
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 32 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 16 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 8 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 4 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 2 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 1 ];
            }
        }
    #endif
    if ( Idx == 0 ){
        for( j=0; j<2; j++){
            vtmp2[ blockIdx.x+j*n ] = temp[ j*(blockSize) ];
        }
    }
}

__global__ void 
magma_dbicgmerge_spmv2_kernel(  
                 int n,
                 double *d_val, 
                 magma_index_t *d_rowptr, 
                 magma_index_t *d_colind,
                 double *s,
                 double *t,
                 double *vtmp
                                            ){

    extern __shared__ double temp[]; 
    int Idx = threadIdx.x;   
    int i   = blockIdx.x * blockDim.x + Idx;
    int j;

    if( i<n ){
        double dot = MAGMA_D_ZERO;
        int start = d_rowptr[ i ];
        int end = d_rowptr[ i+1 ];
        for( j=start; j<end; j++)
            dot += d_val[ j ] * s[ d_colind[j] ];
        t[ i ] =  dot;
    }

    __syncthreads(); 

    // 2 vectors 
    if (i<n){
            double tmp2 = t[i];
            temp[Idx] = s[i] * tmp2;
            temp[Idx+blockDim.x] = tmp2 * tmp2;
    }
    else{
        for( j=0; j<2; j++)
            temp[Idx+j*blockDim.x] =MAGMA_D_MAKE( 0.0, 0.0);
    }
    __syncthreads();
    if ( Idx < 128 ){
        for( j=0; j<2; j++){
            temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 128 ];
        }
    }
    __syncthreads();
    if ( Idx < 64 ){
        for( j=0; j<2; j++){
            temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 64 ];
        }
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ){
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 32 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 16 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 8 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 4 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 2 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 1 ];
                __syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ){
            volatile double *temp2 = temp;
            for( j=0; j<2; j++){
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 32 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 16 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 8 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 4 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 2 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 1 ];
            }
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            for( j=0; j<2; j++){
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 32 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 16 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 8 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 4 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 2 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 1 ];
            }
        }
    #endif
    if ( Idx == 0 ){
        for( j=0; j<2; j++){
            vtmp[ blockIdx.x+j*n ] = temp[ j*blockDim.x ];
        }
    }
}

__global__ void 
magma_dbicgstab_omegakernel(  
                    double *skp ){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if( i==0 ){
        skp[2] = skp[6]/skp[7];
        skp[3] = skp[4];
    }
}

/*  -- MAGMA (version 1.5.0-beta2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2014

    Purpose
    =======

    Merges the second SpmV using CSR with the dot product 
    and the computation of omega

    Arguments
    =========

    int n                               dimension n
    int n                               dimension n
    double *d1              temporary vector
    double *d2              temporary vector
    double *d_val           matrix values
    int *d_rowptr                       matrix row pointer
    int *d_colind                       matrix column indices
    double *d_s             input vector s
    double *d_t             output vector t
    double *skp             array for parameters

    ========================================================================  */

extern "C" magma_int_t
magma_dbicgmerge_spmv2(  
                 magma_d_sparse_matrix A,
                 double *d1,
                 double *d2,
                 double *d_s,
                 double *d_t,
                 double *skp ){

    int n = A.num_rows;
    int local_block_size=256;
    dim3 Bs( local_block_size );
    dim3 Gs( (n+local_block_size-1)/local_block_size );
    dim3 Gs_next;
    int Ms =  2*local_block_size * sizeof( double ); 
    double *aux1 = d1, *aux2 = d2;
    int b = 1;        
    if( A.storage_type == Magma_CSR)
        magma_dbicgmerge_spmv2_kernel<<<Gs, Bs, Ms>>>
                    ( n, A.val, A.row, A.col, d_s, d_t, d1 );
    else
        printf("error: only CSR format supported.\n");

    while( Gs.x > 1 ){
        Gs_next.x = ( Gs.x+Bs.x-1 )/ Bs.x ;
        if( Gs_next.x == 1 ) Gs_next.x = 2;
        magma_dreduce_kernel_spmv2<<< Gs_next.x/2, Bs.x/2, Ms/2 >>> 
                    ( Gs.x, n, aux1, aux2 );
        Gs_next.x = Gs_next.x /2;
        Gs.x = Gs_next.x;
        b = 1 - b;
        if( b ){ aux1 = d1; aux2 = d2; }
        else   { aux2 = d1; aux1 = d2; }
    }


    hipMemcpy( skp+6, aux1, sizeof( double ), 
                                    hipMemcpyDeviceToDevice );
    hipMemcpy( skp+7, aux1+n, sizeof( double ), 
                                    hipMemcpyDeviceToDevice );
    dim3 Bs2( 2 );
    dim3 Gs2( 1 );
    magma_dbicgstab_omegakernel<<<Gs2, Bs2, 0>>>( skp );

   return MAGMA_SUCCESS;
}

/* -------------------------------------------------------------------------- */

__global__ void 
magma_dbicgmerge_xrbeta_kernel(  
                    int n, 
                    double *rr,
                    double *r,
                    double *p,
                    double *s,
                    double *t,
                    double *x, 
                    double *skp,
                    double *vtmp
                                            ){

    extern __shared__ double temp[]; 
    int Idx = threadIdx.x;   
    int i   = blockIdx.x * blockDim.x + Idx;
    int j;

    double alpha=skp[0];
    double omega=skp[2];

    if( i<n ){
        double sl;
        sl = s[i];
        x[i] = x[i] + alpha * p[i] + omega * sl;
        r[i] = sl - omega * t[i];
    }

    __syncthreads(); 

    // 2 vectors 
    if (i<n){
            double tmp2 = r[i];
            temp[Idx] = rr[i] * tmp2;
            temp[Idx+blockDim.x] = tmp2 * tmp2;
    }
    else{
        for( j=0; j<2; j++)
            temp[Idx+j*blockDim.x] =MAGMA_D_MAKE( 0.0, 0.0);
    }
    __syncthreads();
    if ( Idx < 128 ){
        for( j=0; j<2; j++){
            temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 128 ];
        }
    }
    __syncthreads();
    if ( Idx < 64 ){
        for( j=0; j<2; j++){
            temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 64 ];
        }
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ){
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 32 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 16 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 8 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 4 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 2 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 1 ];
                __syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ){
            volatile double *temp2 = temp;
            for( j=0; j<2; j++){
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 32 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 16 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 8 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 4 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 2 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 1 ];
            }
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            for( j=0; j<2; j++){
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 32 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 16 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 8 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 4 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 2 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 1 ];
            }
        }
    #endif
    if ( Idx == 0 ){
        for( j=0; j<2; j++){
            vtmp[ blockIdx.x+j*n ] = temp[ j*blockDim.x ];
        }
    }
}

__global__ void 
magma_dbicgstab_betakernel(  
                    double *skp ){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if( i==0 ){
        double tmp1 = skp[4]/skp[3];
        double tmp2 = skp[0] / skp[2];
        skp[1] =  tmp1*tmp2;
    }
}

/*  -- MAGMA (version 1.5.0-beta2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2014

    Purpose
    =======

    Merges the second SpmV using CSR with the dot product 
    and the computation of omega

    Arguments
    =========

    int n                               dimension n
    int n                               dimension n
    double *d1              temporary vector
    double *d2              temporary vector
    double *d_rr            input vector rr
    double *d_r             input/output vector r
    double *d_p             input vector p
    double *d_s             input vector s
    double *d_t             input vector t
    double *d_x             output vector x
    double *skp             array for parameters

    ========================================================================  */

extern "C" magma_int_t
magma_dbicgmerge_xrbeta(  
                 int n,
                 double *d1,
                 double *d2,
                 double *rr,
                 double *r,
                 double *p,
                 double *s,
                 double *t,
                 double *x, 
                 double *skp ){

    int local_block_size=256;
    dim3 Bs( local_block_size );
    dim3 Gs( (n+local_block_size-1)/local_block_size );
    dim3 Gs_next;
    int Ms =  2*local_block_size * sizeof( double ); 
    double *aux1 = d1, *aux2 = d2;
    int b = 1;        
    magma_dbicgmerge_xrbeta_kernel<<<Gs, Bs, Ms>>>
                    ( n, rr, r, p, s, t, x, skp, d1);  

    while( Gs.x > 1 ){
        Gs_next.x = ( Gs.x+Bs.x-1 )/ Bs.x ;
        if( Gs_next.x == 1 ) Gs_next.x = 2;
        magma_dreduce_kernel_spmv2<<< Gs_next.x/2, Bs.x/2, Ms/2 >>> 
                            ( Gs.x, n, aux1, aux2 );
        Gs_next.x = Gs_next.x /2;
        Gs.x = Gs_next.x;
        b = 1 - b;
        if( b ){ aux1 = d1; aux2 = d2; }
        else   { aux2 = d1; aux1 = d2; }
    }


    hipMemcpy( skp+4, aux1, sizeof( double ), 
                                        hipMemcpyDeviceToDevice );
    hipMemcpy( skp+5, aux1+n, sizeof( double ), 
                                        hipMemcpyDeviceToDevice );
    dim3 Bs2( 2 );
    dim3 Gs2( 1 );
    magma_dbicgstab_betakernel<<<Gs2, Bs2, 0>>>( skp );

   return MAGMA_SUCCESS;
}

/* -------------------------------------------------------------------------- */

