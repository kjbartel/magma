#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.5.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date September 2014

       @generated from zmgeellmv.cu normal z -> c, Wed Sep 17 15:08:43 2014

*/
#include "common_magma.h"

#if (GPUSHMEM < 200)
   #define BLOCK_SIZE 128
#else
   #define BLOCK_SIZE 512
#endif



__global__ void 
cmgeellmv_kernel( int num_rows, 
                  int num_cols,
                  int num_vecs,
                  int num_cols_per_row,
                  magmaFloatComplex alpha, 
                  magmaFloatComplex *d_val, 
                  magma_index_t *d_colind,
                  magmaFloatComplex *d_x,
                  magmaFloatComplex beta, 
                  magmaFloatComplex *d_y)
{
int row = blockDim.x * blockIdx.x + threadIdx.x ;

    extern __shared__ magmaFloatComplex dot[];

    if(row < num_rows ){
        for( int i=0; i<num_vecs; i++)
                dot[ threadIdx.x + i*blockDim.x ] = MAGMA_C_MAKE(0.0, 0.0);
        for ( int n = 0; n < num_cols_per_row ; n ++){
            int col = d_colind [ num_cols_per_row * row + n ];
            magmaFloatComplex val = d_val [ num_cols_per_row * row + n ];
            if( val != 0){
                for( int i=0; i<num_vecs; i++)
                    dot[ threadIdx.x + i*blockDim.x ] += 
                                    val * d_x[col + i * num_cols ];
            }
        }
        for( int i=0; i<num_vecs; i++)
                d_y[ row + i*num_cols ] = dot[ threadIdx.x + i*blockDim.x ] 
                                * alpha + beta * d_y [ row + i * num_cols ];
    }
}





/**
    Purpose
    -------
    
    This routine computes Y = alpha *  A *  X + beta * Y for X and Y sets of 
    num_vec vectors on the GPU. Input format is ELLPACK. 
    
    Arguments
    ---------

    @param
    transA      magma_trans_t
                transposition parameter for A

    @param
    m           magma_int_t
                number of rows in A

    @param
    n           magma_int_t
                number of columns in A 
                              
    @param
    num_vecs    mama_int_t
                number of vectors
                
    @param
    nnz_per_row magma_int_t
                number of elements in the longest row 
                
    @param
    alpha       magmaFloatComplex
                scalar multiplier

    @param
    d_val       magmaFloatComplex*
                array containing values of A in ELLPACK

    @param
    d_colind    magma_int_t*
                columnindices of A in ELLPACK

    @param
    d_x         magmaFloatComplex*
                input vector x

    @param
    beta        magmaFloatComplex
                scalar multiplier

    @param
    d_y         magmaFloatComplex*
                input/output vector y


    @ingroup magmasparse_cblas
    ********************************************************************/

extern "C" magma_int_t
magma_cmgeellmv(   magma_trans_t transA,
                   magma_int_t m, magma_int_t n,
                   magma_int_t num_vecs,
                   magma_int_t nnz_per_row,
                   magmaFloatComplex alpha,
                   magmaFloatComplex *d_val,
                   magma_index_t *d_colind,
                   magmaFloatComplex *d_x,
                   magmaFloatComplex beta,
                   magmaFloatComplex *d_y ){



    dim3 grid( (m+BLOCK_SIZE-1)/BLOCK_SIZE, 1, 1);
    unsigned int MEM_SIZE =  num_vecs* BLOCK_SIZE 
                            * sizeof( magmaFloatComplex ); // num_vecs vectors 
    cmgeellmv_kernel<<< grid, BLOCK_SIZE, MEM_SIZE >>>
        ( m, n, num_vecs, nnz_per_row, alpha, d_val, d_colind, d_x, beta, d_y );


   return MAGMA_SUCCESS;
}



