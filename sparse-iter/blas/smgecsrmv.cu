#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2014

       @generated from zmgecsrmv.cu normal z -> s, Sat Nov 15 19:54:21 2014

*/
#include "common_magma.h"

#if (GPUSHMEM < 200)
   #define BLOCK_SIZE 128
#else
   #define BLOCK_SIZE 512
#endif



__global__ void 
smgecsrmv_kernel( 
    int num_rows, 
    int num_cols, 
    int num_vecs,
    float alpha, 
    magmaFloat_ptr dval, 
    magmaIndex_ptr drowptr, 
    magmaIndex_ptr dcolind,
    magmaFloat_ptr dx,
    float beta, 
    magmaFloat_ptr dy)
{

    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j;
    extern __shared__ float dot[];

    if( row<num_rows ){
        for( int i=0; i<num_vecs; i++ )
                dot[ threadIdx.x+ i*blockDim.x ] = MAGMA_S_MAKE(0.0, 0.0);
        int start = drowptr[ row ] ;
        int end = drowptr[ row+1 ];
        for( j=start; j<end; j++ ){
            int col = dcolind [ j ];
            float val = dval[ j ];
            for( int i=0; i<num_vecs; i++ )
                dot[ threadIdx.x + i*blockDim.x ] += 
                                    val * dx[ col + i*num_cols ];
        }
        for( int i=0; i<num_vecs; i++ )
            dy[ row +i*num_cols ] = alpha * dot[ threadIdx.x + i*blockDim.x ] 
                                             + beta * dy[ row + i*num_cols ];
    }
}



/**
    Purpose
    -------
    
    This routine computes Y = alpha *  A *  X + beta * Y for X and Y sets of 
    num_vec vectors on the GPU. Input format is CSR. 
    
    Arguments
    ---------
    
    @param[in]
    transA      magma_trans_t
                transposition parameter for A

    @param[in]
    m           magma_int_t
                number of rows in A

    @param[in]
    n           magma_int_t
                number of columns in A 
                
    @param[in]
    num_vecs    mama_int_t
                number of vectors
    @param[in]
    alpha       float
                scalar multiplier

    @param[in]
    dval        magmaFloat_ptr
                array containing values of A in CSR

    @param[in]
    drowptr    magma_int_t*
                rowpointer of A in CSR

    @param[in]
    dcolind     magmaIndex_ptr
                columnindices of A in CSR

    @param[in]
    dx          magmaFloat_ptr
                input vector x

    @param[in]
    beta        float
                scalar multiplier

    @param[out]
    dy          magmaFloat_ptr
                input/output vector y

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_sblas
    ********************************************************************/

extern "C" magma_int_t
magma_smgecsrmv(
    magma_trans_t transA,
    magma_int_t m, magma_int_t n,
    magma_int_t num_vecs, 
    float alpha,
    magmaFloat_ptr dval,
    magmaIndex_ptr drowptr,
    magmaIndex_ptr dcolind,
    magmaFloat_ptr dx,
    float beta,
    magmaFloat_ptr dy,
    magma_queue_t queue )
{
    dim3 grid( (m+BLOCK_SIZE-1)/BLOCK_SIZE, 1, 1);
    magma_int_t threads = BLOCK_SIZE;
    unsigned int MEM_SIZE =  num_vecs* BLOCK_SIZE 
                    * sizeof( float ); // num_vecs vectors 
    smgecsrmv_kernel<<< grid, threads, MEM_SIZE >>>
            (m, n, num_vecs, alpha, dval, drowptr, dcolind, dx, beta, dy);

   return MAGMA_SUCCESS;
}



