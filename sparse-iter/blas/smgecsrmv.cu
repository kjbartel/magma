#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.5.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date September 2014

       @generated from zmgecsrmv.cu normal z -> s, Wed Sep 17 15:08:43 2014

*/
#include "common_magma.h"

#if (GPUSHMEM < 200)
   #define BLOCK_SIZE 128
#else
   #define BLOCK_SIZE 512
#endif



__global__ void 
smgecsrmv_kernel( int num_rows, int num_cols, 
                  int num_vecs,
                  float alpha, 
                  float *d_val, 
                  magma_index_t *d_rowptr, 
                  magma_index_t *d_colind,
                  float *d_x,
                  float beta, 
                  float *d_y){

    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j;
    extern __shared__ float dot[];

    if( row<num_rows ){
        for( int i=0; i<num_vecs; i++ )
                dot[ threadIdx.x+ i*blockDim.x ] = MAGMA_S_MAKE(0.0, 0.0);
        int start = d_rowptr[ row ] ;
        int end = d_rowptr[ row+1 ];
        for( j=start; j<end; j++ ){
            int col = d_colind [ j ];
            float val = d_val[ j ];
            for( int i=0; i<num_vecs; i++ )
                dot[ threadIdx.x + i*blockDim.x ] += 
                                    val * d_x[ col + i*num_cols ];
        }
        for( int i=0; i<num_vecs; i++ )
            d_y[ row +i*num_cols ] = alpha * dot[ threadIdx.x + i*blockDim.x ] 
                                             + beta * d_y[ row + i*num_cols ];
    }
}



/**
    Purpose
    -------
    
    This routine computes Y = alpha *  A *  X + beta * Y for X and Y sets of 
    num_vec vectors on the GPU. Input format is CSR. 
    
    Arguments
    ---------
    
    @param
    transA      magma_trans_t
                transposition parameter for A

    @param
    m           magma_int_t
                number of rows in A

    @param
    n           magma_int_t
                number of columns in A 
                
    @param
    num_vecs    mama_int_t
                number of vectors
    @param
    alpha       float
                scalar multiplier

    @param
    d_val       float*
                array containing values of A in CSR

    @param
    d_rowptr    magma_int_t*
                rowpointer of A in CSR

    @param
    d_colind    magma_int_t*
                columnindices of A in CSR

    @param
    d_x         float*
                input vector x

    @param
    beta        float
                scalar multiplier

    @param
    d_y         float*
                input/output vector y


    @ingroup magmasparse_sblas
    ********************************************************************/

extern "C" magma_int_t
magma_smgecsrmv(    magma_trans_t transA,
                    magma_int_t m, magma_int_t n,
                    magma_int_t num_vecs, 
                    float alpha,
                    float *d_val,
                    magma_index_t *d_rowptr,
                    magma_index_t *d_colind,
                    float *d_x,
                    float beta,
                    float *d_y ){

    dim3 grid( (m+BLOCK_SIZE-1)/BLOCK_SIZE, 1, 1);
    unsigned int MEM_SIZE =  num_vecs* BLOCK_SIZE 
                    * sizeof( float ); // num_vecs vectors 
    smgecsrmv_kernel<<< grid, BLOCK_SIZE, MEM_SIZE >>>
            (m, n, num_vecs, alpha, d_val, d_rowptr, d_colind, d_x, beta, d_y);

   return MAGMA_SUCCESS;
}



