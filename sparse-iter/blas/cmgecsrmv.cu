#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2014

       @generated from zmgecsrmv.cu normal z -> c, Sat Nov 15 19:54:21 2014

*/
#include "common_magma.h"

#if (GPUSHMEM < 200)
   #define BLOCK_SIZE 128
#else
   #define BLOCK_SIZE 512
#endif



__global__ void 
cmgecsrmv_kernel( 
    int num_rows, 
    int num_cols, 
    int num_vecs,
    magmaFloatComplex alpha, 
    magmaFloatComplex_ptr dval, 
    magmaIndex_ptr drowptr, 
    magmaIndex_ptr dcolind,
    magmaFloatComplex_ptr dx,
    magmaFloatComplex beta, 
    magmaFloatComplex_ptr dy)
{

    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j;
    extern __shared__ magmaFloatComplex dot[];

    if( row<num_rows ){
        for( int i=0; i<num_vecs; i++ )
                dot[ threadIdx.x+ i*blockDim.x ] = MAGMA_C_MAKE(0.0, 0.0);
        int start = drowptr[ row ] ;
        int end = drowptr[ row+1 ];
        for( j=start; j<end; j++ ){
            int col = dcolind [ j ];
            magmaFloatComplex val = dval[ j ];
            for( int i=0; i<num_vecs; i++ )
                dot[ threadIdx.x + i*blockDim.x ] += 
                                    val * dx[ col + i*num_cols ];
        }
        for( int i=0; i<num_vecs; i++ )
            dy[ row +i*num_cols ] = alpha * dot[ threadIdx.x + i*blockDim.x ] 
                                             + beta * dy[ row + i*num_cols ];
    }
}



/**
    Purpose
    -------
    
    This routine computes Y = alpha *  A *  X + beta * Y for X and Y sets of 
    num_vec vectors on the GPU. Input format is CSR. 
    
    Arguments
    ---------
    
    @param[in]
    transA      magma_trans_t
                transposition parameter for A

    @param[in]
    m           magma_int_t
                number of rows in A

    @param[in]
    n           magma_int_t
                number of columns in A 
                
    @param[in]
    num_vecs    mama_int_t
                number of vectors
    @param[in]
    alpha       magmaFloatComplex
                scalar multiplier

    @param[in]
    dval        magmaFloatComplex_ptr
                array containing values of A in CSR

    @param[in]
    drowptr    magma_int_t*
                rowpointer of A in CSR

    @param[in]
    dcolind     magmaIndex_ptr
                columnindices of A in CSR

    @param[in]
    dx          magmaFloatComplex_ptr
                input vector x

    @param[in]
    beta        magmaFloatComplex
                scalar multiplier

    @param[out]
    dy          magmaFloatComplex_ptr
                input/output vector y

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_cblas
    ********************************************************************/

extern "C" magma_int_t
magma_cmgecsrmv(
    magma_trans_t transA,
    magma_int_t m, magma_int_t n,
    magma_int_t num_vecs, 
    magmaFloatComplex alpha,
    magmaFloatComplex_ptr dval,
    magmaIndex_ptr drowptr,
    magmaIndex_ptr dcolind,
    magmaFloatComplex_ptr dx,
    magmaFloatComplex beta,
    magmaFloatComplex_ptr dy,
    magma_queue_t queue )
{
    dim3 grid( (m+BLOCK_SIZE-1)/BLOCK_SIZE, 1, 1);
    magma_int_t threads = BLOCK_SIZE;
    unsigned int MEM_SIZE =  num_vecs* BLOCK_SIZE 
                    * sizeof( magmaFloatComplex ); // num_vecs vectors 
    cmgecsrmv_kernel<<< grid, threads, MEM_SIZE >>>
            (m, n, num_vecs, alpha, dval, drowptr, dcolind, dx, beta, dy);

   return MAGMA_SUCCESS;
}



