#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.5.0-beta2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2014

       @generated from zgecsrmv.cu normal z -> d, Fri May 30 10:41:36 2014

*/
#include "common_magma.h"

#if (GPUSHMEM < 200)
   #define BLOCK_SIZE 256
#else
   #define BLOCK_SIZE 256
#endif


// CSR-SpMV kernel
__global__ void 
dgecsrmv_kernel( int num_rows, int num_cols, 
                 double alpha, 
                 double *d_val, 
                 magma_index_t *d_rowptr, 
                 magma_index_t *d_colind,
                 double *d_x,
                 double beta, 
                 double *d_y){

    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j;

    if(row<num_rows){
        double dot = MAGMA_D_ZERO;
        int start = d_rowptr[ row ];
        int end = d_rowptr[ row+1 ];
        for( j=start; j<end; j++)
            dot += d_val[ j ] * d_x[ d_colind[j] ];
        d_y[ row ] =  dot *alpha + beta * d_y[ row ];
    }
}

// shifted CSR-SpMV kernel
__global__ void 
dgecsrmv_kernel_shift( int num_rows, int num_cols, 
                       double alpha, 
                       double lambda, 
                       double *d_val, 
                       magma_index_t *d_rowptr, 
                       magma_index_t *d_colind,
                       double *d_x,
                       double beta, 
                       int offset,
                       int blocksize,
                       magma_index_t *add_rows,
                       double *d_y){

    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j;

    if(row<num_rows){
        double dot = MAGMA_D_ZERO;
        int start = d_rowptr[ row ];
        int end = d_rowptr[ row+1 ];
        for( j=start; j<end; j++)
            dot += d_val[ j ] * d_x[ d_colind[j] ];
        if( row<blocksize )
            d_y[ row ] = dot * alpha - lambda 
                        * d_x[ offset+row ] + beta * d_y [ row ];
        else
            d_y[ row ] = dot * alpha - lambda 
                        * d_x[ add_rows[row-blocksize] ] + beta * d_y [ row ];   
    }
}


/*  -- MAGMA (version 1.5.0-beta2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2014

    Purpose
    =======
    
    This routine computes y = alpha *  A *  x + beta * y on the GPU.
    The input format is CSR (val, row, col).
    
    Arguments
    =========

    magma_int_t m                   number of rows in A
    magma_int_t n                   number of columns in A 
    double alpha        scalar multiplier
    double *d_val       array containing values of A in CSR
    magma_int_t *d_rowptr           rowpointer of A in CSR
    magma_int_t *d_colind           columnindices of A in CSR
    double *d_x         input vector x
    double beta         scalar multiplier
    double *d_y         input/output vector y

    ======================================================================    */

extern "C" magma_int_t
magma_dgecsrmv(     magma_trans_t transA,
                    magma_int_t m, magma_int_t n,
                    double alpha,
                    double *d_val,
                    magma_index_t *d_rowptr,
                    magma_index_t *d_colind,
                    double *d_x,
                    double beta,
                    double *d_y ){

    dim3 grid( (m+BLOCK_SIZE-1)/BLOCK_SIZE, 1, 1);

    dgecsrmv_kernel<<< grid, BLOCK_SIZE, 0, magma_stream >>>
                    (m, n, alpha, d_val, d_rowptr, d_colind, d_x, beta, d_y);

    return MAGMA_SUCCESS;
}



/*  -- MAGMA (version 1.5.0-beta2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2014

    Purpose
    =======
    
    This routine computes y = alpha * ( A -lambda I ) * x + beta * y on the GPU.
    It is a shifted version of the CSR-SpMV.
    
    Arguments
    =========

    magma_int_t m                   number of rows in A
    magma_int_t n                   number of columns in A 
    double alpha        scalar multiplier
    double alpha        scalar multiplier
    double *d_val       array containing values of A in CSR
    magma_int_t *d_rowptr           rowpointer of A in CSR
    magma_int_t *d_colind           columnindices of A in CSR
    double *d_x         input vector x
    double beta         scalar multiplier
    double *d_y         input/output vector y

    ======================================================================    */

extern "C" magma_int_t
magma_dgecsrmv_shift( magma_trans_t transA,
                      magma_int_t m, magma_int_t n,
                      double alpha,
                      double lambda,
                      double *d_val,
                      magma_index_t *d_rowptr,
                      magma_index_t *d_colind,
                      double *d_x,
                      double beta,
                      int offset,
                      int blocksize,
                      magma_index_t *add_rows,
                      double *d_y ){

    dim3 grid( (m+BLOCK_SIZE-1)/BLOCK_SIZE, 1, 1);

    dgecsrmv_kernel_shift<<< grid, BLOCK_SIZE, 0, magma_stream >>>
                         (m, n, alpha, lambda, d_val, d_rowptr, d_colind, d_x, 
                                    beta, offset, blocksize, add_rows, d_y);

    return MAGMA_SUCCESS;
}



