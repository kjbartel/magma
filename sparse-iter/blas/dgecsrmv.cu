#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @generated from zgecsrmv.cu normal z -> d, Fri Jan 30 19:00:28 2015

*/
#include "common_magma.h"

#if (GPUSHMEM < 200)
   #define BLOCK_SIZE 256
#else
   #define BLOCK_SIZE 256
#endif


// CSR-SpMV kernel
__global__ void 
dgecsrmv_kernel( 
    int num_rows, 
    int num_cols, 
    double alpha, 
    double * dval, 
    magma_index_t * drowptr, 
    magma_index_t * dcolind,
    double * dx,
    double beta, 
    double * dy)
{

    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j;

    if(row<num_rows){
        double dot = MAGMA_D_ZERO;
        int start = drowptr[ row ];
        int end = drowptr[ row+1 ];
        for( j=start; j<end; j++)
            dot += dval[ j ] * dx[ dcolind[j] ];
        dy[ row ] =  dot *alpha + beta * dy[ row ];
    }
}

// shifted CSR-SpMV kernel
__global__ void 
dgecsrmv_kernel_shift( 
    int num_rows, 
    int num_cols, 
    double alpha, 
    double lambda, 
    double * dval, 
    magma_index_t * drowptr, 
    magma_index_t * dcolind,
    double * dx,
    double beta, 
    int offset,
    int blocksize,
    magma_index_t * addrows,
    double * dy)
{

    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j;

    if(row<num_rows){
        double dot = MAGMA_D_ZERO;
        int start = drowptr[ row ];
        int end = drowptr[ row+1 ];
        for( j=start; j<end; j++)
            dot += dval[ j ] * dx[ dcolind[j] ];
        if( row<blocksize )
            dy[ row ] = dot * alpha - lambda 
                        * dx[ offset+row ] + beta * dy [ row ];
        else
            dy[ row ] = dot * alpha - lambda 
                        * dx[ addrows[row-blocksize] ] + beta * dy [ row ];   
    }
}


/**
    Purpose
    -------
    
    This routine computes y = alpha *  A *  x + beta * y on the GPU.
    The input format is CSR (val, row, col).
    
    Arguments
    ---------
    
    @param[in]
    transA      magma_trans_t
                transposition parameter for A
                
    @param[in]
    m           magma_int_t
                number of rows in A

    @param[in]
    n           magma_int_t
                number of columns in A 

    @param[in]
    alpha       double
                scalar multiplier

    @param[in]
    dval        magmaDouble_ptr
                array containing values of A in CSR

    @param[in]
    drowptr     magmaIndex_ptr
                rowpointer of A in CSR

    @param[in]
    dcolind     magmaIndex_ptr
                columnindices of A in CSR

    @param[in]
    dx          magmaDouble_ptr
                input vector x

    @param[in]
    beta        double
                scalar multiplier

    @param[out]
    dy          magmaDouble_ptr
                input/output vector y

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_dblas
    ********************************************************************/

extern "C" magma_int_t
magma_dgecsrmv(
    magma_trans_t transA,
    magma_int_t m, magma_int_t n,
    double alpha,
    magmaDouble_ptr dval,
    magmaIndex_ptr drowptr,
    magmaIndex_ptr dcolind,
    magmaDouble_ptr dx,
    double beta,
    magmaDouble_ptr dy,
    magma_queue_t queue )
{
    dim3 grid( (m+BLOCK_SIZE-1)/BLOCK_SIZE, 1, 1);
    magma_int_t threads = BLOCK_SIZE;
    dgecsrmv_kernel<<< grid, threads, 0, queue >>>
                    (m, n, alpha, dval, drowptr, dcolind, dx, beta, dy);

    return MAGMA_SUCCESS;
}



/**
    Purpose
    -------
    
    This routine computes y = alpha * ( A -lambda I ) * x + beta * y on the GPU.
    It is a shifted version of the CSR-SpMV.
    
    Arguments
    ---------
    
    @param[in]
    transA      magma_trans_t
                transposition parameter for A

    @param[in]
    m           magma_int_t
                number of rows in A

    @param[in]
    n           magma_int_t
                number of columns in A 

    @param[in]
    alpha       double
                scalar multiplier

    @param[in]
    lambda      double
                scalar multiplier

    @param[in]
    dval        magmaDouble_ptr
                array containing values of A in CSR

    @param[in]
    drowptr     magmaIndex_ptr
                rowpointer of A in CSR

    @param[in]
    dcolind     magmaIndex_ptr
                columnindices of A in CSR

    @param[in]
    dx          magmaDouble_ptr
                input vector x

    @param[in]
    beta        double
                scalar multiplier

    @param[in]
    offset      magma_int_t 
                in case not the main diagonal is scaled
                
    @param[in]
    blocksize   magma_int_t 
                in case of processing multiple vectors  
                
    @param[in]
    addrows     magmaIndex_ptr
                in case the matrixpowerskernel is used
                
    @param[out]
    dy          magmaDouble_ptr
                output vector y  
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_dblas
    ********************************************************************/

extern "C" magma_int_t
magma_dgecsrmv_shift(
    magma_trans_t transA,
    magma_int_t m, magma_int_t n,
    double alpha,
    double lambda,
    magmaDouble_ptr dval,
    magmaIndex_ptr drowptr,
    magmaIndex_ptr dcolind,
    magmaDouble_ptr dx,
    double beta,
    int offset,
    int blocksize,
    magma_index_t * addrows,
    magmaDouble_ptr dy,
    magma_queue_t queue )
{
    dim3 grid( (m+BLOCK_SIZE-1)/BLOCK_SIZE, 1, 1);
    magma_int_t threads = BLOCK_SIZE;
    dgecsrmv_kernel_shift<<< grid, threads, 0, queue >>>
                         (m, n, alpha, lambda, dval, drowptr, dcolind, dx, 
                                    beta, offset, blocksize, addrows, dy);

    return MAGMA_SUCCESS;
}



