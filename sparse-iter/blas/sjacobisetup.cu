#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2015

       @generated from zjacobisetup.cu normal z -> s, Sun May  3 11:22:58 2015
       @author Hartwig Anzt

*/
#include "common_magmasparse.h"

#define BLOCK_SIZE 128


#define PRECISION_s

__global__ void 
svjacobisetup_gpu(  int num_rows, 
                    int num_vecs,
                    float *b, 
                    float *d, 
                    float *c,
                    float *x){

    int row = blockDim.x * blockIdx.x + threadIdx.x ;

    if(row < num_rows ){
        for( int i=0; i<num_vecs; i++ ){
            c[row+i*num_rows] = b[row+i*num_rows] / d[row];
            x[row+i*num_rows] = c[row+i*num_rows];
        }
    }
}





/**
    Purpose
    -------

    Prepares the Jacobi Iteration according to
       x^(k+1) = D^(-1) * b - D^(-1) * (L+U) * x^k
       x^(k+1) =      c     -       M        * x^k.

    Returns the vector c. It calls a GPU kernel

    Arguments
    ---------

    @param[in]
    num_rows    magma_int_t
                number of rows
                
    @param[in]
    b           magma_s_matrix
                RHS b

    @param[in]
    d           magma_s_matrix
                vector with diagonal entries

    @param[out]
    c           magma_s_matrix*
                c = D^(-1) * b

    @param[out]
    x           magma_s_matrix*
                iteration vector
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_sgegpuk
    ********************************************************************/

extern "C" magma_int_t
magma_sjacobisetup_vector_gpu(
    int num_rows, 
    magma_s_matrix b, 
    magma_s_matrix d, 
    magma_s_matrix c,
    magma_s_matrix *x,
    magma_queue_t queue )
{
    dim3 grid( magma_ceildiv( num_rows, BLOCK_SIZE ) );
    int num_vecs = b.num_rows / num_rows;
    magma_int_t threads = BLOCK_SIZE;
    svjacobisetup_gpu<<< grid, threads, 0 >>>
                ( num_rows, num_vecs, b.dval, d.dval, c.dval, x->val );

    return MAGMA_SUCCESS;
}






__global__ void 
sjacobidiagscal_kernel(  int num_rows,
                         int num_vecs, 
                    float *b, 
                    float *d, 
                    float *c){

    int row = blockDim.x * blockIdx.x + threadIdx.x ;

    if(row < num_rows ){
        for( int i=0; i<num_vecs; i++)
            c[row+i*num_rows] = b[row+i*num_rows] * d[row];
    }
}





/**
    Purpose
    -------

    Prepares the Jacobi Iteration according to
       x^(k+1) = D^(-1) * b - D^(-1) * (L+U) * x^k
       x^(k+1) =      c     -       M        * x^k.

    Returns the vector c. It calls a GPU kernel

    Arguments
    ---------

    @param[in]
    num_rows    magma_int_t
                number of rows
                
    @param[in]
    b           magma_s_matrix
                RHS b

    @param[in]
    d           magma_s_matrix
                vector with diagonal entries

    @param[out]
    c           magma_s_matrix*
                c = D^(-1) * b
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_s
    ********************************************************************/

extern "C" magma_int_t
magma_sjacobi_diagscal(
    int num_rows, 
    magma_s_matrix d, 
    magma_s_matrix b, 
    magma_s_matrix *c,
    magma_queue_t queue )
{
    dim3 grid( magma_ceildiv( num_rows, BLOCK_SIZE ));
    int num_vecs = b.num_rows*b.num_cols/num_rows;
    magma_int_t threads = BLOCK_SIZE;
    sjacobidiagscal_kernel<<< grid, threads, 0 >>>( num_rows, num_vecs, b.dval, d.dval, c->val );

    return MAGMA_SUCCESS;
}













__global__ void 
sjacobiupdate_kernel(  int num_rows,
                       int num_cols, 
                    float *t, 
                    float *b, 
                    float *d, 
                    float *x){

    int row = blockDim.x * blockIdx.x + threadIdx.x ;

    if(row < num_rows ){
        for( int i=0; i<num_cols; i++)
            x[row+i*num_rows] += (b[row+i*num_rows]-t[row+i*num_rows]) * d[row];
    }
}


/**
    Purpose
    -------

    Updates the iteration vector x for the Jacobi iteration
    according to
        x=x+d.*(b-t)
    where d is the diagonal of the system matrix A and t=Ax.

    Arguments
    ---------

    @param[in]
    num_rows    magma_int_t
                number of rows
                
    @param[in]
    num_cols    magma_int_t
                number of cols
                
    @param[in]
    t           magma_s_matrix
                t = A*x
                
    @param[in]
    b           magma_s_matrix
                RHS b
                
    @param[in]
    d           magma_s_matrix
                vector with diagonal entries

    @param[out]
    x           magma_s_matrix*
                iteration vector
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_s
    ********************************************************************/

extern "C" magma_int_t
magma_sjacobiupdate(
    magma_s_matrix t, 
    magma_s_matrix b, 
    magma_s_matrix d, 
    magma_s_matrix *x,
    magma_queue_t queue )
{

    dim3 grid( magma_ceildiv( t.num_rows, BLOCK_SIZE ));
    magma_int_t threads = BLOCK_SIZE;
    sjacobiupdate_kernel<<< grid, threads, 0 >>>( t.num_rows, t.num_cols, t.dval, b.dval, d.dval, x->dval );

    return MAGMA_SUCCESS;
}










__global__ void 
sjacobispmvupdate_kernel(  
    int num_rows,
    int num_cols, 
    float * dval, 
    magma_index_t * drowptr, 
    magma_index_t * dcolind,
    float *t, 
    float *b, 
    float *d, 
    float *x ){



    int row = blockDim.x * blockIdx.x + threadIdx.x ;
    int j;

    if(row<num_rows){
        float dot = MAGMA_S_ZERO;
        int start = drowptr[ row ];
        int end = drowptr[ row+1 ];
        for( int i=0; i<num_cols; i++){
            for( j=start; j<end; j++){
                dot += dval[ j ] * x[ dcolind[j]+i*num_rows ];
            }
            x[row+i*num_rows] += (b[row+i*num_rows]-dot) * d[row];
        }
    }
}





/**
    Purpose
    -------

    Updates the iteration vector x for the Jacobi iteration
    according to
        x=x+d.*(b-Ax)


    Arguments
    ---------

    @param[in]
    maxiter     magma_int_t
                number of Jacobi iterations   
                
    @param[in]
    A           magma_s_matrix
                system matrix
                
    @param[in]
    t           magma_s_matrix
                workspace
                
    @param[in]
    b           magma_s_matrix
                RHS b
                
    @param[in]
    d           magma_s_matrix
                vector with diagonal entries

    @param[out]
    x           magma_s_matrix*
                iteration vector
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_s
    ********************************************************************/

extern "C" magma_int_t
magma_sjacobispmvupdate(
    magma_int_t maxiter,
    magma_s_matrix A,
    magma_s_matrix t, 
    magma_s_matrix b, 
    magma_s_matrix d, 
    magma_s_matrix *x,
    magma_queue_t queue )
{

    // local variables
    float c_zero = MAGMA_S_ZERO, c_one = MAGMA_S_ONE;
    dim3 grid( magma_ceildiv( t.num_rows, BLOCK_SIZE ));
    magma_int_t threads = BLOCK_SIZE;

    for( magma_int_t i=0; i<maxiter; i++ ) {
        // distinct routines imply synchronization
        // magma_s_spmv( c_one, A, *x, c_zero, t, queue );                // t =  A * x
        // sjacobiupdate_kernel<<< grid, threads, 0 >>>( t.num_rows, t.num_cols, t.dval, b.dval, d.dval, x->dval );
        // merged in one implies asynchronous update
        sjacobispmvupdate_kernel<<< grid, threads, 0 >>>
            ( t.num_rows, t.num_cols, A.dval, A.drow, A.dcol, t.dval, b.dval, d.dval, x->dval );

    }

    return MAGMA_SUCCESS;
}










