#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @generated from zgeelltmv.cu normal z -> d, Fri Jan 30 19:00:28 2015

*/

#include "common_magma.h"

#if (GPUSHMEM < 200)
   #define BLOCK_SIZE 128
#else
   #define BLOCK_SIZE 512
#endif


// ELL SpMV kernel
//Michael Garland
__global__ void 
dgeelltmv_kernel( 
    int num_rows, 
    int num_cols,
    int num_cols_per_row,
    double alpha, 
    double * dval, 
    magma_index_t * dcolind,
    double * dx,
    double beta, 
    double * dy)
{
    int row = blockDim.x * blockIdx.x + threadIdx.x ;
    if(row < num_rows ){
        double dot = MAGMA_D_MAKE(0.0, 0.0);
        for ( int n = 0; n < num_cols_per_row ; n ++){
            int col = dcolind [ num_rows * n + row ];
            double val = dval [ num_rows * n + row ];
            if( val != 0)
                dot += val * dx[col ];
        }
        dy[ row ] = dot * alpha + beta * dy [ row ];
    }
}

// shifted ELL SpMV kernel
//Michael Garland
__global__ void 
dgeelltmv_kernel_shift( 
    int num_rows, 
    int num_cols,
    int num_cols_per_row,
    double alpha, 
    double lambda, 
    double * dval, 
    magma_index_t * dcolind,
    double * dx,
    double beta, 
    int offset,
    int blocksize,
    magma_index_t * addrows,
    double * dy)
{

    int row = blockDim.x * blockIdx.x + threadIdx.x ;
    if(row < num_rows ){
        double dot = MAGMA_D_MAKE(0.0, 0.0);
        for ( int n = 0; n < num_cols_per_row ; n ++){
            int col = dcolind [ num_rows * n + row ];
            double val = dval [ num_rows * n + row ];
            if( val != 0)
                dot += val * dx[col ];
        }
        if( row<blocksize )
            dy[ row ] = dot * alpha - lambda 
                    * dx[ offset+row ] + beta * dy [ row ];
        else
            dy[ row ] = dot * alpha - lambda 
                    * dx[ addrows[row-blocksize] ] + beta * dy [ row ];            
    }
}




/**
    Purpose
    -------
    
    This routine computes y = alpha *  A^t *  x + beta * y on the GPU.
    Input format is ELL.
    
    Arguments
    ---------
    
    @param[in]
    transA      magma_trans_t
                transposition parameter for A
                
    @param[in]
    m           magma_int_t
                number of rows in A

    @param[in]
    n           magma_int_t
                number of columns in A 
                
    @param[in]
    nnz_per_row magma_int_t
                number of elements in the longest row 

    @param[in]
    alpha       double
                scalar multiplier

    @param[in]
    dval        magmaDouble_ptr
                array containing values of A in ELL

    @param[in]
    dcolind     magmaIndex_ptr
                columnindices of A in ELL

    @param[in]
    dx          magmaDouble_ptr
                input vector x

    @param[in]
    beta        double
                scalar multiplier

    @param[out]
    dy          magmaDouble_ptr
                input/output vector y

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_d
    ********************************************************************/

extern "C" magma_int_t
magma_dgeelltmv(
    magma_trans_t transA,
    magma_int_t m, magma_int_t n,
    magma_int_t nnz_per_row,
    double alpha,
    magmaDouble_ptr dval,
    magmaIndex_ptr dcolind,
    magmaDouble_ptr dx,
    double beta,
    magmaDouble_ptr dy,
    magma_queue_t queue )
{
    dim3 grid( (m+BLOCK_SIZE-1)/BLOCK_SIZE, 1, 1);
    magma_int_t threads = BLOCK_SIZE;
    dgeelltmv_kernel<<< grid, threads, 0, queue >>>
                  ( m, n, nnz_per_row, alpha, dval, dcolind, dx, beta, dy );


   return MAGMA_SUCCESS;
}


/**
    Purpose
    -------
    
    This routine computes y = alpha *( A - lambda I ) * x + beta * y on the GPU.
    Input format is ELL.
    
    Arguments
    ---------

    @param[in]
    transA      magma_trans_t
                transposition parameter for A    

    @param[in]
    m           magma_int_t
                number of rows in A

    @param[in]
    n           magma_int_t
                number of columns in A 
                
    @param[in]
    nnz_per_row magma_int_t
                number of elements in the longest row 

    @param[in]
    alpha       double
                scalar multiplier

    @param[in]
    lambda      double
                scalar multiplier

    @param[in]
    dval        magmaDouble_ptr
                array containing values of A in ELL

    @param[in]
    dcolind     magmaIndex_ptr
                columnindices of A in ELL

    @param[in]
    dx          magmaDouble_ptr
                input vector x

    @param[in]
    beta        double
                scalar multiplier
                
    @param[in]
    offset      magma_int_t 
                in case not the main diagonal is scaled
                
    @param[in]
    blocksize   magma_int_t 
                in case of processing multiple vectors  
                
    @param[in]
    addrows     magmaIndex_ptr
                in case the matrixpowerskernel is used

    @param[out]
    dy          magmaDouble_ptr
                input/output vector y

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_dblas
    ********************************************************************/

extern "C" magma_int_t
magma_dgeelltmv_shift(
    magma_trans_t transA,
    magma_int_t m, magma_int_t n,
    magma_int_t nnz_per_row,
    double alpha,
    double lambda,
    magmaDouble_ptr dval,
    magmaIndex_ptr dcolind,
    magmaDouble_ptr dx,
    double beta,
    int offset,
    int blocksize,
    magmaIndex_ptr addrows,
    magmaDouble_ptr dy,
    magma_queue_t queue )
{
    dim3 grid( (m+BLOCK_SIZE-1)/BLOCK_SIZE, 1, 1);
    magma_int_t threads = BLOCK_SIZE;
    double tmp_shift;
    //magma_dsetvector(1,&lambda,1,&tmp_shift,1); 
    tmp_shift = lambda;
    dgeelltmv_kernel_shift<<< grid, threads, 0, queue >>>
                  ( m, n, nnz_per_row, alpha, tmp_shift, dval, dcolind, dx, 
                            beta, offset, blocksize, addrows, dy );


   return MAGMA_SUCCESS;
}



