#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.5.0-beta3) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date July 2014

       @generated from zmgeelltmv.cu normal z -> d, Fri Jul 18 17:34:28 2014

*/

#include "common_magma.h"

#if (GPUSHMEM < 200)
   #define BLOCK_SIZE 128
#else
   #define BLOCK_SIZE 512
#endif



__global__ void 
dmgeelltmv_kernel( int num_rows, 
                 int num_cols,
                 int num_vecs,
                 int num_cols_per_row,
                 double alpha, 
                 double *d_val, 
                 magma_index_t *d_colind,
                 double *d_x,
                 double beta, 
                 double *d_y)
{
    extern __shared__ double dot[];
    int row = blockDim.x * blockIdx.x + threadIdx.x ;
    if(row < num_rows ){
        for( int i=0; i<num_vecs; i++ )
                dot[ threadIdx.x+ i*blockDim.x ] = MAGMA_D_MAKE(0.0, 0.0);
        for ( int n = 0; n < num_cols_per_row ; n ++){
            int col = d_colind [ num_rows * n + row ];
            double val = d_val [ num_rows * n + row ];
            if( val != 0){
                for( int i=0; i<num_vecs; i++ )
                    dot[ threadIdx.x + i*blockDim.x ] += 
                                        val * d_x[col + i * num_cols ];
            }
        }
        for( int i=0; i<num_vecs; i++ )
                d_y[ row + i*num_cols ] = dot[ threadIdx.x + i*blockDim.x ] 
                                * alpha + beta * d_y [ row + i*num_cols ];
    }
}





/**
    Purpose
    -------
    
    This routine computes Y = alpha *  A *  X + beta * Y for X and Y sets of 
    num_vec vectors on the GPU. Input format is ELL. 
    
    Arguments
    ---------

    @param
    transA      magma_trans_t
                transposition parameter for A

    @param
    m           magma_int_t
                number of rows in A

    @param
    n           magma_int_t
                number of columns in A 
                
    @param
    num_vecs    mama_int_t
                number of vectors
                
    @param
    nnz_per_row magma_int_t
                number of elements in the longest row 
                
    @param
    alpha       double
                scalar multiplier

    @param
    d_val       double*
                array containing values of A in ELL

    @param
    d_colind    magma_int_t*
                columnindices of A in ELL

    @param
    d_x         double*
                input vector x

    @param
    beta        double
                scalar multiplier

    @param
    d_y         double*
                input/output vector y


    @ingroup magmasparse_dblas
    ********************************************************************/

extern "C" magma_int_t
magma_dmgeelltmv(  magma_trans_t transA,
                   magma_int_t m, magma_int_t n,
                   magma_int_t num_vecs,
                   magma_int_t nnz_per_row,
                   double alpha,
                   double *d_val,
                   magma_index_t *d_colind,
                   double *d_x,
                   double beta,
                   double *d_y ){



    dim3 grid( (m+BLOCK_SIZE-1)/BLOCK_SIZE, 1, 1);
    unsigned int MEM_SIZE =  num_vecs* BLOCK_SIZE 
                * sizeof( double ); // num_vecs vectors 
    dmgeelltmv_kernel<<< grid, BLOCK_SIZE, MEM_SIZE >>>
        ( m, n, num_vecs, nnz_per_row, alpha, d_val, d_colind, d_x, beta, d_y );


    return MAGMA_SUCCESS;
}



