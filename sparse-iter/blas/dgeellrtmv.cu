#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @generated from zgeellrtmv.cu normal z -> d, Fri Jan 30 19:00:29 2015

*/

#include "common_magma.h"

//F. Vázquez, G. Ortega, J.J. Fernández, E.M. Garzón, Almeria University
__global__ void 
dgeellrtmv_kernel_32( 
    int num_rows, 
    int num_cols,
    double alpha, 
    double * dval, 
    magma_index_t * dcolind,
    magma_index_t * drowlength,
    double * dx,
    double beta, 
    double * dy,
    int T,
    int alignment )
{
int idx = blockIdx.y * gridDim.x * blockDim.x + 
          blockDim.x * blockIdx.x + threadIdx.x ; // global thread index
int idb = threadIdx.x ;  // local thread index
int idp = idb%T;  // number of threads assigned to one row
int i = idx/T;  // row index

extern __shared__ double shared[];

    if(i < num_rows ){
        double dot = MAGMA_D_MAKE(0.0, 0.0);
        int max_ = (drowlength[i]+T-1)/T;  
            // number of elements each thread handles

        for ( int k = 0; k < max_ ; k++ ){

            // original code in paper (not working for me)
            //double val = dval[ k*(T*alignment)+(i*T)+idp ];  
            //int col = dcolind [ k*(T*alignment)+(i*T)+idp ];    

            // new code (working for me)        
            double val = dval[ k*(T)+(i*alignment)+idp ];
            int col = dcolind [ k*(T)+(i*alignment)+idp ];

            dot += val * dx[ col ];
        }
        shared[idb]  = dot;
        if( idp < 16 ){
            shared[idb]+=shared[idb+16];
            if( idp < 8 ) shared[idb]+=shared[idb+8];
            if( idp < 4 ) shared[idb]+=shared[idb+4];
            if( idp < 2 ) shared[idb]+=shared[idb+2];
            if( idp == 0 ) {
                dy[i] = (shared[idb]+shared[idb+1])*alpha + beta*dy [i];
            }

        }
    }

}

//F. Vázquez, G. Ortega, J.J. Fernández, E.M. Garzón, Almeria University
__global__ void 
dgeellrtmv_kernel_16( 
    int num_rows, 
    int num_cols,
    double alpha, 
    double * dval, 
    magma_index_t * dcolind,
    magma_index_t * drowlength,
    double * dx,
    double beta, 
    double * dy,
    int T,
    int alignment )
{
int idx = blockIdx.y * gridDim.x * blockDim.x + 
          blockDim.x * blockIdx.x + threadIdx.x ; // global thread index
int idb = threadIdx.x ;  // local thread index
int idp = idb%T;  // number of threads assigned to one row
int i = idx/T;  // row index

extern __shared__ double shared[];

    if(i < num_rows ){
        double dot = MAGMA_D_MAKE(0.0, 0.0);
        int max_ = (drowlength[i]+T-1)/T;  
            // number of elements each thread handles

        for ( int k = 0; k < max_ ; k++ ){

            // original code in paper (not working for me)
            //double val = dval[ k*(T*alignment)+(i*T)+idp ];  
            //int col = dcolind [ k*(T*alignment)+(i*T)+idp ];    

            // new code (working for me)        
            double val = dval[ k*(T)+(i*alignment)+idp ];
            int col = dcolind [ k*(T)+(i*alignment)+idp ];

            dot += val * dx[ col ];
        }
        shared[idb]  = dot;
        if( idp < 8 ){
            shared[idb]+=shared[idb+8];
            if( idp < 4 ) shared[idb]+=shared[idb+4];
            if( idp < 2 ) shared[idb]+=shared[idb+2];
            if( idp == 0 ) {
                dy[i] = (shared[idb]+shared[idb+1])*alpha + beta*dy [i];
            }

        }
    }

}

//F. Vázquez, G. Ortega, J.J. Fernández, E.M. Garzón, Almeria University
__global__ void 
dgeellrtmv_kernel_8( 
    int num_rows, 
    int num_cols,
    double alpha, 
    double * dval, 
    magma_index_t * dcolind,
    magma_index_t * drowlength,
    double * dx,
    double beta, 
    double * dy,
    int T,
    int alignment )
{
int idx = blockIdx.y * gridDim.x * blockDim.x + 
          blockDim.x * blockIdx.x + threadIdx.x ; // global thread index
int idb = threadIdx.x ;  // local thread index
int idp = idb%T;  // number of threads assigned to one row
int i = idx/T;  // row index

extern __shared__ double shared[];

    if(i < num_rows ){
        double dot = MAGMA_D_MAKE(0.0, 0.0);
        int max_ = (drowlength[i]+T-1)/T;  
            // number of elements each thread handles

        for ( int k = 0; k < max_ ; k++ ){

            // original code in paper (not working for me)
            //double val = dval[ k*(T*alignment)+(i*T)+idp ];  
            //int col = dcolind [ k*(T*alignment)+(i*T)+idp ];    

            // new code (working for me)        
            double val = dval[ k*(T)+(i*alignment)+idp ];
            int col = dcolind [ k*(T)+(i*alignment)+idp ];

            dot += val * dx[ col ];
        }
        shared[idb]  = dot;
        if( idp < 4 ){
            shared[idb]+=shared[idb+4];
            if( idp < 2 ) shared[idb]+=shared[idb+2];
            if( idp == 0 ) {
                dy[i] = (shared[idb]+shared[idb+1])*alpha + beta*dy [i];
            }

        }
    }

}



/**
    Purpose
    -------
    
    This routine computes y = alpha *  A *  x + beta * y on the GPU.
    Input format is ELLRT. The ideas are taken from 
    "Improving the performance of the sparse matrix
    vector product with GPUs", (CIT 2010), 
    and modified to provide correct values.

    
    Arguments
    ---------

    @param[in]
    transA      magma_trans_t
                transposition parameter for A
    @param[in]
    m           magma_int_t
                number of rows 

    @param[in]
    n           magma_int_t
                number of columns

    @param[in]
    nnz_per_row magma_int_t
                max number of nonzeros in a row

    @param[in]
    alpha       double
                scalar alpha

    @param[in]
    dval        magmaDouble_ptr
                val array

    @param[in]
    dcolind     magmaIndex_ptr
                col indices  

    @param[in]
    drowlength  magmaIndex_ptr
                number of elements in each row

    @param[in]
    dx          magmaDouble_ptr
                input vector x

    @param[in]
    beta        double
                scalar beta

    @param[out]
    dy          magmaDouble_ptr
                output vector y

    @param[in]
    blocksize   magma_int_t
                threads per block

    @param[in]
    alignment   magma_int_t
                threads assigned to each row

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_dblas
    ********************************************************************/

extern "C" magma_int_t
magma_dgeellrtmv(
    magma_trans_t transA,
    magma_int_t m, magma_int_t n,
    magma_int_t nnz_per_row,
    double alpha,
    magmaDouble_ptr dval,
    magmaIndex_ptr dcolind,
    magmaIndex_ptr drowlength,
    magmaDouble_ptr dx,
    double beta,
    magmaDouble_ptr dy,
    magma_int_t alignment,
    magma_int_t blocksize,
    magma_queue_t queue )
{
    int num_blocks = ( (m+blocksize-1)/blocksize);

    magma_int_t num_threads = alignment*blocksize;
    magma_int_t threads = alignment*blocksize;

    int real_row_length = ((int)(nnz_per_row+alignment-1)/alignment)
                            *alignment;

    magma_int_t arch = magma_getdevice_arch();
    if ( arch < 200 && num_threads > 256 )
        printf("error: too much shared memory requested.\n");

    int dimgrid1 = (int) sqrt( (double) num_blocks );
    int dimgrid2 = (num_blocks + dimgrid1 -1 ) / dimgrid1;
    dim3 grid( dimgrid1, dimgrid2, 1);

    int Ms = alignment * blocksize * sizeof( double );
    // printf("launch kernel: %dx%d %d %d\n", grid.x, grid.y, num_threads , Ms);

    if ( alignment == 32 ) {
        dgeellrtmv_kernel_32<<< grid, threads , Ms, queue >>>
                 ( m, n, alpha, dval, dcolind, drowlength, dx, beta, dy, 
                                                 alignment, real_row_length );
    }
    else if ( alignment == 16 ) {
        dgeellrtmv_kernel_16<<< grid, threads , Ms, queue >>>
                 ( m, n, alpha, dval, dcolind, drowlength, dx, beta, dy, 
                                                 alignment, real_row_length );
    }
    else if ( alignment == 8 ) {
        dgeellrtmv_kernel_8<<< grid, threads , Ms, queue >>>
                 ( m, n, alpha, dval, dcolind, drowlength, dx, beta, dy, 
                                                 alignment, real_row_length );
    }
    else {
        printf("error: alignment %d not supported.\n", alignment);
        return MAGMA_ERR_NOT_SUPPORTED;
    }



   return MAGMA_SUCCESS;
}


