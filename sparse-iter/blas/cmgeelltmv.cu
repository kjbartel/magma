#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @generated from zmgeelltmv.cu normal z -> c, Fri Jan 30 19:00:29 2015

*/

#include "common_magma.h"

#if (GPUSHMEM < 200)
   #define BLOCK_SIZE 128
#else
   #define BLOCK_SIZE 512
#endif



__global__ void 
cmgeelltmv_kernel( 
        int num_rows, 
        int num_cols,
        int num_vecs,
        int num_cols_per_row,
        magmaFloatComplex alpha, 
        magmaFloatComplex * dval, 
        magma_index_t * dcolind,
        magmaFloatComplex * dx,
        magmaFloatComplex beta, 
        magmaFloatComplex * dy)
{
    extern __shared__ magmaFloatComplex dot[];
    int row = blockDim.x * blockIdx.x + threadIdx.x ;
    if(row < num_rows ){
        for( int i=0; i<num_vecs; i++ )
                dot[ threadIdx.x+ i*blockDim.x ] = MAGMA_C_MAKE(0.0, 0.0);
        for ( int n = 0; n < num_cols_per_row ; n ++){
            int col = dcolind [ num_rows * n + row ];
            magmaFloatComplex val = dval [ num_rows * n + row ];
            if( val != 0){
                for( int i=0; i<num_vecs; i++ )
                    dot[ threadIdx.x + i*blockDim.x ] += 
                                        val * dx[col + i * num_cols ];
            }
        }
        for( int i=0; i<num_vecs; i++ )
                dy[ row + i*num_cols ] = dot[ threadIdx.x + i*blockDim.x ] 
                                * alpha + beta * dy [ row + i*num_cols ];
    }
}





/**
    Purpose
    -------
    
    This routine computes Y = alpha *  A *  X + beta * Y for X and Y sets of 
    num_vec vectors on the GPU. Input format is ELL. 
    
    Arguments
    ---------

    @param[in]
    transA      magma_trans_t
                transposition parameter for A

    @param[in]
    m           magma_int_t
                number of rows in A

    @param[in]
    n           magma_int_t
                number of columns in A 
                
    @param[in]
    num_vecs    mama_int_t
                number of vectors
                
    @param[in]
    nnz_per_row magma_int_t
                number of elements in the longest row 
                
    @param[in]
    alpha       magmaFloatComplex
                scalar multiplier

    @param[in]
    dval        magmaFloatComplex_ptr
                array containing values of A in ELL

    @param[in]
    dcolind     magmaIndex_ptr
                columnindices of A in ELL

    @param[in]
    dx          magmaFloatComplex_ptr
                input vector x

    @param[in]
    beta        magmaFloatComplex
                scalar multiplier

    @param[out]
    dy          magmaFloatComplex_ptr
                input/output vector y

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_cblas
    ********************************************************************/

extern "C" magma_int_t
magma_cmgeelltmv(
    magma_trans_t transA,
    magma_int_t m, magma_int_t n,
    magma_int_t num_vecs,
    magma_int_t nnz_per_row,
    magmaFloatComplex alpha,
    magmaFloatComplex_ptr dval,
    magmaIndex_ptr dcolind,
    magmaFloatComplex_ptr dx,
    magmaFloatComplex beta,
    magmaFloatComplex_ptr dy,
    magma_queue_t queue )
{
    dim3 grid( (m+BLOCK_SIZE-1)/BLOCK_SIZE, 1, 1);
    magma_int_t threads = BLOCK_SIZE;
    unsigned int MEM_SIZE =  num_vecs* BLOCK_SIZE 
                * sizeof( magmaFloatComplex ); // num_vecs vectors 
    cmgeelltmv_kernel<<< grid, threads, MEM_SIZE, queue >>>
        ( m, n, num_vecs, nnz_per_row, alpha, dval, dcolind, dx, beta, dy );


    return MAGMA_SUCCESS;
}



