#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.5.0-beta3) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date July 2014

       @generated from zmgecsrmv.cu normal z -> d, Fri Jul 18 17:34:28 2014

*/
#include "common_magma.h"

#if (GPUSHMEM < 200)
   #define BLOCK_SIZE 128
#else
   #define BLOCK_SIZE 512
#endif



__global__ void 
dmgecsrmv_kernel( int num_rows, int num_cols, 
                  int num_vecs,
                  double alpha, 
                  double *d_val, 
                  magma_index_t *d_rowptr, 
                  magma_index_t *d_colind,
                  double *d_x,
                  double beta, 
                  double *d_y){

    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j;
    extern __shared__ double dot[];

    if( row<num_rows ){
        for( int i=0; i<num_vecs; i++ )
                dot[ threadIdx.x+ i*blockDim.x ] = MAGMA_D_MAKE(0.0, 0.0);
        int start = d_rowptr[ row ] ;
        int end = d_rowptr[ row+1 ];
        for( j=start; j<end; j++ ){
            int col = d_colind [ j ];
            double val = d_val[ j ];
            for( int i=0; i<num_vecs; i++ )
                dot[ threadIdx.x + i*blockDim.x ] += 
                                    val * d_x[ col + i*num_cols ];
        }
        for( int i=0; i<num_vecs; i++ )
            d_y[ row +i*num_cols ] = alpha * dot[ threadIdx.x + i*blockDim.x ] 
                                             + beta * d_y[ row + i*num_cols ];
    }
}



/**
    Purpose
    -------
    
    This routine computes Y = alpha *  A *  X + beta * Y for X and Y sets of 
    num_vec vectors on the GPU. Input format is CSR. 
    
    Arguments
    ---------
    
    @param
    transA      magma_trans_t
                transposition parameter for A

    @param
    m           magma_int_t
                number of rows in A

    @param
    n           magma_int_t
                number of columns in A 
                
    @param
    num_vecs    mama_int_t
                number of vectors
    @param
    alpha       double
                scalar multiplier

    @param
    d_val       double*
                array containing values of A in CSR

    @param
    d_rowptr    magma_int_t*
                rowpointer of A in CSR

    @param
    d_colind    magma_int_t*
                columnindices of A in CSR

    @param
    d_x         double*
                input vector x

    @param
    beta        double
                scalar multiplier

    @param
    d_y         double*
                input/output vector y


    @ingroup magmasparse_dblas
    ********************************************************************/

extern "C" magma_int_t
magma_dmgecsrmv(    magma_trans_t transA,
                    magma_int_t m, magma_int_t n,
                    magma_int_t num_vecs, 
                    double alpha,
                    double *d_val,
                    magma_index_t *d_rowptr,
                    magma_index_t *d_colind,
                    double *d_x,
                    double beta,
                    double *d_y ){

    dim3 grid( (m+BLOCK_SIZE-1)/BLOCK_SIZE, 1, 1);
    unsigned int MEM_SIZE =  num_vecs* BLOCK_SIZE 
                    * sizeof( double ); // num_vecs vectors 
    dmgecsrmv_kernel<<< grid, BLOCK_SIZE, MEM_SIZE >>>
            (m, n, num_vecs, alpha, d_val, d_rowptr, d_colind, d_x, beta, d_y);

   return MAGMA_SUCCESS;
}



