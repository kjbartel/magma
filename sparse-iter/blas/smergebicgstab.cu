#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.5.0-beta2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2014

       @generated from zmergebicgstab.cu normal z -> s, Fri May 30 10:41:37 2014
       @author Hartwig Anzt

*/
#include "common_magma.h"

#define BLOCK_SIZE 512

#define PRECISION_s


// These routines merge multiple kernels from smergebicgstab into one
// The difference to smergedbicgstab2 is that the SpMV is not merged into the
// kernes. This results in higher flexibility at the price of lower performance.

/* -------------------------------------------------------------------------- */

__global__ void 
magma_sbicgmerge1_kernel(  
                    int n, 
                    float *skp,
                    float *v, 
                    float *r, 
                    float *p ){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    float beta=skp[1];
    float omega=skp[2];
    if( i<n ){
        p[i] =  r[i] + beta * ( p[i] - omega * v[i] );

    }

}

/*  -- MAGMA (version 1.5.0-beta2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2014

    Purpose
    =======

    Mergels multiple operations into one kernel:

    p = beta*p
    p = p-omega*beta*v
    p = p+r
    
    -> p = r + beta * ( p - omega * v ) 

    Arguments
    =========

    int n                               dimension n
    float beta             scalar 
    float omega            scalar
    float *v               input v
    float *r               input r
    float *p               input/output p

    ========================================================================  */

extern "C" int
magma_sbicgmerge1(  int n, 
                    float *skp,
                    float *v, 
                    float *r, 
                    float *p ){

    
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( (n+BLOCK_SIZE-1)/BLOCK_SIZE );
    magma_sbicgmerge1_kernel<<<Gs, Bs, 0>>>( n, skp, v, r, p );

   return MAGMA_SUCCESS;
}

/* -------------------------------------------------------------------------- */

__global__ void 
magma_sbicgmerge2_kernel(  
                    int n, 
                    float *skp, 
                    float *r,
                    float *v, 
                    float *s ){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    float alpha=skp[0];
    if( i<n ){
        s[i] =  r[i] - alpha * v[i] ;
    }

}

/*  -- MAGMA (version 1.5.0-beta2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2014

    Purpose
    =======

    Mergels multiple operations into one kernel:

    s=r
    s=s-alpha*v
        
    -> s = r - alpha * v

    Arguments
    =========

    int n                               dimension n
    float alpha            scalar 
    float *r               input r
    float *v               input v
    float *s               input/output s

    ========================================================================  */

extern "C" int
magma_sbicgmerge2(  int n, 
                    float *skp, 
                    float *r,
                    float *v, 
                    float *s ){

    
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( (n+BLOCK_SIZE-1)/BLOCK_SIZE );

    magma_sbicgmerge2_kernel<<<Gs, Bs, 0>>>( n, skp, r, v, s );

   return MAGMA_SUCCESS;
}

/* -------------------------------------------------------------------------- */

__global__ void 
magma_sbicgmerge3_kernel(  
                    int n, 
                    float *skp, 
                    float *p,
                    float *se,
                    float *t,
                    float *x, 
                    float *r ){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    float alpha=skp[0];
    float omega=skp[2];
    if( i<n ){
        float s;
        s = se[i];
        x[i] = x[i] + alpha * p[i] + omega * s;
        r[i] = s - omega * t[i];
    }

}

/*  -- MAGMA (version 1.5.0-beta2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2014

    Purpose
    =======

    Mergels multiple operations into one kernel:

    x=x+alpha*p
    x=x+omega*s
    r=s
    r=r-omega*t
        
    -> x = x + alpha * p + omega * s
    -> r = s - omega * t

    Arguments
    =========

    int n                               dimension n
    float alpha            scalar 
    float omega            scalar 
    float *p               input p
    float *s               input s
    float *t               input t
    float *x               input/output x
    float *r               input/output r

    ========================================================================  */

extern "C" int
magma_sbicgmerge3(  int n, 
                    float *skp,
                    float *p,
                    float *s,
                    float *t,
                    float *x, 
                    float *r ){

    
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( (n+BLOCK_SIZE-1)/BLOCK_SIZE );
    magma_sbicgmerge3_kernel<<<Gs, Bs, 0>>>( n, skp, p, s, t, x, r );

   return MAGMA_SUCCESS;
}

/* -------------------------------------------------------------------------- */

__global__ void 
magma_sbicgmerge4_kernel_1(  
                    float *skp ){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if( i==0 ){
        float tmp = skp[0];
        skp[0] = skp[4]/tmp;
    }
}

__global__ void 
magma_sbicgmerge4_kernel_2(  
                    float *skp ){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if( i==0 ){
        skp[2] = skp[6]/skp[7];
        skp[3] = skp[4];
    }
}

__global__ void 
magma_sbicgmerge4_kernel_3(  
                    float *skp ){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if( i==0 ){
        float tmp1 = skp[4]/skp[3];
        float tmp2 = skp[0] / skp[2];
        skp[1] =  tmp1*tmp2;
        //skp[1] =  skp[4]/skp[3] * skp[0] / skp[2];

    }
}

/*  -- MAGMA (version 1.5.0-beta2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2014

    Purpose
    =======

    Performs some parameter operations for the BiCGSTAB with scalars on GPU.

    Arguments
    =========

    int type                            kernel type
    float *skp             vector with parameters

    ========================================================================  */

extern "C" int
magma_sbicgmerge4(  int type, 
                    float *skp ){

    dim3 Bs( 2 );
    dim3 Gs( 1 );
    if( type == 1 )
        magma_sbicgmerge4_kernel_1<<<Gs, Bs, 0>>>( skp );
    else if( type == 2 )
        magma_sbicgmerge4_kernel_2<<<Gs, Bs, 0>>>( skp );
    else if( type == 3 )
        magma_sbicgmerge4_kernel_3<<<Gs, Bs, 0>>>( skp );
    else
        printf("error: no kernel called\n");

   return MAGMA_SUCCESS;
}

