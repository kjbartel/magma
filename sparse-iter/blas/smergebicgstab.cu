#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @generated from zmergebicgstab.cu normal z -> s, Fri Jan 30 19:00:29 2015
       @author Hartwig Anzt

*/
#include "common_magma.h"

#define BLOCK_SIZE 512

#define PRECISION_s


// These routines merge multiple kernels from smergebicgstab into one
// The difference to smergedbicgstab2 is that the SpMV is not merged into the
// kernes. This results in higher flexibility at the price of lower performance.

/* -------------------------------------------------------------------------- */

__global__ void
magma_sbicgmerge1_kernel(  
    int n, 
    float * skp,
    float * v, 
    float * r, 
    float * p )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    float beta=skp[1];
    float omega=skp[2];
    if( i<n ){
        p[i] =  r[i] + beta * ( p[i] - omega * v[i] );

    }

}

/**
    Purpose
    -------

    Mergels multiple operations into one kernel:

    p = beta*p
    p = p-omega*beta*v
    p = p+r
    
    -> p = r + beta * ( p - omega * v ) 

    Arguments
    ---------

    @param[in]
    n           int
                dimension n

    @param[in]
    skp         magmaFloat_ptr 
                set of scalar parameters

    @param[in]
    v           magmaFloat_ptr 
                input v

    @param[in]
    r           magmaFloat_ptr 
                input r

    @param[in/out]
    p           magmaFloat_ptr 
                input/output p

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_sgegpuk
    ********************************************************************/

extern "C" int
magma_sbicgmerge1(  
    int n, 
    magmaFloat_ptr skp,
    magmaFloat_ptr v, 
    magmaFloat_ptr r, 
    magmaFloat_ptr p ){

    
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( (n+BLOCK_SIZE-1)/BLOCK_SIZE );
    magma_sbicgmerge1_kernel<<<Gs, Bs, 0>>>( n, skp, v, r, p );

   return MAGMA_SUCCESS;
}

/* -------------------------------------------------------------------------- */

__global__ void
magma_sbicgmerge2_kernel(  
    int n, 
    float * skp, 
    float * r,
    float * v, 
    float * s )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    float alpha=skp[0];
    if( i<n ){
        s[i] =  r[i] - alpha * v[i] ;
    }

}

/**
    Purpose
    -------

    Mergels multiple operations into one kernel:

    s=r
    s=s-alpha*v
        
    -> s = r - alpha * v

    Arguments
    ---------

    @param[in]
    n           int
                dimension n

    @param[in]
    skp         magmaFloat_ptr 
                set of scalar parameters

    @param[in]
    r           magmaFloat_ptr 
                input r

    @param[in]
    v           magmaFloat_ptr 
                input v

    @param[s]
    s           magmaFloat_ptr 
                output s

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_sgegpuk
    ********************************************************************/

extern "C" int
magma_sbicgmerge2(  
    int n, 
    magmaFloat_ptr skp, 
    magmaFloat_ptr r,
    magmaFloat_ptr v, 
    magmaFloat_ptr s )
{

    
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( (n+BLOCK_SIZE-1)/BLOCK_SIZE );

    magma_sbicgmerge2_kernel<<<Gs, Bs, 0>>>( n, skp, r, v, s );

   return MAGMA_SUCCESS;
}

/* -------------------------------------------------------------------------- */

__global__ void
magma_sbicgmerge3_kernel(  
    int n, 
    float * skp, 
    float * p,
    float * se,
    float * t,
    float * x, 
    float * r )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    float alpha=skp[0];
    float omega=skp[2];
    if( i<n ){
        float s;
        s = se[i];
        x[i] = x[i] + alpha * p[i] + omega * s;
        r[i] = s - omega * t[i];
    }

}

/**
    Purpose
    -------

    Mergels multiple operations into one kernel:

    x=x+alpha*p
    x=x+omega*s
    r=s
    r=r-omega*t
        
    -> x = x + alpha * p + omega * s
    -> r = s - omega * t

    Arguments
    ---------

    @param[in]
    n           int
                dimension n

    @param[in]
    skp         magmaFloat_ptr 
                set of scalar parameters

    @param[in]
    p           magmaFloat_ptr 
                input p

    @param[in]
    s           magmaFloat_ptr 
                input s

    @param[in]
    t           magmaFloat_ptr 
                input t

    @param[in/out]
    x           magmaFloat_ptr 
                input/output x

    @param[in/out]
    r           magmaFloat_ptr 
                input/output r

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_sgegpuk
    ********************************************************************/

extern "C" int
magma_sbicgmerge3(  
    int n, 
    magmaFloat_ptr skp,
    magmaFloat_ptr p,
    magmaFloat_ptr s,
    magmaFloat_ptr t,
    magmaFloat_ptr x, 
    magmaFloat_ptr r )
{

    
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( (n+BLOCK_SIZE-1)/BLOCK_SIZE );
    magma_sbicgmerge3_kernel<<<Gs, Bs, 0>>>( n, skp, p, s, t, x, r );

   return MAGMA_SUCCESS;
}

/* -------------------------------------------------------------------------- */

__global__ void
magma_sbicgmerge4_kernel_1(  
    float * skp )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if( i==0 ){
        float tmp = skp[0];
        skp[0] = skp[4]/tmp;
    }
}

__global__ void
magma_sbicgmerge4_kernel_2(  
    float * skp )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if( i==0 ){
        skp[2] = skp[6]/skp[7];
        skp[3] = skp[4];
    }
}

__global__ void
magma_sbicgmerge4_kernel_3(  
    float * skp )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if( i==0 ){
        float tmp1 = skp[4]/skp[3];
        float tmp2 = skp[0] / skp[2];
        skp[1] =  tmp1*tmp2;
        //skp[1] =  skp[4]/skp[3] * skp[0] / skp[2];

    }
}

/**
    Purpose
    -------

    Performs some parameter operations for the BiCGSTAB with scalars on GPU.

    Arguments
    ---------

    @param[in]
    type        int
                kernel type

    @param[in/out]
    skp         magmaFloat_ptr 
                vector with parameters

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_sgegpuk
    ********************************************************************/

extern "C" int
magma_sbicgmerge4(  
    int type, 
    magmaFloat_ptr skp )
{

    dim3 Bs( 1 );
    dim3 Gs( 1 );
    if( type == 1 )
        magma_sbicgmerge4_kernel_1<<<Gs, Bs, 0>>>( skp );
    else if( type == 2 )
        magma_sbicgmerge4_kernel_2<<<Gs, Bs, 0>>>( skp );
    else if( type == 3 )
        magma_sbicgmerge4_kernel_3<<<Gs, Bs, 0>>>( skp );
    else
        printf("error: no kernel called\n");

   return MAGMA_SUCCESS;
}

