#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2014

       @generated from zmgeellmv.cu normal z -> d, Sat Nov 15 19:54:21 2014

*/
#include "common_magma.h"

#if (GPUSHMEM < 200)
   #define BLOCK_SIZE 128
#else
   #define BLOCK_SIZE 512
#endif



__global__ void 
dmgeellmv_kernel( 
    int num_rows, 
    int num_cols,
    int num_vecs,
    int num_cols_per_row,
    double alpha, 
    magmaDouble_ptr dval, 
    magmaIndex_ptr dcolind,
    magmaDouble_ptr dx,
    double beta, 
    magmaDouble_ptr dy)
{
int row = blockDim.x * blockIdx.x + threadIdx.x ;

    extern __shared__ double dot[];

    if(row < num_rows ){
        for( int i=0; i<num_vecs; i++)
                dot[ threadIdx.x + i*blockDim.x ] = MAGMA_D_MAKE(0.0, 0.0);
        for ( int n = 0; n < num_cols_per_row ; n ++){
            int col = dcolind [ num_cols_per_row * row + n ];
            double val = dval [ num_cols_per_row * row + n ];
            if( val != 0){
                for( int i=0; i<num_vecs; i++)
                    dot[ threadIdx.x + i*blockDim.x ] += 
                                    val * dx[col + i * num_cols ];
            }
        }
        for( int i=0; i<num_vecs; i++)
                dy[ row + i*num_cols ] = dot[ threadIdx.x + i*blockDim.x ] 
                                * alpha + beta * dy [ row + i * num_cols ];
    }
}





/**
    Purpose
    -------
    
    This routine computes Y = alpha *  A *  X + beta * Y for X and Y sets of 
    num_vec vectors on the GPU. Input format is ELLPACK. 
    
    Arguments
    ---------

    @param[in]
    transA      magma_trans_t
                transposition parameter for A

    @param[in]
    m           magma_int_t
                number of rows in A

    @param[in]
    n           magma_int_t
                number of columns in A 
                              
    @param[in]
    num_vecs    mama_int_t
                number of vectors
                
    @param[in]
    nnz_per_row magma_int_t
                number of elements in the longest row 
                
    @param[in]
    alpha       double
                scalar multiplier

    @param[in]
    dval        magmaDouble_ptr
                array containing values of A in ELLPACK

    @param[in]
    dcolind     magmaIndex_ptr
                columnindices of A in ELLPACK

    @param[in]
    dx          magmaDouble_ptr
                input vector x

    @param[in]
    beta        double
                scalar multiplier

    @param[out]
    dy          magmaDouble_ptr
                input/output vector y

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_dblas
    ********************************************************************/

extern "C" magma_int_t
magma_dmgeellmv(
    magma_trans_t transA,
    magma_int_t m, magma_int_t n,
    magma_int_t num_vecs,
    magma_int_t nnz_per_row,
    double alpha,
    magmaDouble_ptr dval,
    magmaIndex_ptr dcolind,
    magmaDouble_ptr dx,
    double beta,
    magmaDouble_ptr dy,
    magma_queue_t queue )
{
    dim3 grid( (m+BLOCK_SIZE-1)/BLOCK_SIZE, 1, 1);
    magma_int_t threads = BLOCK_SIZE;
    unsigned int MEM_SIZE =  num_vecs* BLOCK_SIZE 
                            * sizeof( double ); // num_vecs vectors 
    dmgeellmv_kernel<<< grid, threads, MEM_SIZE, queue >>>
        ( m, n, num_vecs, nnz_per_row, alpha, dval, dcolind, dx, beta, dy );


   return MAGMA_SUCCESS;
}



