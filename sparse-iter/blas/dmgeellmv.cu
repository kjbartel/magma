#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.5.0-beta2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2014

       @generated from zmgeellmv.cu normal z -> d, Fri May 30 10:41:37 2014

*/
#include "common_magma.h"

#if (GPUSHMEM < 200)
   #define BLOCK_SIZE 128
#else
   #define BLOCK_SIZE 512
#endif



__global__ void 
dmgeellmv_kernel( int num_rows, 
                  int num_cols,
                  int num_vecs,
                  int num_cols_per_row,
                  double alpha, 
                  double *d_val, 
                  magma_index_t *d_colind,
                  double *d_x,
                  double beta, 
                  double *d_y)
{
int row = blockDim.x * blockIdx.x + threadIdx.x ;

    extern __shared__ double dot[];

    if(row < num_rows ){
        for( int i=0; i<num_vecs; i++)
                dot[ threadIdx.x + i*blockDim.x ] = MAGMA_D_MAKE(0.0, 0.0);
        for ( int n = 0; n < num_cols_per_row ; n ++){
            int col = d_colind [ num_cols_per_row * row + n ];
            double val = d_val [ num_cols_per_row * row + n ];
            if( val != 0){
                for( int i=0; i<num_vecs; i++)
                    dot[ threadIdx.x + i*blockDim.x ] += 
                                    val * d_x[col + i * num_cols ];
            }
        }
        for( int i=0; i<num_vecs; i++)
                d_y[ row + i*num_cols ] = dot[ threadIdx.x + i*blockDim.x ] 
                                * alpha + beta * d_y [ row + i * num_cols ];
    }
}





/*  -- MAGMA (version 1.5.0-beta2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2014

    Purpose
    =======
    
    This routine computes Y = alpha *  A *  X + beta * Y for X and Y sets of 
    num_vec vectors on the GPU. Input format is ELLPACK. 
    
    Arguments
    =========

    magma_int_t m                   number of rows in A
    magma_int_t n                   number of columns in A 
    mama_int_t num_vecs             number of vectors
    double alpha        scalar multiplier
    double *d_val       array containing values of A in ELLPACK
    magma_int_t *d_colind           columnindices of A in ELLPACK
    double *d_x         input vector x
    double beta         scalar multiplier
    double *d_y         input/output vector y

    ======================================================================    */

extern "C" magma_int_t
magma_dmgeellmv(   magma_trans_t transA,
                   magma_int_t m, magma_int_t n,
                   magma_int_t num_vecs,
                   magma_int_t nnz_per_row,
                   double alpha,
                   double *d_val,
                   magma_index_t *d_colind,
                   double *d_x,
                   double beta,
                   double *d_y ){



    dim3 grid( (m+BLOCK_SIZE-1)/BLOCK_SIZE, 1, 1);
    unsigned int MEM_SIZE =  num_vecs* BLOCK_SIZE 
                            * sizeof( double ); // num_vecs vectors 
    dmgeellmv_kernel<<< grid, BLOCK_SIZE, MEM_SIZE >>>
        ( m, n, num_vecs, nnz_per_row, alpha, d_val, d_colind, d_x, beta, d_y );


   return MAGMA_SUCCESS;
}



