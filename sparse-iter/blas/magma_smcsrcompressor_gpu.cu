#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2015

       @generated from magma_zmcsrcompressor_gpu.cu normal z -> s, Sun May  3 11:22:58 2015
       @author Hartwig Anzt

*/
#include "common_magmasparse.h"

#define BLOCK_SIZE1 256
#define BLOCK_SIZE2 1


// copy nonzeros into new structure
__global__ void
magma_smcsrgpu_kernel1( int num_rows,
                 float *A_val,
                 magma_index_t *A_rowptr,
                 magma_index_t *A_colind,
                 float *B_val,
                 magma_index_t *B_rowptr,
                 magma_index_t *B_colind ){

    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j;

    if(row<num_rows){
        float zero = MAGMA_S_ZERO;
        int start = A_rowptr[ row ];
        int new_location = start;
        int end = A_rowptr[ row+1 ];
        for( j=start; j<end; j++ ){
            if( A_val[j] != zero ){
       //         B_val[new_location] = A_val[j];
       //         B_colind[new_location] = A_colind[j];
                new_location++;
            }
        }
        // this is not a correctr rowpointer! this is nn_z in this row!
        B_rowptr[ row ] = new_location-start;
    }
}


// generate a valid rowpointer
__global__ void
magma_smcsrgpu_kernel2( int num_rows,
                 magma_index_t *B_rowptr,
                 magma_index_t *A_rowptr ){

    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    int j, nnz = 0;

    if( idx == 0 ){
    A_rowptr[ 0 ] = nnz;
        for( j=0; j<num_rows; j++ ){
            nnz+=B_rowptr[ j ];
            A_rowptr[ j+1 ] = nnz;
        }
    }
}



// copy new structure into original matrix
__global__ void
magma_smcsrgpu_kernel3( int num_rows,
                 float *B_val,
                 magma_index_t *B_rowptr,
                 magma_index_t *B_colind,
                 magma_index_t *B2_rowptr,
                 float *A_val,
                 magma_index_t *A_rowptr,
                 magma_index_t *A_colind
                                            ){

    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j, new_location;
    
    if(row<num_rows){
    new_location = A_rowptr[ row ];
        int start = B2_rowptr[ row ];
        int end = B2_rowptr[ row+1 ];
        float zero = MAGMA_S_ZERO;
        for( j=start; j<end; j++ ){
            if( A_val[j] != zero ){
                B_val[new_location] = A_val[j];
                B_colind[new_location] = A_colind[j];
                new_location++;
            }
               // A_val[ j ] = B_val[ j ];
               // A_colind[ j ] = B_colind[ j ];
        }
    }
}


/**
    Purpose
    -------

    Removes zeros in a CSR matrix. This is a GPU implementation of the
    CSR compressor.

    Arguments
    ---------

    @param
    A           magma_s_matrix*
                input/output matrix
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_saux
    ********************************************************************/

extern "C" magma_int_t
magma_smcsrcompressor_gpu(
    magma_s_matrix *A,
    magma_queue_t queue )
{
    magma_int_t info = 0;
    magma_s_matrix B={Magma_CSR}, B2={Magma_CSR};
    magma_s_matrix dA={Magma_CSR}, CSRA={Magma_CSR};
    magma_index_t *cputmp = NULL;
    
    if ( A->memory_location == Magma_DEV && A->storage_type == Magma_CSR ) {

        CHECK( magma_index_malloc( &B.drow, A->num_rows + 1 ));
        CHECK( magma_index_malloc( &B2.drow, A->num_rows + 1 ));
        
        magma_index_copyvector( (A->num_rows+1), A->drow, 1, B2.drow, 1 );

        dim3 grid1( magma_ceildiv( A->num_rows, BLOCK_SIZE1 ) );

        // copying the nonzeros into B and write in B.drow how many there are
        magma_smcsrgpu_kernel1<<< grid1, BLOCK_SIZE1, 0, queue >>>
                ( A->num_rows, A->dval, A->drow, A->dcol, B.dval, B.drow, B.dcol );

        // correct the row pointer
        dim3 grid2( 1, 1, 1);
        magma_smcsrgpu_kernel2<<< grid2, BLOCK_SIZE2, 0, queue >>>
                ( A->num_rows, B.drow, A->drow );
        // access the true number of nonzeros

        CHECK( magma_index_malloc_cpu( &cputmp, 1 ));

        magma_index_getvector( 1, A->row+(A->num_rows), 1, cputmp, 1 );
        A->nnz = (magma_int_t) cputmp[0];

        // reallocate with right size
        CHECK( magma_smalloc( &B.dval, A->nnz ));
        CHECK( magma_index_malloc( &B.dcol, A->nnz ));
        
        // copy correct values back
        magma_smcsrgpu_kernel3<<< grid1, BLOCK_SIZE1, 0, queue >>>
                ( A->num_rows, B.dval, B.drow, B.dcol, B2.drow, A->dval, A->drow, A->dcol );

        magma_free( A->dcol );
        magma_free( A->dval );

        A->dcol = B.dcol;
        A->dval = B.dval;


    }
    else {
        magma_storage_t A_storage = A->storage_type;
        magma_location_t A_location = A->memory_location;
        CHECK( magma_smconvert( *A, &CSRA, A->storage_type, Magma_CSR, queue ));
        CHECK( magma_smtransfer( *A, &dA, A->memory_location, Magma_DEV, queue ));

        CHECK( magma_smcsrcompressor_gpu( &dA, queue ));

        magma_smfree( &dA, queue );
        magma_smfree( A, queue );
        CHECK( magma_smtransfer( dA, &CSRA, Magma_DEV, A_location, queue ));
        CHECK( magma_smconvert( CSRA, A, Magma_CSR, A_storage, queue ));
        magma_smfree( &dA, queue );
        magma_smfree( &CSRA, queue );

    }
    
cleanup:
    magma_smfree( &dA, queue );
    magma_smfree( &CSRA, queue );
    magma_free( B2.drow );
    magma_free( B.drow );
    return info;
}


