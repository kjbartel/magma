#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2014

       @generated from zmergebicgstab.cu normal z -> d, Sat Nov 15 19:54:21 2014
       @author Hartwig Anzt

*/
#include "common_magma.h"

#define BLOCK_SIZE 512

#define PRECISION_d


// These routines merge multiple kernels from dmergebicgstab into one
// The difference to dmergedbicgstab2 is that the SpMV is not merged into the
// kernes. This results in higher flexibility at the price of lower performance.

/* -------------------------------------------------------------------------- */

__global__ void
magma_dbicgmerge1_kernel(  
    int n, 
    magmaDouble_ptr skp,
    magmaDouble_ptr v, 
    magmaDouble_ptr r, 
    magmaDouble_ptr p )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    double beta=skp[1];
    double omega=skp[2];
    if( i<n ){
        p[i] =  r[i] + beta * ( p[i] - omega * v[i] );

    }

}

/**
    Purpose
    -------

    Mergels multiple operations into one kernel:

    p = beta*p
    p = p-omega*beta*v
    p = p+r
    
    -> p = r + beta * ( p - omega * v ) 

    Arguments
    ---------

    @param[in]
    n           int
                dimension n

    @param[in]
    skp         magmaDouble_ptr 
                set of scalar parameters

    @param[in]
    v           magmaDouble_ptr 
                input v

    @param[in]
    r           magmaDouble_ptr 
                input r

    @param[in/out]
    p           magmaDouble_ptr 
                input/output p

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_dgegpuk
    ********************************************************************/

extern "C" int
magma_dbicgmerge1(  
    int n, 
    magmaDouble_ptr skp,
    magmaDouble_ptr v, 
    magmaDouble_ptr r, 
    magmaDouble_ptr p ){

    
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( (n+BLOCK_SIZE-1)/BLOCK_SIZE );
    magma_dbicgmerge1_kernel<<<Gs, Bs, 0>>>( n, skp, v, r, p );

   return MAGMA_SUCCESS;
}

/* -------------------------------------------------------------------------- */

__global__ void
magma_dbicgmerge2_kernel(  
    int n, 
    magmaDouble_ptr skp, 
    magmaDouble_ptr r,
    magmaDouble_ptr v, 
    magmaDouble_ptr s )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    double alpha=skp[0];
    if( i<n ){
        s[i] =  r[i] - alpha * v[i] ;
    }

}

/**
    Purpose
    -------

    Mergels multiple operations into one kernel:

    s=r
    s=s-alpha*v
        
    -> s = r - alpha * v

    Arguments
    ---------

    @param[in]
    n           int
                dimension n

    @param[in]
    skp         magmaDouble_ptr 
                set of scalar parameters

    @param[in]
    r           magmaDouble_ptr 
                input r

    @param[in]
    v           magmaDouble_ptr 
                input v

    @param[s]
    s           magmaDouble_ptr 
                output s

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_dgegpuk
    ********************************************************************/

extern "C" int
magma_dbicgmerge2(  
    int n, 
    magmaDouble_ptr skp, 
    magmaDouble_ptr r,
    magmaDouble_ptr v, 
    magmaDouble_ptr s )
{

    
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( (n+BLOCK_SIZE-1)/BLOCK_SIZE );

    magma_dbicgmerge2_kernel<<<Gs, Bs, 0>>>( n, skp, r, v, s );

   return MAGMA_SUCCESS;
}

/* -------------------------------------------------------------------------- */

__global__ void
magma_dbicgmerge3_kernel(  
    int n, 
    magmaDouble_ptr skp, 
    magmaDouble_ptr p,
    magmaDouble_ptr se,
    magmaDouble_ptr t,
    magmaDouble_ptr x, 
    magmaDouble_ptr r )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    double alpha=skp[0];
    double omega=skp[2];
    if( i<n ){
        double s;
        s = se[i];
        x[i] = x[i] + alpha * p[i] + omega * s;
        r[i] = s - omega * t[i];
    }

}

/**
    Purpose
    -------

    Mergels multiple operations into one kernel:

    x=x+alpha*p
    x=x+omega*s
    r=s
    r=r-omega*t
        
    -> x = x + alpha * p + omega * s
    -> r = s - omega * t

    Arguments
    ---------

    @param[in]
    n           int
                dimension n

    @param[in]
    skp         magmaDouble_ptr 
                set of scalar parameters

    @param[in]
    p           magmaDouble_ptr 
                input p

    @param[in]
    s           magmaDouble_ptr 
                input s

    @param[in]
    t           magmaDouble_ptr 
                input t

    @param[in/out]
    x           magmaDouble_ptr 
                input/output x

    @param[in/out]
    r           magmaDouble_ptr 
                input/output r

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_dgegpuk
    ********************************************************************/

extern "C" int
magma_dbicgmerge3(  
    int n, 
    magmaDouble_ptr skp,
    magmaDouble_ptr p,
    magmaDouble_ptr s,
    magmaDouble_ptr t,
    magmaDouble_ptr x, 
    magmaDouble_ptr r )
{

    
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( (n+BLOCK_SIZE-1)/BLOCK_SIZE );
    magma_dbicgmerge3_kernel<<<Gs, Bs, 0>>>( n, skp, p, s, t, x, r );

   return MAGMA_SUCCESS;
}

/* -------------------------------------------------------------------------- */

__global__ void
magma_dbicgmerge4_kernel_1(  
    magmaDouble_ptr skp )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if( i==0 ){
        double tmp = skp[0];
        skp[0] = skp[4]/tmp;
    }
}

__global__ void
magma_dbicgmerge4_kernel_2(  
    magmaDouble_ptr skp )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if( i==0 ){
        skp[2] = skp[6]/skp[7];
        skp[3] = skp[4];
    }
}

__global__ void
magma_dbicgmerge4_kernel_3(  
    magmaDouble_ptr skp )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if( i==0 ){
        double tmp1 = skp[4]/skp[3];
        double tmp2 = skp[0] / skp[2];
        skp[1] =  tmp1*tmp2;
        //skp[1] =  skp[4]/skp[3] * skp[0] / skp[2];

    }
}

/**
    Purpose
    -------

    Performs some parameter operations for the BiCGSTAB with scalars on GPU.

    Arguments
    ---------

    @param[in]
    type        int
                kernel type

    @param[in/out]
    skp         magmaDouble_ptr 
                vector with parameters

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_dgegpuk
    ********************************************************************/

extern "C" int
magma_dbicgmerge4(  
    int type, 
    magmaDouble_ptr skp )
{

    dim3 Bs( 1 );
    dim3 Gs( 1 );
    if( type == 1 )
        magma_dbicgmerge4_kernel_1<<<Gs, Bs, 0>>>( skp );
    else if( type == 2 )
        magma_dbicgmerge4_kernel_2<<<Gs, Bs, 0>>>( skp );
    else if( type == 3 )
        magma_dbicgmerge4_kernel_3<<<Gs, Bs, 0>>>( skp );
    else
        printf("error: no kernel called\n");

   return MAGMA_SUCCESS;
}

