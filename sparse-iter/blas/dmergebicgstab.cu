#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.5.0-beta2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2014

       @generated from zmergebicgstab.cu normal z -> d, Fri May 30 10:41:37 2014
       @author Hartwig Anzt

*/
#include "common_magma.h"

#define BLOCK_SIZE 512

#define PRECISION_d


// These routines merge multiple kernels from dmergebicgstab into one
// The difference to dmergedbicgstab2 is that the SpMV is not merged into the
// kernes. This results in higher flexibility at the price of lower performance.

/* -------------------------------------------------------------------------- */

__global__ void 
magma_dbicgmerge1_kernel(  
                    int n, 
                    double *skp,
                    double *v, 
                    double *r, 
                    double *p ){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    double beta=skp[1];
    double omega=skp[2];
    if( i<n ){
        p[i] =  r[i] + beta * ( p[i] - omega * v[i] );

    }

}

/*  -- MAGMA (version 1.5.0-beta2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2014

    Purpose
    =======

    Mergels multiple operations into one kernel:

    p = beta*p
    p = p-omega*beta*v
    p = p+r
    
    -> p = r + beta * ( p - omega * v ) 

    Arguments
    =========

    int n                               dimension n
    double beta             scalar 
    double omega            scalar
    double *v               input v
    double *r               input r
    double *p               input/output p

    ========================================================================  */

extern "C" int
magma_dbicgmerge1(  int n, 
                    double *skp,
                    double *v, 
                    double *r, 
                    double *p ){

    
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( (n+BLOCK_SIZE-1)/BLOCK_SIZE );
    magma_dbicgmerge1_kernel<<<Gs, Bs, 0>>>( n, skp, v, r, p );

   return MAGMA_SUCCESS;
}

/* -------------------------------------------------------------------------- */

__global__ void 
magma_dbicgmerge2_kernel(  
                    int n, 
                    double *skp, 
                    double *r,
                    double *v, 
                    double *s ){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    double alpha=skp[0];
    if( i<n ){
        s[i] =  r[i] - alpha * v[i] ;
    }

}

/*  -- MAGMA (version 1.5.0-beta2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2014

    Purpose
    =======

    Mergels multiple operations into one kernel:

    s=r
    s=s-alpha*v
        
    -> s = r - alpha * v

    Arguments
    =========

    int n                               dimension n
    double alpha            scalar 
    double *r               input r
    double *v               input v
    double *s               input/output s

    ========================================================================  */

extern "C" int
magma_dbicgmerge2(  int n, 
                    double *skp, 
                    double *r,
                    double *v, 
                    double *s ){

    
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( (n+BLOCK_SIZE-1)/BLOCK_SIZE );

    magma_dbicgmerge2_kernel<<<Gs, Bs, 0>>>( n, skp, r, v, s );

   return MAGMA_SUCCESS;
}

/* -------------------------------------------------------------------------- */

__global__ void 
magma_dbicgmerge3_kernel(  
                    int n, 
                    double *skp, 
                    double *p,
                    double *se,
                    double *t,
                    double *x, 
                    double *r ){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    double alpha=skp[0];
    double omega=skp[2];
    if( i<n ){
        double s;
        s = se[i];
        x[i] = x[i] + alpha * p[i] + omega * s;
        r[i] = s - omega * t[i];
    }

}

/*  -- MAGMA (version 1.5.0-beta2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2014

    Purpose
    =======

    Mergels multiple operations into one kernel:

    x=x+alpha*p
    x=x+omega*s
    r=s
    r=r-omega*t
        
    -> x = x + alpha * p + omega * s
    -> r = s - omega * t

    Arguments
    =========

    int n                               dimension n
    double alpha            scalar 
    double omega            scalar 
    double *p               input p
    double *s               input s
    double *t               input t
    double *x               input/output x
    double *r               input/output r

    ========================================================================  */

extern "C" int
magma_dbicgmerge3(  int n, 
                    double *skp,
                    double *p,
                    double *s,
                    double *t,
                    double *x, 
                    double *r ){

    
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( (n+BLOCK_SIZE-1)/BLOCK_SIZE );
    magma_dbicgmerge3_kernel<<<Gs, Bs, 0>>>( n, skp, p, s, t, x, r );

   return MAGMA_SUCCESS;
}

/* -------------------------------------------------------------------------- */

__global__ void 
magma_dbicgmerge4_kernel_1(  
                    double *skp ){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if( i==0 ){
        double tmp = skp[0];
        skp[0] = skp[4]/tmp;
    }
}

__global__ void 
magma_dbicgmerge4_kernel_2(  
                    double *skp ){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if( i==0 ){
        skp[2] = skp[6]/skp[7];
        skp[3] = skp[4];
    }
}

__global__ void 
magma_dbicgmerge4_kernel_3(  
                    double *skp ){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if( i==0 ){
        double tmp1 = skp[4]/skp[3];
        double tmp2 = skp[0] / skp[2];
        skp[1] =  tmp1*tmp2;
        //skp[1] =  skp[4]/skp[3] * skp[0] / skp[2];

    }
}

/*  -- MAGMA (version 1.5.0-beta2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2014

    Purpose
    =======

    Performs some parameter operations for the BiCGSTAB with scalars on GPU.

    Arguments
    =========

    int type                            kernel type
    double *skp             vector with parameters

    ========================================================================  */

extern "C" int
magma_dbicgmerge4(  int type, 
                    double *skp ){

    dim3 Bs( 2 );
    dim3 Gs( 1 );
    if( type == 1 )
        magma_dbicgmerge4_kernel_1<<<Gs, Bs, 0>>>( skp );
    else if( type == 2 )
        magma_dbicgmerge4_kernel_2<<<Gs, Bs, 0>>>( skp );
    else if( type == 3 )
        magma_dbicgmerge4_kernel_3<<<Gs, Bs, 0>>>( skp );
    else
        printf("error: no kernel called\n");

   return MAGMA_SUCCESS;
}

