#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date January 2015

       @generated from zbajac_csr.cu normal z -> c, Fri Jan 30 19:00:28 2015

*/

#include "common_magma.h"
#include "magmasparse_c.h"
#include "magma.h"


#define PRECISION_c
#define BLOCKSIZE 256


__global__ void
magma_cbajac_csr_ls_kernel(int localiters, int n, 
                            magmaFloatComplex * valD, 
                            magma_index_t * rowD, 
                            magma_index_t * colD, 
                            magmaFloatComplex * valR, 
                            magma_index_t * rowR,
                            magma_index_t * colR, 
                            const magmaFloatComplex *  __restrict__ b,                            
                            magmaFloatComplex * x ){

    int inddiag =  blockIdx.x*blockDim.x;
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    int i, j, start, end;   


    if(index<n){
    
        start=rowR[index];
        end  =rowR[index+1];

        magmaFloatComplex zero = MAGMA_C_MAKE(0.0, 0.0);
        magmaFloatComplex bl, tmp = zero, v = zero; 

#if (__CUDA_ARCH__ >= 350) && (defined(PRECISION_d) || defined(PRECISION_s))
        bl = __ldg( b+index );
#else
        bl = b[index];
#endif

        #pragma unroll
        for( i=start; i<end; i++ )
             v += valR[i] * x[ colR[i] ];

        start=rowD[index];
        end  =rowD[index+1];

        #pragma unroll
        for( i=start; i<end; i++ )
            tmp += valD[i] * x[ colD[i] ];

        v =  bl - v;

        /* add more local iterations */           
        __shared__ magmaFloatComplex local_x[ BLOCKSIZE ];
        local_x[threadIdx.x] = x[index] + ( v - tmp) / (valD[start]);
        __syncthreads();

        #pragma unroll
        for( j=0; j<localiters; j++ )
        {
            tmp = zero;
            #pragma unroll
            for( i=start; i<end; i++ )
                tmp += valD[i] * local_x[ colD[i] - inddiag];

            local_x[threadIdx.x] +=  ( v - tmp) / (valD[start]);
        }
        x[index] = local_x[threadIdx.x];
    }
}



__global__ void
magma_cbajac_csr_kernel(    
    int n, 
    magmaFloatComplex * valD, 
    magma_index_t * rowD, 
    magma_index_t * colD, 
    magmaFloatComplex * valR, 
    magma_index_t * rowR,
    magma_index_t * colR, 
    magmaFloatComplex * b,                                
    magmaFloatComplex * x ){

    int index = blockIdx.x*blockDim.x+threadIdx.x;
    int i, start, end;   

    if(index<n){
        
        magmaFloatComplex zero = MAGMA_C_MAKE(0.0, 0.0);
        magmaFloatComplex bl, tmp = zero, v = zero; 

#if (__CUDA_ARCH__ >= 350) && (defined(PRECISION_d) || defined(PRECISION_s))
        bl = __ldg( b+index );
#else
        bl = b[index];
#endif

        start=rowR[index];
        end  =rowR[index+1];

        #pragma unroll
        for( i=start; i<end; i++ )
             v += valR[i] * x[ colR[i] ];

        v =  bl - v;

        start=rowD[index];
        end  =rowD[index+1];

        #pragma unroll
        for( i=start; i<end; i++ )
            tmp += valD[i] * x[ colD[i] ];

        x[index] = x[index] + ( v - tmp ) / (valD[start]); 
    }
}









/**
    Purpose
    -------
    
    This routine is a block-asynchronous Jacobi iteration performing s
    local Jacobi-updates within the block. Input format is two CSR matrices,
    one containing the diagonal blocks, one containing the rest.

    Arguments
    ---------

    @param[in]
    localiters  magma_int_t
                number of local Jacobi-like updates

    @param[in]
    D           magma_c_sparse_matrix
                input matrix with diagonal blocks

    @param[in]
    R           magma_c_sparse_matrix
                input matrix with non-diagonal parts

    @param[in]
    b           magma_c_vector
                RHS

    @param[in]
    x           magma_c_vector*
                iterate/solution

    
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_cgegpuk
    ********************************************************************/

extern "C" magma_int_t
magma_cbajac_csr(
    magma_int_t localiters,
    magma_c_sparse_matrix D,
    magma_c_sparse_matrix R,
    magma_c_vector b,
    magma_c_vector *x,
    magma_queue_t queue )
{
    int blocksize1 = BLOCKSIZE;
    int blocksize2 = 1;

    int dimgrid1 = ( D.num_rows + blocksize1 -1 ) / blocksize1;
    int dimgrid2 = 1;
    int dimgrid3 = 1;

    dim3 grid( dimgrid1, dimgrid2, dimgrid3 );
    dim3 block( blocksize1, blocksize2, 1 );
    if ( R.nnz > 0 ) { 
        if ( localiters == 1 )
        magma_cbajac_csr_kernel<<< grid, block, 0, queue >>>
            ( D.num_rows, D.dval, D.drow, D.dcol, 
                            R.dval, R.drow, R.dcol, b.dval, x->dval );
        else
            magma_cbajac_csr_ls_kernel<<< grid, block, 0, queue >>>
            ( localiters, D.num_rows, D.dval, D.drow, D.dcol, 
                            R.dval, R.drow, R.dcol, b.dval, x->dval );
    }
    else {
        printf("error: all elements in diagonal block.\n");
    }

    return MAGMA_SUCCESS;
}



