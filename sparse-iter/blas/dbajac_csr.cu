#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.5.0-beta2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2014

       @generated from zbajac_csr.cu normal z -> d, Fri May 30 10:41:37 2014

*/

#include "common_magma.h"
#include "../include/magmasparse_d.h"
#include "../../include/magma.h"


#define PRECISION_d
#define BLOCKSIZE 256


__global__ void 
magma_dbajac_csr_ls_kernel(int localiters, int n, 
                            double *valD, 
                            magma_index_t *rowD, 
                            magma_index_t *colD, 
                            double *valR, 
                            magma_index_t *rowR,
                            magma_index_t *colR, 
                            const double * __restrict__ b,                            
                            double *x ){

    int ind_diag =  blockIdx.x*blockDim.x;
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    int i, j, start, end;   


    if(index<n){
    
        start=rowR[index];
        end  =rowR[index+1];

        double zero = MAGMA_D_MAKE(0.0, 0.0);
        double bl, tmp = zero, v = zero; 

#if (__CUDA_ARCH__ >= 350) && (defined(PRECISION_d) || defined(PRECISION_s))
        bl = __ldg( b+index );
#else
        bl = b[index];
#endif

        #pragma unroll
        for( i=start; i<end; i++ )
             v += valR[i] * x[ colR[i] ];

        start=rowD[index];
        end  =rowD[index+1];

        #pragma unroll
        for( i=start; i<end; i++ )
            tmp += valD[i] * x[ colD[i] ];

        v =  bl - v;

        /* add more local iterations */           
        __shared__ double local_x[ BLOCKSIZE ];
        local_x[threadIdx.x] = x[index] + ( v - tmp) / (valD[start]);
        __syncthreads();

        #pragma unroll
        for( j=0; j<localiters; j++ )
        {
            tmp = zero;
            #pragma unroll
            for( i=start; i<end; i++ )
                tmp += valD[i] * local_x[ colD[i] - ind_diag];

            local_x[threadIdx.x] +=  ( v - tmp) / (valD[start]);
        }
        x[index] = local_x[threadIdx.x];
    }
}



__global__ void 
magma_dbajac_csr_kernel(    int n, 
                            double *valD, 
                            magma_index_t *rowD, 
                            magma_index_t *colD, 
                            double *valR, 
                            magma_index_t *rowR,
                            magma_index_t *colR, 
                            double *b,                                
                            double *x ){

    int index = blockIdx.x*blockDim.x+threadIdx.x;
    int i, start, end;   

    if(index<n){
        
        double zero = MAGMA_D_MAKE(0.0, 0.0);
        double bl, tmp = zero, v = zero; 

#if (__CUDA_ARCH__ >= 350) && (defined(PRECISION_d) || defined(PRECISION_s))
        bl = __ldg( b+index );
#else
        bl = b[index];
#endif

        start=rowR[index];
        end  =rowR[index+1];

        #pragma unroll
        for( i=start; i<end; i++ )
             v += valR[i] * x[ colR[i] ];

        v =  bl - v;

        start=rowD[index];
        end  =rowD[index+1];

        #pragma unroll
        for( i=start; i<end; i++ )
            tmp += valD[i] * x[ colD[i] ];

        x[index] = x[index] + ( v - tmp ) / (valD[start]); 
    }
}









/*  -- MAGMA (version 1.5.0-beta2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2014

    Purpose
    =======
    
    This routine is a block-asynchronous Jacobi iteration performing s
    local Jacobi-updates within the block. Input format is two CSR matrices,
    one containing the diagonal blocks, one containing the rest.

    Arguments
    =========

    magma_int_t localiters              number of local Jacobi-like updates
    magma_d_sparse_matrix D             input matrix with diagonal blocks
    magma_d_sparse_matrix R             input matrix with non-diagonal parts
    magma_d_vector b                    RHS
    magma_d_vector *x                   iterate/solution
    
    ======================================================================    */

extern "C" magma_int_t
magma_dbajac_csr(   magma_int_t localiters,
                    magma_d_sparse_matrix D,
                    magma_d_sparse_matrix R,
                    magma_d_vector b,
                    magma_d_vector *x ){

    int blocksize1 = BLOCKSIZE;
    int blocksize2 = 1;

    int dimgrid1 = ( D.num_rows + blocksize1 -1 ) / blocksize1;
    int dimgrid2 = 1;
    int dimgrid3 = 1;

    dim3 grid( dimgrid1, dimgrid2, dimgrid3 );
    dim3 block( blocksize1, blocksize2, 1 );
    if( R.nnz > 0 ){ 
        if( localiters == 1 )
        magma_dbajac_csr_kernel<<< grid, block, 0, magma_stream >>>
            ( D.num_rows, D.val, D.row, D.col, 
                            R.val, R.row, R.col, b.val, x->val );
        else
            magma_dbajac_csr_ls_kernel<<< grid, block, 0, magma_stream >>>
            ( localiters, D.num_rows, D.val, D.row, D.col, 
                            R.val, R.row, R.col, b.val, x->val );
    }
    else{
        printf("error: all elements in diagonal block.\n");
    }

    return MAGMA_SUCCESS;
}



