#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2014

       @generated from zgecsrmv.cu normal z -> c, Sat Nov 15 19:54:21 2014

*/
#include "common_magma.h"

#if (GPUSHMEM < 200)
   #define BLOCK_SIZE 256
#else
   #define BLOCK_SIZE 256
#endif


// CSR-SpMV kernel
__global__ void 
cgecsrmv_kernel( 
    int num_rows, 
    int num_cols, 
    magmaFloatComplex alpha, 
    magmaFloatComplex_ptr dval, 
    magmaIndex_ptr drowptr, 
    magmaIndex_ptr dcolind,
    magmaFloatComplex_ptr dx,
    magmaFloatComplex beta, 
    magmaFloatComplex_ptr dy)
{

    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j;

    if(row<num_rows){
        magmaFloatComplex dot = MAGMA_C_ZERO;
        int start = drowptr[ row ];
        int end = drowptr[ row+1 ];
        for( j=start; j<end; j++)
            dot += dval[ j ] * dx[ dcolind[j] ];
        dy[ row ] =  dot *alpha + beta * dy[ row ];
    }
}

// shifted CSR-SpMV kernel
__global__ void 
cgecsrmv_kernel_shift( 
    int num_rows, 
    int num_cols, 
    magmaFloatComplex alpha, 
    magmaFloatComplex lambda, 
    magmaFloatComplex_ptr dval, 
    magmaIndex_ptr drowptr, 
    magmaIndex_ptr dcolind,
    magmaFloatComplex_ptr dx,
    magmaFloatComplex beta, 
    int offset,
    int blocksize,
    magmaIndex_ptr addrows,
    magmaFloatComplex_ptr dy)
{

    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j;

    if(row<num_rows){
        magmaFloatComplex dot = MAGMA_C_ZERO;
        int start = drowptr[ row ];
        int end = drowptr[ row+1 ];
        for( j=start; j<end; j++)
            dot += dval[ j ] * dx[ dcolind[j] ];
        if( row<blocksize )
            dy[ row ] = dot * alpha - lambda 
                        * dx[ offset+row ] + beta * dy [ row ];
        else
            dy[ row ] = dot * alpha - lambda 
                        * dx[ addrows[row-blocksize] ] + beta * dy [ row ];   
    }
}


/**
    Purpose
    -------
    
    This routine computes y = alpha *  A *  x + beta * y on the GPU.
    The input format is CSR (val, row, col).
    
    Arguments
    ---------
    
    @param[in]
    transA      magma_trans_t
                transposition parameter for A
                
    @param[in]
    m           magma_int_t
                number of rows in A

    @param[in]
    n           magma_int_t
                number of columns in A 

    @param[in]
    alpha       magmaFloatComplex
                scalar multiplier

    @param[in]
    dval        magmaFloatComplex_ptr
                array containing values of A in CSR

    @param[in]
    drowptr    magma_int_t*
                rowpointer of A in CSR

    @param[in]
    dcolind     magmaIndex_ptr
                columnindices of A in CSR

    @param[in]
    dx          magmaFloatComplex_ptr
                input vector x

    @param[in]
    beta        magmaFloatComplex
                scalar multiplier

    @param[out]
    dy          magmaFloatComplex_ptr
                input/output vector y

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_cblas
    ********************************************************************/

extern "C" magma_int_t
magma_cgecsrmv(
    magma_trans_t transA,
    magma_int_t m, magma_int_t n,
    magmaFloatComplex alpha,
    magmaFloatComplex_ptr dval,
    magmaIndex_ptr drowptr,
    magmaIndex_ptr dcolind,
    magmaFloatComplex_ptr dx,
    magmaFloatComplex beta,
    magmaFloatComplex_ptr dy,
    magma_queue_t queue )
{
    dim3 grid( (m+BLOCK_SIZE-1)/BLOCK_SIZE, 1, 1);
    magma_int_t threads = BLOCK_SIZE;
    cgecsrmv_kernel<<< grid, threads, 0, queue >>>
                    (m, n, alpha, dval, drowptr, dcolind, dx, beta, dy);

    return MAGMA_SUCCESS;
}



/**
    Purpose
    -------
    
    This routine computes y = alpha * ( A -lambda I ) * x + beta * y on the GPU.
    It is a shifted version of the CSR-SpMV.
    
    Arguments
    ---------
    
    @param[in]
    transA      magma_trans_t
                transposition parameter for A

    @param[in]
    m           magma_int_t
                number of rows in A

    @param[in]
    n           magma_int_t
                number of columns in A 

    @param[in]
    alpha       magmaFloatComplex
                scalar multiplier

    @param[in]
    lambda      magmaFloatComplex
                scalar multiplier

    @param[in]
    dval        magmaFloatComplex_ptr
                array containing values of A in CSR

    @param[in]
    drowptr    magma_int_t*
                rowpointer of A in CSR

    @param[in]
    dcolind     magmaIndex_ptr
                columnindices of A in CSR

    @param[in]
    dx          magmaFloatComplex_ptr
                input vector x

    @param[in]
    beta        magmaFloatComplex
                scalar multiplier

    @param[in]
    offset      magma_int_t 
                in case not the main diagonal is scaled
                
    @param[in]
    blocksize   magma_int_t 
                in case of processing multiple vectors  
                
    @param[in]
    addrows     magmaIndex_ptr
                in case the matrixpowerskernel is used
                
    @param[out]
    dy          magmaFloatComplex_ptr
                output vector y  
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_cblas
    ********************************************************************/

extern "C" magma_int_t
magma_cgecsrmv_shift(
    magma_trans_t transA,
    magma_int_t m, magma_int_t n,
    magmaFloatComplex alpha,
    magmaFloatComplex lambda,
    magmaFloatComplex_ptr dval,
    magmaIndex_ptr drowptr,
    magmaIndex_ptr dcolind,
    magmaFloatComplex_ptr dx,
    magmaFloatComplex beta,
    int offset,
    int blocksize,
    magmaIndex_ptr addrows,
    magmaFloatComplex_ptr dy,
    magma_queue_t queue )
{
    dim3 grid( (m+BLOCK_SIZE-1)/BLOCK_SIZE, 1, 1);
    magma_int_t threads = BLOCK_SIZE;
    cgecsrmv_kernel_shift<<< grid, threads, 0, queue >>>
                         (m, n, alpha, lambda, dval, drowptr, dcolind, dx, 
                                    beta, offset, blocksize, addrows, dy);

    return MAGMA_SUCCESS;
}



