#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.5.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date September 2014

       @generated from zgecsrmv.cu normal z -> c, Wed Sep 17 15:08:43 2014

*/
#include "common_magma.h"

#if (GPUSHMEM < 200)
   #define BLOCK_SIZE 256
#else
   #define BLOCK_SIZE 256
#endif


// CSR-SpMV kernel
__global__ void 
cgecsrmv_kernel( int num_rows, int num_cols, 
                 magmaFloatComplex alpha, 
                 magmaFloatComplex *d_val, 
                 magma_index_t *d_rowptr, 
                 magma_index_t *d_colind,
                 magmaFloatComplex *d_x,
                 magmaFloatComplex beta, 
                 magmaFloatComplex *d_y){

    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j;

    if(row<num_rows){
        magmaFloatComplex dot = MAGMA_C_ZERO;
        int start = d_rowptr[ row ];
        int end = d_rowptr[ row+1 ];
        for( j=start; j<end; j++)
            dot += d_val[ j ] * d_x[ d_colind[j] ];
        d_y[ row ] =  dot *alpha + beta * d_y[ row ];
    }
}

// shifted CSR-SpMV kernel
__global__ void 
cgecsrmv_kernel_shift( int num_rows, int num_cols, 
                       magmaFloatComplex alpha, 
                       magmaFloatComplex lambda, 
                       magmaFloatComplex *d_val, 
                       magma_index_t *d_rowptr, 
                       magma_index_t *d_colind,
                       magmaFloatComplex *d_x,
                       magmaFloatComplex beta, 
                       int offset,
                       int blocksize,
                       magma_index_t *add_rows,
                       magmaFloatComplex *d_y){

    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j;

    if(row<num_rows){
        magmaFloatComplex dot = MAGMA_C_ZERO;
        int start = d_rowptr[ row ];
        int end = d_rowptr[ row+1 ];
        for( j=start; j<end; j++)
            dot += d_val[ j ] * d_x[ d_colind[j] ];
        if( row<blocksize )
            d_y[ row ] = dot * alpha - lambda 
                        * d_x[ offset+row ] + beta * d_y [ row ];
        else
            d_y[ row ] = dot * alpha - lambda 
                        * d_x[ add_rows[row-blocksize] ] + beta * d_y [ row ];   
    }
}


/**
    Purpose
    -------
    
    This routine computes y = alpha *  A *  x + beta * y on the GPU.
    The input format is CSR (val, row, col).
    
    Arguments
    ---------
    
    @param
    transA      magma_trans_t
                transposition parameter for A
                
    @param
    m           magma_int_t
                number of rows in A

    @param
    n           magma_int_t
                number of columns in A 

    @param
    alpha       magmaFloatComplex
                scalar multiplier

    @param
    d_val       magmaFloatComplex*
                array containing values of A in CSR

    @param
    d_rowptr    magma_int_t*
                rowpointer of A in CSR

    @param
    d_colind    magma_int_t*
                columnindices of A in CSR

    @param
    d_x         magmaFloatComplex*
                input vector x

    @param
    beta        magmaFloatComplex
                scalar multiplier

    @param
    d_y         magmaFloatComplex*
                input/output vector y


    @ingroup magmasparse_cblas
    ********************************************************************/

extern "C" magma_int_t
magma_cgecsrmv(     magma_trans_t transA,
                    magma_int_t m, magma_int_t n,
                    magmaFloatComplex alpha,
                    magmaFloatComplex *d_val,
                    magma_index_t *d_rowptr,
                    magma_index_t *d_colind,
                    magmaFloatComplex *d_x,
                    magmaFloatComplex beta,
                    magmaFloatComplex *d_y ){

    dim3 grid( (m+BLOCK_SIZE-1)/BLOCK_SIZE, 1, 1);

    cgecsrmv_kernel<<< grid, BLOCK_SIZE, 0, magma_stream >>>
                    (m, n, alpha, d_val, d_rowptr, d_colind, d_x, beta, d_y);

    return MAGMA_SUCCESS;
}



/**
    Purpose
    -------
    
    This routine computes y = alpha * ( A -lambda I ) * x + beta * y on the GPU.
    It is a shifted version of the CSR-SpMV.
    
    Arguments
    ---------
    
    @param
    transA      magma_trans_t
                transposition parameter for A

    @param
    m           magma_int_t
                number of rows in A

    @param
    n           magma_int_t
                number of columns in A 

    @param
    alpha       magmaFloatComplex
                scalar multiplier

    @param
    lambda      magmaFloatComplex
                scalar multiplier

    @param
    d_val       magmaFloatComplex*
                array containing values of A in CSR

    @param
    d_rowptr    magma_int_t*
                rowpointer of A in CSR

    @param
    d_colind    magma_int_t*
                columnindices of A in CSR

    @param
    d_x         magmaFloatComplex*
                input vector x

    @param
    beta        magmaFloatComplex
                scalar multiplier

    @param
    offset      magma_int_t 
                in case not the main diagonal is scaled
                
    @param
    blocksize   magma_int_t 
                in case of processing multiple vectors  
                
    @param
    add_rows    magma_int_t*
                in case the matrixpowerskernel is used
                
    @param
    d_y         magmaFloatComplex*
                output vector y  

    @ingroup magmasparse_cblas
    ********************************************************************/

extern "C" magma_int_t
magma_cgecsrmv_shift( magma_trans_t transA,
                      magma_int_t m, magma_int_t n,
                      magmaFloatComplex alpha,
                      magmaFloatComplex lambda,
                      magmaFloatComplex *d_val,
                      magma_index_t *d_rowptr,
                      magma_index_t *d_colind,
                      magmaFloatComplex *d_x,
                      magmaFloatComplex beta,
                      int offset,
                      int blocksize,
                      magma_index_t *add_rows,
                      magmaFloatComplex *d_y ){

    dim3 grid( (m+BLOCK_SIZE-1)/BLOCK_SIZE, 1, 1);

    cgecsrmv_kernel_shift<<< grid, BLOCK_SIZE, 0, magma_stream >>>
                         (m, n, alpha, lambda, d_val, d_rowptr, d_colind, d_x, 
                                    beta, offset, blocksize, add_rows, d_y);

    return MAGMA_SUCCESS;
}



