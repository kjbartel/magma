#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.5.0-beta2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2014

       @generated from zgesellcmv.cu normal z -> c, Fri May 30 10:41:36 2014

*/
#include "hip/hip_runtime.h"
#include <stdio.h>
#include "common_magma.h"

#if (GPUSHMEM < 200)
   #define BLOCK_SIZE 128
#else
   #define BLOCK_SIZE 512
#endif


#define PRECISION_c


// SELLC SpMV kernel
// see paper by M. KREUTZER, G. HAGER, G WELLEIN, H. FEHSKE A. BISHOP
// A UNIFIED SPARSE MATRIX DATA FORMAT 
// FOR MODERN PROCESSORS WITH WIDE SIMD UNITS
__global__ void 
cgesellcmv_kernel(   int num_rows, 
                     int num_cols,
                     int blocksize,
                     magmaFloatComplex alpha, 
                     magmaFloatComplex *d_val, 
                     magma_index_t *d_colind,
                     magma_index_t *d_rowptr,
                     magmaFloatComplex *d_x,
                     magmaFloatComplex beta, 
                     magmaFloatComplex *d_y)
{
    // threads assigned to rows
    int Idx = blockDim.x * blockIdx.x + threadIdx.x ;
    int offset = d_rowptr[ blockIdx.x ];
    int border = (d_rowptr[ blockIdx.x+1 ]-offset)/blocksize;
    if(Idx < num_rows ){
        magmaFloatComplex dot = MAGMA_C_MAKE(0.0, 0.0);
        for ( int n = 0; n < border; n++){ 
            int col = d_colind [offset+ blocksize * n + threadIdx.x ];
            magmaFloatComplex val = d_val[offset+ blocksize * n + threadIdx.x];
            if( val != 0){
                  dot=dot+val*d_x[col];
            }
        }

        d_y[ Idx ] = dot * alpha + beta * d_y [ Idx ];
    }
}


/*  -- MAGMA (version 1.5.0-beta2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2014

    Purpose
    =======
    
    This routine computes y = alpha *  A^t *  x + beta * y on the GPU.
    Input format is SELLC/SELLP.
    
    Arguments
    =========

    magma_trans_t transA            transpose A?
    magma_int_t m                   number of rows in A
    magma_int_t n                   number of columns in A 
    magma_int_t blocksize           number of rows in one ELL-slice
    magma_int_t slices              number of slices in matrix
    magma_int_t alignment           number of threads assigned to one row (=1)
    magmaFloatComplex alpha        scalar multiplier
    magmaFloatComplex *d_val       array containing values of A in SELLC/P
    magma_int_t *d_colind           columnindices of A in SELLC/P
    magma_int_t *d_rowptr           rowpointer of SELLP
    magmaFloatComplex *d_x         input vector x
    magmaFloatComplex beta         scalar multiplier
    magmaFloatComplex *d_y         input/output vector y

    ======================================================================    */

extern "C" magma_int_t
magma_cgesellcmv(   magma_trans_t transA,
                    magma_int_t m, magma_int_t n,
                    magma_int_t blocksize,
                    magma_int_t slices,
                    magma_int_t alignment,
                    magmaFloatComplex alpha,
                    magmaFloatComplex *d_val,
                    magma_index_t *d_colind,
                    magma_index_t *d_rowptr,
                    magmaFloatComplex *d_x,
                    magmaFloatComplex beta,
                    magmaFloatComplex *d_y ){



   // the kernel can only handle up to 65535 slices 
   // (~2M rows for blocksize 32)
   dim3 grid( slices, 1, 1);

   cgesellcmv_kernel<<< grid, blocksize, 0, magma_stream >>>
   ( m, n, blocksize, alpha,
        d_val, d_colind, d_rowptr, d_x, beta, d_y );

   return MAGMA_SUCCESS;
}

