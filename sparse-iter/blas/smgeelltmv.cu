#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2014

       @generated from zmgeelltmv.cu normal z -> s, Sat Nov 15 19:54:21 2014

*/

#include "common_magma.h"

#if (GPUSHMEM < 200)
   #define BLOCK_SIZE 128
#else
   #define BLOCK_SIZE 512
#endif



__global__ void 
smgeelltmv_kernel( 
        int num_rows, 
        int num_cols,
        int num_vecs,
        int num_cols_per_row,
        float alpha, 
        magmaFloat_ptr dval, 
        magmaIndex_ptr dcolind,
        magmaFloat_ptr dx,
        float beta, 
        magmaFloat_ptr dy)
{
    extern __shared__ float dot[];
    int row = blockDim.x * blockIdx.x + threadIdx.x ;
    if(row < num_rows ){
        for( int i=0; i<num_vecs; i++ )
                dot[ threadIdx.x+ i*blockDim.x ] = MAGMA_S_MAKE(0.0, 0.0);
        for ( int n = 0; n < num_cols_per_row ; n ++){
            int col = dcolind [ num_rows * n + row ];
            float val = dval [ num_rows * n + row ];
            if( val != 0){
                for( int i=0; i<num_vecs; i++ )
                    dot[ threadIdx.x + i*blockDim.x ] += 
                                        val * dx[col + i * num_cols ];
            }
        }
        for( int i=0; i<num_vecs; i++ )
                dy[ row + i*num_cols ] = dot[ threadIdx.x + i*blockDim.x ] 
                                * alpha + beta * dy [ row + i*num_cols ];
    }
}





/**
    Purpose
    -------
    
    This routine computes Y = alpha *  A *  X + beta * Y for X and Y sets of 
    num_vec vectors on the GPU. Input format is ELL. 
    
    Arguments
    ---------

    @param[in]
    transA      magma_trans_t
                transposition parameter for A

    @param[in]
    m           magma_int_t
                number of rows in A

    @param[in]
    n           magma_int_t
                number of columns in A 
                
    @param[in]
    num_vecs    mama_int_t
                number of vectors
                
    @param[in]
    nnz_per_row magma_int_t
                number of elements in the longest row 
                
    @param[in]
    alpha       float
                scalar multiplier

    @param[in]
    dval        magmaFloat_ptr
                array containing values of A in ELL

    @param[in]
    dcolind     magmaIndex_ptr
                columnindices of A in ELL

    @param[in]
    dx          magmaFloat_ptr
                input vector x

    @param[in]
    beta        float
                scalar multiplier

    @param[out]
    dy          magmaFloat_ptr
                input/output vector y

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_sblas
    ********************************************************************/

extern "C" magma_int_t
magma_smgeelltmv(
    magma_trans_t transA,
    magma_int_t m, magma_int_t n,
    magma_int_t num_vecs,
    magma_int_t nnz_per_row,
    float alpha,
    magmaFloat_ptr dval,
    magmaIndex_ptr dcolind,
    magmaFloat_ptr dx,
    float beta,
    magmaFloat_ptr dy,
    magma_queue_t queue )
{
    dim3 grid( (m+BLOCK_SIZE-1)/BLOCK_SIZE, 1, 1);
    magma_int_t threads = BLOCK_SIZE;
    unsigned int MEM_SIZE =  num_vecs* BLOCK_SIZE 
                * sizeof( float ); // num_vecs vectors 
    smgeelltmv_kernel<<< grid, threads, MEM_SIZE, queue >>>
        ( m, n, num_vecs, nnz_per_row, alpha, dval, dcolind, dx, beta, dy );


    return MAGMA_SUCCESS;
}



