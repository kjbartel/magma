#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.5.0-beta3) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date July 2014

       @generated from zmgeelltmv.cu normal z -> s, Fri Jul 18 17:34:28 2014

*/

#include "common_magma.h"

#if (GPUSHMEM < 200)
   #define BLOCK_SIZE 128
#else
   #define BLOCK_SIZE 512
#endif



__global__ void 
smgeelltmv_kernel( int num_rows, 
                 int num_cols,
                 int num_vecs,
                 int num_cols_per_row,
                 float alpha, 
                 float *d_val, 
                 magma_index_t *d_colind,
                 float *d_x,
                 float beta, 
                 float *d_y)
{
    extern __shared__ float dot[];
    int row = blockDim.x * blockIdx.x + threadIdx.x ;
    if(row < num_rows ){
        for( int i=0; i<num_vecs; i++ )
                dot[ threadIdx.x+ i*blockDim.x ] = MAGMA_S_MAKE(0.0, 0.0);
        for ( int n = 0; n < num_cols_per_row ; n ++){
            int col = d_colind [ num_rows * n + row ];
            float val = d_val [ num_rows * n + row ];
            if( val != 0){
                for( int i=0; i<num_vecs; i++ )
                    dot[ threadIdx.x + i*blockDim.x ] += 
                                        val * d_x[col + i * num_cols ];
            }
        }
        for( int i=0; i<num_vecs; i++ )
                d_y[ row + i*num_cols ] = dot[ threadIdx.x + i*blockDim.x ] 
                                * alpha + beta * d_y [ row + i*num_cols ];
    }
}





/**
    Purpose
    -------
    
    This routine computes Y = alpha *  A *  X + beta * Y for X and Y sets of 
    num_vec vectors on the GPU. Input format is ELL. 
    
    Arguments
    ---------

    @param
    transA      magma_trans_t
                transposition parameter for A

    @param
    m           magma_int_t
                number of rows in A

    @param
    n           magma_int_t
                number of columns in A 
                
    @param
    num_vecs    mama_int_t
                number of vectors
                
    @param
    nnz_per_row magma_int_t
                number of elements in the longest row 
                
    @param
    alpha       float
                scalar multiplier

    @param
    d_val       float*
                array containing values of A in ELL

    @param
    d_colind    magma_int_t*
                columnindices of A in ELL

    @param
    d_x         float*
                input vector x

    @param
    beta        float
                scalar multiplier

    @param
    d_y         float*
                input/output vector y


    @ingroup magmasparse_sblas
    ********************************************************************/

extern "C" magma_int_t
magma_smgeelltmv(  magma_trans_t transA,
                   magma_int_t m, magma_int_t n,
                   magma_int_t num_vecs,
                   magma_int_t nnz_per_row,
                   float alpha,
                   float *d_val,
                   magma_index_t *d_colind,
                   float *d_x,
                   float beta,
                   float *d_y ){



    dim3 grid( (m+BLOCK_SIZE-1)/BLOCK_SIZE, 1, 1);
    unsigned int MEM_SIZE =  num_vecs* BLOCK_SIZE 
                * sizeof( float ); // num_vecs vectors 
    smgeelltmv_kernel<<< grid, BLOCK_SIZE, MEM_SIZE >>>
        ( m, n, num_vecs, nnz_per_row, alpha, d_val, d_colind, d_x, beta, d_y );


    return MAGMA_SUCCESS;
}



