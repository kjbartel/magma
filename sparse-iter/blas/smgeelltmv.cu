#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.5.0-beta2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2014

       @generated from zmgeelltmv.cu normal z -> s, Fri May 30 10:41:37 2014

*/

#include "common_magma.h"

#if (GPUSHMEM < 200)
   #define BLOCK_SIZE 128
#else
   #define BLOCK_SIZE 512
#endif



__global__ void 
smgeelltmv_kernel( int num_rows, 
                 int num_cols,
                 int num_vecs,
                 int num_cols_per_row,
                 float alpha, 
                 float *d_val, 
                 magma_index_t *d_colind,
                 float *d_x,
                 float beta, 
                 float *d_y)
{
    extern __shared__ float dot[];
    int row = blockDim.x * blockIdx.x + threadIdx.x ;
    if(row < num_rows ){
        for( int i=0; i<num_vecs; i++ )
                dot[ threadIdx.x+ i*blockDim.x ] = MAGMA_S_MAKE(0.0, 0.0);
        for ( int n = 0; n < num_cols_per_row ; n ++){
            int col = d_colind [ num_rows * n + row ];
            float val = d_val [ num_rows * n + row ];
            if( val != 0){
                for( int i=0; i<num_vecs; i++ )
                    dot[ threadIdx.x + i*blockDim.x ] += 
                                        val * d_x[col + i * num_cols ];
            }
        }
        for( int i=0; i<num_vecs; i++ )
                d_y[ row + i*num_cols ] = dot[ threadIdx.x + i*blockDim.x ] 
                                * alpha + beta * d_y [ row + i*num_cols ];
    }
}





/*  -- MAGMA (version 1.5.0-beta2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2014

    Purpose
    =======
    
    This routine computes Y = alpha *  A *  X + beta * Y for X and Y sets of 
    num_vec vectors on the GPU. Input format is ELL. 
    
    Arguments
    =========

    magma_int_t m                   number of rows in A
    magma_int_t n                   number of columns in A 
    mama_int_t num_vecs             number of vectors
    float alpha        scalar multiplier
    float *d_val       array containing values of A in ELL
    magma_int_t *d_colind           columnindices of A in ELL
    float *d_x         input vector x
    float beta         scalar multiplier
    float *d_y         input/output vector y

    ======================================================================    */

extern "C" magma_int_t
magma_smgeelltmv(  magma_trans_t transA,
                   magma_int_t m, magma_int_t n,
                   magma_int_t num_vecs,
                   magma_int_t nnz_per_row,
                   float alpha,
                   float *d_val,
                   magma_index_t *d_colind,
                   float *d_x,
                   float beta,
                   float *d_y ){



    dim3 grid( (m+BLOCK_SIZE-1)/BLOCK_SIZE, 1, 1);
    unsigned int MEM_SIZE =  num_vecs* BLOCK_SIZE 
                * sizeof( float ); // num_vecs vectors 
    smgeelltmv_kernel<<< grid, BLOCK_SIZE, MEM_SIZE >>>
        ( m, n, num_vecs, nnz_per_row, alpha, d_val, d_colind, d_x, beta, d_y );


    return MAGMA_SUCCESS;
}



