#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2015

       @generated from zjacobisetup.cu normal z -> c, Sun May  3 11:22:58 2015
       @author Hartwig Anzt

*/
#include "common_magmasparse.h"

#define BLOCK_SIZE 128


#define PRECISION_c

__global__ void 
cvjacobisetup_gpu(  int num_rows, 
                    int num_vecs,
                    magmaFloatComplex *b, 
                    magmaFloatComplex *d, 
                    magmaFloatComplex *c,
                    magmaFloatComplex *x){

    int row = blockDim.x * blockIdx.x + threadIdx.x ;

    if(row < num_rows ){
        for( int i=0; i<num_vecs; i++ ){
            c[row+i*num_rows] = b[row+i*num_rows] / d[row];
            x[row+i*num_rows] = c[row+i*num_rows];
        }
    }
}





/**
    Purpose
    -------

    Prepares the Jacobi Iteration according to
       x^(k+1) = D^(-1) * b - D^(-1) * (L+U) * x^k
       x^(k+1) =      c     -       M        * x^k.

    Returns the vector c. It calls a GPU kernel

    Arguments
    ---------

    @param[in]
    num_rows    magma_int_t
                number of rows
                
    @param[in]
    b           magma_c_matrix
                RHS b

    @param[in]
    d           magma_c_matrix
                vector with diagonal entries

    @param[out]
    c           magma_c_matrix*
                c = D^(-1) * b

    @param[out]
    x           magma_c_matrix*
                iteration vector
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_cgegpuk
    ********************************************************************/

extern "C" magma_int_t
magma_cjacobisetup_vector_gpu(
    int num_rows, 
    magma_c_matrix b, 
    magma_c_matrix d, 
    magma_c_matrix c,
    magma_c_matrix *x,
    magma_queue_t queue )
{
    dim3 grid( magma_ceildiv( num_rows, BLOCK_SIZE ) );
    int num_vecs = b.num_rows / num_rows;
    magma_int_t threads = BLOCK_SIZE;
    cvjacobisetup_gpu<<< grid, threads, 0 >>>
                ( num_rows, num_vecs, b.dval, d.dval, c.dval, x->val );

    return MAGMA_SUCCESS;
}






__global__ void 
cjacobidiagscal_kernel(  int num_rows,
                         int num_vecs, 
                    magmaFloatComplex *b, 
                    magmaFloatComplex *d, 
                    magmaFloatComplex *c){

    int row = blockDim.x * blockIdx.x + threadIdx.x ;

    if(row < num_rows ){
        for( int i=0; i<num_vecs; i++)
            c[row+i*num_rows] = b[row+i*num_rows] * d[row];
    }
}





/**
    Purpose
    -------

    Prepares the Jacobi Iteration according to
       x^(k+1) = D^(-1) * b - D^(-1) * (L+U) * x^k
       x^(k+1) =      c     -       M        * x^k.

    Returns the vector c. It calls a GPU kernel

    Arguments
    ---------

    @param[in]
    num_rows    magma_int_t
                number of rows
                
    @param[in]
    b           magma_c_matrix
                RHS b

    @param[in]
    d           magma_c_matrix
                vector with diagonal entries

    @param[out]
    c           magma_c_matrix*
                c = D^(-1) * b
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_c
    ********************************************************************/

extern "C" magma_int_t
magma_cjacobi_diagscal(
    int num_rows, 
    magma_c_matrix d, 
    magma_c_matrix b, 
    magma_c_matrix *c,
    magma_queue_t queue )
{
    dim3 grid( magma_ceildiv( num_rows, BLOCK_SIZE ));
    int num_vecs = b.num_rows*b.num_cols/num_rows;
    magma_int_t threads = BLOCK_SIZE;
    cjacobidiagscal_kernel<<< grid, threads, 0 >>>( num_rows, num_vecs, b.dval, d.dval, c->val );

    return MAGMA_SUCCESS;
}













__global__ void 
cjacobiupdate_kernel(  int num_rows,
                       int num_cols, 
                    magmaFloatComplex *t, 
                    magmaFloatComplex *b, 
                    magmaFloatComplex *d, 
                    magmaFloatComplex *x){

    int row = blockDim.x * blockIdx.x + threadIdx.x ;

    if(row < num_rows ){
        for( int i=0; i<num_cols; i++)
            x[row+i*num_rows] += (b[row+i*num_rows]-t[row+i*num_rows]) * d[row];
    }
}


/**
    Purpose
    -------

    Updates the iteration vector x for the Jacobi iteration
    according to
        x=x+d.*(b-t)
    where d is the diagonal of the system matrix A and t=Ax.

    Arguments
    ---------

    @param[in]
    num_rows    magma_int_t
                number of rows
                
    @param[in]
    num_cols    magma_int_t
                number of cols
                
    @param[in]
    t           magma_c_matrix
                t = A*x
                
    @param[in]
    b           magma_c_matrix
                RHS b
                
    @param[in]
    d           magma_c_matrix
                vector with diagonal entries

    @param[out]
    x           magma_c_matrix*
                iteration vector
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_c
    ********************************************************************/

extern "C" magma_int_t
magma_cjacobiupdate(
    magma_c_matrix t, 
    magma_c_matrix b, 
    magma_c_matrix d, 
    magma_c_matrix *x,
    magma_queue_t queue )
{

    dim3 grid( magma_ceildiv( t.num_rows, BLOCK_SIZE ));
    magma_int_t threads = BLOCK_SIZE;
    cjacobiupdate_kernel<<< grid, threads, 0 >>>( t.num_rows, t.num_cols, t.dval, b.dval, d.dval, x->dval );

    return MAGMA_SUCCESS;
}










__global__ void 
cjacobispmvupdate_kernel(  
    int num_rows,
    int num_cols, 
    magmaFloatComplex * dval, 
    magma_index_t * drowptr, 
    magma_index_t * dcolind,
    magmaFloatComplex *t, 
    magmaFloatComplex *b, 
    magmaFloatComplex *d, 
    magmaFloatComplex *x ){



    int row = blockDim.x * blockIdx.x + threadIdx.x ;
    int j;

    if(row<num_rows){
        magmaFloatComplex dot = MAGMA_C_ZERO;
        int start = drowptr[ row ];
        int end = drowptr[ row+1 ];
        for( int i=0; i<num_cols; i++){
            for( j=start; j<end; j++){
                dot += dval[ j ] * x[ dcolind[j]+i*num_rows ];
            }
            x[row+i*num_rows] += (b[row+i*num_rows]-dot) * d[row];
        }
    }
}





/**
    Purpose
    -------

    Updates the iteration vector x for the Jacobi iteration
    according to
        x=x+d.*(b-Ax)


    Arguments
    ---------

    @param[in]
    maxiter     magma_int_t
                number of Jacobi iterations   
                
    @param[in]
    A           magma_c_matrix
                system matrix
                
    @param[in]
    t           magma_c_matrix
                workspace
                
    @param[in]
    b           magma_c_matrix
                RHS b
                
    @param[in]
    d           magma_c_matrix
                vector with diagonal entries

    @param[out]
    x           magma_c_matrix*
                iteration vector
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_c
    ********************************************************************/

extern "C" magma_int_t
magma_cjacobispmvupdate(
    magma_int_t maxiter,
    magma_c_matrix A,
    magma_c_matrix t, 
    magma_c_matrix b, 
    magma_c_matrix d, 
    magma_c_matrix *x,
    magma_queue_t queue )
{

    // local variables
    magmaFloatComplex c_zero = MAGMA_C_ZERO, c_one = MAGMA_C_ONE;
    dim3 grid( magma_ceildiv( t.num_rows, BLOCK_SIZE ));
    magma_int_t threads = BLOCK_SIZE;

    for( magma_int_t i=0; i<maxiter; i++ ) {
        // distinct routines imply synchronization
        // magma_c_spmv( c_one, A, *x, c_zero, t, queue );                // t =  A * x
        // cjacobiupdate_kernel<<< grid, threads, 0 >>>( t.num_rows, t.num_cols, t.dval, b.dval, d.dval, x->dval );
        // merged in one implies asynchronous update
        cjacobispmvupdate_kernel<<< grid, threads, 0 >>>
            ( t.num_rows, t.num_cols, A.dval, A.drow, A.dcol, t.dval, b.dval, d.dval, x->dval );

    }

    return MAGMA_SUCCESS;
}









