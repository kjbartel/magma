#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.6.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2015

       @generated from zgecsrmv.cu normal z -> s, Sun May  3 11:22:58 2015

*/
#include "common_magma.h"

#define BLOCK_SIZE 256


// CSR-SpMV kernel
__global__ void 
sgecsrmv_kernel( 
    int num_rows, 
    int num_cols, 
    float alpha, 
    float * dval, 
    magma_index_t * drowptr, 
    magma_index_t * dcolind,
    float * dx,
    float beta, 
    float * dy)
{

    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j;

    if(row<num_rows){
        float dot = MAGMA_S_ZERO;
        int start = drowptr[ row ];
        int end = drowptr[ row+1 ];
        for( j=start; j<end; j++)
            dot += dval[ j ] * dx[ dcolind[j] ];
        dy[ row ] =  dot *alpha + beta * dy[ row ];
    }
}

// shifted CSR-SpMV kernel
__global__ void 
sgecsrmv_kernel_shift( 
    int num_rows, 
    int num_cols, 
    float alpha, 
    float lambda, 
    float * dval, 
    magma_index_t * drowptr, 
    magma_index_t * dcolind,
    float * dx,
    float beta, 
    int offset,
    int blocksize,
    magma_index_t * addrows,
    float * dy)
{

    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j;

    if(row<num_rows){
        float dot = MAGMA_S_ZERO;
        int start = drowptr[ row ];
        int end = drowptr[ row+1 ];
        for( j=start; j<end; j++)
            dot += dval[ j ] * dx[ dcolind[j] ];
        if( row<blocksize )
            dy[ row ] = dot * alpha - lambda 
                        * dx[ offset+row ] + beta * dy [ row ];
        else
            dy[ row ] = dot * alpha - lambda 
                        * dx[ addrows[row-blocksize] ] + beta * dy [ row ];   
    }
}


/**
    Purpose
    -------
    
    This routine computes y = alpha *  A *  x + beta * y on the GPU.
    The input format is CSR (val, row, col).
    
    Arguments
    ---------
    
    @param[in]
    transA      magma_trans_t
                transposition parameter for A
                
    @param[in]
    m           magma_int_t
                number of rows in A

    @param[in]
    n           magma_int_t
                number of columns in A 

    @param[in]
    alpha       float
                scalar multiplier

    @param[in]
    dval        magmaFloat_ptr
                array containing values of A in CSR

    @param[in]
    drowptr     magmaIndex_ptr
                rowpointer of A in CSR

    @param[in]
    dcolind     magmaIndex_ptr
                columnindices of A in CSR

    @param[in]
    dx          magmaFloat_ptr
                input vector x

    @param[in]
    beta        float
                scalar multiplier

    @param[out]
    dy          magmaFloat_ptr
                input/output vector y

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_sblas
    ********************************************************************/

extern "C" magma_int_t
magma_sgecsrmv(
    magma_trans_t transA,
    magma_int_t m, magma_int_t n,
    float alpha,
    magmaFloat_ptr dval,
    magmaIndex_ptr drowptr,
    magmaIndex_ptr dcolind,
    magmaFloat_ptr dx,
    float beta,
    magmaFloat_ptr dy,
    magma_queue_t queue )
{
    dim3 grid( magma_ceildiv( m, BLOCK_SIZE ) );
    magma_int_t threads = BLOCK_SIZE;
    sgecsrmv_kernel<<< grid, threads, 0, queue >>>
                    (m, n, alpha, dval, drowptr, dcolind, dx, beta, dy);

    return MAGMA_SUCCESS;
}



/**
    Purpose
    -------
    
    This routine computes y = alpha * ( A -lambda I ) * x + beta * y on the GPU.
    It is a shifted version of the CSR-SpMV.
    
    Arguments
    ---------
    
    @param[in]
    transA      magma_trans_t
                transposition parameter for A

    @param[in]
    m           magma_int_t
                number of rows in A

    @param[in]
    n           magma_int_t
                number of columns in A 

    @param[in]
    alpha       float
                scalar multiplier

    @param[in]
    lambda      float
                scalar multiplier

    @param[in]
    dval        magmaFloat_ptr
                array containing values of A in CSR

    @param[in]
    drowptr     magmaIndex_ptr
                rowpointer of A in CSR

    @param[in]
    dcolind     magmaIndex_ptr
                columnindices of A in CSR

    @param[in]
    dx          magmaFloat_ptr
                input vector x

    @param[in]
    beta        float
                scalar multiplier

    @param[in]
    offset      magma_int_t 
                in case not the main diagonal is scaled
                
    @param[in]
    blocksize   magma_int_t 
                in case of processing multiple vectors  
                
    @param[in]
    addrows     magmaIndex_ptr
                in case the matrixpowerskernel is used
                
    @param[out]
    dy          magmaFloat_ptr
                output vector y  
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_sblas
    ********************************************************************/

extern "C" magma_int_t
magma_sgecsrmv_shift(
    magma_trans_t transA,
    magma_int_t m, magma_int_t n,
    float alpha,
    float lambda,
    magmaFloat_ptr dval,
    magmaIndex_ptr drowptr,
    magmaIndex_ptr dcolind,
    magmaFloat_ptr dx,
    float beta,
    int offset,
    int blocksize,
    magma_index_t * addrows,
    magmaFloat_ptr dy,
    magma_queue_t queue )
{
    dim3 grid( magma_ceildiv( m, BLOCK_SIZE ) );
    magma_int_t threads = BLOCK_SIZE;
    sgecsrmv_kernel_shift<<< grid, threads, 0, queue >>>
                         (m, n, alpha, lambda, dval, drowptr, dcolind, dx, 
                                    beta, offset, blocksize, addrows, dy);

    return MAGMA_SUCCESS;
}



